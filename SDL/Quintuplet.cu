#include "hip/hip_runtime.h"
#ifdef __HIPCC__
#define CUDA_CONST_VAR __device__
#endif
# include "Quintuplet.cuh"
//#ifdef CACHE_ALLOC
#include "allocate.h"
//#endif

SDL::quintuplets::quintuplets()
{
    tripletIndices = nullptr;
    lowerModuleIndices = nullptr;
    nQuintuplets = nullptr;
    innerRadius = nullptr;
    innerRadiusMin = nullptr;
    innerRadiusMax = nullptr;
    bridgeRadius = nullptr;
    bridgeRadiusMin = nullptr;
    bridgeRadiusMax = nullptr;
    outerRadius = nullptr;
    outerRadiusMin = nullptr;
    outerRadiusMax = nullptr;

}

void SDL::quintuplets::freeMemory()
{
    hipFree(tripletIndices);
    hipFree(lowerModuleIndices);
    hipFree(nQuintuplets);
    hipFree(innerRadius);
    hipFree(innerRadiusMin);
    hipFree(innerRadiusMax);
    hipFree(bridgeRadius);
    hipFree(bridgeRadiusMin);
    hipFree(bridgeRadiusMax);
    hipFree(outerRadius);
    hipFree(outerRadiusMin);
    hipFree(outerRadiusMax);
}
void SDL::createQuintupletsInUnifiedMemory(struct SDL::quintuplets& quintupletsInGPU, unsigned int maxQuintuplets, unsigned int nLowerModules)
{
    hipMallocManaged(&quintupletsInGPU.tripletIndices, 2 * maxQuintuplets * nLowerModules * sizeof(unsigned int));
    hipMallocManaged(&quintupletsInGPU.lowerModuleIndices, 5 * maxQuintuplets * nLowerModules * sizeof(unsigned int)); 
    hipMallocManaged(&quintupletsInGPU.innerRadius, maxQuintuplets * nLowerModules * sizeof(float));
    hipMallocManaged(&quintupletsInGPU.innerRadiusMin, maxQuintuplets * nLowerModules * sizeof(float));
    hipMallocManaged(&quintupletsInGPU.innerRadiusMax, maxQuintuplets * nLowerModules * sizeof(float));
    hipMallocManaged(&quintupletsInGPU.bridgeRadius, maxQuintuplets * nLowerModules * sizeof(float));
    hipMallocManaged(&quintupletsInGPU.bridgeRadiusMin, maxQuintuplets * nLowerModules * sizeof(float));
    hipMallocManaged(&quintupletsInGPU.bridgeRadiusMax, maxQuintuplets * nLowerModules * sizeof(float));
    hipMallocManaged(&quintupletsInGPU.outerRadius, maxQuintuplets * nLowerModules * sizeof(float));
    hipMallocManaged(&quintupletsInGPU.outerRadiusMin, maxQuintuplets * nLowerModules * sizeof(float));
    hipMallocManaged(&quintupletsInGPU.outerRadiusMax, maxQuintuplets * nLowerModules * sizeof(float));

    hipMallocManaged(&quintupletsInGPU.nQuintuplets, nLowerModules * sizeof(unsigned int));

#pragma omp parallel for
    for(size_t i = 0; i<nLowerModules;i++)
    {
        quintupletsInGPU.nQuintuplets[i] = 0;
    }

}

__device__ void SDL::addQuintupletToMemory(struct SDL::quintuplets& quintupletsInGPU, unsigned int innerTripletIndex, unsigned int outerTripletIndex, unsigned int lowerModule1, unsigned int lowerModule2, unsigned int lowerModule3, unsigned int lowerModule4, unsigned int lowerModule5, float innerRadius, float innerRadiusMin, float innerRadiusMax, float outerRadius, float outerRadiusMin, float outerRadiusMax, float bridgeRadius, float bridgeRadiusMin, float bridgeRadiusMax, unsigned int quintupletIndex)
{
    quintupletsInGPU.tripletIndices[2 * quintupletIndex] = innerTripletIndex;
    quintupletsInGPU.tripletIndices[2 * quintupletIndex + 1] = outerTripletIndex;

    quintupletsInGPU.innerRadius[quintupletIndex] = innerRadius;
    quintupletsInGPU.innerRadiusMin[quintupletIndex] = innerRadiusMin;
    quintupletsInGPU.innerRadiusMax[quintupletIndex] = innerRadiusMax;
    quintupletsInGPU.outerRadius[quintupletIndex] = outerRadius;
    quintupletsInGPU.outerRadiusMin[quintupletIndex] = outerRadiusMin;
    quintupletsInGPU.outerRadiusMax[quintupletIndex] = outerRadiusMax;
    quintupletsInGPU.bridgeRadius[quintupletIndex] = bridgeRadius;
    quintupletsInGPU.bridgeRadiusMin[quintupletIndex] = bridgeRadiusMin;
    quintupletsInGPU.bridgeRadiusMax[quintupletIndex] = bridgeRadiusMax;

    quintupletsInGPU.lowerModuleIndices[5 * quintupletIndex] = lowerModule1;
    quintupletsInGPU.lowerModuleIndices[5 * quintupletIndex + 1] = lowerModule2;
    quintupletsInGPU.lowerModuleIndices[5 * quintupletIndex + 2] = lowerModule3;
    quintupletsInGPU.lowerModuleIndices[5 * quintupletIndex + 3] = lowerModule4;
    quintupletsInGPU.lowerModuleIndices[5 * quintupletIndex + 4] = lowerModule5;

}

__device__ bool SDL::runQuintupletDefaultAlgo(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::triplets& tripletsInGPU, unsigned int lowerModuleIndex1, unsigned int lowerModuleIndex2, unsigned int lowerModuleIndex3, unsigned int lowerModuleIndex4, unsigned int lowerModuleIndex5, unsigned int innerTripletIndex, unsigned int outerTripletIndex, float& innerRadius, float& innerRadiusMin, float& innerRadiusMax, float& outerRadius, float& outerRadiusMin, float& outerRadiusMax, float& bridgeRadius, float& bridgeRadiusMin, float& bridgeRadiusMax)
{
    bool pass = true;

    if(not T5HasCommonMiniDoublet(tripletsInGPU, segmentsInGPU, innerTripletIndex, outerTripletIndex))
    {
        pass = false;
    }

    unsigned int firstSegmentIndex = tripletsInGPU.segmentIndices[2 * innerTripletIndex];
    unsigned int secondSegmentIndex = tripletsInGPU.segmentIndices[2 * innerTripletIndex + 1];
    unsigned int thirdSegmentIndex = tripletsInGPU.segmentIndices[2 * outerTripletIndex];
    unsigned int fourthSegmentIndex = tripletsInGPU.segmentIndices[2 * outerTripletIndex + 1];
    //apply T4 criteria between segments 1 and 3
    float zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut; //temp stuff
/*    if(not runTrackletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, segmentsInGPU.innerLowerModuleIndices[firstSegmentIndex], segmentsInGPU.outerLowerModuleIndices[firstSegmentIndex], segmentsInGPU.innerLowerModuleIndices[thirdSegmentIndex], segmentsInGPU.outerLowerModuleIndices[thirdSegmentIndex], firstSegmentIndex, thirdSegmentIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut))
    {
        pass = false;
    }
    if(not runTrackletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, segmentsInGPU.innerLowerModuleIndices[firstSegmentIndex], segmentsInGPU.outerLowerModuleIndices[firstSegmentIndex], segmentsInGPU.innerLowerModuleIndices[fourthSegmentIndex], segmentsInGPU.outerLowerModuleIndices[fourthSegmentIndex], firstSegmentIndex, fourthSegmentIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut))
    {
        pass = false;
    }*/

    //radius computation from the three triplet MD anchor hits
    unsigned int innerTripletFirstSegmentAnchorHitIndex = segmentsInGPU.innerMiniDoubletAnchorHitIndices[firstSegmentIndex];
    unsigned int innerTripletSecondSegmentAnchorHitIndex = segmentsInGPU.outerMiniDoubletAnchorHitIndices[firstSegmentIndex]; //same as second segment inner MD anchorhit index
    unsigned int innerTripletThirdSegmentAnchorHitIndex = segmentsInGPU.outerMiniDoubletAnchorHitIndices[secondSegmentIndex]; //same as third segment inner MD anchor hit index

    unsigned int outerTripletSecondSegmentAnchorHitIndex = segmentsInGPU.outerMiniDoubletAnchorHitIndices[thirdSegmentIndex]; //same as fourth segment inner MD anchor hit index
    unsigned int outerTripletThirdSegmentAnchorHitIndex = segmentsInGPU.outerMiniDoubletAnchorHitIndices[fourthSegmentIndex];


    float x1 = hitsInGPU.xs[innerTripletFirstSegmentAnchorHitIndex];
    float x2 = hitsInGPU.xs[innerTripletSecondSegmentAnchorHitIndex];
    float x3 = hitsInGPU.xs[innerTripletThirdSegmentAnchorHitIndex];
    float x4 = hitsInGPU.xs[outerTripletSecondSegmentAnchorHitIndex];
    float x5 = hitsInGPU.xs[outerTripletThirdSegmentAnchorHitIndex];

    float y1 = hitsInGPU.ys[innerTripletFirstSegmentAnchorHitIndex];
    float y2 = hitsInGPU.ys[innerTripletSecondSegmentAnchorHitIndex];
    float y3 = hitsInGPU.ys[innerTripletThirdSegmentAnchorHitIndex];
    float y4 = hitsInGPU.ys[outerTripletSecondSegmentAnchorHitIndex];
    float y5 = hitsInGPU.ys[outerTripletThirdSegmentAnchorHitIndex];

    //construct the arrays
    float x1Vec[] = {x1, x1, x1};
    float y1Vec[] = {y1, y1, y1};
    float x2Vec[] = {x2, x2, x2};
    float y2Vec[] = {y2, y2, y2};
    float x3Vec[] = {x3, x3, x3};
    float y3Vec[] = {y3, y3, y3};
    float x4Vec[] = {x4, x4, x4};
    float y4Vec[] = {y4, y4, y4};
    float x5Vec[] = {x5, x5, x5};
    float y5Vec[] = {y5, y5, y5};

    if(modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex1] == SDL::TwoS)
    {
        x1Vec[1] = hitsInGPU.lowEdgeXs[innerTripletFirstSegmentAnchorHitIndex];
        x1Vec[2] = hitsInGPU.highEdgeXs[innerTripletFirstSegmentAnchorHitIndex];

        y1Vec[1] = hitsInGPU.lowEdgeYs[innerTripletFirstSegmentAnchorHitIndex];
        y1Vec[2] = hitsInGPU.highEdgeYs[innerTripletFirstSegmentAnchorHitIndex];
    }

    if(modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex2] == SDL::TwoS)
    {
        x2Vec[1] = hitsInGPU.lowEdgeXs[innerTripletSecondSegmentAnchorHitIndex];
        x2Vec[2] = hitsInGPU.highEdgeXs[innerTripletSecondSegmentAnchorHitIndex];

        y2Vec[1] = hitsInGPU.lowEdgeYs[innerTripletSecondSegmentAnchorHitIndex];
        y2Vec[2] = hitsInGPU.highEdgeYs[innerTripletSecondSegmentAnchorHitIndex];

    }

    if(modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex3] == SDL::TwoS)
    {
        x3Vec[1] = hitsInGPU.lowEdgeXs[innerTripletThirdSegmentAnchorHitIndex];
        x3Vec[2] = hitsInGPU.highEdgeXs[innerTripletThirdSegmentAnchorHitIndex];

        y3Vec[1] = hitsInGPU.lowEdgeYs[innerTripletThirdSegmentAnchorHitIndex];
        y3Vec[2] = hitsInGPU.highEdgeYs[innerTripletThirdSegmentAnchorHitIndex];
    }

    if(modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex4] == SDL::TwoS)
    {
        x4Vec[1] = hitsInGPU.lowEdgeXs[outerTripletSecondSegmentAnchorHitIndex];
        x4Vec[2] = hitsInGPU.highEdgeXs[outerTripletSecondSegmentAnchorHitIndex];

        y4Vec[1] = hitsInGPU.lowEdgeYs[outerTripletSecondSegmentAnchorHitIndex];
        y4Vec[2] = hitsInGPU.highEdgeYs[outerTripletSecondSegmentAnchorHitIndex];
    }

    if(modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex4] == SDL::TwoS)
    {
        x5Vec[1] = hitsInGPU.lowEdgeXs[outerTripletThirdSegmentAnchorHitIndex];
        x5Vec[2] = hitsInGPU.highEdgeXs[outerTripletThirdSegmentAnchorHitIndex];

        y5Vec[1] = hitsInGPU.lowEdgeYs[outerTripletThirdSegmentAnchorHitIndex];
        y5Vec[2] = hitsInGPU.highEdgeYs[outerTripletThirdSegmentAnchorHitIndex];
    }


    float innerG, innerF; //centers of inner circle
    float outerG, outerF; //centers of outer circle
    float bridgeG, bridgeF;

    innerRadius = computeRadiusFromThreeAnchorHits(x1, y1, x2, y2, x3, y3, innerG, innerF);
    outerRadius = computeRadiusFromThreeAnchorHits(x3, y3, x4, y4, x5, y5, outerG, outerF);
//    bridgeRadius = computeRadiusFromThreeAnchorHits(x1, y1, x3, y3, x5, y5, bridgeG, bridgeF);

    //cross product check   
    float omega1 = (innerG - x1) * (y3 - y1) - (innerF - y1) * (x3 - x1);
    float omega2 = (outerG - x3) * (y5 - y3) - (outerF - y3) * (x5 - x3);

    float temp;
    computeErrorInRadius(x1Vec, y1Vec, x2Vec, y2Vec, x3Vec, y3Vec, innerRadiusMin, innerRadiusMax, omega1);
//    printf("x1 = %f, y1 = %f, x2 = %f, y2 = %f, x3 = %f, y3 = %f, x4 = %f, y4 = %f, x5 = %f, y5 = %f\n",x1,y1,x2,y2,x3,y3,x4,y4,x5,y5);
    printf("(%f,%f),(%f,%f),(%f,%f),(%f,%f),(%f,%f)\n",x1,y1,x2,y2,x3,y3,x4,y4,x5,y5);
     
/*    if(innerRadius < 0.75/(2 * k2Rinv1GeVf))
    {
        pass = false;
    } 

    if(outerRadius < 0.75/(2 * k2Rinv1GeVf))
    {
        pass = false;
    } 

 
    if(omega1 * omega2 < 0)
    {
        pass = false;
    }*/

    return pass;
}

__device__ void SDL::computeErrorInRadius(float* x1Vec, float* y1Vec, float* x2Vec, float* y2Vec, float* x3Vec, float* y3Vec, float& minimumRadius, float& maximumRadius, float& omega1)
{
    //brute force
    float candidateRadius;
    minimumRadius = 123456789;
    maximumRadius = 0;
    float f,g; //placeholders
    float omega2;
    for(size_t i = 0; i < 3; i++)
    {
        for(size_t j = 0; j < 3; j++)
        {
            for(size_t k = 0; k < 3; k++)
            {
               candidateRadius = computeRadiusFromThreeAnchorHits(x1Vec[i], y1Vec[i], x2Vec[j], y2Vec[j], x3Vec[k], y3Vec[k],g,f);
               omega2 = (g - x1Vec[i]) * (y3Vec[k] - y1Vec[i]) - (f - y1Vec[i]) * (x3Vec[k] - x1Vec[i]);
               //first things first
               if(candidateRadius < 0.5/k2Rinv1GeVf) continue;
               if(candidateRadius >= maximumRadius and omega2 * omega1 > 0)
               {
                   maximumRadius = candidateRadius;
               }

               if(candidateRadius <= minimumRadius and omega2 * omega1 > 0)
               {
                   minimumRadius = candidateRadius;
               }
    
            }
        }
    }
}


/*
__device__ float SDL::computeErrorInRadius(float* x1Vec, float* y1Vec, float* x2Vec, float* y2Vec, float* x3Vec, float* y3Vec, float& gError, float& fError)
{
    // Numerical differentiation baby! Scientific computing course put into good use
    float h = x1Vec[0]/100;
    float gUp, gDown;
    float fUp, fDown;

    float dgBydx1, dfBydx1;
    float dRBydx1 = (computeRadiusFromThreeAnchorHits(x1Vec[0] + h, y1Vec[0], x2Vec[0], y2Vec[0], x3Vec[0], y3Vec[0], gUp, fUp) - computeRadiusFromThreeAnchorHits(x1Vec[0] - h, y1Vec[0], x2Vec[0], y2Vec[0], x3Vec[0], y3Vec[0], gDown, fDown))/(2 * h);
    dgBydx1 = (gUp - gDown)/(2 * h);
    dfBydx1 = (fUp - fDown)/(2 * h);

    h = x2Vec[0]/100;
    float dgBydx2, dfBydx2;
    float dRBydx2 = (computeRadiusFromThreeAnchorHits(x1Vec[0], y1Vec[0], x2Vec[0] + h, y2Vec[0], x3Vec[0], y3Vec[0], gUp, fUp) - computeRadiusFromThreeAnchorHits(x1Vec[0], y1Vec[0], x2Vec[0] - h, y2Vec[0], x3Vec[0], y3Vec[0], gDown, fDown))/(2 * h);
    dgBydx2 = (gUp - gDown)/(2 * h);
    dfBydx2 = (fUp - fDown)/(2 * h);

    h = x3Vec[0]/100;
    float dgBydx3, dfBydx3;
    float dRBydx3 = (computeRadiusFromThreeAnchorHits(x1Vec[0], y1Vec[0], x2Vec[0], y2Vec[0], x3Vec[0] + h, y3Vec[0], gUp, fUp) - computeRadiusFromThreeAnchorHits(x1Vec[0], y1Vec[0], x2Vec[0], y2Vec[0], x3Vec[0] - h, y3Vec[0], gDown, fDown))/(2 * h);
    dgBydx3 = (gUp - gDown)/(2 * h);
    dfBydx3 = (fUp - fDown)/(2 * h);

    h = y1Vec[0]/100;
    float dgBydy1, dfBydy1;
    float dRBydy1 = (computeRadiusFromThreeAnchorHits(x1Vec[0], y1Vec[0] + h, x2Vec[0], y2Vec[0], x3Vec[0], y3Vec[0], gUp, fUp) - computeRadiusFromThreeAnchorHits(x1Vec[0], y1Vec[0] - h, x2Vec[0], y2Vec[0], x3Vec[0], y3Vec[0], gDown, fDown))/(2 * h);
    dgBydy1 = (gUp - gDown)/(2 * h);
    dfBydy1 = (fUp - fDown)/(2 * h);

    h = y2Vec[0]/100;
    float dgBydy2, dfBydy2;
    float dRBydy2 = (computeRadiusFromThreeAnchorHits(x1Vec[0], y1Vec[0], x2Vec[0], y2Vec[0] + h, x3Vec[0], y3Vec[0], gUp, fUp) - computeRadiusFromThreeAnchorHits(x1Vec[0], y1Vec[0], x2Vec[0], y2Vec[0] - h, x3Vec[0], y3Vec[0], gDown, fDown))/(2 * h);
    dgBydy2 = (gUp - gDown)/(2 * h);
    dfBydy2 = (fUp - fDown)/(2 * h);

    h = y3Vec[0]/100;
    float dgBydy3, dfBydy3;
    float dRBydy3 = (computeRadiusFromThreeAnchorHits(x1Vec[0], y1Vec[0], x2Vec[0], y2Vec[0], x3Vec[0], y3Vec[0] + h, gUp, fUp) - computeRadiusFromThreeAnchorHits(x1Vec[0], y1Vec[0], x2Vec[0], y2Vec[0], x3Vec[0], y3Vec[0] - h, gDown, fDown))/(2 * h);
    dgBydy3 = (gUp - gDown)/(2 * h);
    dfBydy3 = (fUp - fDown)/(2 * h);


    float radiusError2 = (dRBydx1 * dRBydx1) * (x1Vec[2] - x1Vec[1]) * (x1Vec[2] - x1Vec[1]) + (dRBydx2 * dRBydx2) * (x2Vec[2] - x2Vec[1]) * (x2Vec[2] - x2Vec[1]) + (dRBydx3 * dRBydx3) * (x3Vec[2] - x3Vec[1]) * (x3Vec[2] - x3Vec[1]) + (dRBydy1 * dRBydy1) * (y1Vec[2] - y1Vec[1]) * (y1Vec[2] - y1Vec[1]) + (dRBydy2 * dRBydy2) * (y2Vec[2] - y2Vec[1]) * (y2Vec[2] - y2Vec[1]) + (dRBydy3 * dRBydy3) * (y3Vec[2] - y3Vec[1]) * (y3Vec[2] - y3Vec[1]);

    radiusError2/= 4;

    float gError2 = (dgBydx1 * dgBydx1) * (x1Vec[2] - x1Vec[1]) * (x1Vec[2] - x1Vec[1]) + (dgBydx2 * dgBydx2) * (x2Vec[2] - x2Vec[1]) * (x2Vec[2] - x2Vec[1]) + (dgBydx3 * dgBydx3) * (x3Vec[2] - x3Vec[1]) * (x3Vec[2] - x3Vec[1]) + (dgBydy1 * dgBydy1) * (y1Vec[2] - y1Vec[1]) * (y1Vec[2] - y1Vec[1]) + (dgBydy2 * dgBydy2) * (y2Vec[2] - y2Vec[1]) * (y2Vec[2] - y2Vec[1]) + (dgBydy3 * dgBydy3) * (y3Vec[2] - y3Vec[1]) * (y3Vec[2] - y3Vec[1]);

    gError2/= 4;

    float fError2 = (dfBydx1 * dfBydx1) * (x1Vec[2] - x1Vec[1]) * (x1Vec[2] - x1Vec[1]) + (dfBydx2 * dfBydx2) * (x2Vec[2] - x2Vec[1]) * (x2Vec[2] - x2Vec[1]) + (dfBydx3 * dfBydx3) * (x3Vec[2] - x3Vec[1]) * (x3Vec[2] - x3Vec[1]) + (dfBydy1 * dfBydy1) * (y1Vec[2] - y1Vec[1]) * (y1Vec[2] - y1Vec[1]) + (dfBydy2 * dfBydy2) * (y2Vec[2] - y2Vec[1]) * (y2Vec[2] - y2Vec[1]) + (dfBydy3 * dfBydy3) * (y3Vec[2] - y3Vec[1]) * (y3Vec[2] - y3Vec[1]);

    fError2/= 4;

    gError = sqrtf(gError2);
    fError = sqrtf(fError2);

    return sqrtf(radiusError2);
}*/

__device__ float SDL::computeRadiusFromThreeAnchorHits(float x1, float y1, float x2, float y2, float x3, float y3, float& g, float& f)
{
    float radius = 0;

    //writing manual code for computing radius, which obviously sucks
    //TODO:Use fancy inbuilt libraries like cuBLAS or cuSOLVE for this!
    //(g,f) -> center
    //first anchor hit - (x1,y1), second anchor hit - (x2,y2), third anchor hit - (x3, y3)


    if((y1 - y3) * (x2 - x3) - (x1 - x3) * (y2 - y3) == 0)
    {
        return -1; //WTF man three collinear points!
    }

    float denom = ((y1 - y3) * (x2 - x3) - (x1 - x3) * (y2 - y3));

    g = 0.5 * ((y3 - y2) * (x1 * x1 + y1 * y1) + (y1 - y3) * (x2 * x2 + y2 * y2) + (y2 - y1) * (x3 * x3 + y3 * y3))/denom;

    f = 0.5 * ((x2 - x3) * (x1 * x1 + y1 * y1) + (x3 - x1) * (x2 * x2 + y2 * y2) + (x1 - x2) * (x3 * x3 + y3 * y3))/denom;

    float c = ((x2 * y3 - x3 * y2) * (x1 * x1 + y1 * y1) + (x3 * y1 - x1 * y3) * (x2 * x2 + y2 * y2) + (x1 * y2 - x2 * y1) * (x3 * x3 + y3 * y3))/denom;

    if(g * g + f * f - c < 0)
    {
        printf("FATAL! r^2 < 0!\n");
        return -1;
    }
    
    radius = sqrtf(g * g  + f * f - c);
    return radius;
}

__device__ bool SDL::T5HasCommonMiniDoublet(struct SDL::triplets& tripletsInGPU, struct SDL::segments& segmentsInGPU, unsigned int innerTripletIndex, unsigned int outerTripletIndex)
{
    unsigned int innerOuterSegmentIndex = tripletsInGPU.segmentIndices[2 * innerTripletIndex + 1];
    unsigned int outerInnerSegmentIndex = tripletsInGPU.segmentIndices[2 * outerTripletIndex];
    unsigned int innerOuterOuterMiniDoubletIndex = segmentsInGPU.mdIndices[2 * innerOuterSegmentIndex + 1]; //inner triplet outer segment outer MD index
    unsigned int outerInnerInnerMiniDoubletIndex = segmentsInGPU.mdIndices[2 * outerInnerSegmentIndex]; //outer triplet inner segmnet inner MD index
   

    return (innerOuterOuterMiniDoubletIndex == outerInnerInnerMiniDoubletIndex);
}


