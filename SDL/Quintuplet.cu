#include "hip/hip_runtime.h"
#ifdef __HIPCC__
#define CUDA_CONST_VAR __device__
#endif
# include "Quintuplet.cuh"
#include "allocate.h"
#include "Kernels.cuh"

SDL::quintuplets::quintuplets()
{
    tripletIndices = nullptr;
    lowerModuleIndices = nullptr;
    nQuintuplets = nullptr;
    totOccupancyQuintuplets = nullptr;
    innerRadius = nullptr;
    outerRadius = nullptr;
    isDup = nullptr;
    tightCutFlag = nullptr;
    partOfPT5 = nullptr;
    pt = nullptr;
    layer = nullptr;
    innerG = nullptr;
    innerF = nullptr;
    logicalLayers = nullptr;
    hitIndices = nullptr;
    bridgeRadius = nullptr;
    chiSquared = nullptr;
    rzChiSquared = nullptr;
    nonAnchorChiSquared = nullptr;
}

SDL::quintuplets::~quintuplets()
{
}

void SDL::quintuplets::freeMemoryCache()
{
    int dev;
    hipGetDevice(&dev);
    cms::cuda::free_device(dev,tripletIndices);
    cms::cuda::free_device(dev, lowerModuleIndices);
    cms::cuda::free_device(dev, nQuintuplets);
    cms::cuda::free_device(dev, totOccupancyQuintuplets);
    cms::cuda::free_device(dev, innerRadius);
    cms::cuda::free_device(dev, outerRadius);
    cms::cuda::free_device(dev, partOfPT5);
    cms::cuda::free_device(dev, isDup);
    cms::cuda::free_device(dev, tightCutFlag);
    cms::cuda::free_device(dev, pt);
    cms::cuda::free_device(dev, layer);
    cms::cuda::free_device(dev, innerG);
    cms::cuda::free_device(dev, innerF);
    cms::cuda::free_device(dev, logicalLayers);
    cms::cuda::free_device(dev, hitIndices);
    cms::cuda::free_device(dev, nMemoryLocations);
    cms::cuda::free_device(dev, bridgeRadius);
    cms::cuda::free_device(dev, rzChiSquared);
    cms::cuda::free_device(dev, chiSquared);
    cms::cuda::free_device(dev, nonAnchorChiSquared);
}

void SDL::quintuplets::freeMemory(hipStream_t stream)
{
    hipFree(tripletIndices);
    hipFree(lowerModuleIndices);
    hipFree(nQuintuplets);
    hipFree(totOccupancyQuintuplets);
    hipFree(innerRadius);
    hipFree(outerRadius);
    hipFree(partOfPT5);
    hipFree(isDup);
    hipFree(tightCutFlag);
    hipFree(pt);
    hipFree(layer);
    hipFree(innerG);
    hipFree(innerF);
    hipFree(logicalLayers);
    hipFree(hitIndices);
    hipFree(nMemoryLocations);
    hipFree(bridgeRadius);
    hipFree(rzChiSquared);
    hipFree(chiSquared);
    hipFree(nonAnchorChiSquared);
    hipStreamSynchronize(stream);
}
//TODO:Reuse the track candidate one instead of this!
__global__ void SDL::createEligibleModulesListForQuintupletsGPU(struct modules& modulesInGPU,struct triplets& tripletsInGPU, unsigned int* device_nTotalQuintuplets, hipStream_t stream,struct objectRanges& rangesInGPU)
{
    __shared__ int nEligibleT5Modulesx;
    __shared__ unsigned int nTotalQuintupletsx;
    nTotalQuintupletsx = 0; //start!
    nEligibleT5Modulesx = 0;
    __syncthreads();

    unsigned int occupancy;
    unsigned int category_number, eta_number;
    unsigned int layers, subdets, rings;
    float eta;
    //start filling
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    int np = gridDim.x * blockDim.x;
    for(uint16_t i = gid; i < *modulesInGPU.nLowerModules; i+= np)
    {
        //condition for a quintuple to exist for a module
        //TCs don't exist for layers 5 and 6 barrel, and layers 2,3,4,5 endcap   
        layers = modulesInGPU.layers[i];
        subdets = modulesInGPU.subdets[i];
        rings = modulesInGPU.rings[i];
        eta = modulesInGPU.eta[i];  
        occupancy = 0;

        if (tripletsInGPU.nTriplets[i] == 0) continue;
        if (subdets == SDL::Barrel and layers >= 3) continue;
        if (subdets == SDL::Endcap and layers > 1) continue;

        int nEligibleT5Modules = atomicAdd(&nEligibleT5Modulesx,1);
        if (nEligibleT5Modules < 0) printf("%u\n",nEligibleT5Modules);
        if (layers<=3 && subdets==5) category_number = 0;
        if (layers>=4 && subdets==5) category_number = 1;
        if (layers<=2 && subdets==4 && rings>=11) category_number = 2;
        if (layers>=3 && subdets==4 && rings>=8) category_number = 2;
        if (layers<=2 && subdets==4 && rings<=10) category_number = 3;
        if (layers>=3 && subdets==4 && rings<=7) category_number = 3;
        if (abs(eta)<0.75) eta_number=0;
        if (abs(eta)>0.75 && abs(eta)<1.5) eta_number=1;
        if (abs(eta)>1.5 && abs(eta)<2.25) eta_number=2;
        if (abs(eta)>2.25 && abs(eta)<3) eta_number=3;

        if (category_number == 0 && eta_number == 0) occupancy = 336;
        if (category_number == 0 && eta_number == 1) occupancy = 414;
        if (category_number == 0 && eta_number == 2) occupancy = 231;
        if (category_number == 0 && eta_number == 3) occupancy = 146;
        if (category_number == 3 && eta_number == 1) occupancy = 0;
        if (category_number == 3 && eta_number == 2) occupancy = 191;
        if (category_number == 3 && eta_number == 3) occupancy = 106;

        unsigned int nTotQ = atomicAdd(&nTotalQuintupletsx,occupancy);
        rangesInGPU.quintupletModuleIndices[i] = nTotQ;
        rangesInGPU.indicesOfEligibleT5Modules[nEligibleT5Modules] = i;
    }
    __syncthreads();
    if(threadIdx.x==0){
        *rangesInGPU.nEligibleT5Modules = static_cast<uint16_t>(nEligibleT5Modulesx);
        *device_nTotalQuintuplets = nTotalQuintupletsx;
    }
}

void SDL::createQuintupletsInExplicitMemory(struct SDL::quintuplets& quintupletsInGPU, const unsigned int& nTotalQuintuplets, const uint16_t& nLowerModules, const uint16_t& nEligibleModules,hipStream_t stream)
{
    //unsigned int nMemoryLocations = nEligibleModules * maxQuintuplets;
#ifdef CACHE_ALLOC
 //   hipStream_t stream = 0;
    int dev;
    hipGetDevice(&dev);
    quintupletsInGPU.tripletIndices = (unsigned int*)cms::cuda::allocate_device(dev, 2 * nTotalQuintuplets * sizeof(unsigned int), stream);
    quintupletsInGPU.lowerModuleIndices = (uint16_t*)cms::cuda::allocate_device(dev, 5 * nTotalQuintuplets * sizeof(uint16_t), stream);
    quintupletsInGPU.nQuintuplets = (unsigned int*)cms::cuda::allocate_device(dev, nLowerModules * sizeof(unsigned int), stream);
    quintupletsInGPU.totOccupancyQuintuplets = (unsigned int*)cms::cuda::allocate_device(dev, nLowerModules * sizeof(unsigned int), stream);
    quintupletsInGPU.innerRadius = (FPX*)cms::cuda::allocate_device(dev, nTotalQuintuplets * sizeof(FPX), stream);
    quintupletsInGPU.outerRadius = (FPX*)cms::cuda::allocate_device(dev, nTotalQuintuplets * sizeof(FPX), stream);
    quintupletsInGPU.bridgeRadius = (FPX*)cms::cuda::allocate_device(dev, nTotalQuintuplets * sizeof(float), stream);

    quintupletsInGPU.pt = (FPX*)cms::cuda::allocate_device(dev, nTotalQuintuplets *4* sizeof(FPX), stream);
    quintupletsInGPU.layer = (uint8_t*)cms::cuda::allocate_device(dev, nTotalQuintuplets * sizeof(uint8_t), stream);
    quintupletsInGPU.isDup = (bool*)cms::cuda::allocate_device(dev, nTotalQuintuplets * sizeof(bool), stream);
    quintupletsInGPU.tightCutFlag = (bool*)cms::cuda::allocate_device(dev, nTotalQuintuplets * sizeof(bool), stream);
    quintupletsInGPU.partOfPT5 = (bool*)cms::cuda::allocate_device(dev, nTotalQuintuplets * sizeof(bool), stream);
    quintupletsInGPU.innerG = (float*)cms::cuda::allocate_device(dev, nTotalQuintuplets * sizeof(float), stream);
    quintupletsInGPU.innerF = (float*)cms::cuda::allocate_device(dev, nTotalQuintuplets * sizeof(float), stream);
    quintupletsInGPU.logicalLayers = (uint8_t*)cms::cuda::allocate_device(dev, nTotalQuintuplets * sizeof(uint8_t) * 5, stream);
    quintupletsInGPU.hitIndices = (unsigned int*)cms::cuda::allocate_device(dev, nTotalQuintuplets * sizeof(unsigned int) * 10, stream);
    quintupletsInGPU.nMemoryLocations = (unsigned int*)cms::cuda::allocate_device(dev, sizeof(unsigned int), stream);

    quintupletsInGPU.rzChiSquared = (float*)cms::cuda::allocate_device(dev, nTotalQuintuplets * sizeof(float), stream);
    quintupletsInGPU.chiSquared = (float*)cms::cuda::allocate_device(dev, nTotalQuintuplets * sizeof(float), stream);
    quintupletsInGPU.nonAnchorChiSquared = (float*)cms::cuda::allocate_device(dev, nTotalQuintuplets * sizeof(float), stream);
#else
    hipMalloc(&quintupletsInGPU.tripletIndices, 2 * nTotalQuintuplets * sizeof(unsigned int));
    hipMalloc(&quintupletsInGPU.lowerModuleIndices, 5 * nTotalQuintuplets * sizeof(uint16_t));
    hipMalloc(&quintupletsInGPU.nQuintuplets, nLowerModules * sizeof(unsigned int));
    hipMalloc(&quintupletsInGPU.totOccupancyQuintuplets, nLowerModules * sizeof(unsigned int));
    hipMalloc(&quintupletsInGPU.innerRadius, nTotalQuintuplets * sizeof(FPX));
    hipMalloc(&quintupletsInGPU.outerRadius, nTotalQuintuplets * sizeof(FPX));
    hipMalloc(&quintupletsInGPU.pt, nTotalQuintuplets *4* sizeof(FPX));
    hipMalloc(&quintupletsInGPU.isDup, nTotalQuintuplets * sizeof(bool));
    hipMalloc(&quintupletsInGPU.tightCutFlag, nTotalQuintuplets * sizeof(bool));
    hipMalloc(&quintupletsInGPU.partOfPT5, nTotalQuintuplets * sizeof(bool));
    hipMalloc(&quintupletsInGPU.layer, nTotalQuintuplets * sizeof(uint8_t));
    hipMalloc(&quintupletsInGPU.innerG, nTotalQuintuplets * sizeof(float));
    hipMalloc(&quintupletsInGPU.innerF, nTotalQuintuplets * sizeof(float));
    hipMalloc(&quintupletsInGPU.logicalLayers, nTotalQuintuplets * 5 * sizeof(uint8_t));
    hipMalloc(&quintupletsInGPU.hitIndices, nTotalQuintuplets * 10 * sizeof(unsigned int));
    hipMalloc(&quintupletsInGPU.nMemoryLocations, sizeof(unsigned int));
    hipMalloc(&quintupletsInGPU.bridgeRadius, nTotalQuintuplets * sizeof(float));
    hipMalloc(&quintupletsInGPU.rzChiSquared, nTotalQuintuplets * sizeof(float));
    hipMalloc(&quintupletsInGPU.chiSquared, nTotalQuintuplets * sizeof(float));
    hipMalloc(&quintupletsInGPU.nonAnchorChiSquared, nTotalQuintuplets * sizeof(float));
    hipMalloc(&quintupletsInGPU.nMemoryLocations, sizeof(unsigned int));
#endif
    hipMemsetAsync(quintupletsInGPU.nQuintuplets,0,nLowerModules * sizeof(unsigned int),stream);
    hipMemsetAsync(quintupletsInGPU.totOccupancyQuintuplets,0,nLowerModules * sizeof(unsigned int),stream);
    hipMemsetAsync(quintupletsInGPU.isDup,0,nTotalQuintuplets * sizeof(bool),stream);
    hipMemsetAsync(quintupletsInGPU.tightCutFlag,0,nTotalQuintuplets * sizeof(bool),stream);
    hipMemsetAsync(quintupletsInGPU.partOfPT5,0,nTotalQuintuplets * sizeof(bool),stream);
    hipStreamSynchronize(stream);
    quintupletsInGPU.eta = quintupletsInGPU.pt + nTotalQuintuplets;
    quintupletsInGPU.phi = quintupletsInGPU.pt + 2*nTotalQuintuplets;
    quintupletsInGPU.score_rphisum = quintupletsInGPU.pt + 3*nTotalQuintuplets;
}


__device__ void SDL::addQuintupletToMemory(struct SDL::triplets& tripletsInGPU, struct SDL::quintuplets& quintupletsInGPU, unsigned int innerTripletIndex, unsigned int outerTripletIndex, uint16_t& lowerModule1, uint16_t& lowerModule2, uint16_t& lowerModule3, uint16_t& lowerModule4, uint16_t& lowerModule5, float& innerRadius, float& bridgeRadius, float& outerRadius, float& innerG, float& innerF, float& rzChiSquared, float& rPhiChiSquared, float&
        nonAnchorChiSquared, float pt, float eta, float phi, float scores, uint8_t layer, unsigned int quintupletIndex, bool tightCutFlag)

{
    quintupletsInGPU.tripletIndices[2 * quintupletIndex] = innerTripletIndex;
    quintupletsInGPU.tripletIndices[2 * quintupletIndex + 1] = outerTripletIndex;

    quintupletsInGPU.lowerModuleIndices[5 * quintupletIndex] = lowerModule1;
    quintupletsInGPU.lowerModuleIndices[5 * quintupletIndex + 1] = lowerModule2;
    quintupletsInGPU.lowerModuleIndices[5 * quintupletIndex + 2] = lowerModule3;
    quintupletsInGPU.lowerModuleIndices[5 * quintupletIndex + 3] = lowerModule4;
    quintupletsInGPU.lowerModuleIndices[5 * quintupletIndex + 4] = lowerModule5;
    quintupletsInGPU.innerRadius[quintupletIndex] = __F2H(innerRadius);
    quintupletsInGPU.outerRadius[quintupletIndex] = __F2H(outerRadius);
    quintupletsInGPU.pt[quintupletIndex] = __F2H(pt);
    quintupletsInGPU.eta[quintupletIndex] = __F2H(eta);
    quintupletsInGPU.phi[quintupletIndex] = __F2H(phi);
    quintupletsInGPU.score_rphisum[quintupletIndex] = __F2H(scores);
    quintupletsInGPU.layer[quintupletIndex] = layer;
    quintupletsInGPU.isDup[quintupletIndex] = false;
    quintupletsInGPU.tightCutFlag[quintupletIndex] = tightCutFlag;
    quintupletsInGPU.innerG[quintupletIndex] = innerG;
    quintupletsInGPU.innerF[quintupletIndex] = innerF;
    quintupletsInGPU.logicalLayers[5 * quintupletIndex] = tripletsInGPU.logicalLayers[3 * innerTripletIndex];
    quintupletsInGPU.logicalLayers[5 * quintupletIndex + 1] = tripletsInGPU.logicalLayers[3 * innerTripletIndex + 1];
    quintupletsInGPU.logicalLayers[5 * quintupletIndex + 2] = tripletsInGPU.logicalLayers[3 * innerTripletIndex + 2];
    quintupletsInGPU.logicalLayers[5 * quintupletIndex + 3] = tripletsInGPU.logicalLayers[3 * outerTripletIndex + 1];
    quintupletsInGPU.logicalLayers[5 * quintupletIndex + 4] = tripletsInGPU.logicalLayers[3 * outerTripletIndex + 2];

    quintupletsInGPU.hitIndices[10 * quintupletIndex] = tripletsInGPU.hitIndices[6 * innerTripletIndex];
    quintupletsInGPU.hitIndices[10 * quintupletIndex + 1] = tripletsInGPU.hitIndices[6 * innerTripletIndex + 1];
    quintupletsInGPU.hitIndices[10 * quintupletIndex + 2] = tripletsInGPU.hitIndices[6 * innerTripletIndex + 2];
    quintupletsInGPU.hitIndices[10 * quintupletIndex + 3] = tripletsInGPU.hitIndices[6 * innerTripletIndex + 3];
    quintupletsInGPU.hitIndices[10 * quintupletIndex + 4] = tripletsInGPU.hitIndices[6 * innerTripletIndex + 4];
    quintupletsInGPU.hitIndices[10 * quintupletIndex + 5] = tripletsInGPU.hitIndices[6 * innerTripletIndex + 5];
    quintupletsInGPU.hitIndices[10 * quintupletIndex + 6] = tripletsInGPU.hitIndices[6 * outerTripletIndex + 2];
    quintupletsInGPU.hitIndices[10 * quintupletIndex + 7] = tripletsInGPU.hitIndices[6 * outerTripletIndex + 3];
    quintupletsInGPU.hitIndices[10 * quintupletIndex + 8] = tripletsInGPU.hitIndices[6 * outerTripletIndex + 4];
    quintupletsInGPU.hitIndices[10 * quintupletIndex + 9] = tripletsInGPU.hitIndices[6 * outerTripletIndex + 5];
    quintupletsInGPU.bridgeRadius[quintupletIndex] = bridgeRadius;
    quintupletsInGPU.rzChiSquared[quintupletIndex] = rzChiSquared;
    quintupletsInGPU.chiSquared[quintupletIndex] = rPhiChiSquared;
    quintupletsInGPU.nonAnchorChiSquared[quintupletIndex] = nonAnchorChiSquared;

}

__device__ bool SDL::runQuintupletDefaultAlgo(struct SDL::modules& modulesInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::triplets& tripletsInGPU, uint16_t& lowerModuleIndex1, uint16_t& lowerModuleIndex2, uint16_t& lowerModuleIndex3, uint16_t& lowerModuleIndex4, uint16_t& lowerModuleIndex5, unsigned int& innerTripletIndex, unsigned int& outerTripletIndex, float& innerG, float& innerF, float& innerRadius, float& outerRadius, float& bridgeRadius, float&
        rzChiSquared, float& chiSquared, float& nonAnchorChiSquared, bool& tightCutFlag)
{
    bool pass = true;
    unsigned int firstSegmentIndex = tripletsInGPU.segmentIndices[2 * innerTripletIndex];
    unsigned int secondSegmentIndex = tripletsInGPU.segmentIndices[2 * innerTripletIndex + 1];
    unsigned int thirdSegmentIndex = tripletsInGPU.segmentIndices[2 * outerTripletIndex];
    unsigned int fourthSegmentIndex = tripletsInGPU.segmentIndices[2 * outerTripletIndex + 1];

    unsigned int innerOuterOuterMiniDoubletIndex = segmentsInGPU.mdIndices[2 * secondSegmentIndex + 1]; //inner triplet outer segment outer MD index
    unsigned int outerInnerInnerMiniDoubletIndex = segmentsInGPU.mdIndices[2 * thirdSegmentIndex]; //outer triplet inner segmnet inner MD index

    //this cut reduces the number of candidates by a factor of 3, i.e., 2 out of 3 warps can end right here!
    if (innerOuterOuterMiniDoubletIndex != outerInnerInnerMiniDoubletIndex) return false;
    
    //apply T4 criteria between segments 1 and 3
    float zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta; //temp stuff
    float zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ;

    unsigned int firstMDIndex = segmentsInGPU.mdIndices[2 * firstSegmentIndex];
    unsigned int secondMDIndex = segmentsInGPU.mdIndices[2 * secondSegmentIndex];
    unsigned int thirdMDIndex = segmentsInGPU.mdIndices[2 * secondSegmentIndex + 1];
    unsigned int fourthMDIndex = segmentsInGPU.mdIndices[2 * thirdSegmentIndex + 1];
    unsigned int fifthMDIndex = segmentsInGPU.mdIndices[2 * fourthSegmentIndex + 1];

    pass = pass and runQuintupletDefaultAlgo(modulesInGPU, mdsInGPU, segmentsInGPU, lowerModuleIndex1, lowerModuleIndex2, lowerModuleIndex3, lowerModuleIndex4, firstSegmentIndex, thirdSegmentIndex, firstMDIndex, secondMDIndex, thirdMDIndex, fourthMDIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta, zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ);
    if(not pass) return pass;

    pass = pass and runQuintupletDefaultAlgo(modulesInGPU, mdsInGPU, segmentsInGPU, lowerModuleIndex1, lowerModuleIndex2, lowerModuleIndex4, lowerModuleIndex5, firstSegmentIndex, fourthSegmentIndex, firstMDIndex, secondMDIndex, fourthMDIndex, fifthMDIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta, zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ);
    if(not pass) return pass;

    float x1 = mdsInGPU.anchorX[firstMDIndex];
    float x2 = mdsInGPU.anchorX[secondMDIndex];
    float x3 = mdsInGPU.anchorX[thirdMDIndex];
    float x4 = mdsInGPU.anchorX[fourthMDIndex];
    float x5 = mdsInGPU.anchorX[fifthMDIndex];
    
    float y1 = mdsInGPU.anchorY[firstMDIndex];
    float y2 = mdsInGPU.anchorY[secondMDIndex];
    float y3 = mdsInGPU.anchorY[thirdMDIndex];
    float y4 = mdsInGPU.anchorY[fourthMDIndex];
    float y5 = mdsInGPU.anchorY[fifthMDIndex];

    float bridgeG, bridgeF, outerG, outerF;
    innerRadius = computeRadiusFromThreeAnchorHits(x1, y1, x2, y2, x3, y3, innerG, innerF);
    outerRadius = computeRadiusFromThreeAnchorHits(x3, y3, x4, y4, x5, y5, outerG, outerF);
    bridgeRadius = computeRadiusFromThreeAnchorHits(x1, y1, x3, y3, x5, y5, bridgeG, bridgeF);

    float inner_pt = 2 * k2Rinv1GeVf * innerRadius;

    float residual4, residual5, residual_missing, g, f;
    pass = pass and passT5RZConstraint(modulesInGPU, mdsInGPU, firstMDIndex, secondMDIndex, thirdMDIndex, fourthMDIndex, fifthMDIndex, lowerModuleIndex1, lowerModuleIndex2, lowerModuleIndex3, lowerModuleIndex4, lowerModuleIndex5, rzChiSquared, residual_missing, residual4, residual5, inner_pt, innerRadius, innerG, innerF, tightCutFlag);

    pass = pass & (innerRadius >= 0.95f * ptCut/(2.f * k2Rinv1GeVf));

    float eta = (modulesInGPU.layers[lowerModuleIndex1] == 1) ? mdsInGPU.anchorEta[secondMDIndex] : mdsInGPU.anchorEta[firstMDIndex];

    bool temp;
    if(innerRadius < 1.0/(k2Rinv1GeVf * 2.f))
    {
        temp = (matchRadii_bin1(modulesInGPU, eta, lowerModuleIndex1, lowerModuleIndex2, lowerModuleIndex3, lowerModuleIndex4, lowerModuleIndex5, innerRadius, outerRadius));
//        temp = temp and (matchRadii_inner_v_bridge_bin1(modulesInGPU, eta, lowerModuleIndex1, lowerModuleIndex2, lowerModuleIndex3, lowerModuleIndex4, lowerModuleIndex5, innerRadius, bridgeRadius));
    }
    else if(innerRadius < 1.2/(k2Rinv1GeVf * 2.f))
    {
        temp = (matchRadii_bin2(modulesInGPU, eta, lowerModuleIndex1, lowerModuleIndex2, lowerModuleIndex3, lowerModuleIndex4, lowerModuleIndex5, innerRadius, outerRadius));
 //       temp = temp and (matchRadii_inner_v_bridge_bin2(modulesInGPU, eta, lowerModuleIndex1, lowerModuleIndex2, lowerModuleIndex3, lowerModuleIndex4, lowerModuleIndex5, innerRadius, bridgeRadius));
    }
    else if(innerRadius < 1.5/(k2Rinv1GeVf * 2.f))
    {
        temp = (matchRadii_bin3(modulesInGPU, eta, lowerModuleIndex1, lowerModuleIndex2, lowerModuleIndex3, lowerModuleIndex4, lowerModuleIndex5, innerRadius, outerRadius));
 //       temp = temp and (matchRadii_inner_v_bridge_bin3(modulesInGPU, eta, lowerModuleIndex1, lowerModuleIndex2, lowerModuleIndex3, lowerModuleIndex4, lowerModuleIndex5, innerRadius, bridgeRadius));
    }
    else if(innerRadius < 2.15/(k2Rinv1GeVf * 2.f))
    {
        temp = (matchRadii_bin4(modulesInGPU, eta, lowerModuleIndex1, lowerModuleIndex2, lowerModuleIndex3, lowerModuleIndex4, lowerModuleIndex5, innerRadius, outerRadius));
  //      temp = temp and (matchRadii_inner_v_bridge_bin4(modulesInGPU, eta, lowerModuleIndex1, lowerModuleIndex2, lowerModuleIndex3, lowerModuleIndex4, lowerModuleIndex5, innerRadius, bridgeRadius));
    }
    else if(innerRadius < 5/(k2Rinv1GeVf * 2.f))
    {
        temp = (matchRadii_bin5(modulesInGPU, eta, lowerModuleIndex1, lowerModuleIndex2, lowerModuleIndex3, lowerModuleIndex4, lowerModuleIndex5, innerRadius, outerRadius));
  //      temp = temp and (matchRadii_inner_v_bridge_bin5(modulesInGPU, eta, lowerModuleIndex1, lowerModuleIndex2, lowerModuleIndex3, lowerModuleIndex4, lowerModuleIndex5, innerRadius, bridgeRadius));
    }
    else
    {
         temp = (matchRadii_bin6(modulesInGPU, eta, lowerModuleIndex1, lowerModuleIndex2, lowerModuleIndex3, lowerModuleIndex4, lowerModuleIndex5, innerRadius, outerRadius));  
    //     temp = temp and (matchRadii_inner_v_bridge_bin6(modulesInGPU, eta, lowerModuleIndex1, lowerModuleIndex2, lowerModuleIndex3, lowerModuleIndex4, lowerModuleIndex5, innerRadius, bridgeRadius));  
    }
    pass = pass and temp;
    if(not pass) return pass;

    float xVec[] = {x1, x2, x3, x4, x5};
    float yVec[] = {y1, y2, y3, y4, y5};
    float sigmas[5];
    bool isFlat[5];
    //5 categories for sigmas
    const uint16_t lowerModuleIndices[] = {lowerModuleIndex1, lowerModuleIndex2, lowerModuleIndex3, lowerModuleIndex4, lowerModuleIndex5};

    float regressionG, regressionF, regressionRadius;

    computeSigmasForRegression(modulesInGPU, lowerModuleIndices, sigmas);
    regressionRadius = computeRadiusUsingRegression(5,xVec, yVec, regressionG, regressionF, sigmas, chiSquared);

    //chi squared calibration
  /* 
    if(innerRadius < 5/(k2Rinv1GeVf * 2.f))
    {
        pass = pass and passChiSquared_bin1(modulesInGPU, eta, lowerModuleIndex1, lowerModuleIndex2, lowerModuleIndex3, lowerModuleIndex4, lowerModuleIndex5, chiSquared);
    }
    else
    {
         pass = pass and passChiSquared_bin2(modulesInGPU, eta, lowerModuleIndex1, lowerModuleIndex2, lowerModuleIndex3, lowerModuleIndex4, lowerModuleIndex5, chiSquared);  
    }*/


    //compute the other chisquared
    //non anchor is always shifted for tilted and endcap!
    float nonAnchorSigmas[5];
    float nonAnchorxs[] = { mdsInGPU.outerX[firstMDIndex], mdsInGPU.outerX[secondMDIndex], mdsInGPU.outerX[thirdMDIndex], mdsInGPU.outerX[fourthMDIndex], mdsInGPU.outerX[fifthMDIndex]};
    float nonAnchorys[] = { mdsInGPU.outerY[firstMDIndex], mdsInGPU.outerY[secondMDIndex], mdsInGPU.outerY[thirdMDIndex], mdsInGPU.outerY[fourthMDIndex], mdsInGPU.outerY[fifthMDIndex]};

    computeSigmasForRegression(modulesInGPU, lowerModuleIndices, nonAnchorSigmas);
    nonAnchorChiSquared = computeChiSquared(5, nonAnchorxs, nonAnchorys, nonAnchorSigmas, regressionG, regressionF, regressionRadius);
    return pass;
}


__device__ bool SDL::passChiSquared_bin1(struct SDL::modules& modulesInGPU, float& eta, uint16_t& lowerModuleIndex1, uint16_t& lowerModuleIndex2, uint16_t& lowerModuleIndex3, uint16_t& lowerModuleIndex4, uint16_t& lowerModuleIndex5, float& rPhiChiSquared)
{
    const int layer1 = modulesInGPU.layers[lowerModuleIndex1] + 6 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex1] == SDL::TwoS);
    const int layer2 = modulesInGPU.layers[lowerModuleIndex2] + 6 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex2] == SDL::TwoS);
    const int layer3 = modulesInGPU.layers[lowerModuleIndex3] + 6 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex3] == SDL::TwoS);
    const int layer4 = modulesInGPU.layers[lowerModuleIndex4] + 6 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex4] == SDL::TwoS);
    const int layer5 = modulesInGPU.layers[lowerModuleIndex5] + 6 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex5] == SDL::TwoS);
    if(layer1 == 7 and layer2 == 8 and layer3 == 9 and layer4 == 10 and layer5 == 16)
    {
        return rPhiChiSquared < 24288.676060243808;
    }
    if(layer1 == 7 and layer2 == 8 and layer3 == 9 and layer4 == 15 and layer5 == 16)
    {
        return rPhiChiSquared < 73431.82522418901;
    }
    if(layer1 == 1 and layer2 == 7 and layer3 == 8 and layer4 == 9 and layer5 == 15)
    {
        return rPhiChiSquared < 36440.13849387184;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 7 and layer4 == 8 and layer5 == 14)
    {
        return rPhiChiSquared < 41460.646989336135;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 7 and layer4 == 13 and layer5 == 14)
    {
        return rPhiChiSquared < 44634.23616132676;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 7 and layer5 == 13)
    {
        return rPhiChiSquared < 35120.767889290655;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 12 and layer5 == 13)
    {
        return rPhiChiSquared < 74798.40398647501;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4 and layer5 == 12)
    {
        return rPhiChiSquared < 63360.74267181718;
    }
    if(layer1 == 2 and layer2 == 7 and layer3 == 8 and layer4 == 9 and layer5 == 15)
    {
        return rPhiChiSquared < 15318.055633514412;
    }
    if(layer1 == 2 and layer2 == 7 and layer3 == 8 and layer4 == 14 and layer5 == 15)
    {
        return rPhiChiSquared < 63360.74267181718;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 7 and layer4 == 13 and layer5 == 14)
    {
        return rPhiChiSquared < 57780.35676614198;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 12 and layer5 == 13)
    {
        return rPhiChiSquared < 120809.34770040761;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 5 and layer5 == 12)
    {
        return rPhiChiSquared < 86687.48340013086;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4 and layer5 == 5 and abs(eta) < 0.5)
    {
        return rPhiChiSquared < 52691.453528489714;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 5 and layer5 == 6 and abs(eta) < 0.5)
    {
        return rPhiChiSquared < 171495.4853115175;
    }
    return true;
}
__device__ bool SDL::passChiSquared_bin2(struct SDL::modules& modulesInGPU, float& eta, uint16_t& lowerModuleIndex1, uint16_t& lowerModuleIndex2, uint16_t& lowerModuleIndex3, uint16_t& lowerModuleIndex4, uint16_t& lowerModuleIndex5, float& rPhiChiSquared)
{
    const int layer1 = modulesInGPU.layers[lowerModuleIndex1] + 6 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex1] == SDL::TwoS);
    const int layer2 = modulesInGPU.layers[lowerModuleIndex2] + 6 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex2] == SDL::TwoS);
    const int layer3 = modulesInGPU.layers[lowerModuleIndex3] + 6 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex3] == SDL::TwoS);
    const int layer4 = modulesInGPU.layers[lowerModuleIndex4] + 6 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex4] == SDL::TwoS);
    const int layer5 = modulesInGPU.layers[lowerModuleIndex5] + 6 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex5] == SDL::TwoS);
    if(layer1 == 7 and layer2 == 8 and layer3 == 9 and layer4 == 10 and layer5 == 16)
    {
        return rPhiChiSquared < 70773.11437275149;
    }
    if(layer1 == 7 and layer2 == 8 and layer3 == 9 and layer4 == 15 and layer5 == 16)
    {
        return rPhiChiSquared < 32623.60655594575;
    }
    if(layer1 == 1 and layer2 == 7 and layer3 == 8 and layer4 == 9 and layer5 == 10)
    {
        return rPhiChiSquared < 18762.544296959917;
    }
    if(layer1 == 1 and layer2 == 7 and layer3 == 8 and layer4 == 9 and layer5 == 15)
    {
        return rPhiChiSquared < 104240.49747675558;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 7 and layer4 == 8 and layer5 == 9)
    {
        return rPhiChiSquared < 9840.370498616949;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 7 and layer4 == 8 and layer5 == 14)
    {
        return rPhiChiSquared < 32623.60655594575;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 7 and layer4 == 13 and layer5 == 14)
    {
        return rPhiChiSquared < 202453.2549851752;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 7 and layer5 == 13)
    {
        return rPhiChiSquared < 153534.02786503683;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 12 and layer5 == 13)
    {
        return rPhiChiSquared < 198754.4017493704;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4 and layer5 == 12)
    {
        return rPhiChiSquared < 184622.5408388927;
    }
    if(layer1 == 2 and layer2 == 7 and layer3 == 8 and layer4 == 9 and layer5 == 15)
    {
        return rPhiChiSquared < 37118.2956717501;
    }
    if(layer1 == 2 and layer2 == 7 and layer3 == 8 and layer4 == 14 and layer5 == 15)
    {
        return rPhiChiSquared < 106180.43087194151;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 7 and layer4 == 13 and layer5 == 14)
    {
        return rPhiChiSquared < 198754.4017493704;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 12 and layer5 == 13)
    {
        return rPhiChiSquared < 171495.4853115175;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 5 and layer5 == 12)
    {
        return rPhiChiSquared < 114307.97084446455;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4 and layer5 == 5 and abs(eta) < 0.5)
    {
        return rPhiChiSquared < 41460.646989336135;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4 and layer5 == 5 and abs(eta) > 0.5)
    {
        return rPhiChiSquared < 31442.419291452323;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 5 and layer5 == 6 and abs(eta) < 0.5)
    {
        return rPhiChiSquared < 114307.97084446455;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 5 and layer5 == 6 and abs(eta) > 0.5)
    {
        return rPhiChiSquared < 206220.94450908038;
    }
    return true;
}

/*__device__ float SDL::computeT5RZChiSquared(struct SDL::modules& modulesInGPU, struct SDL::miniDoublets& mdsInGPU, unsigned int firstMDIndex, unsigned int secondMDIndex, unsigned int thirdMDIndex, unsigned int fourthMDIndex, unsigned int fifthMDIndex, uint16_t& lowerModuleIndex1, uint16_t& lowerModuleIndex2, uint16_t& lowerModuleIndex3, uint16_t& lowerModuleIndex4, uint16_t& lowerModuleIndex5)
{
    const float& rt1 = mdsInGPU.anchorRt[firstMDIndex];
    const float& rt2 = mdsInGPU.anchorRt[secondMDIndex];
    const float& rt3 = mdsInGPU.anchorRt[thirdMDIndex];
    const float& rt4 = mdsInGPU.anchorRt[fourthMDIndex];
    const float& rt5 = mdsInGPU.anchorRt[fifthMDIndex];

    const float& z1 = mdsInGPU.anchorZ[firstMDIndex];
    const float& z2 = mdsInGPU.anchorZ[secondMDIndex];
    const float& z3 = mdsInGPU.anchorZ[thirdMDIndex];
    const float& z4 = mdsInGPU.anchorZ[fourthMDIndex];
    const float& z5 = mdsInGPU.anchorZ[fifthMDIndex];

    //following Philip's layer number prescription
    const int layer1 = modulesInGPU.layers[lowerModuleIndex1] + 6 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex1] == SDL::TwoS);
    const int layer2 = modulesInGPU.layers[lowerModuleIndex2] + 6 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex2] == SDL::TwoS);
    const int layer3 = modulesInGPU.layers[lowerModuleIndex3] + 6 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex3] == SDL::TwoS);
    const int layer4 = modulesInGPU.layers[lowerModuleIndex4] + 6 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex4] == SDL::TwoS);
    const int layer5 = modulesInGPU.layers[lowerModuleIndex5] + 6 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex5] == SDL::TwoS);

    const int& moduleType3 = modulesInGPU.moduleType[lowerModuleIndex3];
    const int& moduleType4 = modulesInGPU.moduleType[lowerModuleIndex4];
    const int& moduleType5 = modulesInGPU.moduleType[lowerModuleIndex5];
    const short& subdet3 = modulesInGPU.subdets[lowerModuleIndex3];
    const short& subdet4 = modulesInGPU.subdets[lowerModuleIndex4];
    const short& subdet5 = modulesInGPU.subdets[lowerModuleIndex5];
    const float& drdz3 = modulesInGPU.drdzs[lowerModuleIndex3];
    const float& drdz4 = modulesInGPU.drdzs[lowerModuleIndex4];
    const float& drdz5 = modulesInGPU.drdzs[lowerModuleIndex5];

    const short side3 = modulesInGPU.sides[lowerModuleIndex3];
    const short side4 = modulesInGPU.sides[lowerModuleIndex4];
    const short side5 = modulesInGPU.sides[lowerModuleIndex5];

    //denominator factor for tilted modules : cos theta for < 45 degrees, sin theta for > 45 degrees
//    float projection3 = ((subdet3 == SDL::Endcap) or (side3 == SDL::Center)) ? 1.f : fmaxf(1.f, drdz3)/sqrtf(1+drdz3*drdz3);
    float projection4 = ((subdet4 == SDL::Endcap) or (side4 == SDL::Center)) ? 1.f : fmaxf(1.f, drdz4)/sqrtf(1+drdz4*drdz4);
    float projection5 = ((subdet5 == SDL::Endcap) or (side5 == SDL::Center)) ? 1.f : fmaxf(1.f, drdz5)/sqrtf(1+drdz5*drdz5);

    float slope = (z2-z1)/(rt2-rt1);

    //numerator of chi squared
    float residual4 = (subdet4 == SDL::Barrel and ((side4 == SDL::Center)or (drdz4 < 1))) ? (((z4 - z1) - slope * (rt4 - rt1))) : ((rt4 - rt1) - (z4 - z1)/slope);
    float residual5 = (subdet5 == SDL::Barrel and ((side5 == SDL::Center)or (drdz5 < 1))) ? (((z5 - z1) - slope * (rt5 - rt1))) : ((rt5 - rt1) - (z5 - z1)/slope);

    float denominator4 = (moduleType4 == SDL::PS) ? 0.15f*projection4 : 5.f;
    float denominator5 = (moduleType5 == SDL::PS) ? 0.15f*projection5 : 5.f;

    const float RMSE = sqrtf(0.5 * ((residual5/denominator5) * (residual5/denominator5) + (residual4/denominator4) * (residual4/denominator4)));
    return RMSE;
}*/


__device__ bool SDL::passT5RZConstraint(struct SDL::modules& modulesInGPU, struct SDL::miniDoublets& mdsInGPU, unsigned int firstMDIndex, unsigned int secondMDIndex, unsigned int thirdMDIndex, unsigned int fourthMDIndex, unsigned int fifthMDIndex, uint16_t& lowerModuleIndex1, uint16_t& lowerModuleIndex2, uint16_t& lowerModuleIndex3, uint16_t& lowerModuleIndex4, uint16_t& lowerModuleIndex5, float& rzChiSquared, float& residual_missing, float& residual4, float& residual5, float inner_pt, float innerRadius, float g, float f, bool& tightCutFlag) 
{
    //(g,f) is the center of the circle fitted by the innermost 3 points on x,y coordinates
    const float& rt1 = mdsInGPU.anchorRt[firstMDIndex]/100; //in the unit of m instead of cm
    const float& rt2 = mdsInGPU.anchorRt[secondMDIndex]/100;
    const float& rt3 = mdsInGPU.anchorRt[thirdMDIndex]/100;
    const float& rt4 = mdsInGPU.anchorRt[fourthMDIndex]/100;
    const float& rt5 = mdsInGPU.anchorRt[fifthMDIndex]/100;

    const float& z1 = mdsInGPU.anchorZ[firstMDIndex]/100;
    const float& z2 = mdsInGPU.anchorZ[secondMDIndex]/100;
    const float& z3 = mdsInGPU.anchorZ[thirdMDIndex]/100;
    const float& z4 = mdsInGPU.anchorZ[fourthMDIndex]/100;
    const float& z5 = mdsInGPU.anchorZ[fifthMDIndex]/100;

    //following Philip's layer number prescription
    const int layer1 = modulesInGPU.layers[lowerModuleIndex1] + 6 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex1] == SDL::TwoS);
    const int layer2 = modulesInGPU.layers[lowerModuleIndex2] + 6 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex2] == SDL::TwoS);
    const int layer3 = modulesInGPU.layers[lowerModuleIndex3] + 6 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex3] == SDL::TwoS);
    const int layer4 = modulesInGPU.layers[lowerModuleIndex4] + 6 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex4] == SDL::TwoS);
    const int layer5 = modulesInGPU.layers[lowerModuleIndex5] + 6 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex5] == SDL::TwoS);

    //slope computed using the internal T3s
    const int moduleType1 = modulesInGPU.moduleType[lowerModuleIndex1]; //0 is ps, 1 is 2s
    const int moduleType2 = modulesInGPU.moduleType[lowerModuleIndex2];
    const int moduleType3 = modulesInGPU.moduleType[lowerModuleIndex3];
    const int moduleType4 = modulesInGPU.moduleType[lowerModuleIndex4];
    const int moduleType5 = modulesInGPU.moduleType[lowerModuleIndex5];

    const float& x1 = mdsInGPU.anchorX[firstMDIndex]/100;
    const float& x2 = mdsInGPU.anchorX[secondMDIndex]/100;
    const float& x3 = mdsInGPU.anchorX[thirdMDIndex]/100;
    const float& x4 = mdsInGPU.anchorX[fourthMDIndex]/100;
    const float& x5 = mdsInGPU.anchorX[fifthMDIndex]/100;
    const float& y1 = mdsInGPU.anchorY[firstMDIndex]/100;
    const float& y2 = mdsInGPU.anchorY[secondMDIndex]/100;
    const float& y3 = mdsInGPU.anchorY[thirdMDIndex]/100;
    const float& y4 = mdsInGPU.anchorY[fourthMDIndex]/100;
    const float& y5 = mdsInGPU.anchorY[fifthMDIndex]/100;

    float residual = 0;
    float error = 0;
    float x_center=g/100, y_center=f/100; 
    float x_init=mdsInGPU.anchorX[thirdMDIndex]/100;
    float y_init=mdsInGPU.anchorY[thirdMDIndex]/100;
    float z_init=mdsInGPU.anchorZ[thirdMDIndex]/100;
    float rt_init=mdsInGPU.anchorRt[thirdMDIndex]/100; //use the second MD as initial point

    if (moduleType3==1)  // 1: if MD3 is in 2s layer
    {
        x_init=mdsInGPU.anchorX[secondMDIndex]/100;
        y_init=mdsInGPU.anchorY[secondMDIndex]/100;
        z_init=mdsInGPU.anchorZ[secondMDIndex]/100;
        rt_init=mdsInGPU.anchorRt[secondMDIndex]/100;
    }

    //start from a circle of inner T3.
    // to determine the charge
    int charge=0;
    float slope3c=(y3-y_center)/(x3-x_center);
    float slope1c=(y1-y_center)/(x1-x_center);
    if((y3-y_center)>0 && (y1-y_center)>0) 
    {
        if (slope3c>slope1c) charge=-1; 
        else if (slope3c<slope1c) charge=1;
        if (slope1c>0 && slope3c<0) charge=-1;
        if (slope1c<0 && slope3c>0) charge=1;
    }
    else if((y3-y_center)<0 && (y1-y_center)<0) 
    {
        if (slope3c>slope1c) charge=-1; 
        else if (slope3c<slope1c) charge=1;
        if (slope1c<0 && slope3c>0) charge=1;
        if (slope1c>0 && slope3c<0) charge=-1;
    }
    else if ((y3-y_center)<0 && (y1-y_center)>0)
    {
        if ((x3-x_center)>0 && (x1-x_center)>0) charge = 1;
        else if ((x3-x_center)<0 && (x1-x_center)<0) charge = -1;
    }
    else if ((y3-y_center)>0 && (y1-y_center)<0)
    {
        if ((x3-x_center)>0 && (x1-x_center)>0) charge = -1;
        else if ((x3-x_center)<0 && (x1-x_center)<0) charge = 1;
    }

    float pseudo_phi = atan((y_init-y_center)/(x_init-x_center)); //actually represent pi/2-phi, wrt helix axis z
    float Pt=inner_pt, Px=Pt*abs(sin(pseudo_phi)), Py=Pt*abs(cos(pseudo_phi));

    if (x_init>x_center && y_init>y_center) //1st quad
    {
        if (charge==1) Py=-Py;
        if (charge==-1) Px=-Px;
    }
    if (x_init<x_center && y_init>y_center) //2nd quad
    {
        if (charge==-1) {Px=-Px; Py=-Py;}
    }
    if (x_init<x_center && y_init<y_center) //3rd quad
    {
        if (charge==1) Px=-Px;
        if (charge==-1) Py=-Py;
    }        
    if (x_init>x_center && y_init<y_center) //4th quad
    {
        if (charge==1) {Px=-Px; Py=-Py;}
    }

    if (moduleType3==0){
        if (x4<x3 && x3<x2) Px=-abs(Px);
        if (x4>x3 && x3>x2) Px=abs(Px);
        if (y4<y3 && y3<y2) Py=-abs(Py);
        if (y4>y3 && y3>y2) Py=abs(Py);
    }
    else if(moduleType3==1)
    {
        if (x3<x2 && x2<x1) Px=-abs(Px);
        if (x3>x2 && x2>x1) Px=abs(Px);
        if (y3<y2 && y2<y1) Py=-abs(Py);
        if (y3>y2 && y2>y1) Py=abs(Py);        
    }

    //to get Pz, we use pt/pz=ds/dz, ds is the arclength between MD1 and MD3.
    float AO=sqrt((x1-x_center)*(x1-x_center)+(y1-y_center)*(y1-y_center));
    float BO=sqrt((x_init-x_center)*(x_init-x_center)+(y_init-y_center)*(y_init-y_center));
    float AB=sqrt((x1-x_init)*(x1-x_init)+(y1-y_init)*(y1-y_init)); 
    float dPhi = acos((AO*AO+BO*BO-AB*AB)/(2*AO*BO));
    float ds=innerRadius/100*dPhi;

//    float ds = sqrt((y_init-y1)*(y_init-y1)+(x_init-x1)*(x_init-x1)); //large ds->smallerPz->smaller residual
    float Pz=(z_init-z1)/ds*Pt;
    float p = sqrt(Px*Px+Py*Py+Pz*Pz);

    float B = 3.8112;
    float a = -0.299792*B*charge;

    float zsi, rtsi;
    int layeri, moduleTypei;
    float expectrt4=0,expectrt5=0,expectz4=0, expectz5=0;
    rzChiSquared=0;
    for(size_t i = 2; i < 6; i++)
    {
        if (i==2){
            zsi = z2;
            rtsi = rt2;
            layeri=layer2;
            moduleTypei=moduleType2;
        }
        else if (i==3) {
            zsi = z3;
            rtsi = rt3;
            layeri=layer3;
            moduleTypei=moduleType3;
        }
        else if (i==4){
            zsi = z4;
            rtsi = rt4;
            layeri=layer4;
            moduleTypei=moduleType4;
        }
        else if (i==5){
            zsi = z5;
            rtsi = rt5;
            layeri=layer5;
            moduleTypei=moduleType5;
        }

        if (moduleType3==0) { //0: ps
            if (i==3) continue;
        }
        else{
            if (i==2) continue;
        }

        // calculation is copied from PixelTriplet.cu SDL::computePT3RZChiSquared
        float diffr=0, diffz=0;

        float rou = a/p;
        // for endcap
        float s = (zsi-z_init)*p/Pz;
        float x = x_init + Px/a*sin(rou*s)-Py/a*(1-cos(rou*s));
        float y = y_init + Py/a*sin(rou*s)+Px/a*(1-cos(rou*s));
        diffr = (rtsi-sqrt(x*x+y*y))*100;
        if (i==4) expectrt4=sqrt(x*x+y*y);
        if (i==5) expectrt5=sqrt(x*x+y*y);

        // for barrel
        if (layeri<=6)
        {
            float paraA = rt_init*rt_init + 2*(Px*Px+Py*Py)/(a*a) + 2*(y_init*Px-x_init*Py)/a - rtsi*rtsi;
            float paraB = 2*(x_init*Px+y_init*Py)/a;
            float paraC = 2*(y_init*Px-x_init*Py)/a+2*(Px*Px+Py*Py)/(a*a);
            float A=paraB*paraB+paraC*paraC;
            float B=2*paraA*paraB;
            float C=paraA*paraA-paraC*paraC;
            float sol1 = (-B+sqrt(B*B-4*A*C))/(2*A);
            float sol2 = (-B-sqrt(B*B-4*A*C))/(2*A);
            float solz1 = asin(sol1)/rou*Pz/p+z_init;
            float solz2 = asin(sol2)/rou*Pz/p+z_init;
            float diffz1 = (solz1-zsi)*100;
            float diffz2 = (solz2-zsi)*100;
            if (isnan(diffz1)) diffz = diffz2;
            else if (isnan(diffz2)) diffz = diffz1;
            else {diffz = (fabs(diffz1)<fabs(diffz2)) ? diffz1 : diffz2;}
        }
        residual = (layeri>6) ? diffr : diffz ;

        //PS Modules
        if(moduleTypei == 0)
        {
            error = 0.15f;
        }
        else //2S modules
        {
            error = 5.0f;
        }
        if (i==4) residual4=residual/error;
        if (i==5) residual5=residual/error;

        //check the tilted module, side: PosZ, NegZ, Center(for not tilted)
        float drdz;
        short side, subdets;
        if (i==2){
            drdz=abs(modulesInGPU.drdzs[lowerModuleIndex2]);
            side=modulesInGPU.sides[lowerModuleIndex2];
            subdets=modulesInGPU.subdets[lowerModuleIndex2];
        }
        if (i==3){
            drdz=abs(modulesInGPU.drdzs[lowerModuleIndex3]);
            side=modulesInGPU.sides[lowerModuleIndex3];
            subdets=modulesInGPU.subdets[lowerModuleIndex3];
        }
        if (i==2 || i==3){
            residual = (layeri <= 6 && ((side == SDL::Center) or (drdz < 1))) ? diffz : diffr;
//            residual_missing=residual;
            float projection_missing=1;
        if (drdz<1)
            projection_missing = ((subdets == SDL::Endcap) or (side == SDL::Center)) ? 1.f : 1/sqrt(1+drdz*drdz); // cos(atan(drdz)), if dr/dz<1
        if (drdz>1)
            projection_missing = ((subdets == SDL::Endcap) or (side == SDL::Center)) ? 1.f : drdz/sqrt(1+drdz*drdz);//sin(atan(drdz)), if dr/dz>1
            error=error*projection_missing;
            residual_missing=residual/error;
        }
        rzChiSquared += 12*(residual * residual)/(error * error);
    }
//    rzChiSquared = 12*(residual4 * residual4 + residual5 * residual5 + residual_missing * residual_missing);

//    if (isnan(rzChiSquared)) printf("rzChi2: %f, residual2: %f, inner_pt:%f, pseudo_phi: %f, charge: %i, Px:%f, Py:%f, x1:%f, x2:%f, x3:%f, x4:%f, x5:%f, y1:%f, y2:%f, y3:%f, y4:%f, y5:%f, z1:%f, z2:%f, z3:%f, z4:%f, z5:%f, x_center:%f, y_center:%f, slope1c:%f, slope3c:%f\n", rzChiSquared, residual_missing, inner_pt, pseudo_phi, charge, Px, Py, x1, x2, x3, x4, x5, y1, y2, y3, y4, y5, z1, z2, z3, z4, z5, x_center, y_center, slope1c, slope3c);

//    if(layer1 == 7 and layer2 == 8 and layer3 == 9 and layer4 == 10 and layer5 == 11 and rzChiSquared>100){
//        printf("rt1:%f, rt2:%f, rt3:%f, rt4:%f, rt5:%f\n", rt1, rt2, rt3, rt4, rt5);
//        printf("x1:%f, x2:%f, x3:%f, x4:%f, x5:%f\n", x1, x2, x3, x4, x5);
//        printf("y1:%f, y2:%f, y3:%f, y4:%f, y5:%f\n", y1, y2, y3, y4, y5);
//        printf("z1:%f, z2:%f, z3:%f, z4:%f, z5:%f\n", z1, z2, z3, z4, z5);
//        printf("rt4_ex:%f, rt5_ex:%f\n", expectrt4, expectrt5);
//        printf("z4_ex:%f, z5_ex:%f\n", expectz4, expectz5);
//        printf("residual_missing:%f\n", residual_missing);
//        printf("Pt:%f, Px:%f, Py:%f, Pz:%f, charge: %i, residual_missing: %f, residual4: %f, residual5:%f, moduleType3:%i\n", Pt, Px, Py, Pz, charge, residual_missing, residual4, residual5, moduleType3);
//        if (fabs(rzChiSquared-434.901)<0.01) printf("rzChi2: %f, residual2: %f, residual4: %f, residual5:%f, inner_pt:%f, pseudo_phi: %f, charge: %i, Px:%f, Py:%f, x1:%f, x2:%f, x3:%f, x4:%f, x5:%f, y1:%f, y2:%f, y3:%f, y4:%f, y5:%f, z1:%f, z2:%f, z3:%f, z4:%f, z5:%f, x_center:%f, y_center:%f, slope1c:%f, slope3c:%f\n", rzChiSquared, residual_missing, residual4, residual5, inner_pt, pseudo_phi, charge, Px, Py, x1, x2, x3, x4, x5, y1, y2, y3, y4, y5, z1, z2, z3, z4, z5, x_center, y_center, slope1c, slope3c);
//        printf("residual_missing:%f\n", residual_missing);
//    }

    // when building T5, apply 99% chi2 cuts as default, and add to pT5 collection. But when adding T5 to TC collections, appy 95% cut to reduce the fake rate
    tightCutFlag = false;
    //categories!
    if(layer1 == 1 and layer2 == 2 and layer3 == 3)
    {
        if(layer4 == 4 and layer5 == 5) //11
        {
            if (rzChiSquared < 15.627f) tightCutFlag = true;
            return rzChiSquared < 29.035f; 
        }
        else if(layer4 == 4 and layer5 == 12) //12
        {
            if (rzChiSquared < 14.64f) tightCutFlag = true;
            return rzChiSquared < 23.037f;
        }
        else if(layer4 == 7 and layer5 == 8) //8
        {   
            if (rzChiSquared < 27.824f) tightCutFlag = true;
            return rzChiSquared < 44.247f;
        }
        else if(layer4 == 7 and layer5 == 13) //9
        {
            if (rzChiSquared < 18.145f) tightCutFlag = true;
            return rzChiSquared < 33.752f;
        }
        else if(layer4 == 12 and layer5 == 13) //10
        {
            if (rzChiSquared < 13.308f) tightCutFlag = true;
            return rzChiSquared < 21.213f;
        }
    }
    else if(layer1 == 1 and layer2 == 2 and layer3 == 7)
    {
        if(layer4 == 8 and layer5 == 9) //5
        {
            if (rzChiSquared < 116.148f) tightCutFlag = true;
            return true;
        }
        if(layer4 == 8 and layer5 == 14) //6
        {
            if (rzChiSquared < 19.352f) tightCutFlag = true;
            return rzChiSquared < 52.561f;
        }
        else if(layer4 == 13 and layer5 == 14) //7
        {
            if (rzChiSquared < 10.392f) tightCutFlag = true;
            return rzChiSquared < 13.76f;
        }
    }
    else if(layer1 == 1 and layer2 == 7 and layer3 == 8 and layer4 == 9) 
    {
        if (layer5 == 10) //3
        {
            if (rzChiSquared < 111.390f) tightCutFlag = true;
            return true;
        }
        if (layer5 == 15) //4
        {
            if (rzChiSquared < 18.351f) tightCutFlag = true;
            return rzChiSquared < 37.941f;
        }
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 4)
    {
        if(layer4 == 5 and layer5 == 6) //18
        {
            if (rzChiSquared < 6.065f) tightCutFlag = true;
            return rzChiSquared < 8.803f;
        }
        else if(layer4 == 5 and layer5 == 12) //19
        {
            if (rzChiSquared < 5.693f) tightCutFlag = true;
            return rzChiSquared < 7.930f;
        }

        else if(layer4 == 12 and layer5 == 13) //20
        {
            if (rzChiSquared < 5.473f) tightCutFlag = true;
            return rzChiSquared < 7.626f;
        }
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 7) 
    {
        if(layer4 == 8 and layer5 == 14) //16
        {
            if (rzChiSquared < 23.730f) tightCutFlag = true;
            return rzChiSquared < 23.748f;
        }
        if(layer4 == 13 and layer5 == 14) //17
        {
            if (rzChiSquared < 10.772f) tightCutFlag = true;
            return rzChiSquared < 17.945f;
        }
    }

    else if(layer1 == 2 and layer2 == 7 and layer3 == 8)
    {
        if(layer4 == 9 and layer5 == 15) //14
        {
            if (rzChiSquared < 24.662f) tightCutFlag = true;
            return rzChiSquared < 41.036f;
        }
        else if(layer4 == 14 and layer5 == 15) //15
        {
            if (rzChiSquared < 8.866f) tightCutFlag = true;
            return rzChiSquared < 14.092f;
        }
    }

    else if(layer1 == 7 and layer2 == 8 and layer3 == 9 and layer4 == 15 and layer5 == 16) //2
    {
        if (rzChiSquared < 7.992f) tightCutFlag = true;
        return rzChiSquared < 11.622f;
    }

    else if(layer1 == 7 and layer2 == 8 and layer3 == 9 and layer4 == 10 and layer5 == 11) //0
    {
        if (rzChiSquared < 94.470f) tightCutFlag = true;
        return true;
    }

    else if(layer1 == 7 and layer2 == 8 and layer3 == 9 and layer4 == 10 and layer5 == 16) //1
    {
        if (rzChiSquared < 22.099f) tightCutFlag = true;
        return rzChiSquared < 37.956f;
    }
    return true;
}


//90% constraint
/*__device__ bool SDL::passChiSquaredConstraint(struct SDL::modules& modulesInGPU, uint16_t& lowerModuleIndex1, uint16_t& lowerModuleIndex2, uint16_t& lowerModuleIndex3, uint16_t& lowerModuleIndex4, uint16_t& lowerModuleIndex5, float& chiSquared)
{
    //following Philip's layer number prescription
    const int layer1 = modulesInGPU.layers[lowerModuleIndex1] + 6 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex1] == SDL::TwoS);
    const int layer2 = modulesInGPU.layers[lowerModuleIndex2] + 6 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex2] == SDL::TwoS);
    const int layer3 = modulesInGPU.layers[lowerModuleIndex3] + 6 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex3] == SDL::TwoS);
    const int layer4 = modulesInGPU.layers[lowerModuleIndex4] + 6 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex4] == SDL::TwoS);
    const int layer5 = modulesInGPU.layers[lowerModuleIndex5] + 6 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex5] == SDL::TwoS);

    if(layer1 == 7 and layer2 == 8 and layer3 == 9)
    {
        if(layer4 == 10 and layer5 == 11)
        {
            return chiSquared < 0.01788f;
        }
        else if(layer4 == 10 and layer5 == 16)
        {
            return chiSquared < 0.04725f;
        }
        else if(layer4 == 15 and layer5 == 16)
        {
            return chiSquared < 0.04725f;
        }
    }
    else if(layer1 == 1 and layer2 == 7 and layer3 == 8)
    {
        if(layer4 == 9 and layer5 == 10)
        {       
            return chiSquared < 0.01788f;
        }   
        else if(layer4 == 9 and layer5 == 15)
        {
            return chiSquared < 0.08234f;
        }
    }
    else if(layer1 == 1 and layer2 == 2 and layer3 == 7)
    {
        if(layer4 == 8 and layer5 == 9)
        {
            return chiSquared < 0.02360f;
        }
        else if(layer4 == 8 and layer5 == 14)
        {
            return chiSquared < 0.07167f;
        }
        else if(layer4 == 13 and layer5 == 14)
        {   
            return chiSquared < 0.08234f;
        }
    }
    else if(layer1 == 1 and layer2 == 2 and layer3 == 3)
    {
        if(layer4 == 7 and layer5 == 8)
        {
            return chiSquared < 0.01026f;
        }
        else if(layer4 == 7 and layer5 == 13)
        {
            return chiSquared < 0.06238f;
        }
        else if(layer4 == 12 and layer5 == 13)
        {
            return chiSquared < 0.06238f;
        }
    }
    else if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4)
    {
        if(layer5 == 12)
        {
            return chiSquared < 0.09461f;
        }
        else if(layer5 == 5)
        {
            return chiSquared < 0.04725f;
        }
    }
    else if(layer1 == 2 and layer2 == 7 and layer3 == 8)
    {
        if(layer4 == 9 and layer5 == 10)
        {
            return chiSquared < 0.00512f;
        }
        if(layer4 == 9 and layer5 == 15)
        {
            return chiSquared < 0.04112f;
        }
        else if(layer4 == 14 and layer5 == 15)
        {
            return chiSquared < 0.06238f;
        }
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 7)
    {
        if(layer4 == 8 and layer5 == 14)
        {
            return chiSquared < 0.07167f;
        }
        else if(layer4 == 13 and layer5 == 14)
        {
            return chiSquared < 0.06238f;
        }
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 4)
    {
        if(layer4 == 12 and layer5 == 13)
        {
            return chiSquared < 0.10870f;
        }
        else if(layer4 == 5 and layer5 == 12)
        {
            return chiSquared < 0.10870f;
        }
        else if(layer4 == 5 and layer5 == 6)
        {
            return chiSquared < 0.08234f;
        }
    }
    else if(layer1 == 3 and layer2 == 7 and layer3 == 8 and layer4 == 14 and layer5 == 15)
    {
        return chiSquared < 0.09461f;
    }
    else if(layer1 == 3 and layer2 == 4 and layer3 == 5 and layer4 == 12 and layer5 == 13)
    {
        return chiSquared < 0.09461f;
    }

    return true;
}*/

__device__ bool SDL::matchRadii_bin1(struct SDL::modules& modulesInGPU, float& eta, uint16_t& lowerModuleIndex1, uint16_t& lowerModuleIndex2, uint16_t& lowerModuleIndex3, uint16_t& lowerModuleIndex4, uint16_t& lowerModuleIndex5, float& innerRadius, float& outerRadius)
{
    const int layer1 = modulesInGPU.layers[lowerModuleIndex1] + 6 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex1] == SDL::TwoS);
    const int layer2 = modulesInGPU.layers[lowerModuleIndex2] + 6 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex2] == SDL::TwoS);
    const int layer3 = modulesInGPU.layers[lowerModuleIndex3] + 6 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex3] == SDL::TwoS);
    const int layer4 = modulesInGPU.layers[lowerModuleIndex4] + 6 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex4] == SDL::TwoS);
    const int layer5 = modulesInGPU.layers[lowerModuleIndex5] + 6 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex5] == SDL::TwoS);
    if(layer1 == 7 and layer2 == 8 and layer3 == 9 and layer4 == 10 and layer5 == 11)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.23523523523523526;
    }
    if(layer1 == 1 and layer2 == 7 and layer3 == 8 and layer4 == 9 and layer5 == 10)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.3153153153153153;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 7 and layer4 == 8 and layer5 == 9)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.2052052052052052;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 7 and layer5 == 8)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.12512512512512514;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4 and layer5 == 12)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.4754754754754755;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4 and layer5 == 5 and abs(eta) < 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.1051051051051051;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4 and layer5 == 5 and abs(eta) > 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.1051051051051051;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 5 and layer5 == 6 and abs(eta) < 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.08508508508508508;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 5 and layer5 == 6 and abs(eta) > 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.08508508508508508;
    }
    return true;
}
__device__ bool SDL::matchRadii_bin2(struct SDL::modules& modulesInGPU, float& eta, uint16_t& lowerModuleIndex1, uint16_t& lowerModuleIndex2, uint16_t& lowerModuleIndex3, uint16_t& lowerModuleIndex4, uint16_t& lowerModuleIndex5, float& innerRadius, float& outerRadius)
{
    const int layer1 = modulesInGPU.layers[lowerModuleIndex1] + 6 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex1] == SDL::TwoS);
    const int layer2 = modulesInGPU.layers[lowerModuleIndex2] + 6 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex2] == SDL::TwoS);
    const int layer3 = modulesInGPU.layers[lowerModuleIndex3] + 6 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex3] == SDL::TwoS);
    const int layer4 = modulesInGPU.layers[lowerModuleIndex4] + 6 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex4] == SDL::TwoS);
    const int layer5 = modulesInGPU.layers[lowerModuleIndex5] + 6 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex5] == SDL::TwoS);
    if(layer1 == 7 and layer2 == 8 and layer3 == 9 and layer4 == 10 and layer5 == 11)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.24524524524524527;
    }
    if(layer1 == 1 and layer2 == 7 and layer3 == 8 and layer4 == 9 and layer5 == 10)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.34534534534534533;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 7 and layer4 == 8 and layer5 == 9)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.2052052052052052;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 7 and layer4 == 8 and layer5 == 14)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.5855855855855856;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4 and layer5 == 5 and abs(eta) < 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.15515515515515516;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4 and layer5 == 5 and abs(eta) > 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.15515515515515516;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 5 and layer5 == 6 and abs(eta) > 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.17517517517517517;
    }
    return true;
}
__device__ bool SDL::matchRadii_bin3(struct SDL::modules& modulesInGPU, float& eta, uint16_t& lowerModuleIndex1, uint16_t& lowerModuleIndex2, uint16_t& lowerModuleIndex3, uint16_t& lowerModuleIndex4, uint16_t& lowerModuleIndex5, float& innerRadius, float& outerRadius)
{
    const int layer1 = modulesInGPU.layers[lowerModuleIndex1] + 6 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex1] == SDL::TwoS);
    const int layer2 = modulesInGPU.layers[lowerModuleIndex2] + 6 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex2] == SDL::TwoS);
    const int layer3 = modulesInGPU.layers[lowerModuleIndex3] + 6 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex3] == SDL::TwoS);
    const int layer4 = modulesInGPU.layers[lowerModuleIndex4] + 6 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex4] == SDL::TwoS);
    const int layer5 = modulesInGPU.layers[lowerModuleIndex5] + 6 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex5] == SDL::TwoS);
    if(layer1 == 7 and layer2 == 8 and layer3 == 9 and layer4 == 10 and layer5 == 11)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.3053053053053053;
    }
    if(layer1 == 7 and layer2 == 8 and layer3 == 9 and layer4 == 15 and layer5 == 16)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.7857857857857857;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 7 and layer4 == 8 and layer5 == 9)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.3353353353353353;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 7 and layer5 == 8)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.13513513513513514;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4 and layer5 == 5 and abs(eta) < 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.17517517517517517;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4 and layer5 == 5 and abs(eta) > 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.16516516516516516;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 5 and layer5 == 6 and abs(eta) < 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.16516516516516516;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 5 and layer5 == 6 and abs(eta) > 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.19519519519519518;
    }
    return true;
}
__device__ bool SDL::matchRadii_bin4(struct SDL::modules& modulesInGPU, float& eta, uint16_t& lowerModuleIndex1, uint16_t& lowerModuleIndex2, uint16_t& lowerModuleIndex3, uint16_t& lowerModuleIndex4, uint16_t& lowerModuleIndex5, float& innerRadius, float& outerRadius)
{
    const int layer1 = modulesInGPU.layers[lowerModuleIndex1] + 6 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex1] == SDL::TwoS);
    const int layer2 = modulesInGPU.layers[lowerModuleIndex2] + 6 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex2] == SDL::TwoS);
    const int layer3 = modulesInGPU.layers[lowerModuleIndex3] + 6 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex3] == SDL::TwoS);
    const int layer4 = modulesInGPU.layers[lowerModuleIndex4] + 6 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex4] == SDL::TwoS);
    const int layer5 = modulesInGPU.layers[lowerModuleIndex5] + 6 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex5] == SDL::TwoS);
    if(layer1 == 7 and layer2 == 8 and layer3 == 9 and layer4 == 10 and layer5 == 11)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.3853853853853854;
    }
    if(layer1 == 7 and layer2 == 8 and layer3 == 9 and layer4 == 15 and layer5 == 16)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.7757757757757757;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 7 and layer4 == 8 and layer5 == 9)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.2852852852852853;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 7 and layer4 == 13 and layer5 == 14)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 1.1961961961961962;
    }
    if(layer1 == 2 and layer2 == 7 and layer3 == 8 and layer4 == 14 and layer5 == 15)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.9459459459459458;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 7 and layer4 == 8 and layer5 == 14)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.7257257257257257;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4 and layer5 == 5 and abs(eta) < 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.19519519519519518;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4 and layer5 == 5 and abs(eta) > 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.2052052052052052;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 5 and layer5 == 6 and abs(eta) > 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.18518518518518517;
    }
    return true;
}
__device__ bool SDL::matchRadii_bin5(struct SDL::modules& modulesInGPU, float& eta, uint16_t& lowerModuleIndex1, uint16_t& lowerModuleIndex2, uint16_t& lowerModuleIndex3, uint16_t& lowerModuleIndex4, uint16_t& lowerModuleIndex5, float& innerRadius, float& outerRadius)
{
    const int layer1 = modulesInGPU.layers[lowerModuleIndex1] + 6 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex1] == SDL::TwoS);
    const int layer2 = modulesInGPU.layers[lowerModuleIndex2] + 6 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex2] == SDL::TwoS);
    const int layer3 = modulesInGPU.layers[lowerModuleIndex3] + 6 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex3] == SDL::TwoS);
    const int layer4 = modulesInGPU.layers[lowerModuleIndex4] + 6 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex4] == SDL::TwoS);
    const int layer5 = modulesInGPU.layers[lowerModuleIndex5] + 6 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex5] == SDL::TwoS);
    if(layer1 == 7 and layer2 == 8 and layer3 == 9 and layer4 == 10 and layer5 == 11)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.5455455455455456;
    }
    if(layer1 == 7 and layer2 == 8 and layer3 == 9 and layer4 == 15 and layer5 == 16)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.9459459459459458;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 7 and layer4 == 8 and layer5 == 9)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.4854854854854855;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 7 and layer4 == 13 and layer5 == 14)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 1.2662662662662663;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 7 and layer5 == 8)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.05505505505505505;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 12 and layer5 == 13)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 1.0860860860860861;
    }
    if(layer1 == 2 and layer2 == 7 and layer3 == 8 and layer4 == 14 and layer5 == 15)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 1.2262262262262262;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 7 and layer4 == 8 and layer5 == 14)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.39539539539539537;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 12 and layer5 == 13)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 1.2562562562562563;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4 and layer5 == 5 and abs(eta) < 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.2052052052052052;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4 and layer5 == 5 and abs(eta) > 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.2752752752752753;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 5 and layer5 == 6 and abs(eta) > 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.2052052052052052;
    }
    return true;
}
__device__ bool SDL::matchRadii_bin6(struct SDL::modules& modulesInGPU, float& eta, uint16_t& lowerModuleIndex1, uint16_t& lowerModuleIndex2, uint16_t& lowerModuleIndex3, uint16_t& lowerModuleIndex4, uint16_t& lowerModuleIndex5, float& innerRadius, float& outerRadius)
{
    const int layer1 = modulesInGPU.layers[lowerModuleIndex1] + 6 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex1] == SDL::TwoS);
    const int layer2 = modulesInGPU.layers[lowerModuleIndex2] + 6 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex2] == SDL::TwoS);
    const int layer3 = modulesInGPU.layers[lowerModuleIndex3] + 6 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex3] == SDL::TwoS);
    const int layer4 = modulesInGPU.layers[lowerModuleIndex4] + 6 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex4] == SDL::TwoS);
    const int layer5 = modulesInGPU.layers[lowerModuleIndex5] + 6 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex5] == SDL::TwoS);
    if(layer1 == 7 and layer2 == 8 and layer3 == 9 and layer4 == 15 and layer5 == 16)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 1.7517517517517518;
    }
    if(layer1 == 1 and layer2 == 7 and layer3 == 8 and layer4 == 9 and layer5 == 15)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 6.356356356356356;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 7 and layer4 == 8 and layer5 == 9)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 2.1521521521521523;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 7 and layer4 == 8 and layer5 == 14)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 5.7557557557557555;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 7 and layer4 == 13 and layer5 == 14)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 6.456456456456456;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 7 and layer5 == 13)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 2.4524524524524525;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 12 and layer5 == 13)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 5.8558558558558556;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4 and layer5 == 12)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 1.4514514514514514;
    }
    if(layer1 == 2 and layer2 == 7 and layer3 == 8 and layer4 == 9 and layer5 == 15)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 4.854854854854855;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 7 and layer4 == 13 and layer5 == 14)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 3.6536536536536532;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 12 and layer5 == 13)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 5.7557557557557555;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 5 and layer5 == 12)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 1.1511511511511512;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4 and layer5 == 5 and abs(eta) > 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.35035035035035034;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 5 and layer5 == 6 and abs(eta) < 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.45045045045045046;
    }
    return true;
}

__device__ bool SDL::matchRadii_bin1_tight(struct SDL::modules& modulesInGPU, float& eta, uint16_t& lowerModuleIndex1, uint16_t& lowerModuleIndex2, uint16_t& lowerModuleIndex3, uint16_t& lowerModuleIndex4, uint16_t& lowerModuleIndex5, float& innerRadius, float& outerRadius)
{
    const int layer1 = modulesInGPU.layers[lowerModuleIndex1] + 6 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex1] == SDL::TwoS);
    const int layer2 = modulesInGPU.layers[lowerModuleIndex2] + 6 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex2] == SDL::TwoS);
    const int layer3 = modulesInGPU.layers[lowerModuleIndex3] + 6 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex3] == SDL::TwoS);
    const int layer4 = modulesInGPU.layers[lowerModuleIndex4] + 6 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex4] == SDL::TwoS);
    const int layer5 = modulesInGPU.layers[lowerModuleIndex5] + 6 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex5] == SDL::TwoS);
    if(layer1 == 7 and layer2 == 8 and layer3 == 9 and layer4 == 10 and layer5 == 11)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.14514514514514515;
    }
    if(layer1 == 7 and layer2 == 8 and layer3 == 9 and layer4 == 10 and layer5 == 16)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.5155155155155156;
    }
    if(layer1 == 7 and layer2 == 8 and layer3 == 9 and layer4 == 15 and layer5 == 16)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.4854854854854855;
    }
    if(layer1 == 1 and layer2 == 7 and layer3 == 8 and layer4 == 9 and layer5 == 10)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.17517517517517517;
    }
    if(layer1 == 1 and layer2 == 7 and layer3 == 8 and layer4 == 9 and layer5 == 15)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.5855855855855856;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 7 and layer4 == 8 and layer5 == 9)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.11511511511511512;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 7 and layer4 == 8 and layer5 == 14)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.5755755755755756;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 7 and layer5 == 8)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.1051051051051051;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4 and layer5 == 12)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.2952952952952953;
    }
    if(layer1 == 2 and layer2 == 7 and layer3 == 8 and layer4 == 9 and layer5 == 15)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.5155155155155156;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 7 and layer4 == 8 and layer5 == 14)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.5355355355355356;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 5 and layer5 == 12)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.3253253253253253;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4 and layer5 == 5 and abs(eta) < 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.035035035035035036;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4 and layer5 == 5 and abs(eta) > 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.04504504504504504;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 5 and layer5 == 6 and abs(eta) < 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.035035035035035036;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 5 and layer5 == 6 and abs(eta) > 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.035035035035035036;
    }
    return true;
}
__device__ bool SDL::matchRadii_bin2_tight(struct SDL::modules& modulesInGPU, float& eta, uint16_t& lowerModuleIndex1, uint16_t& lowerModuleIndex2, uint16_t& lowerModuleIndex3, uint16_t& lowerModuleIndex4, uint16_t& lowerModuleIndex5, float& innerRadius, float& outerRadius)
{
    const int layer1 = modulesInGPU.layers[lowerModuleIndex1] + 6 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex1] == SDL::TwoS);
    const int layer2 = modulesInGPU.layers[lowerModuleIndex2] + 6 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex2] == SDL::TwoS);
    const int layer3 = modulesInGPU.layers[lowerModuleIndex3] + 6 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex3] == SDL::TwoS);
    const int layer4 = modulesInGPU.layers[lowerModuleIndex4] + 6 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex4] == SDL::TwoS);
    const int layer5 = modulesInGPU.layers[lowerModuleIndex5] + 6 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex5] == SDL::TwoS);
    if(layer1 == 7 and layer2 == 8 and layer3 == 9 and layer4 == 10 and layer5 == 11)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.14514514514514515;
    }
    if(layer1 == 7 and layer2 == 8 and layer3 == 9 and layer4 == 10 and layer5 == 16)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.5355355355355356;
    }
    if(layer1 == 7 and layer2 == 8 and layer3 == 9 and layer4 == 15 and layer5 == 16)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.5555555555555556;
    }
    if(layer1 == 1 and layer2 == 7 and layer3 == 8 and layer4 == 9 and layer5 == 10)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.18518518518518517;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 7 and layer4 == 8 and layer5 == 9)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.11511511511511512;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 7 and layer4 == 8 and layer5 == 14)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.5855855855855856;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4 and layer5 == 12)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.3153153153153153;
    }
    if(layer1 == 2 and layer2 == 7 and layer3 == 8 and layer4 == 9 and layer5 == 15)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.5155155155155156;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 7 and layer4 == 8 and layer5 == 14)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.5255255255255256;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 5 and layer5 == 12)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.35535535535535534;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4 and layer5 == 5 and abs(eta) < 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.04504504504504504;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4 and layer5 == 5 and abs(eta) > 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.04504504504504504;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 5 and layer5 == 6 and abs(eta) < 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.035035035035035036;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 5 and layer5 == 6 and abs(eta) > 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.04504504504504504;
    }
    return true;
}
__device__ bool SDL::matchRadii_bin3_tight(struct SDL::modules& modulesInGPU, float& eta, uint16_t& lowerModuleIndex1, uint16_t& lowerModuleIndex2, uint16_t& lowerModuleIndex3, uint16_t& lowerModuleIndex4, uint16_t& lowerModuleIndex5, float& innerRadius, float& outerRadius)
{
    const int layer1 = modulesInGPU.layers[lowerModuleIndex1] + 6 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex1] == SDL::TwoS);
    const int layer2 = modulesInGPU.layers[lowerModuleIndex2] + 6 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex2] == SDL::TwoS);
    const int layer3 = modulesInGPU.layers[lowerModuleIndex3] + 6 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex3] == SDL::TwoS);
    const int layer4 = modulesInGPU.layers[lowerModuleIndex4] + 6 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex4] == SDL::TwoS);
    const int layer5 = modulesInGPU.layers[lowerModuleIndex5] + 6 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex5] == SDL::TwoS);
    if(layer1 == 7 and layer2 == 8 and layer3 == 9 and layer4 == 10 and layer5 == 11)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.16516516516516516;
    }
    if(layer1 == 7 and layer2 == 8 and layer3 == 9 and layer4 == 10 and layer5 == 16)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.5555555555555556;
    }
    if(layer1 == 7 and layer2 == 8 and layer3 == 9 and layer4 == 15 and layer5 == 16)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.5455455455455456;
    }
    if(layer1 == 1 and layer2 == 7 and layer3 == 8 and layer4 == 9 and layer5 == 10)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.22522522522522526;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 7 and layer4 == 8 and layer5 == 9)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.12512512512512514;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 7 and layer4 == 13 and layer5 == 14)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.8858858858858859;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 7 and layer5 == 8)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.1051051051051051;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 12 and layer5 == 13)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.5555555555555556;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4 and layer5 == 12)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.35535535535535534;
    }
    if(layer1 == 2 and layer2 == 7 and layer3 == 8 and layer4 == 14 and layer5 == 15)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.6956956956956957;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 5 and layer5 == 12)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.43543543543543545;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4 and layer5 == 5 and abs(eta) < 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.04504504504504504;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4 and layer5 == 5 and abs(eta) > 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.04504504504504504;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 5 and layer5 == 6 and abs(eta) < 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.035035035035035036;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 5 and layer5 == 6 and abs(eta) > 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.035035035035035036;
    }
    return true;
}
__device__ bool SDL::matchRadii_bin4_tight(struct SDL::modules& modulesInGPU, float& eta, uint16_t& lowerModuleIndex1, uint16_t& lowerModuleIndex2, uint16_t& lowerModuleIndex3, uint16_t& lowerModuleIndex4, uint16_t& lowerModuleIndex5, float& innerRadius, float& outerRadius)
{
    const int layer1 = modulesInGPU.layers[lowerModuleIndex1] + 6 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex1] == SDL::TwoS);
    const int layer2 = modulesInGPU.layers[lowerModuleIndex2] + 6 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex2] == SDL::TwoS);
    const int layer3 = modulesInGPU.layers[lowerModuleIndex3] + 6 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex3] == SDL::TwoS);
    const int layer4 = modulesInGPU.layers[lowerModuleIndex4] + 6 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex4] == SDL::TwoS);
    const int layer5 = modulesInGPU.layers[lowerModuleIndex5] + 6 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex5] == SDL::TwoS);
    if(layer1 == 7 and layer2 == 8 and layer3 == 9 and layer4 == 10 and layer5 == 11)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.17517517517517517;
    }
    if(layer1 == 7 and layer2 == 8 and layer3 == 9 and layer4 == 15 and layer5 == 16)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.5255255255255256;
    }
    if(layer1 == 1 and layer2 == 7 and layer3 == 8 and layer4 == 9 and layer5 == 10)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.2752752752752753;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 7 and layer4 == 8 and layer5 == 9)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.12512512512512514;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 7 and layer4 == 13 and layer5 == 14)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.8858858858858859;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 12 and layer5 == 13)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.5655655655655656;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4 and layer5 == 12)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.39539539539539537;
    }
    if(layer1 == 2 and layer2 == 7 and layer3 == 8 and layer4 == 14 and layer5 == 15)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.6656656656656657;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 7 and layer4 == 8 and layer5 == 14)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.6156156156156156;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 7 and layer4 == 13 and layer5 == 14)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 1.006006006006006;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 12 and layer5 == 13)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.8058058058058057;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 5 and layer5 == 12)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.4854854854854855;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4 and layer5 == 5 and abs(eta) < 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.04504504504504504;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4 and layer5 == 5 and abs(eta) > 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.04504504504504504;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 5 and layer5 == 6 and abs(eta) < 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.035035035035035036;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 5 and layer5 == 6 and abs(eta) > 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.035035035035035036;
    }
    return true;
}
__device__ bool SDL::matchRadii_bin5_tight(struct SDL::modules& modulesInGPU, float& eta, uint16_t& lowerModuleIndex1, uint16_t& lowerModuleIndex2, uint16_t& lowerModuleIndex3, uint16_t& lowerModuleIndex4, uint16_t& lowerModuleIndex5, float& innerRadius, float& outerRadius)
{
    const int layer1 = modulesInGPU.layers[lowerModuleIndex1] + 6 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex1] == SDL::TwoS);
    const int layer2 = modulesInGPU.layers[lowerModuleIndex2] + 6 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex2] == SDL::TwoS);
    const int layer3 = modulesInGPU.layers[lowerModuleIndex3] + 6 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex3] == SDL::TwoS);
    const int layer4 = modulesInGPU.layers[lowerModuleIndex4] + 6 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex4] == SDL::TwoS);
    const int layer5 = modulesInGPU.layers[lowerModuleIndex5] + 6 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex5] == SDL::TwoS);
    if(layer1 == 7 and layer2 == 8 and layer3 == 9 and layer4 == 10 and layer5 == 11)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.24524524524524527;
    }
    if(layer1 == 7 and layer2 == 8 and layer3 == 9 and layer4 == 10 and layer5 == 16)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.7257257257257257;
    }
    if(layer1 == 7 and layer2 == 8 and layer3 == 9 and layer4 == 15 and layer5 == 16)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.5755755755755756;
    }
    if(layer1 == 1 and layer2 == 7 and layer3 == 8 and layer4 == 9 and layer5 == 10)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.4454454454454454;
    }
    if(layer1 == 1 and layer2 == 7 and layer3 == 8 and layer4 == 9 and layer5 == 15)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.7357357357357357;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 7 and layer4 == 8 and layer5 == 9)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.16516516516516516;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 7 and layer4 == 8 and layer5 == 14)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.8158158158158157;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 7 and layer4 == 13 and layer5 == 14)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.8958958958958958;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 7 and layer5 == 8)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.05505505505505505;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 7 and layer5 == 13)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.8458458458458458;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 12 and layer5 == 13)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.5855855855855856;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4 and layer5 == 12)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.4054054054054054;
    }
    if(layer1 == 2 and layer2 == 7 and layer3 == 8 and layer4 == 9 and layer5 == 15)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.7957957957957957;
    }
    if(layer1 == 2 and layer2 == 7 and layer3 == 8 and layer4 == 14 and layer5 == 15)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.7357357357357357;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 7 and layer4 == 8 and layer5 == 14)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.39539539539539537;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 7 and layer4 == 13 and layer5 == 14)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 1.006006006006006;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 12 and layer5 == 13)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.8558558558558558;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 5 and layer5 == 12)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.5555555555555556;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4 and layer5 == 5 and abs(eta) < 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.04504504504504504;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4 and layer5 == 5 and abs(eta) > 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.05505505505505505;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 5 and layer5 == 6 and abs(eta) < 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.035035035035035036;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 5 and layer5 == 6 and abs(eta) > 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.04504504504504504;
    }
    return true;
}
__device__ bool SDL::matchRadii_bin6_tight(struct SDL::modules& modulesInGPU, float& eta, uint16_t& lowerModuleIndex1, uint16_t& lowerModuleIndex2, uint16_t& lowerModuleIndex3, uint16_t& lowerModuleIndex4, uint16_t& lowerModuleIndex5, float& innerRadius, float& outerRadius)
{
    const int layer1 = modulesInGPU.layers[lowerModuleIndex1] + 6 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex1] == SDL::TwoS);
    const int layer2 = modulesInGPU.layers[lowerModuleIndex2] + 6 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex2] == SDL::TwoS);
    const int layer3 = modulesInGPU.layers[lowerModuleIndex3] + 6 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex3] == SDL::TwoS);
    const int layer4 = modulesInGPU.layers[lowerModuleIndex4] + 6 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex4] == SDL::TwoS);
    const int layer5 = modulesInGPU.layers[lowerModuleIndex5] + 6 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex5] == SDL::TwoS);
    if(layer1 == 7 and layer2 == 8 and layer3 == 9 and layer4 == 10 and layer5 == 11)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.6506506506506506;
    }
    if(layer1 == 7 and layer2 == 8 and layer3 == 9 and layer4 == 10 and layer5 == 16)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 2.3523523523523524;
    }
    if(layer1 == 7 and layer2 == 8 and layer3 == 9 and layer4 == 15 and layer5 == 16)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.9509509509509511;
    }
    if(layer1 == 1 and layer2 == 7 and layer3 == 8 and layer4 == 9 and layer5 == 10)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 3.8538538538538543;
    }
    if(layer1 == 1 and layer2 == 7 and layer3 == 8 and layer4 == 9 and layer5 == 15)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.9509509509509511;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 7 and layer4 == 8 and layer5 == 9)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.5505505505505506;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 7 and layer4 == 8 and layer5 == 14)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.9509509509509511;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 7 and layer4 == 13 and layer5 == 14)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 3.5535535535535536;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 7 and layer5 == 13)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 1.0510510510510511;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 12 and layer5 == 13)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.9509509509509511;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4 and layer5 == 12)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.5505505505505506;
    }
    if(layer1 == 2 and layer2 == 7 and layer3 == 8 and layer4 == 9 and layer5 == 15)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 2.052052052052052;
    }
    if(layer1 == 2 and layer2 == 7 and layer3 == 8 and layer4 == 14 and layer5 == 15)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 2.5525525525525525;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 7 and layer4 == 13 and layer5 == 14)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 1.6516516516516515;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 12 and layer5 == 13)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.9509509509509511;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 5 and layer5 == 12)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.7507507507507507;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4 and layer5 == 5 and abs(eta) < 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.15015015015015015;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4 and layer5 == 5 and abs(eta) > 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.15015015015015015;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 5 and layer5 == 6 and abs(eta) < 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.050050050050050046;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 5 and layer5 == 6 and abs(eta) > 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/outerRadius))/(1.0/innerRadius) < 0.050050050050050046;
    }
    return true;
}



__device__ bool SDL::matchRadii_inner_v_bridge_bin1(struct SDL::modules& modulesInGPU, float& eta, uint16_t& lowerModuleIndex1, uint16_t& lowerModuleIndex2, uint16_t& lowerModuleIndex3, uint16_t& lowerModuleIndex4, uint16_t& lowerModuleIndex5, float& innerRadius, float& bridgeRadius)
{
    const int layer1 = modulesInGPU.layers[lowerModuleIndex1] + 6 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex1] == SDL::TwoS);
    const int layer2 = modulesInGPU.layers[lowerModuleIndex2] + 6 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex2] == SDL::TwoS);
    const int layer3 = modulesInGPU.layers[lowerModuleIndex3] + 6 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex3] == SDL::TwoS);
    const int layer4 = modulesInGPU.layers[lowerModuleIndex4] + 6 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex4] == SDL::TwoS);
    const int layer5 = modulesInGPU.layers[lowerModuleIndex5] + 6 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex5] == SDL::TwoS);
    if(layer1 == 7 and layer2 == 8 and layer3 == 9 and layer4 == 10 and layer5 == 16)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.3053053053053053;
    }
    if(layer1 == 7 and layer2 == 8 and layer3 == 9 and layer4 == 15 and layer5 == 16)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.24524524524524527;
    }
    if(layer1 == 1 and layer2 == 7 and layer3 == 8 and layer4 == 9 and layer5 == 15)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.22522522522522526;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 7 and layer4 == 8 and layer5 == 9)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.15515515515515516;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 7 and layer4 == 8 and layer5 == 14)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.23523523523523526;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 7 and layer4 == 13 and layer5 == 14)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.15515515515515516;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 7 and layer5 == 13)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.2552552552552553;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 12 and layer5 == 13)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.16516516516516516;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4 and layer5 == 12)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.12512512512512514;
    }
    if(layer1 == 2 and layer2 == 7 and layer3 == 8 and layer4 == 9 and layer5 == 15)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.3253253253253253;
    }
    if(layer1 == 2 and layer2 == 7 and layer3 == 8 and layer4 == 14 and layer5 == 15)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.22522522522522526;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 7 and layer4 == 8 and layer5 == 14)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.2552552552552553;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 7 and layer4 == 13 and layer5 == 14)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.19519519519519518;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 12 and layer5 == 13)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.18518518518518517;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 5 and layer5 == 12)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.11511511511511512;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4 and layer5 == 5 and abs(eta) < 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.05505505505505505;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4 and layer5 == 5 and abs(eta) > 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.05505505505505505;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 5 and layer5 == 6 and abs(eta) < 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.05505505505505505;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 5 and layer5 == 6 and abs(eta) > 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.05505505505505505;
    }
    return true;
}
__device__ bool SDL::matchRadii_inner_v_bridge_bin2(struct SDL::modules& modulesInGPU, float& eta, uint16_t& lowerModuleIndex1, uint16_t& lowerModuleIndex2, uint16_t& lowerModuleIndex3, uint16_t& lowerModuleIndex4, uint16_t& lowerModuleIndex5, float& innerRadius, float& bridgeRadius)
{
    const int layer1 = modulesInGPU.layers[lowerModuleIndex1] + 6 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex1] == SDL::TwoS);
    const int layer2 = modulesInGPU.layers[lowerModuleIndex2] + 6 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex2] == SDL::TwoS);
    const int layer3 = modulesInGPU.layers[lowerModuleIndex3] + 6 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex3] == SDL::TwoS);
    const int layer4 = modulesInGPU.layers[lowerModuleIndex4] + 6 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex4] == SDL::TwoS);
    const int layer5 = modulesInGPU.layers[lowerModuleIndex5] + 6 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex5] == SDL::TwoS);
    if(layer1 == 7 and layer2 == 8 and layer3 == 9 and layer4 == 10 and layer5 == 16)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.2952952952952953;
    }
    if(layer1 == 7 and layer2 == 8 and layer3 == 9 and layer4 == 15 and layer5 == 16)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.24524524524524527;
    }
    if(layer1 == 1 and layer2 == 7 and layer3 == 8 and layer4 == 9 and layer5 == 15)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.24524524524524527;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 7 and layer4 == 8 and layer5 == 14)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.23523523523523526;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 7 and layer4 == 13 and layer5 == 14)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.16516516516516516;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 7 and layer5 == 8)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.06506506506506507;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 7 and layer5 == 13)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.24524524524524527;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 12 and layer5 == 13)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.14514514514514515;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4 and layer5 == 12)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.12512512512512514;
    }
    if(layer1 == 2 and layer2 == 7 and layer3 == 8 and layer4 == 9 and layer5 == 15)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.3253253253253253;
    }
    if(layer1 == 2 and layer2 == 7 and layer3 == 8 and layer4 == 14 and layer5 == 15)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.22522522522522526;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 7 and layer4 == 13 and layer5 == 14)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.23523523523523526;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 12 and layer5 == 13)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.16516516516516516;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 5 and layer5 == 12)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.13513513513513514;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4 and layer5 == 5 and abs(eta) < 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.07507507507507508;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4 and layer5 == 5 and abs(eta) > 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.06506506506506507;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 5 and layer5 == 6 and abs(eta) > 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.07507507507507508;
    }
    return true;
}
__device__ bool SDL::matchRadii_inner_v_bridge_bin3(struct SDL::modules& modulesInGPU, float& eta, uint16_t& lowerModuleIndex1, uint16_t& lowerModuleIndex2, uint16_t& lowerModuleIndex3, uint16_t& lowerModuleIndex4, uint16_t& lowerModuleIndex5, float& innerRadius, float& bridgeRadius)
{
    const int layer1 = modulesInGPU.layers[lowerModuleIndex1] + 6 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex1] == SDL::TwoS);
    const int layer2 = modulesInGPU.layers[lowerModuleIndex2] + 6 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex2] == SDL::TwoS);
    const int layer3 = modulesInGPU.layers[lowerModuleIndex3] + 6 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex3] == SDL::TwoS);
    const int layer4 = modulesInGPU.layers[lowerModuleIndex4] + 6 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex4] == SDL::TwoS);
    const int layer5 = modulesInGPU.layers[lowerModuleIndex5] + 6 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex5] == SDL::TwoS);
    if(layer1 == 7 and layer2 == 8 and layer3 == 9 and layer4 == 10 and layer5 == 16)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.3053053053053053;
    }
    if(layer1 == 7 and layer2 == 8 and layer3 == 9 and layer4 == 15 and layer5 == 16)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.2552552552552553;
    }
    if(layer1 == 1 and layer2 == 7 and layer3 == 8 and layer4 == 9 and layer5 == 15)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.2552552552552553;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 7 and layer4 == 8 and layer5 == 14)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.2552552552552553;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 7 and layer4 == 13 and layer5 == 14)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.19519519519519518;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 12 and layer5 == 13)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.24524524524524527;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4 and layer5 == 12)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.17517517517517517;
    }
    if(layer1 == 2 and layer2 == 7 and layer3 == 8 and layer4 == 9 and layer5 == 15)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.35535535535535534;
    }
    if(layer1 == 2 and layer2 == 7 and layer3 == 8 and layer4 == 14 and layer5 == 15)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.2552552552552553;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 7 and layer4 == 8 and layer5 == 14)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.21521521521521522;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 7 and layer4 == 13 and layer5 == 14)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.24524524524524527;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 12 and layer5 == 13)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.21521521521521522;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 5 and layer5 == 12)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.14514514514514515;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4 and layer5 == 5 and abs(eta) < 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.08508508508508508;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4 and layer5 == 5 and abs(eta) > 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.08508508508508508;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 5 and layer5 == 6 and abs(eta) < 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.08508508508508508;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 5 and layer5 == 6 and abs(eta) > 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.08508508508508508;
    }
    return true;
}
__device__ bool SDL::matchRadii_inner_v_bridge_bin4(struct SDL::modules& modulesInGPU, float& eta, uint16_t& lowerModuleIndex1, uint16_t& lowerModuleIndex2, uint16_t& lowerModuleIndex3, uint16_t& lowerModuleIndex4, uint16_t& lowerModuleIndex5, float& innerRadius, float& bridgeRadius)
{
    const int layer1 = modulesInGPU.layers[lowerModuleIndex1] + 6 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex1] == SDL::TwoS);
    const int layer2 = modulesInGPU.layers[lowerModuleIndex2] + 6 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex2] == SDL::TwoS);
    const int layer3 = modulesInGPU.layers[lowerModuleIndex3] + 6 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex3] == SDL::TwoS);
    const int layer4 = modulesInGPU.layers[lowerModuleIndex4] + 6 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex4] == SDL::TwoS);
    const int layer5 = modulesInGPU.layers[lowerModuleIndex5] + 6 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex5] == SDL::TwoS);
    if(layer1 == 7 and layer2 == 8 and layer3 == 9 and layer4 == 10 and layer5 == 11)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.2652652652652653;
    }
    if(layer1 == 7 and layer2 == 8 and layer3 == 9 and layer4 == 10 and layer5 == 16)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.35535535535535534;
    }
    if(layer1 == 7 and layer2 == 8 and layer3 == 9 and layer4 == 15 and layer5 == 16)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.2752752752752753;
    }
    if(layer1 == 1 and layer2 == 7 and layer3 == 8 and layer4 == 9 and layer5 == 15)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.2952952952952953;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 7 and layer4 == 8 and layer5 == 14)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.2752752752752753;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 7 and layer4 == 13 and layer5 == 14)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.2052052052052052;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 7 and layer5 == 13)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.2852852852852853;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 12 and layer5 == 13)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.18518518518518517;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4 and layer5 == 12)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.16516516516516516;
    }
    if(layer1 == 2 and layer2 == 7 and layer3 == 8 and layer4 == 9 and layer5 == 15)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.4454454454454454;
    }
    if(layer1 == 2 and layer2 == 7 and layer3 == 8 and layer4 == 14 and layer5 == 15)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.23523523523523526;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 7 and layer4 == 8 and layer5 == 14)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.23523523523523526;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 7 and layer4 == 13 and layer5 == 14)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.2552552552552553;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 12 and layer5 == 13)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.2052052052052052;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 5 and layer5 == 12)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.19519519519519518;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4 and layer5 == 5 and abs(eta) < 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.1051051051051051;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4 and layer5 == 5 and abs(eta) > 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.09509509509509509;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 5 and layer5 == 6 and abs(eta) < 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.12512512512512514;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 5 and layer5 == 6 and abs(eta) > 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.07507507507507508;
    }
    return true;
}
__device__ bool SDL::matchRadii_inner_v_bridge_bin5(struct SDL::modules& modulesInGPU, float& eta, uint16_t& lowerModuleIndex1, uint16_t& lowerModuleIndex2, uint16_t& lowerModuleIndex3, uint16_t& lowerModuleIndex4, uint16_t& lowerModuleIndex5, float& innerRadius, float& bridgeRadius)
{
    const int layer1 = modulesInGPU.layers[lowerModuleIndex1] + 6 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex1] == SDL::TwoS);
    const int layer2 = modulesInGPU.layers[lowerModuleIndex2] + 6 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex2] == SDL::TwoS);
    const int layer3 = modulesInGPU.layers[lowerModuleIndex3] + 6 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex3] == SDL::TwoS);
    const int layer4 = modulesInGPU.layers[lowerModuleIndex4] + 6 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex4] == SDL::TwoS);
    const int layer5 = modulesInGPU.layers[lowerModuleIndex5] + 6 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex5] == SDL::TwoS);
    if(layer1 == 7 and layer2 == 8 and layer3 == 9 and layer4 == 10 and layer5 == 11)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.4054054054054054;
    }
    if(layer1 == 7 and layer2 == 8 and layer3 == 9 and layer4 == 10 and layer5 == 16)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.6356356356356356;
    }
    if(layer1 == 7 and layer2 == 8 and layer3 == 9 and layer4 == 15 and layer5 == 16)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.34534534534534533;
    }
    if(layer1 == 1 and layer2 == 7 and layer3 == 8 and layer4 == 9 and layer5 == 15)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.3253253253253253;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 7 and layer4 == 8 and layer5 == 14)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.35535535535535534;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 7 and layer4 == 13 and layer5 == 14)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.16516516516516516;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 7 and layer5 == 13)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.35535535535535534;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 12 and layer5 == 13)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.2052052052052052;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4 and layer5 == 12)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.21521521521521522;
    }
    if(layer1 == 2 and layer2 == 7 and layer3 == 8 and layer4 == 9 and layer5 == 15)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.6456456456456456;
    }
    if(layer1 == 2 and layer2 == 7 and layer3 == 8 and layer4 == 14 and layer5 == 15)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.2752752752752753;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 7 and layer4 == 13 and layer5 == 14)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.6656656656656657;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 12 and layer5 == 13)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.23523523523523526;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 5 and layer5 == 12)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.16516516516516516;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4 and layer5 == 5 and abs(eta) < 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.09509509509509509;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4 and layer5 == 5 and abs(eta) > 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.09509509509509509;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 5 and layer5 == 6 and abs(eta) < 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.14514514514514515;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 4 and layer4 == 5 and layer5 == 6 and abs(eta) > 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.09509509509509509;
    }
    return true;
}
__device__ bool SDL::matchRadii_inner_v_bridge_bin6(struct SDL::modules& modulesInGPU, float& eta, uint16_t& lowerModuleIndex1, uint16_t& lowerModuleIndex2, uint16_t& lowerModuleIndex3, uint16_t& lowerModuleIndex4, uint16_t& lowerModuleIndex5, float& innerRadius, float& bridgeRadius)
{
    const int layer1 = modulesInGPU.layers[lowerModuleIndex1] + 6 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex1] == SDL::TwoS);
    const int layer2 = modulesInGPU.layers[lowerModuleIndex2] + 6 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex2] == SDL::TwoS);
    const int layer3 = modulesInGPU.layers[lowerModuleIndex3] + 6 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex3] == SDL::TwoS);
    const int layer4 = modulesInGPU.layers[lowerModuleIndex4] + 6 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex4] == SDL::TwoS);
    const int layer5 = modulesInGPU.layers[lowerModuleIndex5] + 6 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex5] == SDL::TwoS);
    if(layer1 == 7 and layer2 == 8 and layer3 == 9 and layer4 == 15 and layer5 == 16)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 1.9519519519519521;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 7 and layer4 == 13 and layer5 == 14)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.2502502502502503;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 7 and layer5 == 8)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.050050050050050046;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 12 and layer5 == 13)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.5505505505505506;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4 and layer5 == 12)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.45045045045045046;
    }
    if(layer1 == 2 and layer2 == 3 and layer3 == 7 and layer4 == 13 and layer5 == 14)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 1.2512512512512513;
    }
    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4 and layer5 == 5 and abs(eta) < 0.5)
    {
        return fabsf((1.0/innerRadius) - (1.0/bridgeRadius))/(1.0/innerRadius) < 0.35035035035035034;
    }
    return true;
}


__device__ float SDL::computeRadiusFromThreeAnchorHits(float x1, float y1, float x2, float y2, float x3, float y3, float& g, float& f)
{
    float radius = 0.f;

    //writing manual code for computing radius, which obviously sucks
    //TODO:Use fancy inbuilt libraries like cuBLAS or cuSOLVE for this!
    //(g,f) -> center
    //first anchor hit - (x1,y1), second anchor hit - (x2,y2), third anchor hit - (x3, y3)

    /*
    if((y1 - y3) * (x2 - x3) - (x1 - x3) * (y2 - y3) == 0)
    {
        return -1; //WTF man three collinear points!
    }
    */

    float denomInv = 1.0f/((y1 - y3) * (x2 - x3) - (x1 - x3) * (y2 - y3));

    float xy1sqr = x1 * x1 + y1 * y1;

    float xy2sqr = x2 * x2 + y2 * y2;

    float xy3sqr = x3 * x3 + y3 * y3;

    g = 0.5f * ((y3 - y2) * xy1sqr + (y1 - y3) * xy2sqr + (y2 - y1) * xy3sqr) * denomInv;

    f = 0.5f * ((x2 - x3) * xy1sqr + (x3 - x1) * xy2sqr + (x1 - x2) * xy3sqr) * denomInv;

    float c = ((x2 * y3 - x3 * y2) * xy1sqr + (x3 * y1 - x1 * y3) * xy2sqr + (x1 * y2 - x2 * y1) * xy3sqr) * denomInv;

    if(((y1 - y3) * (x2 - x3) - (x1 - x3) * (y2 - y3) == 0) || (g * g + f * f - c < 0))
    {
        printf("three collinear points or FATAL! r^2 < 0!\n");
	radius = -1.f;
    }
    else
      radius = sqrtf(g * g  + f * f - c);

    return radius;
}

__device__ bool SDL::T5HasCommonMiniDoublet(struct SDL::triplets& tripletsInGPU, struct SDL::segments& segmentsInGPU, unsigned int innerTripletIndex, unsigned int outerTripletIndex)
{
    unsigned int innerOuterSegmentIndex = tripletsInGPU.segmentIndices[2 * innerTripletIndex + 1];
    unsigned int outerInnerSegmentIndex = tripletsInGPU.segmentIndices[2 * outerTripletIndex];
    unsigned int innerOuterOuterMiniDoubletIndex = segmentsInGPU.mdIndices[2 * innerOuterSegmentIndex + 1]; //inner triplet outer segment outer MD index
    unsigned int outerInnerInnerMiniDoubletIndex = segmentsInGPU.mdIndices[2 * outerInnerSegmentIndex]; //outer triplet inner segmnet inner MD index


    return (innerOuterOuterMiniDoubletIndex == outerInnerInnerMiniDoubletIndex);
}


__device__ void SDL::computeSigmasForRegression(SDL::modules& modulesInGPU, const uint16_t* lowerModuleIndices, float* sigmas)
{
    ModuleType moduleType;
    short moduleSubdet, moduleSide;

    for(size_t i=0; i <5; i++)
    {
        moduleType = modulesInGPU.moduleType[lowerModuleIndices[i]];
        moduleSubdet = modulesInGPU.subdets[lowerModuleIndices[i]];
        moduleSide = modulesInGPU.sides[lowerModuleIndices[i]];
        float& drdz = modulesInGPU.drdzs[lowerModuleIndices[i]];
        
        //category 1 - barrel PS flat
        if(moduleSubdet == Barrel and moduleType == PS and moduleSide == Center)
        {
            sigmas[i] = 1.f;
        }
        //category 2 - barrel 2S flat
        else if(moduleSubdet == Barrel and moduleType == TwoS)
        {
            sigmas[i] = 1.f;
        }
        //category 3 - barrel PS tilted
        else if(moduleSubdet == Barrel and moduleType == PS and moduleSide != Center)
        {
            sigmas[i] = (0.075/0.0006) * drdz/sqrt(1+drdz*drdz);
        }
        //category 4 - endcap PS
        else if(moduleSubdet == Endcap and moduleType == PS)
        {
            sigmas[i] = 0.075/0.0006;
        }
        //category 5 - endcap 2S
        else if(moduleSubdet == Endcap and moduleType == TwoS)
        {
            sigmas[i] = 2.5/0.0006;
        }
    }
}


/*__device__ void SDL::computeSigmasForRegression(SDL::modules& modulesInGPU, const uint16_t* lowerModuleIndices, float* delta1, float* delta2, float* slopes, bool* isFlat, int nPoints, bool anchorHits) 
{
   bool anchorHits required to deal with a weird edge case wherein 
     the hits ultimately used in the regression are anchor hits, but the
     lower modules need not all be Pixel Modules (in case of PS). Similarly,
     when we compute the chi squared for the non-anchor hits, the "partner module"
     need not always be a PS strip module, but all non-anchor hits sit on strip 
     modules.
    
    ModuleType moduleType;
    short moduleSubdet, moduleSide;
    float inv1 = 0.01f/0.009f;
    float inv2 = 0.15f/0.009f;
    float inv3 = 2.4f/0.009f;
    for(size_t i=0; i<nPoints; i++)
    {
        moduleType = modulesInGPU.moduleType[lowerModuleIndices[i]];
        moduleSubdet = modulesInGPU.subdets[lowerModuleIndices[i]];
        moduleSide = modulesInGPU.sides[lowerModuleIndices[i]];
        float& drdz = modulesInGPU.drdzs[lowerModuleIndices[i]];
        slopes[i] = modulesInGPU.slopes[lowerModuleIndices[i]]; 
        //category 1 - barrel PS flat
        if(moduleSubdet == Barrel and moduleType == PS and moduleSide == Center)
        {
            delta1[i] = inv1;//1.1111f;//0.01;
            delta2[i] = inv1;//1.1111f;//0.01;
            slopes[i] = -999.f;
            isFlat[i] = true;
        }

        //category 2 - barrel 2S
        else if(moduleSubdet == Barrel and moduleType == TwoS)
        {
            delta1[i] = 1.f;//0.009;
            delta2[i] = 1.f;//0.009;
            slopes[i] = -999.f;
            isFlat[i] = true;
        }

        //category 3 - barrel PS tilted
        else if(moduleSubdet == Barrel and moduleType == PS and moduleSide != Center)
        {

            //delta1[i] = 0.01;
            delta1[i] = inv1;//1.1111f;//0.01;
            isFlat[i] = false;

            if(anchorHits)
            {
                delta2[i] = (inv2 * drdz/sqrtf(1 + drdz * drdz));
            }
            else
            {
                delta2[i] = (inv3 * drdz/sqrtf(1 + drdz * drdz));
            }
        }
        //category 4 - endcap PS
        else if(moduleSubdet == Endcap and moduleType == PS)
        {
            delta1[i] = inv1;//1.1111f;//0.01;
            isFlat[i] = false;

            //despite the type of the module layer of the lower module index,
            all anchor hits are on the pixel side and all non-anchor hits are
            on the strip side!
            if(anchorHits)
            {
                delta2[i] = inv2;//16.6666f;//0.15f;
            }
            else
            {
                delta2[i] = inv3;//266.666f;//2.4f;
            }
        }

        //category 5 - endcap 2S
        else if(moduleSubdet == Endcap and moduleType == TwoS)
        {
            delta1[i] = 1.f;//0.009;
            delta2[i] = 500.f*inv1;//555.5555f;//5.f;
            isFlat[i] = false;
        }
        else
        {
            printf("ERROR!!!!! I SHOULDN'T BE HERE!!!! subdet = %d, type = %d, side = %d\n", moduleSubdet, moduleType, moduleSide);
        }
    }
}*/

__device__ float SDL::computeRadiusUsingRegression(int nPoints, float* xs, float* ys, float& g, float& f, float* sigmas, float& chiSquared)
{
    float radius = 0.f;

    //some extra variables
    //the two variables will be caled x1 and x2, and y (which is x^2 + y^2)

    float sigmaX1Squared = 0.f;
    float sigmaX2Squared = 0.f;
    float sigmaX1X2 = 0.f; 
    float sigmaX1y = 0.f; 
    float sigmaX2y = 0.f;
    float sigmaY = 0.f;
    float sigmaX1 = 0.f;
    float sigmaX2 = 0.f;
    float sigmaOne = 0.f;

    float xPrime, yPrime, absArctanSlope, angleM;
    for(size_t i = 0; i < nPoints; i++)
    {
        //computing sigmas is a very tricky affair
        //if the module is tilted or endcap, we need to use the slopes properly!

/*        absArctanSlope = ((slopes[i] != 123456789) ? fabs(atanf(slopes[i])) : 0.5f*float(M_PI)); // Since C++ can't represent infinity, SDL_INF = 123456789 was used to represent infinity in the data table

        if(xs[i] > 0 and ys[i] > 0)
        {
            angleM = 0.5f*float(M_PI) - absArctanSlope;
        }
        else if(xs[i] < 0 and ys[i] > 0)
        {
            angleM = absArctanSlope + 0.5f*float(M_PI);
        }
        else if(xs[i] < 0 and ys[i] < 0)
        {
            angleM = -(absArctanSlope + 0.5f*float(M_PI));
        }
        else if(xs[i] > 0 and ys[i] < 0)
        {
            angleM = -(0.5f*float(M_PI) - absArctanSlope);
        }

        if(not isFlat[i])
        {
            xPrime = xs[i] * cosf(angleM) + ys[i] * sinf(angleM);
            yPrime = ys[i] * cosf(angleM) - xs[i] * sinf(angleM);
        }
        else
        {
            xPrime = xs[i];
            yPrime = ys[i];
        }
        float sigma = 1;//2 * sqrtf((xPrime * delta1[i]) * (xPrime * delta1[i]) + (yPrime * delta2[i]) * (yPrime * delta2[i]));*/
        float sigma = 1.f;

        sigmaX1Squared += (xs[i] * xs[i])/(sigma * sigma);
        sigmaX2Squared += (ys[i] * ys[i])/(sigma * sigma);
        sigmaX1X2 += (xs[i] * ys[i])/(sigma * sigma);
        sigmaX1y += (xs[i] * (xs[i] * xs[i] + ys[i] * ys[i]))/(sigma * sigma);
        sigmaX2y += (ys[i] * (xs[i] * xs[i] + ys[i] * ys[i]))/(sigma * sigma);
        sigmaY += (xs[i] * xs[i] + ys[i] * ys[i])/(sigma * sigma);
        sigmaX1 += xs[i]/(sigma * sigma);
        sigmaX2 += ys[i]/(sigma * sigma);
        sigmaOne += 1.0f/(sigma * sigma);
    }
    float denominator = (sigmaX1X2 - sigmaX1 * sigmaX2) * (sigmaX1X2 - sigmaX1 * sigmaX2) - (sigmaX1Squared - sigmaX1 * sigmaX1) * (sigmaX2Squared - sigmaX2 * sigmaX2);

    float twoG = ((sigmaX2y - sigmaX2 * sigmaY) * (sigmaX1X2 - sigmaX1 * sigmaX2) - (sigmaX1y - sigmaX1 * sigmaY) * (sigmaX2Squared - sigmaX2 * sigmaX2)) / denominator;
    float twoF = ((sigmaX1y - sigmaX1 * sigmaY) * (sigmaX1X2 - sigmaX1 * sigmaX2) - (sigmaX2y - sigmaX2 * sigmaY) * (sigmaX1Squared - sigmaX1 * sigmaX1)) / denominator;

    float c = -(sigmaY - twoG * sigmaX1 - twoF * sigmaX2)/sigmaOne;
    g = 0.5f*twoG;
    f = 0.5f*twoF;
    if(g * g + f * f - c < 0)
    {
        printf("FATAL! r^2 < 0!\n");
        return -1;
    }
    
    radius = sqrtf(g * g  + f * f - c);
    //compute chi squared
    chiSquared = 0.f;
    for(size_t i = 0; i < nPoints; i++)
    {
       chiSquared +=  (xs[i] * xs[i] + ys[i] * ys[i] - twoG * xs[i] - twoF * ys[i] + c) * (xs[i] * xs[i] + ys[i] * ys[i] - twoG * xs[i] - twoF * ys[i] + c) / (sigmas[i] * sigmas[i]);
    }
    return radius;
}

__device__ float SDL::computeChiSquared(int nPoints, float* xs, float* ys, float* sigmas, float g, float f, float radius)
{
    // given values of (g, f, radius) and a set of points (and its uncertainties)
    //compute chi squared
    float c = g*g + f*f - radius*radius;
    float chiSquared = 0.f;
    float sigma;
    for(size_t i = 0; i < nPoints; i++)
    {
/*        absArctanSlope = ((slopes[i] != 123456789) ? fabs(atanf(slopes[i])) : 0.5f*float(M_PI)); // Since C++ can't represent infinity, SDL_INF = 123456789 was used to represent infinity in the data table
        if(xs[i] > 0 and ys[i] > 0)
        {
            angleM = 0.5f*float(M_PI) - absArctanSlope;
        }
        else if(xs[i] < 0 and ys[i] > 0)
        {
            angleM = absArctanSlope + 0.5f*float(M_PI);
        }
        else if(xs[i] < 0 and ys[i] < 0)
        {
            angleM = -(absArctanSlope + 0.5f*float(M_PI));
        }
        else if(xs[i] > 0 and ys[i] < 0)
        {
            angleM = -(0.5f*float(M_PI) - absArctanSlope);
        }

        if(not isFlat[i])
        {
            xPrime = xs[i] * cosf(angleM) + ys[i] * sinf(angleM);
            yPrime = ys[i] * cosf(angleM) - xs[i] * sinf(angleM);
        }
        else
        {
            xPrime = xs[i];
            yPrime = ys[i];
        }
        sigma = 1;//2 * sqrtf((xPrime * delta1[i]) * (xPrime * delta1[i]) + (yPrime * delta2[i]) * (yPrime * delta2[i]));*/
        chiSquared +=  (xs[i] * xs[i] + ys[i] * ys[i] - 2 * g * xs[i] - 2 * f * ys[i] + c) * (xs[i] * xs[i] + ys[i] * ys[i] - 2 * g * xs[i] - 2 * f * ys[i] + c) / (sigmas[i] * sigmas[i]);
    }
    return chiSquared; 
}

__global__ void SDL::createQuintupletsInGPUv2(struct SDL::modules& modulesInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::triplets& tripletsInGPU, struct SDL::quintuplets& quintupletsInGPU, struct SDL::objectRanges& rangesInGPU, uint16_t nEligibleT5Modules)
{
    int gidy = blockIdx.y * blockDim.y + threadIdx.y;
    int npy = gridDim.y * blockDim.y;
    int gidx = blockIdx.x * blockDim.x + threadIdx.x;
    int npx = gridDim.x * blockDim.x;
    int gidz = blockIdx.z * blockDim.z + threadIdx.z;
    int npz = gridDim.z * blockDim.z;

    for (int iter=gidz; iter < nEligibleT5Modules; iter+=npz){
      uint16_t lowerModule1 = rangesInGPU.indicesOfEligibleT5Modules[iter];


      unsigned int nInnerTriplets = tripletsInGPU.nTriplets[lowerModule1];
      for( unsigned int innerTripletArrayIndex =gidy; innerTripletArrayIndex < nInnerTriplets; innerTripletArrayIndex+=npy){

      unsigned int innerTripletIndex = rangesInGPU.tripletModuleIndices[lowerModule1] + innerTripletArrayIndex;
      uint16_t lowerModule2 = tripletsInGPU.lowerModuleIndices[3 * innerTripletIndex + 1];
      uint16_t lowerModule3 = tripletsInGPU.lowerModuleIndices[3 * innerTripletIndex + 2];
      unsigned int nOuterTriplets = tripletsInGPU.nTriplets[lowerModule3];
        for (int outerTripletArrayIndex=gidx; outerTripletArrayIndex < nOuterTriplets; outerTripletArrayIndex+=npx)
        {
            unsigned int outerTripletIndex = rangesInGPU.tripletModuleIndices[lowerModule3] + outerTripletArrayIndex;
            uint16_t lowerModule4 = tripletsInGPU.lowerModuleIndices[3 * outerTripletIndex + 1];
            uint16_t lowerModule5 = tripletsInGPU.lowerModuleIndices[3 * outerTripletIndex + 2];

            float innerRadius, outerRadius, bridgeRadius, innerG, innerF, rzChiSquared, chiSquared, nonAnchorChiSquared; //required for making distributions
            bool tightCutFlag;
            bool success = runQuintupletDefaultAlgo(modulesInGPU, mdsInGPU, segmentsInGPU, tripletsInGPU, lowerModule1, lowerModule2, lowerModule3, lowerModule4, lowerModule5, innerTripletIndex, outerTripletIndex, innerG, innerF, innerRadius, outerRadius,  bridgeRadius, rzChiSquared, chiSquared, nonAnchorChiSquared, tightCutFlag);

            if(success)
            {
                short layer2_adjustment;
                int layer = modulesInGPU.layers[lowerModule1];
                if(layer == 1)
                {
                    layer2_adjustment = 1;
                } //get upper segment to be in second layer
                else if(layer == 2)
                {
                    layer2_adjustment = 0;
                } // get lower segment to be in second layer
                else
                {
                    return;
                } // ignore anything else TODO: move this to start, before object is made (faster)
                unsigned int totOccupancyQuintuplets = atomicAdd(&quintupletsInGPU.totOccupancyQuintuplets[lowerModule1], 1);
                if(totOccupancyQuintuplets >= (rangesInGPU.quintupletModuleIndices[lowerModule1 + 1] - rangesInGPU.quintupletModuleIndices[lowerModule1]))
                {
#ifdef Warnings
                    printf("Quintuplet excess alert! Module index = %d\n", lowerModule1);
#endif
                }
                else
                {
                    unsigned int quintupletModuleIndex = atomicAdd(&quintupletsInGPU.nQuintuplets[lowerModule1], 1);
                    //this if statement should never get executed!
                    if(rangesInGPU.quintupletModuleIndices[lowerModule1] == -1)
                    {
                        printf("Quintuplets : no memory for module at module index = %d\n", lowerModule1);
                    }
                    else
                    {
                        unsigned int quintupletIndex = rangesInGPU.quintupletModuleIndices[lowerModule1] +  quintupletModuleIndex;
                        float phi = mdsInGPU.anchorPhi[segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*innerTripletIndex+layer2_adjustment]]];
                        float eta = mdsInGPU.anchorEta[segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*innerTripletIndex+layer2_adjustment]]];
                        float pt = (innerRadius+outerRadius)*3.8f*1.602f/(2*100*5.39f);
                        float scores = chiSquared + nonAnchorChiSquared;
                        addQuintupletToMemory(tripletsInGPU, quintupletsInGPU, innerTripletIndex, outerTripletIndex, lowerModule1, lowerModule2, lowerModule3, lowerModule4, lowerModule5, innerRadius, bridgeRadius, outerRadius, innerG, innerF, rzChiSquared, chiSquared, nonAnchorChiSquared, pt,eta,phi,scores,layer,quintupletIndex, tightCutFlag);

                        tripletsInGPU.partOfT5[quintupletsInGPU.tripletIndices[2 * quintupletIndex]] = true;
                        tripletsInGPU.partOfT5[quintupletsInGPU.tripletIndices[2 * quintupletIndex + 1]] = true;
                    }
                }
            }
        }
      }
    }
}

__device__ bool SDL::runQuintupletDefaultAlgoBBBB(struct SDL::modules& modulesInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, uint16_t& innerInnerLowerModuleIndex, uint16_t& innerOuterLowerModuleIndex, uint16_t& outerInnerLowerModuleIndex, uint16_t& outerOuterLowerModuleIndex, unsigned int& innerSegmentIndex, unsigned int& outerSegmentIndex, unsigned int& firstMDIndex, unsigned int& secondMDIndex, unsigned int& thirdMDIndex,
        unsigned int& fourthMDIndex, float& zOut, float& rtOut, float& deltaPhiPos, float& dPhi, float& betaIn, float&
        betaOut, float& pt_beta, float& zLo, float& zHi, float& zLoPointed, float& zHiPointed, float& sdlCut, float& betaInCut, float& betaOutCut, float& deltaBetaCut)
{
    bool pass = true;

    bool isPS_InLo = (modulesInGPU.moduleType[innerInnerLowerModuleIndex] == SDL::PS);
    bool isPS_OutLo = (modulesInGPU.moduleType[outerInnerLowerModuleIndex] == SDL::PS);

    float rt_InLo = mdsInGPU.anchorRt[firstMDIndex];
    float rt_InOut = mdsInGPU.anchorRt[secondMDIndex];
    float rt_OutLo = mdsInGPU.anchorRt[thirdMDIndex];

    float z_InLo = mdsInGPU.anchorZ[firstMDIndex];
    float z_InOut = mdsInGPU.anchorZ[secondMDIndex];
    float z_OutLo = mdsInGPU.anchorZ[thirdMDIndex];

    float alpha1GeV_OutLo = asinf(fminf(rt_OutLo * SDL::k2Rinv1GeVf / SDL::ptCut, SDL::sinAlphaMax));

    float rtRatio_OutLoInLo = rt_OutLo / rt_InLo; // Outer segment beginning rt divided by inner segment beginning rt;
    float dzDrtScale = tanf(alpha1GeV_OutLo) / alpha1GeV_OutLo; // The track can bend in r-z plane slightly
    float zpitch_InLo = (isPS_InLo ? SDL::pixelPSZpitch : SDL::strip2SZpitch);
    float zpitch_OutLo = (isPS_OutLo ? SDL::pixelPSZpitch : SDL::strip2SZpitch);

    zHi = z_InLo + (z_InLo + SDL::deltaZLum) * (rtRatio_OutLoInLo - 1.f) * (z_InLo < 0.f ? 1.f : dzDrtScale) + (zpitch_InLo + zpitch_OutLo);
    zLo = z_InLo + (z_InLo - SDL::deltaZLum) * (rtRatio_OutLoInLo - 1.f) * (z_InLo > 0.f ? 1.f : dzDrtScale) - (zpitch_InLo + zpitch_OutLo);


    //Cut 1 - z compatibility
    zOut = z_OutLo;
    rtOut = rt_OutLo;
    pass = pass and ((z_OutLo >= zLo) & (z_OutLo <= zHi));
    if(not pass) return pass;

    float drt_OutLo_InLo = (rt_OutLo - rt_InLo);
    float r3_InLo = sqrtf(z_InLo * z_InLo + rt_InLo * rt_InLo);
    float drt_InSeg = rt_InOut - rt_InLo;
    float dz_InSeg = z_InOut - z_InLo;
    float dr3_InSeg = sqrtf(rt_InOut * rt_InOut + z_InOut * z_InOut) - sqrtf(rt_InLo * rt_InLo + z_InLo * z_InLo);

    float coshEta = dr3_InSeg/drt_InSeg;
    float dzErr = (zpitch_InLo + zpitch_OutLo) * (zpitch_InLo + zpitch_OutLo) * 2.f;

    float sdlThetaMulsF = 0.015f * sqrtf(0.1f + 0.2f * (rt_OutLo - rt_InLo) / 50.f) * sqrtf(r3_InLo / rt_InLo);
    float sdlMuls = sdlThetaMulsF * 3.f / SDL::ptCut * 4.f; // will need a better guess than x4?
    dzErr += sdlMuls * sdlMuls * drt_OutLo_InLo * drt_OutLo_InLo / 3.f * coshEta * coshEta; //sloppy
    dzErr = sqrtf(dzErr);

    // Constructing upper and lower bound
    const float dzMean = dz_InSeg / drt_InSeg * drt_OutLo_InLo;
    const float zWindow = dzErr / drt_InSeg * drt_OutLo_InLo + (zpitch_InLo + zpitch_OutLo); //FIXME for SDL::ptCut lower than ~0.8 need to add curv path correction
    zLoPointed = z_InLo + dzMean * (z_InLo > 0.f ? 1.f : dzDrtScale) - zWindow;
    zHiPointed = z_InLo + dzMean * (z_InLo < 0.f ? 1.f : dzDrtScale) + zWindow;

    // Cut #2: Pointed Z (Inner segment two MD points to outer segment inner MD)
    pass =  pass and ((z_OutLo >= zLoPointed) & (z_OutLo <= zHiPointed));
    if(not pass) return pass;

    float sdlPVoff = 0.1f/rt_OutLo;
    sdlCut = alpha1GeV_OutLo + sqrtf(sdlMuls * sdlMuls + sdlPVoff * sdlPVoff);

    deltaPhiPos = SDL::deltaPhi(mdsInGPU.anchorX[secondMDIndex], mdsInGPU.anchorY[secondMDIndex], mdsInGPU.anchorX[fourthMDIndex], mdsInGPU.anchorY[fourthMDIndex]);
    // Cut #3: FIXME:deltaPhiPos can be tighter
    pass = pass and (fabsf(deltaPhiPos) <= sdlCut);
    if(not pass) return pass;

    float midPointX = 0.5f*(mdsInGPU.anchorX[firstMDIndex] + mdsInGPU.anchorX[thirdMDIndex]);
    float midPointY = 0.5f* (mdsInGPU.anchorY[firstMDIndex] + mdsInGPU.anchorY[thirdMDIndex]);
    float midPointZ = 0.5f*(mdsInGPU.anchorZ[firstMDIndex] + mdsInGPU.anchorZ[thirdMDIndex]);
    float diffX = mdsInGPU.anchorX[thirdMDIndex] - mdsInGPU.anchorX[firstMDIndex];
    float diffY = mdsInGPU.anchorY[thirdMDIndex] - mdsInGPU.anchorY[firstMDIndex];
    float diffZ = mdsInGPU.anchorZ[thirdMDIndex] - mdsInGPU.anchorZ[firstMDIndex];

    dPhi = SDL::deltaPhi(midPointX, midPointY, diffX, diffY);

    // Cut #4: deltaPhiChange
    pass = pass and (fabsf(dPhi) <= sdlCut);
    //lots of array accesses below. Cut here!
    if(not pass) return pass;

    // First obtaining the raw betaIn and betaOut values without any correction and just purely based on the mini-doublet hit positions

    float alpha_InLo  = __H2F(segmentsInGPU.dPhiChanges[innerSegmentIndex]);
    float alpha_OutLo = __H2F(segmentsInGPU.dPhiChanges[outerSegmentIndex]);

    bool isEC_lastLayer = modulesInGPU.subdets[outerOuterLowerModuleIndex] == SDL::Endcap and modulesInGPU.moduleType[outerOuterLowerModuleIndex] == SDL::TwoS;

    float alpha_OutUp,alpha_OutUp_highEdge,alpha_OutUp_lowEdge;

    alpha_OutUp = SDL::deltaPhi(mdsInGPU.anchorX[fourthMDIndex], mdsInGPU.anchorY[fourthMDIndex], mdsInGPU.anchorX[fourthMDIndex] - mdsInGPU.anchorX[thirdMDIndex], mdsInGPU.anchorY[fourthMDIndex] - mdsInGPU.anchorY[thirdMDIndex]);

    alpha_OutUp_highEdge = alpha_OutUp;
    alpha_OutUp_lowEdge = alpha_OutUp;

    float tl_axis_x = mdsInGPU.anchorX[fourthMDIndex] - mdsInGPU.anchorX[firstMDIndex];
    float tl_axis_y = mdsInGPU.anchorY[fourthMDIndex] - mdsInGPU.anchorY[firstMDIndex];
    float tl_axis_z = mdsInGPU.anchorZ[fourthMDIndex] - mdsInGPU.anchorZ[firstMDIndex];
    float tl_axis_highEdge_x = tl_axis_x;
    float tl_axis_highEdge_y = tl_axis_y;
    float tl_axis_lowEdge_x = tl_axis_x;
    float tl_axis_lowEdge_y = tl_axis_y;

    betaIn = alpha_InLo - SDL::deltaPhi(mdsInGPU.anchorX[firstMDIndex], mdsInGPU.anchorY[firstMDIndex], tl_axis_x, tl_axis_y);

    float betaInRHmin = betaIn;
    float betaInRHmax = betaIn;
    betaOut = -alpha_OutUp + SDL::deltaPhi(mdsInGPU.anchorX[fourthMDIndex], mdsInGPU.anchorY[fourthMDIndex], tl_axis_x, tl_axis_y);

    float betaOutRHmin = betaOut;
    float betaOutRHmax = betaOut;

    if(isEC_lastLayer)
    {
        alpha_OutUp_highEdge = SDL::deltaPhi(mdsInGPU.anchorHighEdgeX[fourthMDIndex], mdsInGPU.anchorHighEdgeY[fourthMDIndex], mdsInGPU.anchorHighEdgeX[fourthMDIndex] - mdsInGPU.anchorX[thirdMDIndex], mdsInGPU.anchorHighEdgeY[fourthMDIndex] - mdsInGPU.anchorY[thirdMDIndex]);
        alpha_OutUp_lowEdge = SDL::deltaPhi(mdsInGPU.anchorLowEdgeX[fourthMDIndex], mdsInGPU.anchorLowEdgeY[fourthMDIndex], mdsInGPU.anchorLowEdgeX[fourthMDIndex] - mdsInGPU.anchorX[thirdMDIndex], mdsInGPU.anchorLowEdgeY[fourthMDIndex] - mdsInGPU.anchorY[thirdMDIndex]);

        tl_axis_highEdge_x = mdsInGPU.anchorHighEdgeX[fourthMDIndex] - mdsInGPU.anchorX[firstMDIndex];
        tl_axis_highEdge_y = mdsInGPU.anchorHighEdgeY[fourthMDIndex] - mdsInGPU.anchorY[firstMDIndex];
        tl_axis_lowEdge_x = mdsInGPU.anchorLowEdgeX[fourthMDIndex] - mdsInGPU.anchorX[firstMDIndex];
        tl_axis_lowEdge_y = mdsInGPU.anchorLowEdgeY[fourthMDIndex] - mdsInGPU.anchorY[firstMDIndex];


        betaOutRHmin = -alpha_OutUp_highEdge + SDL::deltaPhi(mdsInGPU.anchorHighEdgeX[fourthMDIndex], mdsInGPU.anchorHighEdgeY[fourthMDIndex], tl_axis_highEdge_x, tl_axis_highEdge_y);
        betaOutRHmax = -alpha_OutUp_lowEdge + SDL::deltaPhi(mdsInGPU.anchorLowEdgeX[fourthMDIndex], mdsInGPU.anchorLowEdgeY[fourthMDIndex], tl_axis_lowEdge_x, tl_axis_lowEdge_y);
    }

    //beta computation
    float drt_tl_axis = sqrtf(tl_axis_x * tl_axis_x + tl_axis_y * tl_axis_y);
    float drt_tl_lowEdge = sqrtf(tl_axis_lowEdge_x * tl_axis_lowEdge_x + tl_axis_lowEdge_y * tl_axis_lowEdge_y);
    float drt_tl_highEdge = sqrtf(tl_axis_highEdge_x * tl_axis_highEdge_x + tl_axis_highEdge_y * tl_axis_highEdge_y);

    float corrF = 1.f;
    //innerOuterAnchor - innerInnerAnchor
    const float rt_InSeg = sqrtf((mdsInGPU.anchorX[secondMDIndex] - mdsInGPU.anchorX[firstMDIndex]) * (mdsInGPU.anchorX[secondMDIndex] - mdsInGPU.anchorX[firstMDIndex]) + (mdsInGPU.anchorY[secondMDIndex] - mdsInGPU.anchorY[firstMDIndex]) * (mdsInGPU.anchorY[secondMDIndex] - mdsInGPU.anchorY[firstMDIndex]));
    betaInCut = asinf(fminf((-rt_InSeg * corrF + drt_tl_axis) * SDL::k2Rinv1GeVf / SDL::ptCut, SDL::sinAlphaMax)) + (0.02f / drt_InSeg);

    //Cut #5: first beta cut
    pass = pass and (fabsf(betaInRHmin) < betaInCut);
    if(not pass) return pass;

    float betaAv = 0.5f * (betaIn + betaOut);
    pt_beta = drt_tl_axis * SDL::k2Rinv1GeVf/sinf(betaAv);
    int lIn = 5;
    int lOut = isEC_lastLayer ? 11 : 5;
    float sdOut_dr = sqrtf((mdsInGPU.anchorX[fourthMDIndex] - mdsInGPU.anchorX[thirdMDIndex]) * (mdsInGPU.anchorX[fourthMDIndex] - mdsInGPU.anchorX[thirdMDIndex]) + (mdsInGPU.anchorY[fourthMDIndex] - mdsInGPU.anchorY[thirdMDIndex]) * (mdsInGPU.anchorY[fourthMDIndex] - mdsInGPU.anchorY[thirdMDIndex]));
    float sdOut_d = mdsInGPU.anchorRt[fourthMDIndex] - mdsInGPU.anchorRt[thirdMDIndex];

    const float diffDr = fabsf(rt_InSeg - sdOut_dr) / fabsf(rt_InSeg + sdOut_dr);

    SDL::runDeltaBetaIterationsT5(betaIn, betaOut, betaAv, pt_beta, rt_InSeg, sdOut_dr, drt_tl_axis, lIn);

    const float betaInMMSF = (fabsf(betaInRHmin + betaInRHmax) > 0) ? (2.f * betaIn / fabsf(betaInRHmin + betaInRHmax)) : 0.f; //mean value of min,max is the old betaIn
    const float betaOutMMSF = (fabsf(betaOutRHmin + betaOutRHmax) > 0) ? (2.f * betaOut / fabsf(betaOutRHmin + betaOutRHmax)) : 0.f;
    betaInRHmin *= betaInMMSF;
    betaInRHmax *= betaInMMSF;
    betaOutRHmin *= betaOutMMSF;
    betaOutRHmax *= betaOutMMSF;

    const float dBetaMuls = sdlThetaMulsF * 4.f / fminf(fabsf(pt_beta), SDL::pt_betaMax); //need to confimm the range-out value of 7 GeV


    const float alphaInAbsReg = fmaxf(fabsf(alpha_InLo), asinf(fminf(rt_InLo * SDL::k2Rinv1GeVf / 3.0f, SDL::sinAlphaMax)));
    const float alphaOutAbsReg = fmaxf(fabs(alpha_OutLo), asinf(fminf(rt_OutLo * SDL::k2Rinv1GeVf / 3.0f, SDL::sinAlphaMax)));
    const float dBetaInLum = lIn < 11 ? 0.0f : fabsf(alphaInAbsReg*SDL::deltaZLum / z_InLo);
    const float dBetaOutLum = lOut < 11 ? 0.0f : fabsf(alphaOutAbsReg*SDL::deltaZLum / z_OutLo);
    const float dBetaLum2 = (dBetaInLum + dBetaOutLum) * (dBetaInLum + dBetaOutLum);
    const float sinDPhi = sinf(dPhi);

    const float dBetaRIn2 = 0; // TODO-RH
    // const float dBetaROut2 = 0; // TODO-RH
    float dBetaROut = 0;
    if(isEC_lastLayer)
    {
        dBetaROut = (sqrtf(mdsInGPU.anchorHighEdgeX[fourthMDIndex] * mdsInGPU.anchorHighEdgeX[fourthMDIndex] + mdsInGPU.anchorHighEdgeY[fourthMDIndex] * mdsInGPU.anchorHighEdgeY[fourthMDIndex]) - sqrtf(mdsInGPU.anchorLowEdgeX[fourthMDIndex] * mdsInGPU.anchorLowEdgeX[fourthMDIndex] + mdsInGPU.anchorLowEdgeY[fourthMDIndex] * mdsInGPU.anchorLowEdgeY[fourthMDIndex])) * sinDPhi / drt_tl_axis;
    }

    const float dBetaROut2 =  dBetaROut * dBetaROut;

    betaOutCut = asinf(fminf(drt_tl_axis*SDL::k2Rinv1GeVf / SDL::ptCut, SDL::sinAlphaMax)) //FIXME: need faster version
        + (0.02f / sdOut_d) + sqrtf(dBetaLum2 + dBetaMuls*dBetaMuls);

    //Cut #6: The real beta cut
    pass =  pass and ((fabsf(betaOut) < betaOutCut));
    if(not pass) return pass;

    float pt_betaIn = drt_tl_axis * SDL::k2Rinv1GeVf/sinf(betaIn);
    float pt_betaOut = drt_tl_axis * SDL::k2Rinv1GeVf / sinf(betaOut);
    float dBetaRes = 0.02f/fminf(sdOut_d,drt_InSeg);
    float dBetaCut2 = (dBetaRes*dBetaRes * 2.0f + dBetaMuls * dBetaMuls + dBetaLum2 + dBetaRIn2 + dBetaROut2
            + 0.25f * (fabsf(betaInRHmin - betaInRHmax) + fabsf(betaOutRHmin - betaOutRHmax)) * (fabsf(betaInRHmin - betaInRHmax) + fabsf(betaOutRHmin - betaOutRHmax)));

    float dBeta = betaIn - betaOut;
    deltaBetaCut = sqrtf(dBetaCut2);
    pass = pass and (dBeta * dBeta <= dBetaCut2);

    return pass;
}

__device__ bool SDL::runQuintupletDefaultAlgoBBEE(struct SDL::modules& modulesInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, uint16_t& innerInnerLowerModuleIndex, uint16_t& innerOuterLowerModuleIndex, uint16_t& outerInnerLowerModuleIndex, uint16_t& outerOuterLowerModuleIndex, unsigned int& innerSegmentIndex, unsigned int& outerSegmentIndex, unsigned int& firstMDIndex, unsigned int& secondMDIndex, unsigned int& thirdMDIndex,
        unsigned int& fourthMDIndex, float& zOut, float& rtOut, float& deltaPhiPos, float& dPhi, float& betaIn, float&
        betaOut, float& pt_beta, float& zLo, float& rtLo, float& rtHi, float& sdlCut, float& betaInCut, float& betaOutCut, float& deltaBetaCut, float& kZ)
{
    bool pass = true;
    bool isPS_InLo = (modulesInGPU.moduleType[innerInnerLowerModuleIndex] == SDL::PS);
    bool isPS_OutLo = (modulesInGPU.moduleType[outerInnerLowerModuleIndex] == SDL::PS);

    float rt_InLo = mdsInGPU.anchorRt[firstMDIndex];
    float rt_InOut = mdsInGPU.anchorRt[secondMDIndex];
    float rt_OutLo = mdsInGPU.anchorRt[thirdMDIndex];

    float z_InLo = mdsInGPU.anchorZ[firstMDIndex];
    float z_InOut = mdsInGPU.anchorZ[secondMDIndex];
    float z_OutLo = mdsInGPU.anchorZ[thirdMDIndex];

    float alpha1GeV_OutLo = asinf(fminf(rt_OutLo * SDL::k2Rinv1GeVf / SDL::ptCut, SDL::sinAlphaMax));

    float rtRatio_OutLoInLo = rt_OutLo / rt_InLo; // Outer segment beginning rt divided by inner segment beginning rt;
    float dzDrtScale = tanf(alpha1GeV_OutLo) / alpha1GeV_OutLo; // The track can bend in r-z plane slightly
    float zpitch_InLo = (isPS_InLo ? SDL::pixelPSZpitch : SDL::strip2SZpitch);
    float zpitch_OutLo = (isPS_OutLo ? SDL::pixelPSZpitch : SDL::strip2SZpitch);
    float zGeom = zpitch_InLo + zpitch_OutLo;

    zLo = z_InLo + (z_InLo - SDL::deltaZLum) * (rtRatio_OutLoInLo - 1.f) * (z_InLo > 0.f ? 1.f : dzDrtScale) - zGeom;

    // Cut #0: Preliminary (Only here in endcap case)
    pass = pass and (z_InLo * z_OutLo > 0);
    if(not pass) return pass;

    float dLum = copysignf(SDL::deltaZLum, z_InLo);
    bool isOutSgInnerMDPS = modulesInGPU.moduleType[outerInnerLowerModuleIndex] == SDL::PS;
    float rtGeom1 = isOutSgInnerMDPS ? SDL::pixelPSZpitch : SDL::strip2SZpitch;
    float zGeom1 = copysignf(zGeom,z_InLo);
    rtLo = rt_InLo * (1.f + (z_OutLo - z_InLo - zGeom1) / (z_InLo + zGeom1 + dLum) / dzDrtScale) - rtGeom1; //slope correction only on the lower end
    zOut = z_OutLo;
    rtOut = rt_OutLo;

    //Cut #1: rt condition
    pass =  pass and (rtOut >= rtLo);
    if(not pass) return pass;

    float zInForHi = z_InLo - zGeom1 - dLum;
    if(zInForHi * z_InLo < 0)
    {
        zInForHi = copysignf(0.1f,z_InLo);
    }
    rtHi = rt_InLo * (1.f + (z_OutLo - z_InLo + zGeom1) / zInForHi) + rtGeom1;

    //Cut #2: rt condition
    pass =  pass and ((rt_OutLo >= rtLo) & (rt_OutLo <= rtHi));
    if(not pass) return pass;

    float rIn = sqrtf(z_InLo * z_InLo + rt_InLo * rt_InLo);
    const float drtSDIn = rt_InOut - rt_InLo;
    const float dzSDIn = z_InOut - z_InLo;
    const float dr3SDIn = sqrtf(rt_InOut * rt_InOut + z_InOut * z_InOut) - sqrtf(rt_InLo * rt_InLo + z_InLo * z_InLo);

    const float coshEta = dr3SDIn / drtSDIn; //direction estimate
    const float dzOutInAbs = fabsf(z_OutLo - z_InLo);
    const float multDzDr = dzOutInAbs * coshEta / (coshEta * coshEta - 1.f);
    const float zGeom1_another = SDL::pixelPSZpitch; //What's this?
    kZ = (z_OutLo - z_InLo) / dzSDIn;
    float drtErr = zGeom1_another * zGeom1_another * drtSDIn * drtSDIn / dzSDIn / dzSDIn * (1.f - 2.f * kZ + 2.f * kZ * kZ); //Notes:122316
    const float sdlThetaMulsF = 0.015f * sqrtf(0.1f + 0.2f * (rt_OutLo - rt_InLo) / 50.f) * sqrtf(rIn / rt_InLo);
    const float sdlMuls = sdlThetaMulsF * 3.f / SDL::ptCut * 4.f; //will need a better guess than x4?
    drtErr += sdlMuls * sdlMuls * multDzDr * multDzDr / 3.f * coshEta * coshEta; //sloppy: relative muls is 1/3 of total muls
    drtErr = sqrtf(drtErr);
    const float drtMean = drtSDIn * dzOutInAbs / fabsf(dzSDIn); //
    const float rtWindow = drtErr + rtGeom1;
    const float rtLo_another = rt_InLo + drtMean / dzDrtScale - rtWindow;
    const float rtHi_another = rt_InLo + drtMean + rtWindow;

    //Cut #3: rt-z pointed
    pass =  pass and ((kZ >= 0) & (rtOut >= rtLo) & (rtOut <= rtHi));
    if(not pass) return pass;

    const float sdlPVoff = 0.1f / rt_OutLo;
    sdlCut = alpha1GeV_OutLo + sqrtf(sdlMuls * sdlMuls + sdlPVoff*sdlPVoff);


    deltaPhiPos = SDL::deltaPhi(mdsInGPU.anchorX[secondMDIndex], mdsInGPU.anchorY[secondMDIndex], mdsInGPU.anchorX[fourthMDIndex], mdsInGPU.anchorY[fourthMDIndex]);


    //Cut #4: deltaPhiPos can be tighter
    pass =  pass and (fabsf(deltaPhiPos) <= sdlCut);
    if(not pass) return pass;

    float midPointX = 0.5f*(mdsInGPU.anchorX[firstMDIndex] + mdsInGPU.anchorX[thirdMDIndex]);
    float midPointY = 0.5f* (mdsInGPU.anchorY[firstMDIndex] + mdsInGPU.anchorY[thirdMDIndex]);
    float midPointZ = 0.5f*(mdsInGPU.anchorZ[firstMDIndex] + mdsInGPU.anchorZ[thirdMDIndex]);
    float diffX = mdsInGPU.anchorX[thirdMDIndex] - mdsInGPU.anchorX[firstMDIndex];
    float diffY = mdsInGPU.anchorY[thirdMDIndex] - mdsInGPU.anchorY[firstMDIndex];
    float diffZ = mdsInGPU.anchorZ[thirdMDIndex] - mdsInGPU.anchorZ[firstMDIndex];

    dPhi = SDL::deltaPhi(midPointX, midPointY, diffX, diffY);
    // Cut #5: deltaPhiChange
    pass =  pass and (fabsf(dPhi) <= sdlCut);
    if(not pass) return pass;

    float sdIn_alpha     = __H2F(segmentsInGPU.dPhiChanges[innerSegmentIndex]);
    float sdIn_alpha_min = __H2F(segmentsInGPU.dPhiChangeMins[innerSegmentIndex]);
    float sdIn_alpha_max = __H2F(segmentsInGPU.dPhiChangeMaxs[innerSegmentIndex]);
    float sdOut_alpha = sdIn_alpha; //weird

    float sdOut_alphaOut = SDL::deltaPhi(mdsInGPU.anchorX[fourthMDIndex], mdsInGPU.anchorY[fourthMDIndex], mdsInGPU.anchorX[fourthMDIndex] - mdsInGPU.anchorX[thirdMDIndex], mdsInGPU.anchorY[fourthMDIndex] - mdsInGPU.anchorY[thirdMDIndex]);

    float sdOut_alphaOut_min = SDL::phi_mpi_pi(__H2F(segmentsInGPU.dPhiChangeMins[outerSegmentIndex]) - __H2F(segmentsInGPU.dPhiMins[outerSegmentIndex]));
    float sdOut_alphaOut_max = SDL::phi_mpi_pi(__H2F(segmentsInGPU.dPhiChangeMaxs[outerSegmentIndex]) - __H2F(segmentsInGPU.dPhiMaxs[outerSegmentIndex]));

    float tl_axis_x = mdsInGPU.anchorX[fourthMDIndex] - mdsInGPU.anchorX[firstMDIndex];
    float tl_axis_y = mdsInGPU.anchorY[fourthMDIndex] - mdsInGPU.anchorY[firstMDIndex];
    float tl_axis_z = mdsInGPU.anchorZ[fourthMDIndex] - mdsInGPU.anchorZ[firstMDIndex];

    betaIn = sdIn_alpha - SDL::deltaPhi(mdsInGPU.anchorX[firstMDIndex], mdsInGPU.anchorY[firstMDIndex], tl_axis_x, tl_axis_y);

    float betaInRHmin = betaIn;
    float betaInRHmax = betaIn;
    betaOut = -sdOut_alphaOut + SDL::deltaPhi(mdsInGPU.anchorX[fourthMDIndex], mdsInGPU.anchorY[fourthMDIndex], tl_axis_x, tl_axis_y);

    float betaOutRHmin = betaOut;
    float betaOutRHmax = betaOut;

    bool isEC_secondLayer = (modulesInGPU.subdets[innerOuterLowerModuleIndex] == SDL::Endcap) and (modulesInGPU.moduleType[innerOuterLowerModuleIndex] == SDL::TwoS);

    if(isEC_secondLayer)
    {
        betaInRHmin = betaIn - sdIn_alpha_min + sdIn_alpha;
        betaInRHmax = betaIn - sdIn_alpha_max + sdIn_alpha;
    }

    betaOutRHmin = betaOut - sdOut_alphaOut_min + sdOut_alphaOut;
    betaOutRHmax = betaOut - sdOut_alphaOut_max + sdOut_alphaOut;

    float swapTemp;
    if(fabsf(betaOutRHmin) > fabsf(betaOutRHmax))
    {
        swapTemp = betaOutRHmin;
        betaOutRHmin = betaOutRHmax;
        betaOutRHmax = swapTemp;
    }

    if(fabsf(betaInRHmin) > fabsf(betaInRHmax))
    {
        swapTemp = betaInRHmin;
        betaInRHmin = betaInRHmax;
        betaInRHmax = swapTemp;
    }

    float sdIn_dr = sqrtf((mdsInGPU.anchorX[secondMDIndex] - mdsInGPU.anchorX[firstMDIndex]) * (mdsInGPU.anchorX[secondMDIndex] - mdsInGPU.anchorX[firstMDIndex]) + (mdsInGPU.anchorY[secondMDIndex] - mdsInGPU.anchorY[firstMDIndex]) * (mdsInGPU.anchorY[secondMDIndex] - mdsInGPU.anchorY[firstMDIndex]));
    float sdIn_d = rt_InOut - rt_InLo;

    float dr = sqrtf(tl_axis_x * tl_axis_x + tl_axis_y * tl_axis_y);
    const float corrF = 1.f;
    betaInCut = asinf(fminf((-sdIn_dr * corrF + dr) * SDL::k2Rinv1GeVf / SDL::ptCut, SDL::sinAlphaMax)) + (0.02f / sdIn_d);

    //Cut #6: first beta cut
    pass =  pass and (fabsf(betaInRHmin) < betaInCut);
    if(not pass) return pass;

    float betaAv = 0.5f * (betaIn + betaOut);
    pt_beta = dr * SDL::k2Rinv1GeVf / sinf(betaAv);

    float lIn = 5;
    float lOut = 11;

    float sdOut_dr = sqrtf((mdsInGPU.anchorX[fourthMDIndex] - mdsInGPU.anchorX[thirdMDIndex]) * (mdsInGPU.anchorX[fourthMDIndex] - mdsInGPU.anchorX[thirdMDIndex]) + (mdsInGPU.anchorY[fourthMDIndex] - mdsInGPU.anchorY[thirdMDIndex]) * (mdsInGPU.anchorY[fourthMDIndex] - mdsInGPU.anchorY[thirdMDIndex]));
    float sdOut_d = mdsInGPU.anchorRt[fourthMDIndex] - mdsInGPU.anchorRt[thirdMDIndex];

    SDL::runDeltaBetaIterationsT5(betaIn, betaOut, betaAv, pt_beta, sdIn_dr, sdOut_dr, dr, lIn);

     const float betaInMMSF = (fabsf(betaInRHmin + betaInRHmax) > 0) ? (2.f * betaIn / fabsf(betaInRHmin + betaInRHmax)) : 0.; //mean value of min,max is the old betaIn
    const float betaOutMMSF = (fabsf(betaOutRHmin + betaOutRHmax) > 0) ? (2.f * betaOut / fabsf(betaOutRHmin + betaOutRHmax)) : 0.;
    betaInRHmin *= betaInMMSF;
    betaInRHmax *= betaInMMSF;
    betaOutRHmin *= betaOutMMSF;
    betaOutRHmax *= betaOutMMSF;

    const float dBetaMuls = sdlThetaMulsF * 4.f / fminf(fabsf(pt_beta), SDL::pt_betaMax); //need to confirm the range-out value of 7 GeV

    const float alphaInAbsReg = fmaxf(fabsf(sdIn_alpha), asinf(fminf(rt_InLo * SDL::k2Rinv1GeVf / 3.0f, SDL::sinAlphaMax)));
    const float alphaOutAbsReg = fmaxf(fabsf(sdOut_alpha), asinf(fminf(rt_OutLo * SDL::k2Rinv1GeVf / 3.0f, SDL::sinAlphaMax)));
    const float dBetaInLum = lIn < 11 ? 0.0f : fabsf(alphaInAbsReg*SDL::deltaZLum / z_InLo);
    const float dBetaOutLum = lOut < 11 ? 0.0f : fabsf(alphaOutAbsReg*SDL::deltaZLum / z_OutLo);
    const float dBetaLum2 = (dBetaInLum + dBetaOutLum) * (dBetaInLum + dBetaOutLum);
    const float sinDPhi = sinf(dPhi);

    const float dBetaRIn2 = 0; // TODO-RH
    // const float dBetaROut2 = 0; // TODO-RH
    float dBetaROut = 0;
    if(modulesInGPU.moduleType[outerOuterLowerModuleIndex] == SDL::TwoS)
    {
        dBetaROut = (sqrtf(mdsInGPU.anchorHighEdgeX[fourthMDIndex] * mdsInGPU.anchorHighEdgeX[fourthMDIndex] + mdsInGPU.anchorHighEdgeY[fourthMDIndex] * mdsInGPU.anchorHighEdgeY[fourthMDIndex]) - sqrtf(mdsInGPU.anchorLowEdgeX[fourthMDIndex] * mdsInGPU.anchorLowEdgeX[fourthMDIndex] + mdsInGPU.anchorLowEdgeY[fourthMDIndex] * mdsInGPU.anchorLowEdgeY[fourthMDIndex])) * sinDPhi / dr;
    }

    const float dBetaROut2 = dBetaROut * dBetaROut;
    betaOutCut = asinf(fminf(dr*SDL::k2Rinv1GeVf / SDL::ptCut, SDL::sinAlphaMax)) //FIXME: need faster version
        + (0.02f / sdOut_d) + sqrtf(dBetaLum2 + dBetaMuls*dBetaMuls);

    //Cut #6: The real beta cut
    pass =  pass and (fabsf(betaOut) < betaOutCut);
    if(not pass) return pass;

    float pt_betaIn = dr * SDL::k2Rinv1GeVf/sinf(betaIn);
    float pt_betaOut = dr * SDL::k2Rinv1GeVf / sinf(betaOut);
    float dBetaRes = 0.02f/fminf(sdOut_d,sdIn_d);
    float dBetaCut2 = (dBetaRes*dBetaRes * 2.0f + dBetaMuls * dBetaMuls + dBetaLum2 + dBetaRIn2 + dBetaROut2
            + 0.25f * (fabsf(betaInRHmin - betaInRHmax) + fabsf(betaOutRHmin - betaOutRHmax)) * (fabsf(betaInRHmin - betaInRHmax) + fabsf(betaOutRHmin - betaOutRHmax)));
    float dBeta = betaIn - betaOut;
    deltaBetaCut = sqrtf(dBetaCut2);
    //Cut #7: Cut on dBet
    pass =  pass and (dBeta * dBeta <= dBetaCut2);

    return pass;
}

__device__ bool SDL::runQuintupletDefaultAlgoEEEE(struct SDL::modules& modulesInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, uint16_t& innerInnerLowerModuleIndex, uint16_t& innerOuterLowerModuleIndex, uint16_t& outerInnerLowerModuleIndex, uint16_t& outerOuterLowerModuleIndex, unsigned int& innerSegmentIndex, unsigned int& outerSegmentIndex, unsigned int& firstMDIndex, unsigned int& secondMDIndex, unsigned int& thirdMDIndex,
        unsigned int& fourthMDIndex, float& zOut, float& rtOut, float& deltaPhiPos, float& dPhi, float& betaIn, float&
        betaOut, float& pt_beta, float& zLo, float& rtLo, float& rtHi, float& sdlCut, float& betaInCut, float& betaOutCut, float& deltaBetaCut, float& kZ)
{
    bool pass = true;

    bool isPS_InLo = (modulesInGPU.moduleType[innerInnerLowerModuleIndex] == SDL::PS);
    bool isPS_OutLo = (modulesInGPU.moduleType[outerInnerLowerModuleIndex] == SDL::PS);

    float rt_InLo = mdsInGPU.anchorRt[firstMDIndex];
    float rt_InOut = mdsInGPU.anchorRt[secondMDIndex];
    float rt_OutLo = mdsInGPU.anchorRt[thirdMDIndex];

    float z_InLo = mdsInGPU.anchorZ[firstMDIndex];
    float z_InOut = mdsInGPU.anchorZ[secondMDIndex];
    float z_OutLo = mdsInGPU.anchorZ[thirdMDIndex];

    float alpha1GeV_OutLo = asinf(fminf(rt_OutLo * SDL::k2Rinv1GeVf / SDL::ptCut, SDL::sinAlphaMax));

    float rtRatio_OutLoInLo = rt_OutLo / rt_InLo; // Outer segment beginning rt divided by inner segment beginning rt;
    float dzDrtScale = tanf(alpha1GeV_OutLo) / alpha1GeV_OutLo; // The track can bend in r-z plane slightly
    float zpitch_InLo = (isPS_InLo ? SDL::pixelPSZpitch : SDL::strip2SZpitch);
    float zpitch_OutLo = (isPS_OutLo ? SDL::pixelPSZpitch : SDL::strip2SZpitch);
    float zGeom = zpitch_InLo + zpitch_OutLo;

    zLo = z_InLo + (z_InLo - SDL::deltaZLum) * (rtRatio_OutLoInLo - 1.f) * (z_InLo > 0.f ? 1.f : dzDrtScale) - zGeom; //slope-correction only on outer end

    // Cut #0: Preliminary (Only here in endcap case)
    pass =  pass and ((z_InLo * z_OutLo) > 0);
    if(not pass) return pass;

    float dLum = copysignf(SDL::deltaZLum, z_InLo);
    bool isOutSgInnerMDPS = modulesInGPU.moduleType[outerInnerLowerModuleIndex] == SDL::PS;
    bool isInSgInnerMDPS = modulesInGPU.moduleType[innerInnerLowerModuleIndex] == SDL::PS;

    float rtGeom = (isInSgInnerMDPS and isOutSgInnerMDPS) ? 2.f * SDL::pixelPSZpitch : (isInSgInnerMDPS or isOutSgInnerMDPS) ? SDL::pixelPSZpitch + SDL::strip2SZpitch : 2.f * SDL::strip2SZpitch;

    float zGeom1 = copysignf(zGeom,z_InLo);
    float dz = z_OutLo - z_InLo;
    rtLo = rt_InLo * (1.f + dz / (z_InLo + dLum) / dzDrtScale) - rtGeom; //slope correction only on the lower end

    zOut = z_OutLo;
    rtOut = rt_OutLo;

    //Cut #1: rt condition

    rtHi = rt_InLo * (1.f + dz / (z_InLo - dLum)) + rtGeom;

    pass =  pass and ((rtOut >= rtLo) & (rtOut <= rtHi));
    if(not pass) return pass;

    bool isInSgOuterMDPS = modulesInGPU.moduleType[innerOuterLowerModuleIndex] == SDL::PS;

    float drOutIn = rtOut - rt_InLo;
    const float drtSDIn = rt_InOut - rt_InLo;
    const float dzSDIn = z_InOut - z_InLo;
    const float dr3SDIn = sqrtf(rt_InOut * rt_InOut + z_InOut * z_InOut) - sqrtf(rt_InLo * rt_InLo + z_InLo * z_InLo);
    float coshEta = dr3SDIn / drtSDIn; //direction estimate
    float dzOutInAbs =  fabsf(z_OutLo - z_InLo);
    float multDzDr = dzOutInAbs * coshEta / (coshEta * coshEta - 1.f);

    kZ = (z_OutLo - z_InLo) / dzSDIn;
    float sdlThetaMulsF = 0.015f * sqrtf(0.1f + 0.2f * (rt_OutLo - rt_InLo) / 50.f);

    float sdlMuls = sdlThetaMulsF * 3.f / SDL::ptCut * 4.f; //will need a better guess than x4?

    float drtErr = sqrtf(SDL::pixelPSZpitch * SDL::pixelPSZpitch * 2.f / (dzSDIn * dzSDIn) * (dzOutInAbs * dzOutInAbs) + sdlMuls * sdlMuls * multDzDr * multDzDr / 3.f * coshEta * coshEta);

    float drtMean = drtSDIn * dzOutInAbs/fabsf(dzSDIn);
    float rtWindow = drtErr + rtGeom;
    float rtLo_point = rt_InLo + drtMean / dzDrtScale - rtWindow;
    float rtHi_point = rt_InLo + drtMean + rtWindow;

    // Cut #3: rt-z pointed
    // https://github.com/slava77/cms-tkph2-ntuple/blob/superDoubletLinked-91X-noMock/doubletAnalysis.C#L3765

    if (isInSgInnerMDPS and isInSgOuterMDPS) // If both PS then we can point
    {
        pass =  pass and (kZ >= 0 and rtOut >= rtLo_point and rtOut <= rtHi_point);
        if(not pass) return pass;
    }

    float sdlPVoff = 0.1f/rtOut;
    sdlCut = alpha1GeV_OutLo + sqrtf(sdlMuls * sdlMuls + sdlPVoff * sdlPVoff);

    deltaPhiPos = SDL::deltaPhi(mdsInGPU.anchorX[secondMDIndex], mdsInGPU.anchorY[secondMDIndex], mdsInGPU.anchorX[fourthMDIndex], mdsInGPU.anchorY[fourthMDIndex]);

    pass =  pass and (fabsf(deltaPhiPos) <= sdlCut);
    if(not pass) return pass;

    float midPointX = 0.5f*(mdsInGPU.anchorX[firstMDIndex] + mdsInGPU.anchorX[thirdMDIndex]);
    float midPointY = 0.5f* (mdsInGPU.anchorY[firstMDIndex] + mdsInGPU.anchorY[thirdMDIndex]);
    float midPointZ = 0.5f*(mdsInGPU.anchorZ[firstMDIndex] + mdsInGPU.anchorZ[thirdMDIndex]);
    float diffX = mdsInGPU.anchorX[thirdMDIndex] - mdsInGPU.anchorX[firstMDIndex];
    float diffY = mdsInGPU.anchorY[thirdMDIndex] - mdsInGPU.anchorY[firstMDIndex];
    float diffZ = mdsInGPU.anchorZ[thirdMDIndex] - mdsInGPU.anchorZ[firstMDIndex];

    dPhi = SDL::deltaPhi(midPointX, midPointY, diffX, diffY);

    // Cut #5: deltaPhiChange
    pass =  pass and ((fabsf(dPhi) <= sdlCut));
    if(not pass) return pass;

    float sdIn_alpha = __H2F(segmentsInGPU.dPhiChanges[innerSegmentIndex]);
    float sdOut_alpha = sdIn_alpha; //weird
    float sdOut_dPhiPos = SDL::deltaPhi(mdsInGPU.anchorX[thirdMDIndex], mdsInGPU.anchorY[thirdMDIndex], mdsInGPU.anchorX[fourthMDIndex], mdsInGPU.anchorY[fourthMDIndex]);

    float sdOut_dPhiChange = __H2F(segmentsInGPU.dPhiChanges[outerSegmentIndex]);
    float sdOut_dPhiChange_min = __H2F(segmentsInGPU.dPhiChangeMins[outerSegmentIndex]);
    float sdOut_dPhiChange_max = __H2F(segmentsInGPU.dPhiChangeMaxs[outerSegmentIndex]);

    float sdOut_alphaOutRHmin = SDL::phi_mpi_pi(sdOut_dPhiChange_min - sdOut_dPhiPos);
    float sdOut_alphaOutRHmax = SDL::phi_mpi_pi(sdOut_dPhiChange_max - sdOut_dPhiPos);
    float sdOut_alphaOut = SDL::phi_mpi_pi(sdOut_dPhiChange - sdOut_dPhiPos);

    float tl_axis_x = mdsInGPU.anchorX[fourthMDIndex] - mdsInGPU.anchorX[firstMDIndex];
    float tl_axis_y = mdsInGPU.anchorY[fourthMDIndex] - mdsInGPU.anchorY[firstMDIndex];
    float tl_axis_z = mdsInGPU.anchorZ[fourthMDIndex] - mdsInGPU.anchorZ[firstMDIndex];

    betaIn = sdIn_alpha - SDL::deltaPhi(mdsInGPU.anchorX[firstMDIndex], mdsInGPU.anchorY[firstMDIndex], tl_axis_x, tl_axis_y);

    float sdIn_alphaRHmin = __H2F(segmentsInGPU.dPhiChangeMins[innerSegmentIndex]);
    float sdIn_alphaRHmax = __H2F(segmentsInGPU.dPhiChangeMaxs[innerSegmentIndex]);
    float betaInRHmin = betaIn + sdIn_alphaRHmin - sdIn_alpha;
    float betaInRHmax = betaIn + sdIn_alphaRHmax - sdIn_alpha;

    betaOut = -sdOut_alphaOut + SDL::deltaPhi(mdsInGPU.anchorX[fourthMDIndex], mdsInGPU.anchorY[fourthMDIndex], tl_axis_x, tl_axis_y);

    float betaOutRHmin = betaOut - sdOut_alphaOutRHmin + sdOut_alphaOut;
    float betaOutRHmax = betaOut - sdOut_alphaOutRHmax + sdOut_alphaOut;

    float swapTemp;
    if(fabsf(betaOutRHmin) > fabsf(betaOutRHmax))
    {
        swapTemp = betaOutRHmin;
        betaOutRHmin = betaOutRHmax;
        betaOutRHmax = swapTemp;
    }

    if(fabsf(betaInRHmin) > fabsf(betaInRHmax))
    {
        swapTemp = betaInRHmin;
        betaInRHmin = betaInRHmax;
        betaInRHmax = swapTemp;
    }
    float sdIn_dr = sqrtf((mdsInGPU.anchorX[secondMDIndex] - mdsInGPU.anchorX[firstMDIndex]) * (mdsInGPU.anchorX[secondMDIndex] - mdsInGPU.anchorX[firstMDIndex]) + (mdsInGPU.anchorY[secondMDIndex] - mdsInGPU.anchorY[firstMDIndex]) * (mdsInGPU.anchorY[secondMDIndex] - mdsInGPU.anchorY[firstMDIndex]));
    float sdIn_d = rt_InOut - rt_InLo;

    float dr = sqrtf(tl_axis_x * tl_axis_x + tl_axis_y * tl_axis_y);
    const float corrF = 1.f;
    betaInCut = asinf(fminf((-sdIn_dr * corrF + dr) * SDL::k2Rinv1GeVf / SDL::ptCut, SDL::sinAlphaMax)) + (0.02f / sdIn_d);

    //Cut #6: first beta cut
    pass =  pass and (fabsf(betaInRHmin) < betaInCut);
    if(not pass) return pass;

    float betaAv = 0.5f * (betaIn + betaOut);
    pt_beta = dr * SDL::k2Rinv1GeVf / sinf(betaAv);


    int lIn= 11; //endcap
    int lOut = 13; //endcap

    float sdOut_dr = sqrtf((mdsInGPU.anchorX[fourthMDIndex] - mdsInGPU.anchorX[thirdMDIndex]) * (mdsInGPU.anchorX[fourthMDIndex] - mdsInGPU.anchorX[thirdMDIndex]) + (mdsInGPU.anchorY[fourthMDIndex] - mdsInGPU.anchorY[thirdMDIndex]) * (mdsInGPU.anchorY[fourthMDIndex] - mdsInGPU.anchorY[thirdMDIndex]));
    float sdOut_d = mdsInGPU.anchorRt[fourthMDIndex] - mdsInGPU.anchorRt[thirdMDIndex];

    float diffDr = fabsf(sdIn_dr - sdOut_dr)/fabs(sdIn_dr + sdOut_dr);

    SDL::runDeltaBetaIterationsT5(betaIn, betaOut, betaAv, pt_beta, sdIn_dr, sdOut_dr, dr, lIn);

     const float betaInMMSF = (fabsf(betaInRHmin + betaInRHmax) > 0) ? (2.f * betaIn / fabsf(betaInRHmin + betaInRHmax)) : 0.; //mean value of min,max is the old betaIn
    const float betaOutMMSF = (fabsf(betaOutRHmin + betaOutRHmax) > 0) ? (2.f * betaOut / fabsf(betaOutRHmin + betaOutRHmax)) : 0.;
    betaInRHmin *= betaInMMSF;
    betaInRHmax *= betaInMMSF;
    betaOutRHmin *= betaOutMMSF;
    betaOutRHmax *= betaOutMMSF;

    const float dBetaMuls = sdlThetaMulsF * 4.f / fminf(fabsf(pt_beta), SDL::pt_betaMax); //need to confirm the range-out value of 7 GeV

    const float alphaInAbsReg = fmaxf(fabsf(sdIn_alpha), asinf(fminf(rt_InLo * SDL::k2Rinv1GeVf / 3.0f, SDL::sinAlphaMax)));
    const float alphaOutAbsReg = fmaxf(fabsf(sdOut_alpha), asinf(fminf(rt_OutLo * SDL::k2Rinv1GeVf / 3.0f, SDL::sinAlphaMax)));
    const float dBetaInLum = lIn < 11 ? 0.0f : fabsf(alphaInAbsReg*SDL::deltaZLum / z_InLo);
    const float dBetaOutLum = lOut < 11 ? 0.0f : fabsf(alphaOutAbsReg*SDL::deltaZLum / z_OutLo);
    const float dBetaLum2 = (dBetaInLum + dBetaOutLum) * (dBetaInLum + dBetaOutLum);
    const float sinDPhi = sinf(dPhi);

    const float dBetaRIn2 = 0; // TODO-RH
    // const float dBetaROut2 = 0; // TODO-RH
    float dBetaROut2 = 0;//TODO-RH
    betaOutCut = asinf(fminf(dr*SDL::k2Rinv1GeVf / SDL::ptCut, SDL::sinAlphaMax)) //FIXME: need faster version
        + (0.02f / sdOut_d) + sqrtf(dBetaLum2 + dBetaMuls*dBetaMuls);

    //Cut #6: The real beta cut
    pass =  pass and (fabsf(betaOut) < betaOutCut);
    if(not pass) return pass;

    float pt_betaIn = dr * SDL::k2Rinv1GeVf/sinf(betaIn);
    float pt_betaOut = dr * SDL::k2Rinv1GeVf / sinf(betaOut);
    float dBetaRes = 0.02f/fminf(sdOut_d,sdIn_d);
    float dBetaCut2 = (dBetaRes*dBetaRes * 2.0f + dBetaMuls * dBetaMuls + dBetaLum2 + dBetaRIn2 + dBetaROut2
            + 0.25f * (fabsf(betaInRHmin - betaInRHmax) + fabsf(betaOutRHmin - betaOutRHmax)) * (fabsf(betaInRHmin - betaInRHmax) + fabsf(betaOutRHmin - betaOutRHmax)));
    float dBeta = betaIn - betaOut;
    //Cut #7: Cut on dBeta
    deltaBetaCut = sqrtf(dBetaCut2);

    pass =  pass and (dBeta * dBeta <= dBetaCut2);

    return pass;
}
__device__ bool SDL::runQuintupletDefaultAlgo(struct SDL::modules& modulesInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, uint16_t& innerInnerLowerModuleIndex, uint16_t& innerOuterLowerModuleIndex, uint16_t& outerInnerLowerModuleIndex, uint16_t& outerOuterLowerModuleIndex, unsigned int& innerSegmentIndex, unsigned int& outerSegmentIndex, unsigned int& firstMDIndex, unsigned int& secondMDIndex, unsigned int& thirdMDIndex, unsigned int& fourthMDIndex, float& zOut, float& rtOut, float& deltaPhiPos, float& deltaPhi, float& betaIn, float&
        betaOut, float& pt_beta, float& zLo, float& zHi, float& rtLo, float& rtHi, float& zLoPointed, float& zHiPointed, float& sdlCut, float& betaInCut, float& betaOutCut, float& deltaBetaCut, float& kZ)
{

    bool pass = false;

    zLo = -999;
    zHi = -999;
    rtLo = -999;
    rtHi = -999;
    zLoPointed = -999;
    zHiPointed = -999;
    kZ = -999;
    betaInCut = -999;

    short innerInnerLowerModuleSubdet = modulesInGPU.subdets[innerInnerLowerModuleIndex];
    short innerOuterLowerModuleSubdet = modulesInGPU.subdets[innerOuterLowerModuleIndex];
    short outerInnerLowerModuleSubdet = modulesInGPU.subdets[outerInnerLowerModuleIndex];
    short outerOuterLowerModuleSubdet = modulesInGPU.subdets[outerOuterLowerModuleIndex];

    if(innerInnerLowerModuleSubdet == SDL::Barrel
            and innerOuterLowerModuleSubdet == SDL::Barrel
            and outerInnerLowerModuleSubdet == SDL::Barrel
            and outerOuterLowerModuleSubdet == SDL::Barrel)
    {
        return runQuintupletDefaultAlgoBBBB(modulesInGPU,mdsInGPU,segmentsInGPU,innerInnerLowerModuleIndex,innerOuterLowerModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,innerSegmentIndex,outerSegmentIndex,firstMDIndex, secondMDIndex, thirdMDIndex, fourthMDIndex, zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta, zLo, zHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut);
    }

    else if(innerInnerLowerModuleSubdet == SDL::Barrel
            and innerOuterLowerModuleSubdet == SDL::Barrel
            and outerInnerLowerModuleSubdet == SDL::Endcap
            and outerOuterLowerModuleSubdet == SDL::Endcap)
    {
       return runQuintupletDefaultAlgoBBEE(modulesInGPU,mdsInGPU,segmentsInGPU,innerInnerLowerModuleIndex,innerOuterLowerModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,innerSegmentIndex,outerSegmentIndex,firstMDIndex, secondMDIndex, thirdMDIndex, fourthMDIndex, zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta, zLo, rtLo, rtHi, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ);
    }


    else if(innerInnerLowerModuleSubdet == SDL::Barrel
            and innerOuterLowerModuleSubdet == SDL::Barrel
            and outerInnerLowerModuleSubdet == SDL::Barrel
            and outerOuterLowerModuleSubdet == SDL::Endcap)
    {
        return runQuintupletDefaultAlgoBBBB(modulesInGPU,mdsInGPU,segmentsInGPU,innerInnerLowerModuleIndex,innerOuterLowerModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,innerSegmentIndex,outerSegmentIndex,firstMDIndex, secondMDIndex, thirdMDIndex, fourthMDIndex, zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta,zLo, zHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut);

    }

    else if(innerInnerLowerModuleSubdet == SDL::Barrel
            and innerOuterLowerModuleSubdet == SDL::Endcap
            and outerInnerLowerModuleSubdet == SDL::Endcap
            and outerOuterLowerModuleSubdet == SDL::Endcap)
    {
        return runQuintupletDefaultAlgoBBEE(modulesInGPU,mdsInGPU,segmentsInGPU,innerInnerLowerModuleIndex,innerOuterLowerModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,innerSegmentIndex,outerSegmentIndex, firstMDIndex, secondMDIndex, thirdMDIndex, fourthMDIndex, zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta, zLo, rtLo, rtHi, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ);

    }

    else if(innerInnerLowerModuleSubdet == SDL::Endcap
            and innerOuterLowerModuleSubdet == SDL::Endcap
            and outerInnerLowerModuleSubdet == SDL::Endcap
            and outerOuterLowerModuleSubdet == SDL::Endcap)
    {
        return runQuintupletDefaultAlgoEEEE(modulesInGPU,mdsInGPU,segmentsInGPU,innerInnerLowerModuleIndex,innerOuterLowerModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,innerSegmentIndex,outerSegmentIndex, firstMDIndex, secondMDIndex, thirdMDIndex, fourthMDIndex, zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta, zLo, rtLo, rtHi, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ);
    }

    return pass;
}
__device__ void SDL::runDeltaBetaIterationsT5(float& betaIn, float& betaOut, float& betaAv, float & pt_beta, float sdIn_dr, float sdOut_dr, float dr, float lIn)
{
    if (lIn == 0)
    {
        betaOut += copysign(asinf(fminf(sdOut_dr * SDL::k2Rinv1GeVf / fabsf(pt_beta), SDL::sinAlphaMax)), betaOut);
        return;
    }

    if (betaIn * betaOut > 0.f and (fabsf(pt_beta) < 4.f * SDL::pt_betaMax or (lIn >= 11 and fabsf(pt_beta) < 8.f * SDL::pt_betaMax)))   //and the pt_beta is well-defined; less strict for endcap-endcap
    {

        const float betaInUpd  = betaIn + copysignf(asinf(fminf(sdIn_dr * SDL::k2Rinv1GeVf / fabsf(pt_beta), SDL::sinAlphaMax)), betaIn); //FIXME: need a faster version
        const float betaOutUpd = betaOut + copysignf(asinf(fminf(sdOut_dr * SDL::k2Rinv1GeVf / fabsf(pt_beta), SDL::sinAlphaMax)), betaOut); //FIXME: need a faster version
        betaAv = 0.5f * (betaInUpd + betaOutUpd);

        //1st update
        //pt_beta = dr * k2Rinv1GeVf / sinf(betaAv); //get a better pt estimate
        const float pt_beta_inv = 1.f/fabsf(dr * k2Rinv1GeVf / sinf(betaAv)); //get a better pt estimate

        betaIn  += copysignf(asinf(fminf(sdIn_dr * SDL::k2Rinv1GeVf *pt_beta_inv, SDL::sinAlphaMax)), betaIn); //FIXME: need a faster version
        betaOut += copysignf(asinf(fminf(sdOut_dr * SDL::k2Rinv1GeVf *pt_beta_inv, SDL::sinAlphaMax)), betaOut); //FIXME: need a faster version
        //update the av and pt
        betaAv = 0.5f * (betaIn + betaOut);
        //2nd update
        pt_beta = dr * SDL::k2Rinv1GeVf / sinf(betaAv); //get a better pt estimate
    }
    else if (lIn < 11 && fabsf(betaOut) < 0.2f * fabsf(betaIn) && fabsf(pt_beta) < 12.f * SDL::pt_betaMax)   //use betaIn sign as ref
    {

        const float pt_betaIn = dr * k2Rinv1GeVf / sinf(betaIn);

        const float betaInUpd  = betaIn + copysignf(asinf(fminf(sdIn_dr * SDL::k2Rinv1GeVf / fabsf(pt_betaIn), SDL::sinAlphaMax)), betaIn); //FIXME: need a faster version
        const float betaOutUpd = betaOut + copysignf(asinf(fminf(sdOut_dr * SDL::k2Rinv1GeVf / fabsf(pt_betaIn), SDL::sinAlphaMax)), betaIn); //FIXME: need a faster version
        betaAv = (fabsf(betaOut) > 0.2f * fabsf(betaIn)) ? (0.5f * (betaInUpd + betaOutUpd)) : betaInUpd;

        //1st update
        pt_beta = dr * SDL::k2Rinv1GeVf / sin(betaAv); //get a better pt estimate
        betaIn  += copysignf(asinf(fminf(sdIn_dr * SDL::k2Rinv1GeVf / fabsf(pt_beta), SDL::sinAlphaMax)), betaIn); //FIXME: need a faster version
        betaOut += copysignf(asinf(fminf(sdOut_dr * SDL::k2Rinv1GeVf / fabsf(pt_beta), SDL::sinAlphaMax)), betaIn); //FIXME: need a faster version
        //update the av and pt
        betaAv = 0.5f * (betaIn + betaOut);
        //2nd update
        pt_beta = dr * SDL::k2Rinv1GeVf / sin(betaAv); //get a better pt estimate

    }
}

__device__ bool SDL::checkIntervalOverlap(const float& firstMin, const float& firstMax, const float& secondMin, const float& secondMax)
{
    return ((firstMin <= secondMin) & (secondMin < firstMax)) |  ((secondMin < firstMin) & (firstMin < secondMax));
}
