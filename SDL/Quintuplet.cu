#include "hip/hip_runtime.h"
#ifdef __HIPCC__
#define CUDA_CONST_VAR __device__
#endif
# include "Quintuplet.cuh"
//#ifdef CACHE_ALLOC
#include "allocate.h"
//#endif

SDL::quintuplets::quintuplets()
{
    tripletIndices = nullptr;
    lowerModuleIndices = nullptr;
    nQuintuplets = nullptr;
    innerRadius = nullptr;
    innerRadiusMin = nullptr;
    innerRadiusMax = nullptr;
    bridgeRadius = nullptr;
    bridgeRadiusMin = nullptr;
    bridgeRadiusMax = nullptr;
    outerRadius = nullptr;
    outerRadiusMin = nullptr;
    outerRadiusMax = nullptr;

}

void SDL::quintuplets::freeMemory()
{
    hipFree(tripletIndices);
    hipFree(lowerModuleIndices);
    hipFree(nQuintuplets);
    hipFree(innerRadius);
    hipFree(innerRadiusMin);
    hipFree(innerRadiusMax);
    hipFree(bridgeRadius);
    hipFree(bridgeRadiusMin);
    hipFree(bridgeRadiusMax);
    hipFree(outerRadius);
    hipFree(outerRadiusMin);
    hipFree(outerRadiusMax);
}
void SDL::createQuintupletsInUnifiedMemory(struct SDL::quintuplets& quintupletsInGPU, unsigned int maxQuintuplets, unsigned int nLowerModules)
{
    hipMallocManaged(&quintupletsInGPU.tripletIndices, 2 * maxQuintuplets * nLowerModules * sizeof(unsigned int));
    hipMallocManaged(&quintupletsInGPU.lowerModuleIndices, 5 * maxQuintuplets * nLowerModules * sizeof(unsigned int)); 

    hipMallocManaged(&quintupletsInGPU.nQuintuplets, nLowerModules * sizeof(unsigned int));


    hipMallocManaged(&quintupletsInGPU.innerRadius, maxQuintuplets * nLowerModules * sizeof(float));
    hipMallocManaged(&quintupletsInGPU.innerRadiusMin, maxQuintuplets * nLowerModules * sizeof(float));
    hipMallocManaged(&quintupletsInGPU.innerRadiusMax, maxQuintuplets * nLowerModules * sizeof(float));
    hipMallocManaged(&quintupletsInGPU.bridgeRadius, maxQuintuplets * nLowerModules * sizeof(float));
    hipMallocManaged(&quintupletsInGPU.bridgeRadiusMin, maxQuintuplets * nLowerModules * sizeof(float));
    hipMallocManaged(&quintupletsInGPU.bridgeRadiusMax, maxQuintuplets * nLowerModules * sizeof(float));
    hipMallocManaged(&quintupletsInGPU.outerRadius, maxQuintuplets * nLowerModules * sizeof(float));
    hipMallocManaged(&quintupletsInGPU.outerRadiusMin, maxQuintuplets * nLowerModules * sizeof(float));
    hipMallocManaged(&quintupletsInGPU.outerRadiusMax, maxQuintuplets * nLowerModules * sizeof(float));
    hipMallocManaged(&quintupletsInGPU.innerRadiusMin2S, maxQuintuplets * nLowerModules * sizeof(float));
    hipMallocManaged(&quintupletsInGPU.innerRadiusMax2S, maxQuintuplets * nLowerModules * sizeof(float));
    hipMallocManaged(&quintupletsInGPU.bridgeRadiusMin2S, maxQuintuplets * nLowerModules * sizeof(float));
    hipMallocManaged(&quintupletsInGPU.bridgeRadiusMax2S, maxQuintuplets * nLowerModules * sizeof(float));
    hipMallocManaged(&quintupletsInGPU.outerRadiusMin2S, maxQuintuplets * nLowerModules * sizeof(float));
    hipMallocManaged(&quintupletsInGPU.outerRadiusMax2S, maxQuintuplets * nLowerModules * sizeof(float));

    hipMallocManaged(&quintupletsInGPU.nQuintuplets, nLowerModules * sizeof(unsigned int));

#pragma omp parallel for
    for(size_t i = 0; i<nLowerModules;i++)
    {
        quintupletsInGPU.nQuintuplets[i] = 0;
    }

}

__device__ void SDL::addQuintupletToMemory(struct SDL::quintuplets& quintupletsInGPU, unsigned int innerTripletIndex, unsigned int outerTripletIndex, unsigned int lowerModule1, unsigned int lowerModule2, unsigned int lowerModule3, unsigned int lowerModule4, unsigned int lowerModule5, float innerRadius, float innerRadiusMin, float innerRadiusMax, float outerRadius, float outerRadiusMin, float outerRadiusMax, float bridgeRadius, float bridgeRadiusMin, float bridgeRadiusMax,
        float innerRadiusMin2S, float innerRadiusMax2S, float bridgeRadiusMin2S, float bridgeRadiusMax2S, float outerRadiusMin2S, float outerRadiusMax2S,unsigned int quintupletIndex)
{
    quintupletsInGPU.tripletIndices[2 * quintupletIndex] = innerTripletIndex;
    quintupletsInGPU.tripletIndices[2 * quintupletIndex + 1] = outerTripletIndex;

    quintupletsInGPU.innerRadius[quintupletIndex] = innerRadius;
    quintupletsInGPU.innerRadiusMin[quintupletIndex] = innerRadiusMin;
    quintupletsInGPU.innerRadiusMax[quintupletIndex] = innerRadiusMax;
    quintupletsInGPU.outerRadius[quintupletIndex] = outerRadius;
    quintupletsInGPU.outerRadiusMin[quintupletIndex] = outerRadiusMin;
    quintupletsInGPU.outerRadiusMax[quintupletIndex] = outerRadiusMax;
    quintupletsInGPU.bridgeRadius[quintupletIndex] = bridgeRadius;
    quintupletsInGPU.bridgeRadiusMin[quintupletIndex] = bridgeRadiusMin;
    quintupletsInGPU.bridgeRadiusMax[quintupletIndex] = bridgeRadiusMax;
    quintupletsInGPU.innerRadiusMin2S[quintupletIndex] = innerRadiusMin2S;
    quintupletsInGPU.innerRadiusMax2S[quintupletIndex] = innerRadiusMax2S;
    quintupletsInGPU.bridgeRadiusMin2S[quintupletIndex] = bridgeRadiusMin2S;
    quintupletsInGPU.bridgeRadiusMax2S[quintupletIndex] = bridgeRadiusMax2S;
    quintupletsInGPU.outerRadiusMin2S[quintupletIndex] = outerRadiusMin2S;
    quintupletsInGPU.outerRadiusMax2S[quintupletIndex] = outerRadiusMax2S;

    quintupletsInGPU.lowerModuleIndices[5 * quintupletIndex] = lowerModule1;
    quintupletsInGPU.lowerModuleIndices[5 * quintupletIndex + 1] = lowerModule2;
    quintupletsInGPU.lowerModuleIndices[5 * quintupletIndex + 2] = lowerModule3;
    quintupletsInGPU.lowerModuleIndices[5 * quintupletIndex + 3] = lowerModule4;
    quintupletsInGPU.lowerModuleIndices[5 * quintupletIndex + 4] = lowerModule5;
}

__device__ bool SDL::runQuintupletDefaultAlgo(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::triplets& tripletsInGPU, unsigned int lowerModuleIndex1, unsigned int lowerModuleIndex2, unsigned int lowerModuleIndex3, unsigned int lowerModuleIndex4, unsigned int lowerModuleIndex5, unsigned int innerTripletIndex, unsigned int outerTripletIndex, float& innerRadius, float& innerRadiusMin, float&
    innerRadiusMax, float& outerRadius, float& outerRadiusMin, float& outerRadiusMax, float& bridgeRadius, float& bridgeRadiusMin, float& bridgeRadiusMax, float& innerRadiusMin2S, float& innerRadiusMax2S, float& bridgeRadiusMin2S, float& bridgeRadiusMax2S, float& outerRadiusMin2S, float& outerRadiusMax2S)
{
    bool pass = true;

    if(not T5HasCommonMiniDoublet(tripletsInGPU, segmentsInGPU, innerTripletIndex, outerTripletIndex))
    {
        pass = false;
    }

    unsigned int firstSegmentIndex = tripletsInGPU.segmentIndices[2 * innerTripletIndex];
    unsigned int secondSegmentIndex = tripletsInGPU.segmentIndices[2 * innerTripletIndex + 1];
    unsigned int thirdSegmentIndex = tripletsInGPU.segmentIndices[2 * outerTripletIndex];
    unsigned int fourthSegmentIndex = tripletsInGPU.segmentIndices[2 * outerTripletIndex + 1];

    //apply T4 criteria between segments 1 and 3
    float zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut; //temp stuff
    if(not runTrackletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, segmentsInGPU.innerLowerModuleIndices[firstSegmentIndex], segmentsInGPU.outerLowerModuleIndices[firstSegmentIndex], segmentsInGPU.innerLowerModuleIndices[thirdSegmentIndex], segmentsInGPU.outerLowerModuleIndices[thirdSegmentIndex], firstSegmentIndex, thirdSegmentIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut))
    {
        pass = false;
    }
    if(not runTrackletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, segmentsInGPU.innerLowerModuleIndices[firstSegmentIndex], segmentsInGPU.outerLowerModuleIndices[firstSegmentIndex], segmentsInGPU.innerLowerModuleIndices[fourthSegmentIndex], segmentsInGPU.outerLowerModuleIndices[fourthSegmentIndex], firstSegmentIndex, fourthSegmentIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut))
    {
        pass = false;
    }


/*    unsigned int firstMDIndex = segmentsInGPU.mdIndices[2 * firstSegmentIndex];
    unsigned int secondMDIndex = segmentsInGPU.mdIndices[2 * secondSegmentIndex];
    unsigned int thirdMDIndex = segmentsInGPU.mdIndices[2 * secondSegmentIndex + 1];
    unsigned int fourthMDIndex = segmentsInGPU.mdIndices[2 * thirdSegmentIndex + 1];
    unsigned int fifthMDIndex = segmentsInGPU.mdIndices[2 * fourthSegmentIndex + 1];

    unsigned int firstMDLowerHitIndex = mdsInGPU.hitIndices[2 * firstMDIndex];
    unsigned int firstMDUpperHitIndex = mdsInGPU.hitIndices[2 * firstMDIndex + 1];
    unsigned int secondMDLowerHitIndex = mdsInGPU.hitIndices[2 * secondMDIndex];
    unsigned int secondMDUpperHitIndex = mdsInGPU.hitIndices[2 * secondMDIndex + 1];
    unsigned int thirdMDLowerHitIndex = mdsInGPU.hitIndices[2 * thirdMDIndex];
    unsigned int thirdMDUpperHitIndex = mdsInGPU.hitIndices[2 * thirdMDIndex + 1];
    unsigned int fourthMDLowerHitIndex = mdsInGPU.hitIndices[2 * fourthMDIndex];
    unsigned int fourthMDUpperHitIndex = mdsInGPU.hitIndices[2 * fourthMDIndex + 1];
    unsigned int fifthMDLowerHitIndex = mdsInGPU.hitIndices[2 * fifthMDIndex];
    unsigned int fifthMDUpperHitIndex = mdsInGPU.hitIndices[2 * fifthMDIndex + 1];*/

    //radius computation from the three triplet MD anchor hits
    unsigned int innerTripletFirstSegmentAnchorHitIndex = segmentsInGPU.innerMiniDoubletAnchorHitIndices[firstSegmentIndex];
    unsigned int innerTripletSecondSegmentAnchorHitIndex = segmentsInGPU.outerMiniDoubletAnchorHitIndices[firstSegmentIndex]; //same as second segment inner MD anchorhit index
    unsigned int innerTripletThirdSegmentAnchorHitIndex = segmentsInGPU.outerMiniDoubletAnchorHitIndices[secondSegmentIndex]; //same as third segment inner MD anchor hit index

    unsigned int outerTripletSecondSegmentAnchorHitIndex = segmentsInGPU.outerMiniDoubletAnchorHitIndices[thirdSegmentIndex]; //same as fourth segment inner MD anchor hit index
    unsigned int outerTripletThirdSegmentAnchorHitIndex = segmentsInGPU.outerMiniDoubletAnchorHitIndices[fourthSegmentIndex];

    float x1 = hitsInGPU.xs[innerTripletFirstSegmentAnchorHitIndex];
    float x2 = hitsInGPU.xs[innerTripletSecondSegmentAnchorHitIndex];
    float x3 = hitsInGPU.xs[innerTripletThirdSegmentAnchorHitIndex];
    float x4 = hitsInGPU.xs[outerTripletSecondSegmentAnchorHitIndex];
    float x5 = hitsInGPU.xs[outerTripletThirdSegmentAnchorHitIndex];

    float y1 = hitsInGPU.ys[innerTripletFirstSegmentAnchorHitIndex];
    float y2 = hitsInGPU.ys[innerTripletSecondSegmentAnchorHitIndex];
    float y3 = hitsInGPU.ys[innerTripletThirdSegmentAnchorHitIndex];
    float y4 = hitsInGPU.ys[outerTripletSecondSegmentAnchorHitIndex];
    float y5 = hitsInGPU.ys[outerTripletThirdSegmentAnchorHitIndex];


    //construct the arrays
    float x1Vec[] = {x1, x1, x1};
    float y1Vec[] = {y1, y1, y1};
    float x2Vec[] = {x2, x2, x2};
    float y2Vec[] = {y2, y2, y2};
    float x3Vec[] = {x3, x3, x3};
    float y3Vec[] = {y3, y3, y3};
    float x4Vec[] = {x4, x4, x4};
    float y4Vec[] = {y4, y4, y4};
    float x5Vec[] = {x5, x5, x5};
    float y5Vec[] = {y5, y5, y5};

    if(modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex1] == SDL::TwoS)
    {
        x1Vec[1] = hitsInGPU.lowEdgeXs[innerTripletFirstSegmentAnchorHitIndex];
        x1Vec[2] = hitsInGPU.highEdgeXs[innerTripletFirstSegmentAnchorHitIndex];

        y1Vec[1] = hitsInGPU.lowEdgeYs[innerTripletFirstSegmentAnchorHitIndex];
        y1Vec[2] = hitsInGPU.highEdgeYs[innerTripletFirstSegmentAnchorHitIndex];
    }

    if(modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex2] == SDL::TwoS)
    {
        x2Vec[1] = hitsInGPU.lowEdgeXs[innerTripletSecondSegmentAnchorHitIndex];
        x2Vec[2] = hitsInGPU.highEdgeXs[innerTripletSecondSegmentAnchorHitIndex];

        y2Vec[1] = hitsInGPU.lowEdgeYs[innerTripletSecondSegmentAnchorHitIndex];
        y2Vec[2] = hitsInGPU.highEdgeYs[innerTripletSecondSegmentAnchorHitIndex];

    }

    if(modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex3] == SDL::TwoS)
    {
        x3Vec[1] = hitsInGPU.lowEdgeXs[innerTripletThirdSegmentAnchorHitIndex];
        x3Vec[2] = hitsInGPU.highEdgeXs[innerTripletThirdSegmentAnchorHitIndex];

        y3Vec[1] = hitsInGPU.lowEdgeYs[innerTripletThirdSegmentAnchorHitIndex];
        y3Vec[2] = hitsInGPU.highEdgeYs[innerTripletThirdSegmentAnchorHitIndex];
    }

    if(modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex4] == SDL::TwoS)
    {
        x4Vec[1] = hitsInGPU.lowEdgeXs[outerTripletSecondSegmentAnchorHitIndex];
        x4Vec[2] = hitsInGPU.highEdgeXs[outerTripletSecondSegmentAnchorHitIndex];

        y4Vec[1] = hitsInGPU.lowEdgeYs[outerTripletSecondSegmentAnchorHitIndex];
        y4Vec[2] = hitsInGPU.highEdgeYs[outerTripletSecondSegmentAnchorHitIndex];
    }

    if(modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex4] == SDL::TwoS)
    {
        x5Vec[1] = hitsInGPU.lowEdgeXs[outerTripletThirdSegmentAnchorHitIndex];
        x5Vec[2] = hitsInGPU.highEdgeXs[outerTripletThirdSegmentAnchorHitIndex];

        y5Vec[1] = hitsInGPU.lowEdgeYs[outerTripletThirdSegmentAnchorHitIndex];
        y5Vec[2] = hitsInGPU.highEdgeYs[outerTripletThirdSegmentAnchorHitIndex];
    }


    float innerG, innerF; //centers of inner circle
    float outerG, outerF; //centers of outer circle
    float bridgeG, bridgeF;

    innerRadius = computeRadiusFromThreeAnchorHits(x1, y1, x2, y2, x3, y3, innerG, innerF);
    outerRadius = computeRadiusFromThreeAnchorHits(x3, y3, x4, y4, x5, y5, outerG, outerF);
    bridgeRadius = computeRadiusFromThreeAnchorHits(x2, y2, x3, y3, x4, y4, bridgeG, bridgeF);

    float omega1 = (innerG - x1Vec[0]) * (y3Vec[0] - y1Vec[0]) - (innerF - y1Vec[0]) * (x3Vec[0] - x1Vec[0]);

    computeErrorInRadius(x1Vec, y1Vec, x2Vec, y2Vec, x3Vec, y3Vec, innerRadiusMin2S, innerRadiusMax2S, omega1);
    computeErrorInRadius(x2Vec, y2Vec, x3Vec, y3Vec, x4Vec, y4Vec, bridgeRadiusMin2S, bridgeRadiusMax2S, omega1);
    computeErrorInRadius(x3Vec, y3Vec, x4Vec, y4Vec, x5Vec, y5Vec, outerRadiusMin2S, outerRadiusMax2S, omega1);

    /*
        innerRadius : (BBBBB, BBBBE, BBBEE) => common factor (BBB), BBEEEE, BEEEE, EEEEE
        bridgeRadius : (BBBBB, BBBBE) => common factor (BBB), BBBEE, BBEEE, EEEEE
        outerRadius : BBBBB, BBBBE, BBBEE, BBEEEE, EEEEE
        In addition, compute endcap uncertainties!!!!!!!!!!!!
    */   

    if(innerRadius < 0.95/(2 * k2Rinv1GeVf))
    {
        pass = false;
    } 
   
    //split by category
    bool tempPass;
    if(modulesInGPU.subdets[lowerModuleIndex1] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex2] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex3] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex4] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex5] == SDL::Barrel)
    {
       tempPass = matchRadiiBBBBB(innerRadius, bridgeRadius, outerRadius, innerRadiusMin, innerRadiusMax, bridgeRadiusMin, bridgeRadiusMax, outerRadiusMin, outerRadiusMax);    
    }
    else if(modulesInGPU.subdets[lowerModuleIndex1] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex2] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex3] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex4] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap)
    {
        tempPass = matchRadiiBBBBE(innerRadius, bridgeRadius, outerRadius, innerRadiusMin2S, innerRadiusMax2S, bridgeRadiusMin2S, bridgeRadiusMax2S, outerRadiusMin2S, outerRadiusMax2S, innerRadiusMin, innerRadiusMax, bridgeRadiusMin, bridgeRadiusMax, outerRadiusMin, outerRadiusMax);
    }
    else if(modulesInGPU.subdets[lowerModuleIndex1] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex2] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex3] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap and modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap)
    {
        tempPass = matchRadiiBBBEE(innerRadius, bridgeRadius, outerRadius,innerRadiusMin2S, innerRadiusMax2S, bridgeRadiusMin2S, bridgeRadiusMax2S, outerRadiusMin2S, outerRadiusMax2S, innerRadiusMin, innerRadiusMax, bridgeRadiusMin, bridgeRadiusMax, outerRadiusMin, outerRadiusMax);
    }
    else if(modulesInGPU.subdets[lowerModuleIndex1] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex2] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap and modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap)
    {
        tempPass = matchRadiiBBEEE(innerRadius, bridgeRadius, outerRadius, innerRadiusMin2S, innerRadiusMax2S, bridgeRadiusMin2S, bridgeRadiusMax2S, outerRadiusMin2S, outerRadiusMax2S, innerRadiusMin, innerRadiusMax, bridgeRadiusMin, bridgeRadiusMax, outerRadiusMin, outerRadiusMax);
    }
    else if(modulesInGPU.subdets[lowerModuleIndex1] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap and modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap)
    {
        tempPass = matchRadiiBEEEE(innerRadius, bridgeRadius, outerRadius, innerRadiusMin2S, innerRadiusMax2S, bridgeRadiusMin2S, bridgeRadiusMax2S, outerRadiusMin2S, outerRadiusMax2S, innerRadiusMin, innerRadiusMax, bridgeRadiusMin, bridgeRadiusMax, outerRadiusMin, outerRadiusMax);
    }
    else    
    {
        tempPass = matchRadiiEEEEE(innerRadius, bridgeRadius, outerRadius, innerRadiusMin2S, innerRadiusMax2S, bridgeRadiusMin2S, bridgeRadiusMax2S, outerRadiusMin2S, outerRadiusMax2S,innerRadiusMin, innerRadiusMax, bridgeRadiusMin, bridgeRadiusMax, outerRadiusMin, outerRadiusMax);
    }


    //pass = true;
    return pass;
}

__device__ bool SDL::checkIntervalOverlap(const float& firstMin, const float& firstMax, const float& secondMin, const float& secondMax)
{
    if(firstMin < secondMin)
    {
        return secondMin < firstMax;
    }
    else
    {
        return firstMin < secondMax;
    }
}

__device__ bool SDL::matchRadiiBBBBB(const float& innerRadius, const float& bridgeRadius, const float& outerRadius, float& innerRadiusMin, float& innerRadiusMax, float& bridgeRadiusMin, float& bridgeRadiusMax, float& outerRadiusMin, float& outerRadiusMax)
{
    bool pass = true;
    float innerInvRadiusErrorBound =  0.1512;
    float bridgeInvRadiusErrorBound = 0.1781;
    float outerInvRadiusErrorBound = 0.1840;

    innerRadiusMin = innerRadius/(1 + innerInvRadiusErrorBound);
    innerRadiusMax = innerInvRadiusErrorBound < 1 ? innerRadius/(1 - innerInvRadiusErrorBound) : 123456789;

    bridgeRadiusMin = bridgeRadius/(1 + bridgeInvRadiusErrorBound);
    bridgeRadiusMax = bridgeInvRadiusErrorBound < 1 ? bridgeRadius/(1 - bridgeInvRadiusErrorBound) : 123456789;

    outerRadiusMin = outerRadius/(1 + outerInvRadiusErrorBound);
    outerRadiusMax = outerInvRadiusErrorBound < 1 ? outerRadius/(1 - outerInvRadiusErrorBound) : 123456789;
    
    if(not checkIntervalOverlap(innerRadiusMin, innerRadiusMax, bridgeRadiusMin, bridgeRadiusMax))
    {
        pass = false;
    }
    
    if(not checkIntervalOverlap(innerRadiusMin, innerRadiusMax, outerRadiusMin, outerRadiusMax))
    {
        pass = false;
    }
    
    return pass;
}

__device__ bool SDL::matchRadiiBBBBE(const float& innerRadius, const float& bridgeRadius, const float& outerRadius, const float& innerRadiusMin2S, const float& innerRadiusMax2S, const float& bridgeRadiusMin2S, const float& bridgeRadiusMax2S, const float& outerRadiusMin2S, const float& outerRadiusMax2S, float& innerRadiusMin, float& innerRadiusMax, float& bridgeRadiusMin, float& bridgeRadiusMax, float& outerRadiusMin, float& outerRadiusMax)
{
    bool pass = true;

    float innerInvRadiusErrorBound =  0.1781;
    float bridgeInvRadiusErrorBound = 0.2167;
    float outerInvRadiusErrorBound = 1.1116;

    innerRadiusMin = innerRadius/(1 + innerInvRadiusErrorBound);
    innerRadiusMax = innerInvRadiusErrorBound < 1 ? innerRadius/(1 - innerInvRadiusErrorBound) : 123456789; //large number signifying infty

    bridgeRadiusMin = bridgeRadius/(1 + bridgeInvRadiusErrorBound);
    bridgeRadiusMax = bridgeInvRadiusErrorBound < 1 ? bridgeRadius/(1 - bridgeInvRadiusErrorBound) : 123456789;

    outerRadiusMin = outerRadius/(1 + outerInvRadiusErrorBound);
    outerRadiusMax = outerInvRadiusErrorBound < 1 ? outerRadius/(1 - outerInvRadiusErrorBound) : 123456789;

    if(not checkIntervalOverlap(innerRadiusMin, innerRadiusMax, bridgeRadiusMin, bridgeRadiusMax))
    {
        pass = false;
    }

    return pass;
}

__device__ bool SDL::matchRadiiBBBEE(const float& innerRadius, const float& bridgeRadius, const float& outerRadius, const float& innerRadiusMin2S, const float& innerRadiusMax2S, const float& bridgeRadiusMin2S, const float& bridgeRadiusMax2S, const float& outerRadiusMin2S, const float& outerRadiusMax2S, float& innerRadiusMin, float& innerRadiusMax, float& bridgeRadiusMin, float& bridgeRadiusMax, float& outerRadiusMin, float& outerRadiusMax)
{
    bool pass = false;

    float innerInvRadiusErrorBound =  0.1840;
    float bridgeInvRadiusErrorBound = 0.5971;
    float outerInvRadiusErrorBound = 11.7102;

    innerRadiusMin = innerRadius/(1 + innerInvRadiusErrorBound);
    innerRadiusMax = innerInvRadiusErrorBound < 1 ? innerRadius/(1 - innerInvRadiusErrorBound) : 123456789;

    bridgeRadiusMin = bridgeRadius/(1 + bridgeInvRadiusErrorBound);
    bridgeRadiusMax = bridgeInvRadiusErrorBound < 1 ? bridgeRadius/(1 - bridgeInvRadiusErrorBound) : 123456789;

    outerRadiusMin = outerRadius/(1 + outerInvRadiusErrorBound);
    outerRadiusMax = outerInvRadiusErrorBound < 1 ? outerRadius/(1 - outerInvRadiusErrorBound) : 123456789;

    if(not checkIntervalOverlap(innerRadiusMin, innerRadiusMax, fminf(bridgeRadiusMin, bridgeRadiusMin2S), fmaxf(bridgeRadiusMax, bridgeRadiusMax2S)))
    {
        pass = false;
    }

    if(not checkIntervalOverlap(innerRadiusMin, innerRadiusMax, fminf(outerRadiusMin, outerRadiusMin2S), fmaxf(outerRadiusMax, outerRadiusMax2S)))
    {
        pass = false;
    }


    return pass;
}

__device__ bool SDL::matchRadiiBBEEE(const float& innerRadius, const float& bridgeRadius, const float& outerRadius, const float& innerRadiusMin2S, const float& innerRadiusMax2S, const float& bridgeRadiusMin2S, const float& bridgeRadiusMax2S, const float& outerRadiusMin2S, const float& outerRadiusMax2S, float& innerRadiusMin, float& innerRadiusMax, float& bridgeRadiusMin, float& bridgeRadiusMax, float& outerRadiusMin, float& outerRadiusMax)
{
    bool pass = false;

    float innerInvRadiusErrorBound =  0.6376;
    float bridgeInvRadiusErrorBound = 2.1381;
    float outerInvRadiusErrorBound = 20.4179;

    innerRadiusMin = innerRadius/(1 + innerInvRadiusErrorBound);
    innerRadiusMax = innerInvRadiusErrorBound < 1 ? innerRadius/(1 - innerInvRadiusErrorBound) : 123456789;

    bridgeRadiusMin = bridgeRadius/(1 + bridgeInvRadiusErrorBound);
    bridgeRadiusMax = bridgeInvRadiusErrorBound < 1 ? bridgeRadius/(1 - bridgeInvRadiusErrorBound) : 123456789;

    outerRadiusMin = outerRadius/(1 + outerInvRadiusErrorBound);
    outerRadiusMax = outerInvRadiusErrorBound < 1 ? outerRadius/(1 - outerInvRadiusErrorBound) : 123456789;

    if(not checkIntervalOverlap(innerRadiusMin, innerRadiusMax, fminf(bridgeRadiusMin, bridgeRadiusMin2S), fmaxf(bridgeRadiusMax, bridgeRadiusMax2S)))
    {
        pass = false;
    }

    if(not checkIntervalOverlap(innerRadiusMin, innerRadiusMax, fminf(outerRadiusMin, outerRadiusMin2S), fmaxf(outerRadiusMax, outerRadiusMax2S)))
    {
        pass = false;
    }

    return pass;
}

__device__ bool SDL::matchRadiiBEEEE(const float& innerRadius, const float& bridgeRadius, const float& outerRadius, const float& innerRadiusMin2S, const float& innerRadiusMax2S, const float& bridgeRadiusMin2S, const float& bridgeRadiusMax2S, const float& outerRadiusMin2S, const float& outerRadiusMax2S, float& innerRadiusMin, float& innerRadiusMax, float& bridgeRadiusMin, float& bridgeRadiusMax, float& outerRadiusMin, float& outerRadiusMax) 
{
    bool pass = false;

    float innerInvRadiusErrorBound =  1.9382;
    float bridgeInvRadiusErrorBound = 3.7280;
    float outerInvRadiusErrorBound = 5.7030;

    innerRadiusMin = innerRadius/(1 + innerInvRadiusErrorBound);
    innerRadiusMax = innerInvRadiusErrorBound < 1 ? innerRadius/(1 - innerInvRadiusErrorBound) : 123456789;

    bridgeRadiusMin = bridgeRadius/(1 + bridgeInvRadiusErrorBound);
    bridgeRadiusMax = bridgeInvRadiusErrorBound < 1 ? bridgeRadius/(1 - bridgeInvRadiusErrorBound) : 123456789;

    outerRadiusMin = outerRadius/(1 + outerInvRadiusErrorBound);
    outerRadiusMax = outerInvRadiusErrorBound < 1 ? outerRadius/(1 - outerInvRadiusErrorBound) : 123456789;

    return pass;
}

__device__ bool SDL::matchRadiiEEEEE(const float& innerRadius, const float& bridgeRadius, const float& outerRadius, const float& innerRadiusMin2S, const float& innerRadiusMax2S, const float& bridgeRadiusMin2S, const float& bridgeRadiusMax2S, const float& outerRadiusMin2S, const float& outerRadiusMax2S, float& innerRadiusMin, float& innerRadiusMax, float& bridgeRadiusMin, float& bridgeRadiusMax, float& outerRadiusMin, float& outerRadiusMax)
{
    bool pass = false;

    float innerInvRadiusErrorBound =  1.9382;
    float bridgeInvRadiusErrorBound = 2.2091;
    float outerInvRadiusErrorBound = 7.4084;

    innerRadiusMin = innerRadius/(1 + innerInvRadiusErrorBound);
    innerRadiusMax = innerInvRadiusErrorBound < 1 ? innerRadius/(1 - innerInvRadiusErrorBound) : 123456789;

    bridgeRadiusMin = bridgeRadius/(1 + bridgeInvRadiusErrorBound);
    bridgeRadiusMax = bridgeInvRadiusErrorBound < 1 ? bridgeRadius/(1 - bridgeInvRadiusErrorBound) : 123456789;

    outerRadiusMin = outerRadius/(1 + outerInvRadiusErrorBound);
    outerRadiusMax = outerInvRadiusErrorBound < 1 ? outerRadius/(1 - outerInvRadiusErrorBound) : 123456789;

    if(not checkIntervalOverlap(fminf(innerRadiusMin, innerRadiusMin2S), fmaxf(innerRadiusMax, innerRadiusMax2S),  fminf(bridgeRadiusMin, bridgeRadiusMin2S), fmaxf(bridgeRadiusMax, bridgeRadiusMax2S)))
    {
        pass = false;
    }

    if(not checkIntervalOverlap(fminf(innerRadiusMin, innerRadiusMin2S), fmaxf(innerRadiusMax, innerRadiusMax2S),  fminf(outerRadiusMin, outerRadiusMin2S), fmaxf(outerRadiusMax, outerRadiusMax2S)))
    {
        pass = false;
    }

    return pass;
}

__device__ void SDL::computeErrorInRadius(float* x1Vec, float* y1Vec, float* x2Vec, float* y2Vec, float* x3Vec, float* y3Vec, float& minimumRadius, float& maximumRadius, float& omega1)
{
    //brute force
    float candidateRadius;
    minimumRadius = 0;
    maximumRadius = 0;
    float f,g; //placeholders
    float omega2;
    for(size_t i = 0; i < 3; i++)
    {
        for(size_t j = 0; j < 3; j++)
        {
            for(size_t k = 0; k < 3; k++)
            {
               candidateRadius = computeRadiusFromThreeAnchorHits(x1Vec[i], y1Vec[i], x2Vec[j], y2Vec[j], x3Vec[k], y3Vec[k],g,f);
               if(i == 0 and j == 0 and k == 0)
               {
                   minimumRadius = candidateRadius;
                   maximumRadius = candidateRadius;
                   continue;
               }
               omega2 = (g - x1Vec[i]) * (y3Vec[k] - y1Vec[i]) - (f - y1Vec[i]) * (x3Vec[k] - x1Vec[i]);
               //first things first
               if(candidateRadius < 0.95/k2Rinv1GeVf) continue;
               if(candidateRadius >= maximumRadius and omega2 * omega1 > 0)
               {
                   maximumRadius = candidateRadius;
               }

               if(candidateRadius <= minimumRadius and omega2 * omega1 > 0)
               {
                   minimumRadius = candidateRadius;
               }
    
            }
        }
    }
}


/*
__device__ float SDL::computeErrorInRadius(float* x1Vec, float* y1Vec, float* x2Vec, float* y2Vec, float* x3Vec, float* y3Vec, float& gError, float& fError)
{
    // Numerical differentiation baby! Scientific computing course put into good use
    float h = x1Vec[0]/100;
    float gUp, gDown;
    float fUp, fDown;

    float dgBydx1, dfBydx1;
    float dRBydx1 = (computeRadiusFromThreeAnchorHits(x1Vec[0] + h, y1Vec[0], x2Vec[0], y2Vec[0], x3Vec[0], y3Vec[0], gUp, fUp) - computeRadiusFromThreeAnchorHits(x1Vec[0] - h, y1Vec[0], x2Vec[0], y2Vec[0], x3Vec[0], y3Vec[0], gDown, fDown))/(2 * h);
    dgBydx1 = (gUp - gDown)/(2 * h);
    dfBydx1 = (fUp - fDown)/(2 * h);

    h = x2Vec[0]/100;
    float dgBydx2, dfBydx2;
    float dRBydx2 = (computeRadiusFromThreeAnchorHits(x1Vec[0], y1Vec[0], x2Vec[0] + h, y2Vec[0], x3Vec[0], y3Vec[0], gUp, fUp) - computeRadiusFromThreeAnchorHits(x1Vec[0], y1Vec[0], x2Vec[0] - h, y2Vec[0], x3Vec[0], y3Vec[0], gDown, fDown))/(2 * h);
    dgBydx2 = (gUp - gDown)/(2 * h);
    dfBydx2 = (fUp - fDown)/(2 * h);

    h = x3Vec[0]/100;
    float dgBydx3, dfBydx3;
    float dRBydx3 = (computeRadiusFromThreeAnchorHits(x1Vec[0], y1Vec[0], x2Vec[0], y2Vec[0], x3Vec[0] + h, y3Vec[0], gUp, fUp) - computeRadiusFromThreeAnchorHits(x1Vec[0], y1Vec[0], x2Vec[0], y2Vec[0], x3Vec[0] - h, y3Vec[0], gDown, fDown))/(2 * h);
    dgBydx3 = (gUp - gDown)/(2 * h);
    dfBydx3 = (fUp - fDown)/(2 * h);

    h = y1Vec[0]/100;
    float dgBydy1, dfBydy1;
    float dRBydy1 = (computeRadiusFromThreeAnchorHits(x1Vec[0], y1Vec[0] + h, x2Vec[0], y2Vec[0], x3Vec[0], y3Vec[0], gUp, fUp) - computeRadiusFromThreeAnchorHits(x1Vec[0], y1Vec[0] - h, x2Vec[0], y2Vec[0], x3Vec[0], y3Vec[0], gDown, fDown))/(2 * h);
    dgBydy1 = (gUp - gDown)/(2 * h);
    dfBydy1 = (fUp - fDown)/(2 * h);

    h = y2Vec[0]/100;
    float dgBydy2, dfBydy2;
    float dRBydy2 = (computeRadiusFromThreeAnchorHits(x1Vec[0], y1Vec[0], x2Vec[0], y2Vec[0] + h, x3Vec[0], y3Vec[0], gUp, fUp) - computeRadiusFromThreeAnchorHits(x1Vec[0], y1Vec[0], x2Vec[0], y2Vec[0] - h, x3Vec[0], y3Vec[0], gDown, fDown))/(2 * h);
    dgBydy2 = (gUp - gDown)/(2 * h);
    dfBydy2 = (fUp - fDown)/(2 * h);

    h = y3Vec[0]/100;
    float dgBydy3, dfBydy3;
    float dRBydy3 = (computeRadiusFromThreeAnchorHits(x1Vec[0], y1Vec[0], x2Vec[0], y2Vec[0], x3Vec[0], y3Vec[0] + h, gUp, fUp) - computeRadiusFromThreeAnchorHits(x1Vec[0], y1Vec[0], x2Vec[0], y2Vec[0], x3Vec[0], y3Vec[0] - h, gDown, fDown))/(2 * h);
    dgBydy3 = (gUp - gDown)/(2 * h);
    dfBydy3 = (fUp - fDown)/(2 * h);


    float radiusError2 = (dRBydx1 * dRBydx1) * (x1Vec[2] - x1Vec[1]) * (x1Vec[2] - x1Vec[1]) + (dRBydx2 * dRBydx2) * (x2Vec[2] - x2Vec[1]) * (x2Vec[2] - x2Vec[1]) + (dRBydx3 * dRBydx3) * (x3Vec[2] - x3Vec[1]) * (x3Vec[2] - x3Vec[1]) + (dRBydy1 * dRBydy1) * (y1Vec[2] - y1Vec[1]) * (y1Vec[2] - y1Vec[1]) + (dRBydy2 * dRBydy2) * (y2Vec[2] - y2Vec[1]) * (y2Vec[2] - y2Vec[1]) + (dRBydy3 * dRBydy3) * (y3Vec[2] - y3Vec[1]) * (y3Vec[2] - y3Vec[1]);

    radiusError2/= 4;

    float gError2 = (dgBydx1 * dgBydx1) * (x1Vec[2] - x1Vec[1]) * (x1Vec[2] - x1Vec[1]) + (dgBydx2 * dgBydx2) * (x2Vec[2] - x2Vec[1]) * (x2Vec[2] - x2Vec[1]) + (dgBydx3 * dgBydx3) * (x3Vec[2] - x3Vec[1]) * (x3Vec[2] - x3Vec[1]) + (dgBydy1 * dgBydy1) * (y1Vec[2] - y1Vec[1]) * (y1Vec[2] - y1Vec[1]) + (dgBydy2 * dgBydy2) * (y2Vec[2] - y2Vec[1]) * (y2Vec[2] - y2Vec[1]) + (dgBydy3 * dgBydy3) * (y3Vec[2] - y3Vec[1]) * (y3Vec[2] - y3Vec[1]);

    gError2/= 4;

    float fError2 = (dfBydx1 * dfBydx1) * (x1Vec[2] - x1Vec[1]) * (x1Vec[2] - x1Vec[1]) + (dfBydx2 * dfBydx2) * (x2Vec[2] - x2Vec[1]) * (x2Vec[2] - x2Vec[1]) + (dfBydx3 * dfBydx3) * (x3Vec[2] - x3Vec[1]) * (x3Vec[2] - x3Vec[1]) + (dfBydy1 * dfBydy1) * (y1Vec[2] - y1Vec[1]) * (y1Vec[2] - y1Vec[1]) + (dfBydy2 * dfBydy2) * (y2Vec[2] - y2Vec[1]) * (y2Vec[2] - y2Vec[1]) + (dfBydy3 * dfBydy3) * (y3Vec[2] - y3Vec[1]) * (y3Vec[2] - y3Vec[1]);

    fError2/= 4;

    gError = sqrtf(gError2);
    fError = sqrtf(fError2);

    return sqrtf(radiusError2);
}*/


__device__ float SDL::computeRadiusUsingRegression(int nPoints, float* xs, float* ys, float& g, float& f)
{
    float radius = 0;

    //3 variable linear regression
    //http://faculty.cas.usf.edu/mbrannick/regression/Part3/Reg2.html

    //some extra variables
    //the two variables will be caled x1 and x2, and y (which is x^2 + y^2)
    float sigmaX1Squared = 0.f;
    float sigmaX2Squared = 0.f;
    float sigmaX1X2 = 0.f; 
    float sigmaX1y = 0.f; 
    float sigmaX2y = 0.f;
    float ybar = 0.f;
    float x1bar = 0.f;
    float x2bar = 0.f;

    for(size_t i = 0; i < nPoints; i++)
    {
        sigmaX1Squared += xs[i] * xs[i];
        sigmaX2Squared += ys[i] * ys[i];
        sigmaX1X2 += xs[i] * ys[i];
        sigmaX1y += xs[i] * (xs[i] * xs[i] + ys[i] * ys[i]);
        sigmaX2y += ys[i] * (xs[i] * xs[i] + ys[i] * ys[i]);
        ybar += (xs[i] * xs[i] + ys[i] * ys[i]);
        x1bar += (xs[i]);
        x2bar += (ys[i]);
    }
    ybar /= nPoints;
    x1bar /= nPoints;
    x2bar /= nPoints;
    float varX1 = (sigmaX1Squared / nPoints) - (x1bar) * (x1bar);
    float varX2 = (sigmaX2Squared / nPoints) - (x2bar) * (x2bar);
    float covX1X2 = (sigmaX1X2 / nPoints) - (x1bar) * (x2bar);
    float covX1y = sigmaX1y / nPoints - x1bar * ybar;
    float covX2y = sigmaX2y / nPoints - x2bar * ybar;
    float denom = varX1 * varX2 - covX1X2 * covX1X2;
    
    g = (varX2 * covX1y - covX1X2 * covX2y)/(2 * denom);
    f = (varX1 * covX2y - covX1X2 * covX1y)/(2 * denom); 
    float c = -(ybar - 2 * g * x1bar - 2 * f * x2bar);
    if(g * g + f * f - c < 0)
    {
        printf("FATAL! r^2 < 0!\n");
        return -1;
    }
    
    radius = sqrtf(g * g  + f * f - c);
    return radius;

}
__device__ float SDL::computeRadiusFromThreeAnchorHits(float x1, float y1, float x2, float y2, float x3, float y3, float& g, float& f)
{
    float radius = 0;

    //writing manual code for computing radius, which obviously sucks
    //TODO:Use fancy inbuilt libraries like cuBLAS or cuSOLVE for this!
    //(g,f) -> center
    //first anchor hit - (x1,y1), second anchor hit - (x2,y2), third anchor hit - (x3, y3)


    if((y1 - y3) * (x2 - x3) - (x1 - x3) * (y2 - y3) == 0)
    {
        return -1; //WTF man three collinear points!
    }

    float denom = ((y1 - y3) * (x2 - x3) - (x1 - x3) * (y2 - y3));

    g = 0.5 * ((y3 - y2) * (x1 * x1 + y1 * y1) + (y1 - y3) * (x2 * x2 + y2 * y2) + (y2 - y1) * (x3 * x3 + y3 * y3))/denom;

    f = 0.5 * ((x2 - x3) * (x1 * x1 + y1 * y1) + (x3 - x1) * (x2 * x2 + y2 * y2) + (x1 - x2) * (x3 * x3 + y3 * y3))/denom;

    float c = ((x2 * y3 - x3 * y2) * (x1 * x1 + y1 * y1) + (x3 * y1 - x1 * y3) * (x2 * x2 + y2 * y2) + (x1 * y2 - x2 * y1) * (x3 * x3 + y3 * y3))/denom;

    if(g * g + f * f - c < 0)
    {
        printf("FATAL! r^2 < 0!\n");
        return -1;
    }
    
    radius = sqrtf(g * g  + f * f - c);
    return radius;
}

__device__ bool SDL::T5HasCommonMiniDoublet(struct SDL::triplets& tripletsInGPU, struct SDL::segments& segmentsInGPU, unsigned int innerTripletIndex, unsigned int outerTripletIndex)
{
    unsigned int innerOuterSegmentIndex = tripletsInGPU.segmentIndices[2 * innerTripletIndex + 1];
    unsigned int outerInnerSegmentIndex = tripletsInGPU.segmentIndices[2 * outerTripletIndex];
    unsigned int innerOuterOuterMiniDoubletIndex = segmentsInGPU.mdIndices[2 * innerOuterSegmentIndex + 1]; //inner triplet outer segment outer MD index
    unsigned int outerInnerInnerMiniDoubletIndex = segmentsInGPU.mdIndices[2 * outerInnerSegmentIndex]; //outer triplet inner segmnet inner MD index
   

    return (innerOuterOuterMiniDoubletIndex == outerInnerInnerMiniDoubletIndex);
}


