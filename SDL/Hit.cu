#include "hip/hip_runtime.h"
# include "Hit.cuh"
# include "allocate.h"
#ifdef __HIPCC__
#define CUDA_CONST_VAR __device__
#endif

SDL::hits::hits()
{
    nHits = nullptr;
//    n2SHits = nullptr;
    xs = nullptr;
    ys = nullptr;
    zs = nullptr;
    moduleIndices = nullptr;
    rts = nullptr;
    phis = nullptr;
    etas = nullptr;
//    edge2SMap = nullptr;
    highEdgeXs = nullptr;
    highEdgeYs = nullptr;
    lowEdgeXs = nullptr;
    lowEdgeYs = nullptr;
}

SDL::hits::~hits()
{
}
//FIXME:New array!
void SDL::createHitsInUnifiedMemory(struct hits& hitsInGPU,unsigned int nMaxHits,unsigned int nMax2SHits,hipStream_t stream)
{
#ifdef CACHE_ALLOC
//    hipStream_t stream=0;
    hitsInGPU.xs = (float*)cms::cuda::allocate_managed(nMaxHits*sizeof(float),stream);
    hitsInGPU.ys = (float*)cms::cuda::allocate_managed(nMaxHits*sizeof(float),stream);
    hitsInGPU.zs = (float*)cms::cuda::allocate_managed(nMaxHits*sizeof(float),stream);

    hitsInGPU.rts = (float*)cms::cuda::allocate_managed(nMaxHits*sizeof(float),stream);
    hitsInGPU.phis = (float*)cms::cuda::allocate_managed(nMaxHits*sizeof(float),stream);
    hitsInGPU.etas = (float*)cms::cuda::allocate_managed(nMaxHits*sizeof(float),stream);

    hitsInGPU.moduleIndices = (unsigned int*)cms::cuda::allocate_managed(nMaxHits*sizeof(unsigned int),stream);
    hitsInGPU.idxs = (unsigned int*)cms::cuda::allocate_managed(nMaxHits*sizeof(unsigned int),stream);

    hitsInGPU.highEdgeXs = (float*)cms::cuda::allocate_managed(nMaxHits*sizeof(float),stream);
    hitsInGPU.highEdgeYs = (float*)cms::cuda::allocate_managed(nMaxHits*sizeof(float),stream);
    hitsInGPU.lowEdgeXs = (float*)cms::cuda::allocate_managed(nMaxHits*sizeof(float),stream);
    hitsInGPU.lowEdgeYs = (float*)cms::cuda::allocate_managed(nMaxHits*sizeof(float),stream);

    hitsInGPU.nHits = (unsigned int*)cms::cuda::allocate_managed(sizeof(unsigned int),stream);
#else
    //nMaxHits and nMax2SHits are the maximum possible numbers
    hipMallocManaged(&hitsInGPU.xs, nMaxHits * sizeof(float));
    hipMallocManaged(&hitsInGPU.ys, nMaxHits * sizeof(float));
    hipMallocManaged(&hitsInGPU.zs, nMaxHits * sizeof(float));
    hipMallocManaged(&hitsInGPU.moduleIndices, nMaxHits * sizeof(unsigned int));
    //TODO:This dude (idxs) is not used in the GPU at all. It is only used for simhit matching to make efficiency plots
    //We can even skip this one later
    hipMallocManaged(&hitsInGPU.idxs, nMaxHits * sizeof(unsigned int));

    hipMallocManaged(&hitsInGPU.rts, nMaxHits * sizeof(float));
    hipMallocManaged(&hitsInGPU.phis, nMaxHits * sizeof(float));
    hipMallocManaged(&hitsInGPU.etas, nMaxHits * sizeof(float));

    hipMallocManaged(&hitsInGPU.highEdgeXs, nMaxHits * sizeof(float));
    hipMallocManaged(&hitsInGPU.highEdgeYs, nMaxHits * sizeof(float));
    hipMallocManaged(&hitsInGPU.lowEdgeXs, nMaxHits * sizeof(float));
    hipMallocManaged(&hitsInGPU.lowEdgeYs, nMaxHits * sizeof(float));

    //counters
    hipMallocManaged(&hitsInGPU.nHits, sizeof(unsigned int));
#endif
    *hitsInGPU.nHits = 0;
}
void SDL::createHitsInExplicitMemory(struct hits& hitsInGPU, unsigned int nMaxHits,hipStream_t stream)
{
#ifdef CACHE_ALLOC
 //   hipStream_t stream=0;
    int dev;
    hipGetDevice(&dev);
    hitsInGPU.xs = (float*)cms::cuda::allocate_device(dev,nMaxHits*sizeof(float),stream);
    hitsInGPU.ys = (float*)cms::cuda::allocate_device(dev,nMaxHits*sizeof(float),stream);
    hitsInGPU.zs = (float*)cms::cuda::allocate_device(dev,nMaxHits*sizeof(float),stream);

    hitsInGPU.rts = (float*)cms::cuda::allocate_device(dev,nMaxHits*sizeof(float),stream);
    hitsInGPU.phis = (float*)cms::cuda::allocate_device(dev,nMaxHits*sizeof(float),stream);
    hitsInGPU.etas = (float*)cms::cuda::allocate_device(dev,nMaxHits*sizeof(float),stream);

    hitsInGPU.moduleIndices = (unsigned int*)cms::cuda::allocate_device(dev,nMaxHits*sizeof(unsigned int),stream);
    hitsInGPU.idxs = (unsigned int*)cms::cuda::allocate_device(dev,nMaxHits*sizeof(unsigned int),stream);

    hitsInGPU.highEdgeXs = (float*)cms::cuda::allocate_device(dev,nMaxHits*sizeof(float),stream);
    hitsInGPU.highEdgeYs = (float*)cms::cuda::allocate_device(dev,nMaxHits*sizeof(float),stream);
    hitsInGPU.lowEdgeXs = (float*)cms::cuda::allocate_device(dev,nMaxHits*sizeof(float),stream);
    hitsInGPU.lowEdgeYs = (float*)cms::cuda::allocate_device(dev,nMaxHits*sizeof(float),stream);

    hitsInGPU.nHits = (unsigned int*)cms::cuda::allocate_device(dev,sizeof(unsigned int),stream);
#else
    //hipMallocAsync(&hitsInGPU.xs, nMaxHits * sizeof(float),stream);
    //hipMallocAsync(&hitsInGPU.ys, nMaxHits * sizeof(float),stream);
    //hipMallocAsync(&hitsInGPU.zs, nMaxHits * sizeof(float),stream);

    //hipMallocAsync(&hitsInGPU.moduleIndices, nMaxHits * sizeof(unsigned int),stream);
    //hipMallocAsync(&hitsInGPU.idxs, nMaxHits * sizeof(unsigned int),stream);

    //hipMallocAsync(&hitsInGPU.rts, nMaxHits * sizeof(float),stream);
    //hipMallocAsync(&hitsInGPU.phis, nMaxHits * sizeof(float),stream);
    //hipMallocAsync(&hitsInGPU.etas, nMaxHits * sizeof(float),stream);

    //hipMallocAsync(&hitsInGPU.highEdgeXs, nMaxHits * sizeof(float),stream);
    //hipMallocAsync(&hitsInGPU.highEdgeYs, nMaxHits * sizeof(float),stream);
    //hipMallocAsync(&hitsInGPU.lowEdgeXs, nMaxHits * sizeof(float),stream);
    //hipMallocAsync(&hitsInGPU.lowEdgeYs, nMaxHits * sizeof(float),stream);

    ////countersAsync
    //hipMallocAsync(&hitsInGPU.nHits, sizeof(unsigned int),stream);
    hipMalloc(&hitsInGPU.xs, nMaxHits * sizeof(float));
    hipMalloc(&hitsInGPU.ys, nMaxHits * sizeof(float));
    hipMalloc(&hitsInGPU.zs, nMaxHits * sizeof(float));

    hipMalloc(&hitsInGPU.moduleIndices, nMaxHits * sizeof(unsigned int));
    hipMalloc(&hitsInGPU.idxs, nMaxHits * sizeof(unsigned int));

    hipMalloc(&hitsInGPU.rts, nMaxHits * sizeof(float));
    hipMalloc(&hitsInGPU.phis, nMaxHits * sizeof(float));
    hipMalloc(&hitsInGPU.etas, nMaxHits * sizeof(float));

    hipMalloc(&hitsInGPU.highEdgeXs, nMaxHits * sizeof(float));
    hipMalloc(&hitsInGPU.highEdgeYs, nMaxHits * sizeof(float));
    hipMalloc(&hitsInGPU.lowEdgeXs, nMaxHits * sizeof(float));
    hipMalloc(&hitsInGPU.lowEdgeYs, nMaxHits * sizeof(float));

    //counters
    hipMalloc(&hitsInGPU.nHits, sizeof(unsigned int));
#endif
    hipMemsetAsync(hitsInGPU.nHits,0,sizeof(unsigned int),stream);
    hipStreamSynchronize(stream);
}

//__global__ void SDL::addHitToMemoryGPU(struct hits& hitsInCPU, struct modules& modulesInGPU, float x, float y, float z, unsigned int detId, unsigned int idxInNtuple,unsigned int moduleIndex,float phis,struct objectRanges& rangesInGPU) // TODO: delete I don't think this function is used any more.
//{
//    unsigned int idx = *(hitsInCPU.nHits);
////    unsigned int idxEdge2S = *(hitsInCPU.n2SHits);
//
//    hitsInCPU.xs[idx] = x;
//    hitsInCPU.ys[idx] = y;
//    hitsInCPU.zs[idx] = z;
//    hitsInCPU.rts[idx] = sqrt(x*x + y*y);
//    hitsInCPU.phis[idx] = phi(x,y,z);
//    hitsInCPU.idxs[idx] = idxInNtuple;
// //   unsigned int moduleIndex = (*detIdToIndex)[detId];
//    hitsInCPU.moduleIndices[idx] = moduleIndex;
//    if(modulesInGPU.subdets[moduleIndex] == Endcap and modulesInGPU.moduleType[moduleIndex] == TwoS)
//    {
//        float xhigh, yhigh, xlow, ylow;
//        //getEdgeHits(detId,x,y,xhigh,yhigh,xlow,ylow);
//        getEdgeHitsK(phis,x,y,xhigh,yhigh,xlow,ylow);
//        //hitsInCPU.edge2SMap[idx] = idxEdge2S;
//        //hitsInCPU.highEdgeXs[idxEdge2S] = xhigh; // due to changes to support the explicit version
//        //hitsInCPU.highEdgeYs[idxEdge2S] = yhigh;
//        //hitsInCPU.lowEdgeXs[idxEdge2S] = xlow;
//        //hitsInCPU.lowEdgeYs[idxEdge2S] = ylow;
//        hitsInCPU.highEdgeXs[idx] = xhigh;
//        hitsInCPU.highEdgeYs[idx] = yhigh;
//        hitsInCPU.lowEdgeXs[idx] = xlow;
//        hitsInCPU.lowEdgeYs[idx] = ylow;
//
//        //(*hitsInCPU.n2SHits)++;
//    }
////    else
////    {
////        hitsInCPU.edge2SMap[idx] = -1;
////    }
//
//    //set the hit ranges appropriately in the modules struct
//
//    //start the index rolling if the module is encountered for the first time
//    if(rangesInGPU.hitRanges[moduleIndex * 2] == -1)
//    {
//        rangesInGPU.hitRanges[moduleIndex * 2] = idx;
//    }
//    //always update the end index
//    rangesInGPU.hitRanges[moduleIndex * 2 + 1] = idx;
//    //printf("ranges: %u %u\n",idx,rangesInGPU.hitRanges[moduleIndex * 2]);
//    (*hitsInCPU.nHits)++;
//}
//void SDL::addHitToMemory(struct hits& hitsInGPU, struct modules& modulesInGPU, float x, float y, float z, unsigned int detId, unsigned int idxInNtuple,hipStream_t stream,struct objectRanges& rangesInGPU)
//{ //TODO: I don't think this used either
//    unsigned int idx = *(hitsInGPU.nHits);
////    unsigned int idxEdge2S = *(hitsInCPU.n2SHits);
//
//    hitsInGPU.xs[idx] = x;
//    hitsInGPU.ys[idx] = y;
//    hitsInGPU.zs[idx] = z;
//    hitsInGPU.rts[idx] = sqrt(x*x + y*y);
//    hitsInGPU.phis[idx] = phi(x,y,z);
//    hitsInGPU.idxs[idx] = idxInNtuple;
//    unsigned int moduleIndex = (*detIdToIndex)[detId];
//    hitsInGPU.moduleIndices[idx] = moduleIndex;
//
//    unsigned int nModules;
//    hipMemcpyAsync(&nModules,modulesInGPU.nModules,sizeof(unsigned int),hipMemcpyDeviceToHost,stream);
//
//    ModuleType* module_moduleType;
//    hipHostMalloc(&module_moduleType, nModules* sizeof(ModuleType));
//    hipMemcpyAsync(module_moduleType,modulesInGPU.moduleType,nModules*sizeof(ModuleType),hipMemcpyDeviceToHost,stream);
//    short* module_subdets;
//    hipHostMalloc(&module_subdets, nModules* sizeof(short));
//    hipMemcpyAsync(module_subdets,modulesInGPU.subdets,nModules*sizeof(short),hipMemcpyDeviceToHost,stream);
//    int* module_hitRanges;
//    hipHostMalloc(&module_hitRanges, nModules* 2*sizeof(int));
//    hipMemcpyAsync(module_hitRanges,rangesInGPU.hitRanges,nModules*2*sizeof(int),hipMemcpyDeviceToHost,stream);
//    hipStreamSynchronize(stream);
//
//    if(module_subdets[moduleIndex] == Endcap and module_moduleType[moduleIndex] == TwoS)
//    {
//        float xhigh, yhigh, xlow, ylow;
//        getEdgeHits(detId,x,y,xhigh,yhigh,xlow,ylow);
//        //hitsInCPU.edge2SMap[idx] = idxEdge2S;
//        //hitsInCPU.highEdgeXs[idxEdge2S] = xhigh; // due to changes to support the explicit version
//        //hitsInCPU.highEdgeYs[idxEdge2S] = yhigh;
//        //hitsInCPU.lowEdgeXs[idxEdge2S] = xlow;
//        //hitsInCPU.lowEdgeYs[idxEdge2S] = ylow;
//        hitsInGPU.highEdgeXs[idx] = xhigh;
//        hitsInGPU.highEdgeYs[idx] = yhigh;
//        hitsInGPU.lowEdgeXs[idx] = xlow;
//        hitsInGPU.lowEdgeYs[idx] = ylow;
//
//        //(*hitsInCPU.n2SHits)++;
//    }
////    else
////    {
////        hitsInCPU.edge2SMap[idx] = -1;
////    }
//
//    //set the hit ranges appropriately in the modules struct
//
//    //start the index rolling if the module is encountered for the first time
//    if(module_hitRanges[moduleIndex * 2] == -1)
//    {
//        module_hitRanges[moduleIndex * 2] = idx;
//    }
//    //always update the end index
//    module_hitRanges[moduleIndex * 2 + 1] = idx;
//    printf("ranges: %u %u\n",idx,rangesInGPU.hitRanges[moduleIndex * 2]);
//    hipMemcpyAsync(rangesInGPU.hitRanges,module_hitRanges,nModules*2*sizeof(int),hipMemcpyHostToDevice,stream);
//    hipStreamSynchronize(stream);
//    hipHostFree(module_moduleType);
//    hipHostFree(module_subdets);
//    hipHostFree(module_hitRanges);
//   (*hitsInGPU.nHits)++;
//}
__global__ void SDL::addHitToMemoryKernel(struct hits& hitsInGPU, struct modules& modulesInGPU,const float* x,const  float* y,const  float* z, const unsigned int* moduleIndex,const float* phis, const int loopsize)
{
  for (unsigned int ihit = blockIdx.x*blockDim.x + threadIdx.x; ihit <loopsize; ihit += blockDim.x*gridDim.x)
  //if(ihit < loopsize)
  {
      unsigned int idx = ihit;//*(hitsInGPU.nHits);

      hitsInGPU.xs[idx] = x[ihit];
      hitsInGPU.ys[idx] = y[ihit];
      hitsInGPU.zs[idx] = z[ihit];
      hitsInGPU.rts[idx] = sqrt(x[ihit]*x[ihit] + y[ihit]*y[ihit]);
      hitsInGPU.phis[idx] = phi(x[ihit],y[ihit],z[ihit]);
      hitsInGPU.moduleIndices[idx] = moduleIndex[ihit];
      hitsInGPU.idxs[idx] = ihit;
    //  if(modulesInGPU.subdets[moduleIndex[ihit]] == Endcap && modulesInGPU.moduleType[moduleIndex[ihit]] == TwoS)
    //  {
    //      float xhigh, yhigh, xlow, ylow;
    //      getEdgeHitsK(phis[ihit],x[ihit],y[ihit],xhigh,yhigh,xlow,ylow);
    ////      hitsInGPU.edge2SMap[idx] = idxEdge2S;
    //      //hitsInGPU.highEdgeXs[idxEdge2S] = xhigh;
    //      //hitsInGPU.highEdgeYs[idxEdge2S] = yhigh;
    //      //hitsInGPU.lowEdgeXs[idxEdge2S] = xlow;
    //      //hitsInGPU.lowEdgeYs[idxEdge2S] = ylow;
    //      hitsInGPU.highEdgeXs[idx] = xhigh;
    //      hitsInGPU.highEdgeYs[idx] = yhigh;
    //      hitsInGPU.lowEdgeXs[idx] = xlow;
    //      hitsInGPU.lowEdgeYs[idx] = ylow;

    //   //   (*hitsInGPU.n2SHits)++;
    //  }

    //  //set the hit ranges appropriately in the modules struct

    //  //start the index rolling if the module is encountered for the first time
    //  if(modulesInGPU.hitRanges[moduleIndex[ihit] * 2] == -1)
    //  {
    //      modulesInGPU.hitRanges[moduleIndex[ihit] * 2] = idx;
    //  }
    //  //always update the end index
    //  modulesInGPU.hitRanges[moduleIndex[ihit] * 2 + 1] = idx;
  }
}
//__global__ void SDL::checkHits(struct hits& hitsInGPU, const int loopsize){
//  //for (unsigned int ihit = blockIdx.x*blockDim.x + threadIdx.x; ihit <loopsize; ihit += blockDim.x*gridDim.x)
//  for (int ihit = 0; ihit <loopsize; ihit ++ )
//  {
//    printf("checkHits: %d %f %f %f %f %f %u %u %f %f %f %f\n",ihit,hitsInGPU.xs[ihit],hitsInGPU.ys[ihit],hitsInGPU.zs[ihit],hitsInGPU.rts[ihit],hitsInGPU.phis[ihit],hitsInGPU.moduleIndices[ihit],hitsInGPU.idxs[ihit],hitsInGPU.highEdgeXs[ihit],hitsInGPU.highEdgeYs[ihit],hitsInGPU.lowEdgeXs[ihit],hitsInGPU.lowEdgeYs[ihit]);
//  }
//}

/*
float SDL::ATan2(float y, float x)
{
    if (x != 0) return  atan2(y, x);
    if (y == 0) return  0;
    if (y >  0) return  M_PI / 2;
    else        return -M_PI / 2;
}

//TODO:Check if cuda atan2f will work here
float SDL::phi(float x, float y, float z)
{
    return phi_mpi_pi(M_PI + ATan2(-y, -x));
}


float SDL::phi_mpi_pi(float x)
{
    if (isnan(x))
    {
       printf("phi_mpi_pi() function called with NaN\n");
        return x;
    }

    while (x >= M_PI)
        x -= 2. * M_PI;

    while (x < -M_PI)
        x += 2. * M_PI;

    return x;
}

float SDL::deltaPhi(float x1, float y1, float z1, float x2, float y2, float z2)
{
    float phi1 = phi(x1,y1,z1);
    float phi2 = phi(x2,y2,z2);
    return phi_mpi_pi((phi2 - phi1));
}

float SDL::deltaPhiChange(float x1, float y1, float z1, float x2, float y2, float z2)
{
    return deltaPhi(x1,y1,z1,x2-x1, y2-y1, z2-z1);
}
*/

__device__ void SDL::getEdgeHitsK(float phi,float x, float y, float& xhigh, float& yhigh, float& xlow, float& ylow)
{
//    float phi = endcapGeometry.getCentroidPhi(detId);
    xhigh = x + 2.5 * cos(phi);
    yhigh = y + 2.5 * sin(phi);
    xlow = x - 2.5 * cos(phi);
    ylow = y - 2.5 * sin(phi);
}
void SDL::getEdgeHits(unsigned int detId,float x, float y, float& xhigh, float& yhigh, float& xlow, float& ylow)
{
    float phi = endcapGeometry.getCentroidPhi(detId);
    xhigh = x + 2.5 * cos(phi);
    yhigh = y + 2.5 * sin(phi);
    xlow = x - 2.5 * cos(phi);
    ylow = y - 2.5 * sin(phi);
}

void SDL::printHit(struct hits& hitsInGPU, struct modules& modulesInGPU, unsigned int hitIndex)
{
    std::cout << "Hit(x=" << hitsInGPU.xs[hitIndex] << ", y=" << hitsInGPU.ys[hitIndex] << ", z=" << hitsInGPU.zs[hitIndex] << ", rt=" << hitsInGPU.rts[hitIndex] << ", phi=" << hitsInGPU.phis[hitIndex] <<", module subdet = "<<modulesInGPU.subdets[hitsInGPU.moduleIndices[hitIndex]]<<", module layer = "<< modulesInGPU.layers[hitsInGPU.moduleIndices[hitIndex]]<<", module ring = "<< modulesInGPU.rings[hitsInGPU.moduleIndices[hitIndex]]<<" )"<<std::endl;
}


void SDL::hits::freeMemoryCache()
{
#ifdef Explicit_Hit
    int dev;
    hipGetDevice(&dev);
    cms::cuda::free_device(dev,nHits);
    cms::cuda::free_device(dev,xs);
    cms::cuda::free_device(dev,ys);
    cms::cuda::free_device(dev,zs);
    cms::cuda::free_device(dev,moduleIndices);
    cms::cuda::free_device(dev,rts);
    cms::cuda::free_device(dev,idxs);
    cms::cuda::free_device(dev,phis);
    cms::cuda::free_device(dev,etas);

    cms::cuda::free_device(dev,highEdgeXs);
    cms::cuda::free_device(dev,highEdgeYs);
    cms::cuda::free_device(dev,lowEdgeXs);
    cms::cuda::free_device(dev,lowEdgeYs);
#else
    cms::cuda::free_managed(nHits);
    cms::cuda::free_managed(xs);
    cms::cuda::free_managed(ys);
    cms::cuda::free_managed(zs);
    cms::cuda::free_managed(moduleIndices);
    cms::cuda::free_managed(rts);
    cms::cuda::free_managed(idxs);
    cms::cuda::free_managed(phis);
    cms::cuda::free_managed(etas);

    cms::cuda::free_managed(highEdgeXs);
    cms::cuda::free_managed(highEdgeYs);
    cms::cuda::free_managed(lowEdgeXs);
    cms::cuda::free_managed(lowEdgeYs);
#endif
}
void SDL::hits::freeMemory(hipStream_t stream)
{
    //hipFreeAsync(nHits,stream);
    //hipFreeAsync(xs,stream);
    //hipFreeAsync(ys,stream);
    //hipFreeAsync(zs,stream);
    //hipFreeAsync(moduleIndices,stream);
    //hipFreeAsync(rts,stream);
    //hipFreeAsync(idxs,stream);
    //hipFreeAsync(phis,stream);
    //hipFreeAsync(etas,stream);

    //hipFreeAsync(highEdgeXs,stream);
    //hipFreeAsync(highEdgeYs,stream);
    //hipFreeAsync(lowEdgeXs,stream);
    //hipFreeAsync(lowEdgeYs,stream);
    hipFree(nHits);
    hipFree(xs);
    hipFree(ys);
    hipFree(zs);
    hipFree(moduleIndices);
    hipFree(rts);
    hipFree(idxs);
    hipFree(phis);
    hipFree(etas);

    hipFree(highEdgeXs);
    hipFree(highEdgeYs);
    hipFree(lowEdgeXs);
    hipFree(lowEdgeYs);
    hipStreamSynchronize(stream);
}
