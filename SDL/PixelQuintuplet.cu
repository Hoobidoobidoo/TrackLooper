#include "hip/hip_runtime.h"
#ifdef __HIPCC__
#define CUDA_CONST_VAR __device__
#endif
# include "PixelQuintuplet.cuh"
#include "allocate.h"

SDL::pixelQuintuplets::pixelQuintuplets()
{
    pT3Indices = nullptr;
    T5Indices = nullptr;
    nPixelQuintuplets = nullptr;
}

SDL::pixelQuintuplets::~pixelQuintuplets()
{
}

void SDL::pixelQuintuplets::freeMemory()
{
    hipFree(pT3Indices);
    hipFree(T5Indices);
    hipFree(nPixelQuintuplets);
}

void SDL::createPixelQuintupletsInUnifiedMemory(struct SDL::pixelQuintuplets& pixelQuintupletsInGPU, unsigned int maxPixelQuintuplets)
{
    hipMallocManaged(&pixelQuintupletsInGPU.pT3Indices, maxPixelQuintuplets * sizeof(unsigned int));
    hipMallocManaged(&pixelQuintupletsInGPU.T5Indices, maxPixelQuintuplets * sizeof(unsigned int));
    hipMallocManaged(&pixelQuintupletsInGPU.nPixelQuintuplets, sizeof(unsigned int));
#ifdef CUT_VALUE_DEBUG
    hipMallocManaged(&pixelQuintupletsInGPU.rzChiSquared, maxPixelQuintuplets * sizeof(unsigned int));
    hipMallocManaged(&pixelQuintupletsInGPU.rPhiChiSquared, maxPixelQuintuplets * sizeof(unsigned int));
    hipMallocManaged(&pixelQuintupletsInGPU.rPhiChiSquaredInwards, maxPixelQuintuplets * sizeof(unsigned int));
#endif

    hipMemset(pixelQuintupletsInGPU.nPixelQuintuplets, 0, sizeof(unsigned int));
}

void SDL::createPixelQuintupletsInExplicitMemory(struct SDL::pixelQuintuplets& pixelQuintupletsInGPU, unsigned int maxPixelQuintuplets)
{
    hipMalloc(&pixelQuintupletsInGPU.pT3Indices, maxPixelQuintuplets * sizeof(unsigned int));
    hipMalloc(&pixelQuintupletsInGPU.T5Indices, maxPixelQuintuplets * sizeof(unsigned int));
    hipMalloc(&pixelQuintupletsInGPU.nPixelQuintuplets, sizeof(unsigned int));

    hipMemset(pixelQuintupletsInGPU.nPixelQuintuplets, 0, sizeof(unsigned int));
}

#ifdef CUT_VALUE_DEBUG
__device__ void SDL::addPixelQuintupletToMemory(struct pixelQuintuplets& pixelQuintupletsInGPU, unsigned int pT3Index, unsigned int T5Index, unsigned int pixelQuintupletIndex, float& rzChiSquared, float& rPhiChiSquared, float& rPhiChiSquaredInwards)
#else
__device__ void SDL::addPixelQuintupletToMemory(struct pixelQuintuplets& pixelQuintupletsInGPU, unsigned int pT3Index, unsigned int T5Index, unsigned int pixelQuintupletIndex)
#endif
{
    pixelQuintupletsInGPU.pT3Indices[pixelQuintupletIndex] = pT3Index;
    pixelQuintupletsInGPU.T5Indices[pixelQuintupletIndex] = T5Index;
#ifdef CUT_VALUE_DEBUG
    pixelQuintupletsInGPU.rzChiSquared[pixelQuintupletIndex] = rzChiSquared;
#endif
}

__device__ bool SDL::runPixelQuintupletDefaultAlgo(struct modules& modulesInGPU, struct hits& hitsInGPU, struct miniDoublets& mdsInGPU, struct segments& segmentsInGPU, struct triplets& tripletsInGPU, struct quintuplets& quintupletsInGPU, struct pixelTriplets& pixelTripletsInGPU, unsigned int& pixelTripletIndex, unsigned int& quintupletIndex, float& rzChiSquared, float& rPhiChiSquared, float& rPhiChiSquaredInwards)
{
    bool pass = true;
    
    unsigned int pT3OuterT3Index = pixelTripletsInGPU.tripletIndices[pixelTripletIndex];
    unsigned int pT3InnerSegmentIndex = pixelTripletsInGPU.pixelSegmentIndices[pixelTripletIndex];

    unsigned int T5InnerT3Index = quintupletsInGPU.tripletIndices[2 * quintupletIndex];
    unsigned int T5OuterT3Index = quintupletsInGPU.tripletIndices[2 * quintupletIndex + 1];

    unsigned int firstSegmentIndex = tripletsInGPU.segmentIndices[2 * T5InnerT3Index];
    unsigned int secondSegmentIndex = tripletsInGPU.segmentIndices[2 * T5InnerT3Index + 1];
    unsigned int thirdSegmentIndex = tripletsInGPU.segmentIndices[2 * T5OuterT3Index];
    unsigned int fourthSegmentIndex = tripletsInGPU.segmentIndices[2 * T5OuterT3Index + 1];

    unsigned int pixelInnerMDIndex = segmentsInGPU.mdIndices[2 * pT3InnerSegmentIndex];
    unsigned int pixelOuterMDIndex = segmentsInGPU.mdIndices[2 * pT3InnerSegmentIndex + 1];

    unsigned int pixelAnchorHitIndex1 = mdsInGPU.hitIndices[2 * pixelInnerMDIndex];
    unsigned int pixelNonAnchorHitIndex1 = mdsInGPU.hitIndices[2 * pixelInnerMDIndex + 1];
    unsigned int pixelAnchorHitIndex2 = mdsInGPU.hitIndices[2 * pixelOuterMDIndex];
    unsigned int pixelNonAnchorHitIndex2 = mdsInGPU.hitIndices[2 * pixelOuterMDIndex + 1];

    unsigned int anchorHitIndex1 = segmentsInGPU.innerMiniDoubletAnchorHitIndices[firstSegmentIndex];
    unsigned int anchorHitIndex2 = segmentsInGPU.outerMiniDoubletAnchorHitIndices[firstSegmentIndex]; //same as second segment inner MD anchorhit index
    unsigned int anchorHitIndex3 = segmentsInGPU.outerMiniDoubletAnchorHitIndices[secondSegmentIndex]; //same as third segment inner MD anchor hit index

    unsigned int anchorHitIndex4 = segmentsInGPU.outerMiniDoubletAnchorHitIndices[thirdSegmentIndex]; //same as fourth segment inner MD anchor hit index
    unsigned int anchorHitIndex5 = segmentsInGPU.outerMiniDoubletAnchorHitIndices[fourthSegmentIndex];

    unsigned int lowerModuleIndex1 = quintupletsInGPU.lowerModuleIndices[5 * quintupletIndex];
    unsigned int lowerModuleIndex2 = quintupletsInGPU.lowerModuleIndices[5 * quintupletIndex + 1];
    unsigned int lowerModuleIndex3 = quintupletsInGPU.lowerModuleIndices[5 * quintupletIndex + 2];
    unsigned int lowerModuleIndex4 = quintupletsInGPU.lowerModuleIndices[5 * quintupletIndex + 3];
    unsigned int lowerModuleIndex5 = quintupletsInGPU.lowerModuleIndices[5 * quintupletIndex + 4];

    unsigned int lowerModuleIndices[] = {lowerModuleIndex1, lowerModuleIndex2, lowerModuleIndex3, lowerModuleIndex4, lowerModuleIndex5};
    unsigned int anchorHits[] = {anchorHitIndex1, anchorHitIndex2, anchorHitIndex3, anchorHitIndex4, anchorHitIndex5};
    unsigned int pixelHits[] = {pixelAnchorHitIndex1, pixelNonAnchorHitIndex1, pixelAnchorHitIndex2, pixelNonAnchorHitIndex2};

    //cut 1 -> common T3
    pass = pass & (pT3OuterT3Index == T5InnerT3Index);

    rzChiSquared = computePT5RZChiSquared(modulesInGPU, hitsInGPU, pixelAnchorHitIndex1, pixelAnchorHitIndex2, anchorHits, lowerModuleIndices);

    rPhiChiSquared = computePT5RPhiChiSquared(modulesInGPU, hitsInGPU, pixelHits, anchorHits, lowerModuleIndices);

    rPhiChiSquaredInwards = computePT5RPhiChiSquaredInwards(modulesInGPU, hitsInGPU, quintupletsInGPU, quintupletIndex, pixelHits);

    //other cuts will be filled here!
    return pass;
}


__device__ float SDL::computePT5RPhiChiSquared(struct modules& modulesInGPU, struct hits& hitsInGPU, unsigned int* pixelHits, unsigned int* anchorHits, unsigned int* lowerModuleIndices)
{
    /*
       Compute circle parameters from 3 pixel hits, and then use them to compute the chi squared for the outer hits
    */
    float g, f;
    unsigned int pixelAnchorHitIndex1 = pixelHits[0];
    unsigned int pixelNonAnchorHitIndex1 = pixelHits[1];
    unsigned int pixelAnchorHitIndex2 = pixelHits[2];

    float radius = computeRadiusFromThreeAnchorHits(hitsInGPU.xs[pixelAnchorHitIndex1], hitsInGPU.ys[pixelAnchorHitIndex1], hitsInGPU.xs[pixelNonAnchorHitIndex1], hitsInGPU.ys[pixelNonAnchorHitIndex1], hitsInGPU.xs[pixelAnchorHitIndex2], hitsInGPU.ys[pixelAnchorHitIndex2], g, f);

    float delta1[5], delta2[5], slopes[5];
    bool isFlat[5];
    float xs[5];
    float ys[5];
    float chiSquared = 0;
    for(size_t i = 0; i < 5; i++)
    {
        xs[i] = hitsInGPU.xs[anchorHits[i]];
        ys[i] = hitsInGPU.ys[anchorHits[i]];
    }

    computeSigmasForRegression(modulesInGPU, lowerModuleIndices, delta1, delta2, slopes, isFlat);
    chiSquared = computeChiSquared(5, xs, ys, delta1, delta2, slopes, isFlat, g, f, radius);

    return chiSquared;
}

__device__ float SDL::computePT5RPhiChiSquaredInwards(struct modules& modulesInGPU, struct hits& hitsInGPU, struct quintuplets& quintupletsInGPU, unsigned int quintupletIndex, unsigned int* pixelHits)
{
    /*Using the computed regression center and radius, compute the chi squared for the pixels*/
    float g = quintupletsInGPU.regressionG[quintupletIndex];
    float f = quintupletsInGPU.regressionF[quintupletIndex];
    float r = quintupletsInGPU.regressionRadius[quintupletIndex];
    float x, y;
    float chiSquared = 0;   
    int nPoints = (pixelHits[3] == pixelHits[2]) ? 3 : 4;
    for(size_t i = 0; i < nPoints; i++)
    {
        x = hitsInGPU.xs[pixelHits[i]];
        y = hitsInGPU.ys[pixelHits[i]];
        float residual = (x - g) * (x -g) + (y - f) * (y - f) - r * r;
        chiSquared += residual * residual;
    }
    chiSquared /= nPoints;
    return chiSquared;
}

__device__ float SDL::computePT5RZChiSquared(struct modules& modulesInGPU, struct hits& hitsInGPU, unsigned int& pixelAnchorHitIndex1, unsigned int& pixelAnchorHitIndex2, unsigned int* anchorHits, unsigned int* lowerModuleIndices)
{
    //use the two anchor hits of the pixel segment to compute the slope
    //then compute the pseudo chi squared of the five outer hits

    float& rtPix1 = hitsInGPU.rts[pixelAnchorHitIndex1];
    float& rtPix2 = hitsInGPU.rts[pixelAnchorHitIndex2];
    float& zPix1 = hitsInGPU.zs[pixelAnchorHitIndex1];
    float& zPix2 = hitsInGPU.zs[pixelAnchorHitIndex2];
    float slope = (zPix2 - zPix1)/(rtPix2 - rtPix1);
    float rtAnchor, zAnchor;
    float residual = 0;
    float error = 0;
    //hardcoded array indices!!!
    float RMSE = 0;
    float drdz;
    for(size_t i = 0; i < 5; i++)
    {
        unsigned int& anchorHitIndex = anchorHits[i];
        unsigned int& lowerModuleIndex = lowerModuleIndices[i];
        rtAnchor = hitsInGPU.rts[anchorHitIndex];
        zAnchor = hitsInGPU.zs[anchorHitIndex];

        const int moduleType = modulesInGPU.moduleType[lowerModuleIndex];
        const int moduleSide = modulesInGPU.sides[lowerModuleIndex];
        const int moduleLayerType = modulesInGPU.moduleLayerType[lowerModuleIndex];
        const int layer = modulesInGPU.layers[lowerModuleIndex] + 6 * (modulesInGPU.subdets[lowerModuleIndex] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex] == SDL::TwoS);
        
        residual = (layer <= 6) ?  (zAnchor - zPix1) - slope * (rtAnchor - rtPix1) : (rtAnchor - rtPix1) - (zAnchor - zPix1)/slope;
        
        //PS Modules
        if(moduleType == 0)
        {
            error = 0.15;
        }
        else //2S modules
        {
            error = 5.0;
        }

        //special dispensation to tilted PS modules!
        if(moduleType == 0 and layer <= 6 and moduleSide != Center)
        {
            if(moduleLayerType == Strip)
            {
                drdz = modulesInGPU.drdzs[lowerModuleIndex];
            }
            else
            {
                drdz = modulesInGPU.drdzs[modulesInGPU.partnerModuleIndex(lowerModuleIndex)];
            }

            error *= 1/sqrtf(1 + drdz * drdz);
        }
        RMSE += (residual * residual)/(error * error);
    }

    RMSE = sqrtf(0.2 * RMSE);
    return RMSE;
}

