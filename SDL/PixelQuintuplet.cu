#include "hip/hip_runtime.h"
#ifdef __HIPCC__
#define CUDA_CONST_VAR __device__
#endif
# include "PixelQuintuplet.cuh"
#include "allocate.h"

SDL::pixelQuintuplets::pixelQuintuplets()
{
    pixelIndices = nullptr;
    T5Indices = nullptr;
    nPixelQuintuplets = nullptr;
    isDup = nullptr;
    score = nullptr;
    logicalLayers = nullptr;
    hitIndices = nullptr;
    lowerModuleIndices = nullptr;
}

SDL::pixelQuintuplets::~pixelQuintuplets()
{
}

void SDL::pixelQuintuplets::freeMemory()
{
    hipFree(pixelIndices);
    hipFree(T5Indices);
    hipFree(nPixelQuintuplets);
    hipFree(isDup);
    hipFree(score);
    hipFree(logicalLayers);
    hipFree(hitIndices);
    hipFree(lowerModuleIndices);
    hipFree(eta);
    hipFree(phi);
#ifdef CUT_VALUE_DEBUG
    hipFree(rzChiSquared);
    hipFree(rPhiChiSquared);
    hipFree(rPhiChiSquaredInwards);
#endif
}

void SDL::createPixelQuintupletsInUnifiedMemory(struct SDL::pixelQuintuplets& pixelQuintupletsInGPU, unsigned int maxPixelQuintuplets)
{
    hipMallocManaged(&pixelQuintupletsInGPU.pixelIndices, maxPixelQuintuplets * sizeof(unsigned int));
    hipMallocManaged(&pixelQuintupletsInGPU.T5Indices, maxPixelQuintuplets * sizeof(unsigned int));
    hipMallocManaged(&pixelQuintupletsInGPU.nPixelQuintuplets, sizeof(unsigned int));
    hipMallocManaged(&pixelQuintupletsInGPU.isDup, maxPixelQuintuplets * sizeof(bool));
    hipMallocManaged(&pixelQuintupletsInGPU.score, maxPixelQuintuplets * sizeof(float));
    hipMallocManaged(&pixelQuintupletsInGPU.logicalLayers, maxPixelQuintuplets * 7 * sizeof(unsigned int));
    hipMallocManaged(&pixelQuintupletsInGPU.lowerModuleIndices, maxPixelQuintuplets * 7 * sizeof(unsigned int));
    hipMallocManaged(&pixelQuintupletsInGPU.hitIndices, maxPixelQuintuplets * 14 * sizeof(unsigned int));
    hipMallocManaged(&pixelQuintupletsInGPU.eta, maxPixelQuintuplets * sizeof(float));
    hipMallocManaged(&pixelQuintupletsInGPU.phi, maxPixelQuintuplets * sizeof(float));
#ifdef CUT_VALUE_DEBUG
    hipMallocManaged(&pixelQuintupletsInGPU.rzChiSquared, maxPixelQuintuplets * sizeof(unsigned int));
    hipMallocManaged(&pixelQuintupletsInGPU.rPhiChiSquared, maxPixelQuintuplets * sizeof(unsigned int));
    hipMallocManaged(&pixelQuintupletsInGPU.rPhiChiSquaredInwards, maxPixelQuintuplets * sizeof(unsigned int));
#endif

    hipMemset(pixelQuintupletsInGPU.nPixelQuintuplets, 0, sizeof(unsigned int));
}

void SDL::createPixelQuintupletsInExplicitMemory(struct SDL::pixelQuintuplets& pixelQuintupletsInGPU, unsigned int maxPixelQuintuplets)
{
    hipMalloc(&pixelQuintupletsInGPU.pixelIndices, maxPixelQuintuplets * sizeof(unsigned int));
    hipMalloc(&pixelQuintupletsInGPU.T5Indices, maxPixelQuintuplets * sizeof(unsigned int));
    hipMalloc(&pixelQuintupletsInGPU.nPixelQuintuplets, sizeof(unsigned int));
    hipMalloc(&pixelQuintupletsInGPU.isDup, maxPixelQuintuplets * sizeof(bool));
    hipMalloc(&pixelQuintupletsInGPU.score, maxPixelQuintuplets * sizeof(float));
    hipMalloc(&pixelQuintupletsInGPU.eta, maxPixelQuintuplets * sizeof(float));
    hipMalloc(&pixelQuintupletsInGPU.phi, maxPixelQuintuplets * sizeof(float));

    hipMalloc(&pixelQuintupletsInGPU.logicalLayers, maxPixelQuintuplets * 7 *sizeof(unsigned int));
    hipMalloc(&pixelQuintupletsInGPU.hitIndices, maxPixelQuintuplets * 14 * sizeof(unsigned int));
    hipMalloc(&pixelQuintupletsInGPU.lowerModuleIndices, maxPixelQuintuplets * 7 * sizeof(unsigned int));

    hipMemset(pixelQuintupletsInGPU.nPixelQuintuplets, 0, sizeof(unsigned int));
}

__device__ void SDL::rmPixelQuintupletToMemory(struct pixelQuintuplets& pixelQuintupletsInGPU, unsigned int pixelQuintupletIndex)
{

    pixelQuintupletsInGPU.isDup[pixelQuintupletIndex] = 1;
}
#ifdef CUT_VALUE_DEBUG
__device__ void SDL::addPixelQuintupletToMemory(struct modules& modulesInGPU, struct miniDoublets& mdsInGPU, struct segments& segmentsInGPU, struct quintuplets& quintupletsInGPU, struct pixelQuintuplets& pixelQuintupletsInGPU, unsigned int pixelIndex, unsigned int T5Index, unsigned int pixelQuintupletIndex, float& rzChiSquared, float& rPhiChiSquared, float& rPhiChiSquaredInwards, float score, float eta, float phi)
#else
__device__ void SDL::addPixelQuintupletToMemory(struct modules& modulesInGPU, struct miniDoublets& mdsInGPU, struct segments& segmentsInGPU, struct quintuplets& quintupletsInGPU, struct pixelQuintuplets& pixelQuintupletsInGPU, unsigned int pixelIndex, unsigned int T5Index, unsigned int pixelQuintupletIndex, float score, float eta, float phi)
#endif
{
    pixelQuintupletsInGPU.pixelIndices[pixelQuintupletIndex] = pixelIndex;
    pixelQuintupletsInGPU.T5Indices[pixelQuintupletIndex] = T5Index;
    pixelQuintupletsInGPU.isDup[pixelQuintupletIndex] = 0;
    pixelQuintupletsInGPU.score[pixelQuintupletIndex] = score;

    pixelQuintupletsInGPU.logicalLayers[7 * pixelQuintupletIndex] = 0;
    pixelQuintupletsInGPU.logicalLayers[7 * pixelQuintupletIndex + 1] = 0;
    pixelQuintupletsInGPU.logicalLayers[7 * pixelQuintupletIndex + 2] = quintupletsInGPU.logicalLayers[T5Index * 5];
    pixelQuintupletsInGPU.logicalLayers[7 * pixelQuintupletIndex + 3] = quintupletsInGPU.logicalLayers[T5Index * 5 + 1];
    pixelQuintupletsInGPU.logicalLayers[7 * pixelQuintupletIndex + 4] = quintupletsInGPU.logicalLayers[T5Index * 5 + 2];
    pixelQuintupletsInGPU.logicalLayers[7 * pixelQuintupletIndex + 5] = quintupletsInGPU.logicalLayers[T5Index * 5 + 3];
    pixelQuintupletsInGPU.logicalLayers[7 * pixelQuintupletIndex + 6] = quintupletsInGPU.logicalLayers[T5Index * 5 + 4];

    pixelQuintupletsInGPU.lowerModuleIndices[7 * pixelQuintupletIndex] = segmentsInGPU.innerMiniDoubletAnchorHitIndices[pixelIndex];
    pixelQuintupletsInGPU.lowerModuleIndices[7 * pixelQuintupletIndex + 1] = segmentsInGPU.outerMiniDoubletAnchorHitIndices[pixelIndex];
    pixelQuintupletsInGPU.lowerModuleIndices[7 * pixelQuintupletIndex + 2] = quintupletsInGPU.lowerModuleIndices[T5Index * 5];
    pixelQuintupletsInGPU.lowerModuleIndices[7 * pixelQuintupletIndex + 3] = quintupletsInGPU.lowerModuleIndices[T5Index * 5 + 1];
    pixelQuintupletsInGPU.lowerModuleIndices[7 * pixelQuintupletIndex + 4] = quintupletsInGPU.lowerModuleIndices[T5Index * 5 + 2];
    pixelQuintupletsInGPU.lowerModuleIndices[7 * pixelQuintupletIndex + 5] = quintupletsInGPU.lowerModuleIndices[T5Index * 5 + 3];
    pixelQuintupletsInGPU.lowerModuleIndices[7 * pixelQuintupletIndex + 6] = quintupletsInGPU.lowerModuleIndices[T5Index * 5 + 4];


    unsigned int pixelInnerMD = segmentsInGPU.mdIndices[2 * pixelIndex];
    unsigned int pixelOuterMD = segmentsInGPU.mdIndices[2 * pixelIndex + 1];

    pixelQuintupletsInGPU.hitIndices[14 * pixelQuintupletIndex] = mdsInGPU.hitIndices[2 * pixelInnerMD];
    pixelQuintupletsInGPU.hitIndices[14 * pixelQuintupletIndex + 1] = mdsInGPU.hitIndices[2 * pixelInnerMD + 1];
    pixelQuintupletsInGPU.hitIndices[14 * pixelQuintupletIndex + 2] = mdsInGPU.hitIndices[2 * pixelOuterMD];
    pixelQuintupletsInGPU.hitIndices[14 * pixelQuintupletIndex + 3] = mdsInGPU.hitIndices[2 * pixelOuterMD + 1];

    pixelQuintupletsInGPU.hitIndices[14 * pixelQuintupletIndex + 4] = quintupletsInGPU.hitIndices[10 * T5Index];
    pixelQuintupletsInGPU.hitIndices[14 * pixelQuintupletIndex + 5] = quintupletsInGPU.hitIndices[10 * T5Index + 1];
    pixelQuintupletsInGPU.hitIndices[14 * pixelQuintupletIndex + 6] = quintupletsInGPU.hitIndices[10 * T5Index + 2];
    pixelQuintupletsInGPU.hitIndices[14 * pixelQuintupletIndex + 7] = quintupletsInGPU.hitIndices[10 * T5Index + 3];
    pixelQuintupletsInGPU.hitIndices[14 * pixelQuintupletIndex + 8] = quintupletsInGPU.hitIndices[10 * T5Index + 4];
    pixelQuintupletsInGPU.hitIndices[14 * pixelQuintupletIndex + 9] = quintupletsInGPU.hitIndices[10 * T5Index + 5];
    pixelQuintupletsInGPU.hitIndices[14 * pixelQuintupletIndex + 10] = quintupletsInGPU.hitIndices[10 * T5Index + 6];
    pixelQuintupletsInGPU.hitIndices[14 * pixelQuintupletIndex + 11] = quintupletsInGPU.hitIndices[10 * T5Index + 7];
    pixelQuintupletsInGPU.hitIndices[14 * pixelQuintupletIndex + 12] = quintupletsInGPU.hitIndices[10 * T5Index + 8];
    pixelQuintupletsInGPU.hitIndices[14 * pixelQuintupletIndex + 13] = quintupletsInGPU.hitIndices[10 * T5Index + 9];

    pixelQuintupletsInGPU.eta[pixelQuintupletIndex] = eta;
    pixelQuintupletsInGPU.phi[pixelQuintupletIndex] = phi;
    
#ifdef CUT_VALUE_DEBUG
    pixelQuintupletsInGPU.rzChiSquared[pixelQuintupletIndex] = rzChiSquared;
    pixelQuintupletsInGPU.rPhiChiSquared[pixelQuintupletIndex] = rPhiChiSquared;
    pixelQuintupletsInGPU.rPhiChiSquaredInwards[pixelQuintupletIndex] = rPhiChiSquaredInwards;
#endif
}

__device__ bool SDL::runPixelQuintupletDefaultAlgo(struct modules& modulesInGPU, struct hits& hitsInGPU, struct miniDoublets& mdsInGPU, struct segments& segmentsInGPU, struct triplets& tripletsInGPU, struct quintuplets& quintupletsInGPU, unsigned int& pixelSegmentIndex, unsigned int& quintupletIndex, float& rzChiSquared, float& rPhiChiSquared, float& rPhiChiSquaredInwards)
{
    bool pass = true;
    
    unsigned int pixelModuleIndex = segmentsInGPU.innerLowerModuleIndices[pixelSegmentIndex];

    unsigned int pixelSegmentArrayIndex = pixelSegmentIndex - (600 * pixelModuleIndex);

    unsigned int T5InnerT3Index = quintupletsInGPU.tripletIndices[2 * quintupletIndex];
    unsigned int T5OuterT3Index = quintupletsInGPU.tripletIndices[2 * quintupletIndex + 1];

    unsigned int firstSegmentIndex = tripletsInGPU.segmentIndices[2 * T5InnerT3Index];
    unsigned int secondSegmentIndex = tripletsInGPU.segmentIndices[2 * T5InnerT3Index + 1];
    unsigned int thirdSegmentIndex = tripletsInGPU.segmentIndices[2 * T5OuterT3Index];
    unsigned int fourthSegmentIndex = tripletsInGPU.segmentIndices[2 * T5OuterT3Index + 1];

    unsigned int pixelInnerMDIndex = segmentsInGPU.mdIndices[2 * pixelSegmentIndex];
    unsigned int pixelOuterMDIndex = segmentsInGPU.mdIndices[2 * pixelSegmentIndex + 1];

    unsigned int pixelAnchorHitIndex1 = mdsInGPU.hitIndices[2 * pixelInnerMDIndex];
    unsigned int pixelNonAnchorHitIndex1 = mdsInGPU.hitIndices[2 * pixelInnerMDIndex + 1];
    unsigned int pixelAnchorHitIndex2 = mdsInGPU.hitIndices[2 * pixelOuterMDIndex];
    unsigned int pixelNonAnchorHitIndex2 = mdsInGPU.hitIndices[2 * pixelOuterMDIndex + 1];

    unsigned int anchorHitIndex1 = segmentsInGPU.innerMiniDoubletAnchorHitIndices[firstSegmentIndex];
    unsigned int anchorHitIndex2 = segmentsInGPU.outerMiniDoubletAnchorHitIndices[firstSegmentIndex]; //same as second segment inner MD anchorhit index
    unsigned int anchorHitIndex3 = segmentsInGPU.outerMiniDoubletAnchorHitIndices[secondSegmentIndex]; //same as third segment inner MD anchor hit index

    unsigned int anchorHitIndex4 = segmentsInGPU.outerMiniDoubletAnchorHitIndices[thirdSegmentIndex]; //same as fourth segment inner MD anchor hit index
    unsigned int anchorHitIndex5 = segmentsInGPU.outerMiniDoubletAnchorHitIndices[fourthSegmentIndex];

    unsigned int lowerModuleIndex1 = quintupletsInGPU.lowerModuleIndices[5 * quintupletIndex];
    unsigned int lowerModuleIndex2 = quintupletsInGPU.lowerModuleIndices[5 * quintupletIndex + 1];
    unsigned int lowerModuleIndex3 = quintupletsInGPU.lowerModuleIndices[5 * quintupletIndex + 2];
    unsigned int lowerModuleIndex4 = quintupletsInGPU.lowerModuleIndices[5 * quintupletIndex + 3];
    unsigned int lowerModuleIndex5 = quintupletsInGPU.lowerModuleIndices[5 * quintupletIndex + 4];

    unsigned int lowerModuleIndices[] = {lowerModuleIndex1, lowerModuleIndex2, lowerModuleIndex3, lowerModuleIndex4, lowerModuleIndex5};
    unsigned int anchorHits[] = {anchorHitIndex1, anchorHitIndex2, anchorHitIndex3, anchorHitIndex4, anchorHitIndex5};
    unsigned int pixelHits[] = {pixelAnchorHitIndex1, pixelAnchorHitIndex2};
    
    float pixelRadius, pixelRadiusError, tripletRadius, rPhiChiSquaredTemp, rzChiSquaredTemp, rPhiChiSquaredInwardsTemp;

    pass = pass & runPixelTripletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, tripletsInGPU, pixelSegmentIndex, T5InnerT3Index, pixelRadius, pixelRadiusError, tripletRadius, rzChiSquaredTemp, rPhiChiSquaredTemp, rPhiChiSquaredInwardsTemp, false);

    rzChiSquared = computePT5RZChiSquared(modulesInGPU, hitsInGPU, pixelAnchorHitIndex1, pixelAnchorHitIndex2, anchorHits, lowerModuleIndices);

    rPhiChiSquared = computePT5RPhiChiSquared(modulesInGPU, hitsInGPU, segmentsInGPU, pixelSegmentArrayIndex, anchorHits, lowerModuleIndices);

    rPhiChiSquaredInwards = computePT5RPhiChiSquaredInwards(modulesInGPU, hitsInGPU, quintupletsInGPU, quintupletIndex, pixelHits);

    if(segmentsInGPU.circleRadius[pixelSegmentArrayIndex] < 5.0/(2 * k2Rinv1GeVf))
    {
        pass = pass & passPT5RZChiSquaredCuts(modulesInGPU, lowerModuleIndex1, lowerModuleIndex2, lowerModuleIndex3, lowerModuleIndex4, lowerModuleIndex5, rzChiSquared);

        pass = pass & passPT5RPhiChiSquaredCuts(modulesInGPU, lowerModuleIndex1, lowerModuleIndex2, lowerModuleIndex3, lowerModuleIndex4, lowerModuleIndex5, rPhiChiSquared);
    }
    
    if(quintupletsInGPU.regressionRadius[quintupletIndex] < 5.0/(2 * k2Rinv1GeVf))
    {
        pass = pass & passPT5RPhiChiSquaredInwardsCuts(modulesInGPU, lowerModuleIndex1, lowerModuleIndex2, lowerModuleIndex3, lowerModuleIndex4, lowerModuleIndex5, rPhiChiSquaredInwards);
    }

    //other cuts will be filled here!
    return pass;
}

__device__ bool SDL::passPT5RPhiChiSquaredCuts(struct modules& modulesInGPU, unsigned int lowerModuleIndex1, unsigned int lowerModuleIndex2, unsigned int lowerModuleIndex3, unsigned int lowerModuleIndex4, unsigned int lowerModuleIndex5, float rPhiChiSquared)
{
    const int layer1 = modulesInGPU.layers[lowerModuleIndex1] + 6 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex1] == SDL::TwoS);
    const int layer2 = modulesInGPU.layers[lowerModuleIndex2] + 6 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex2] == SDL::TwoS);
    const int layer3 = modulesInGPU.layers[lowerModuleIndex3] + 6 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex3] == SDL::TwoS);
    const int layer4 = modulesInGPU.layers[lowerModuleIndex4] + 6 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex4] == SDL::TwoS);
    const int layer5 = modulesInGPU.layers[lowerModuleIndex5] + 6 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex5] == SDL::TwoS);

    if(layer1 == 1 and layer2 == 2 and layer3 == 3)
    {
        if(layer4 == 12 and layer5 == 13)
        {
            return rPhiChiSquared < 48.921;
        }
        else if(layer4 == 4 and layer5 == 12)
        {
            return rPhiChiSquared < 97.948;
        }
        else if(layer4 == 4 and layer5 == 5)
        {
            return rPhiChiSquared < 129.3;
        }
        else if(layer4 == 7 and layer5 == 13)
        {
            return rPhiChiSquared < 56.21;
        }
        else if(layer4 == 7 and layer5 == 8)
        {
            return rPhiChiSquared < 74.198;
        }
    }
    else if(layer1 == 1 and layer2 == 2 and layer3 == 7)
    {
        if(layer4 == 13 and layer5 == 14)
        {
            return rPhiChiSquared < 21.265;
        }
        else if(layer4 == 8 and layer5 == 14)
        {
            return rPhiChiSquared < 37.058;
        }
        else if(layer4 == 8 and layer5 == 9)
        {
            return rPhiChiSquared < 42.578;
        }
    }
    else if(layer1 == 1 and layer2 == 7 and layer3 == 8)
    {
        if(layer4 == 9 and layer5 == 10)
        {
            return rPhiChiSquared < 32.253;
        }
        else if(layer4 == 9 and layer5 == 15)
        {
            return rPhiChiSquared < 37.058;
        }
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 4)
    {
        if(layer4 == 12 and layer5 == 13)
        {
            return rPhiChiSquared < 97.947;
        }
        else if(layer4 == 5 and layer5 == 12)
        {
            return rPhiChiSquared < 129.3;
        }
        else if(layer4 == 5 and layer5 == 6)
        {
            return rPhiChiSquared < 170.68;
        }
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 7)
    {
        if(layer4 == 13 and layer5 == 14)
        {   
            return rPhiChiSquared < 48.92;
        }
        else if(layer4 == 8 and layer5 == 14)
        {
            return rPhiChiSquared < 74.2;
        }
    }
    else if(layer1 == 2 and layer2 == 7 and layer3 == 8)
    {
        if(layer4 == 14 and layer5 == 15)
        {
            return rPhiChiSquared < 42.58;
        }
        else if(layer4 == 9 and layer5 == 10)
        {
            return rPhiChiSquared < 37.06;
        }
        else if(layer4 == 9 and layer5 == 15)
        {
            return rPhiChiSquared < 48.92;
        }
    }
    else if(layer1 == 3 and layer2 == 7 and layer3 == 8 and layer4 == 14 and layer5 == 15)
    {
        return rPhiChiSquared < 85.25;
    }
    else if(layer1 == 7 and layer2 == 8 and layer3 == 9)
    {
        if(layer4 == 10 and layer5 == 11)
        {
            return rPhiChiSquared < 42.58;
        }
        else if(layer4 == 10 and layer5 == 16)
        {
            return rPhiChiSquared < 37.06;
        }
        else if(layer4 == 15 and layer5 == 16)
        {
            return rPhiChiSquared < 37.06;
        }
    }
    return true;
}



__device__ bool SDL::passPT5RPhiChiSquaredInwardsCuts(struct modules& modulesInGPU, unsigned int lowerModuleIndex1, unsigned int lowerModuleIndex2, unsigned int lowerModuleIndex3, unsigned int lowerModuleIndex4, unsigned int lowerModuleIndex5, float rPhiChiSquared)
{
    const int layer1 = modulesInGPU.layers[lowerModuleIndex1] + 6 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex1] == SDL::TwoS);
    const int layer2 = modulesInGPU.layers[lowerModuleIndex2] + 6 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex2] == SDL::TwoS);
    const int layer3 = modulesInGPU.layers[lowerModuleIndex3] + 6 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex3] == SDL::TwoS);
    const int layer4 = modulesInGPU.layers[lowerModuleIndex4] + 6 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex4] == SDL::TwoS);
    const int layer5 = modulesInGPU.layers[lowerModuleIndex5] + 6 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex5] == SDL::TwoS);

    if(layer1 == 1 and layer2 == 2 and layer3 == 3)
    {
        if(layer4 == 12 and layer5 == 13)
        {
            return rPhiChiSquared < 451.141;
        }
        else if(layer4 == 4 and layer5 == 12)
        {
            return rPhiChiSquared < 786.173;
        }
        else if(layer4 == 4 and layer5 == 5)
        {
            return rPhiChiSquared < 595.545;
        }
        else if(layer4 == 7 and layer5 == 13)
        {
            return rPhiChiSquared < 581.339;
        }
        else if(layer4 == 7 and layer5 == 8)
        {
            return rPhiChiSquared < 112.537;
        }
    }
    else if(layer1 == 1 and layer2 == 2 and layer3 == 7)
    {
        if(layer4 == 13 and layer5 == 14)
        {
            return rPhiChiSquared < 225.322;
        }
        else if(layer4 == 8 and layer5 == 14)
        {
            return rPhiChiSquared < 1192.402;
        }
        else if(layer4 == 8 and layer5 == 9)
        {
            return rPhiChiSquared < 786.173;
        }
    }
    else if(layer1 == 1 and layer2 == 7 and layer3 == 8)
    {
        if(layer4 == 9 and layer5 == 10)
        {
            return rPhiChiSquared < 1037.817;
        }
        else if(layer4 == 9 and layer5 == 15)
        {
            return rPhiChiSquared < 1808.536;
        }
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 4)
    {
        if(layer4 == 12 and layer5 == 13)
        {
            return rPhiChiSquared < 684.253;
        }
        else if(layer4 == 5 and layer5 == 12)
        {
            return rPhiChiSquared < 684.253;
        }
        else if(layer4 == 5 and layer5 == 6)
        {
            return rPhiChiSquared < 684.253;
        }
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 7)
    {
        if(layer4 == 13 and layer5 == 14)
        {   
            return rPhiChiSquared < 451.141;
        }
        else if(layer4 == 8 and layer5 == 14)
        {
            return rPhiChiSquared < 518.34;
        }
    }
    else if(layer1 == 2 and layer2 == 7 and layer3 == 8)
    {
        if(layer4 == 14 and layer5 == 15)
        {
            return rPhiChiSquared < 2077.92;
        }
        else if(layer4 == 9 and layer5 == 10)
        {
            return rPhiChiSquared < 74.20;
        }
        else if(layer4 == 9 and layer5 == 15)
        {
            return rPhiChiSquared < 1808.536;
        }
    }
    else if(layer1 == 3 and layer2 == 7 and layer3 == 8 and layer4 == 14 and layer5 == 15)
    {
        return rPhiChiSquared < 786.173;
    }
    else if(layer1 == 7 and layer2 == 8 and layer3 == 9)
    {
        if(layer4 == 10 and layer5 == 11)
        {
            return rPhiChiSquared < 1574.076;
        }
        else if(layer4 == 10 and layer5 == 16)
        {
            return rPhiChiSquared < 5492.11;
        }
        else if(layer4 == 15 and layer5 == 16)
        {
            return rPhiChiSquared < 2743.037;
        }
    }
    return true;
}

__device__ float SDL::computePT5RPhiChiSquared(struct modules& modulesInGPU, struct hits& hitsInGPU, struct segments& segmentsInGPU, unsigned int pixelSegmentArrayIndex, unsigned int* anchorHits, unsigned int* lowerModuleIndices)
{
    /*
       Compute circle parameters from 3 pixel hits, and then use them to compute the chi squared for the outer hits
    */

    float g = segmentsInGPU.circleCenterX[pixelSegmentArrayIndex];
    float f = segmentsInGPU.circleCenterY[pixelSegmentArrayIndex];
    float radius = segmentsInGPU.circleRadius[pixelSegmentArrayIndex];
    float delta1[5], delta2[5], slopes[5];
    bool isFlat[5];
    float xs[5];
    float ys[5];
    float chiSquared = 0;
    for(size_t i = 0; i < 5; i++)
    {
        xs[i] = hitsInGPU.xs[anchorHits[i]];
        ys[i] = hitsInGPU.ys[anchorHits[i]];
    }

    computeSigmasForRegression(modulesInGPU, lowerModuleIndices, delta1, delta2, slopes, isFlat);
    chiSquared = computeChiSquared(5, xs, ys, delta1, delta2, slopes, isFlat, g, f, radius);

    return chiSquared;
}

__device__ bool SDL::passPT5RZChiSquaredCuts(struct modules& modulesInGPU, unsigned int lowerModuleIndex1, unsigned int lowerModuleIndex2, unsigned int lowerModuleIndex3, unsigned int lowerModuleIndex4, unsigned int lowerModuleIndex5, float& rzChiSquared)
{
    const int layer1 = modulesInGPU.layers[lowerModuleIndex1] + 6 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex1] == SDL::TwoS);
    const int layer2 = modulesInGPU.layers[lowerModuleIndex2] + 6 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex2] == SDL::TwoS);
    const int layer3 = modulesInGPU.layers[lowerModuleIndex3] + 6 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex3] == SDL::TwoS);
    const int layer4 = modulesInGPU.layers[lowerModuleIndex4] + 6 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex4] == SDL::TwoS);
    const int layer5 = modulesInGPU.layers[lowerModuleIndex5] + 6 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex5] == SDL::TwoS);

    
    if(layer1 == 1 and layer2 == 2 and layer3 == 3)
    {
        if(layer4 == 12 and layer5 == 13)
        {
            return rzChiSquared < 451.141;
        }
        else if(layer4 == 4 and layer5 == 12)
        {
            return rzChiSquared < 392.654;
        }
        else if(layer4 == 4 and layer5 == 5)
        {
            return rzChiSquared < 225.322;
        }
        else if(layer4 == 7 and layer5 == 13)
        {
            return rzChiSquared < 595.546;
        }
        else if(layer4 == 7 and layer5 == 8)
        {
            return rzChiSquared < 196.111;
        }
    }
    else if(layer1 == 1 and layer2 == 2 and layer3 == 7)
    {
        if(layer4 == 13 and layer5 == 14)
        {
            return rzChiSquared < 297.446;
        }
        else if(layer4 == 8 and layer5 == 14)
        {   
            return rzChiSquared < 451.141;
        }
        else if(layer4 == 8 and layer5 == 9)
        {
            return rzChiSquared < 518.339;
        }
    }
    else if(layer1 == 1 and layer2 == 7 and layer3 == 8)
    {
        if(layer4 == 9 and layer5 == 10)
        {
            return rzChiSquared < 341.75;
        }
        else if(layer4 == 9 and layer5 == 15)
        {
            return rzChiSquared < 341.75;
        }
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 4)
    {
        if(layer4 == 12 and layer5 == 13)
        {
            return rzChiSquared < 392.655;
        }
        else if(layer4 == 5 and layer5 == 12)
        {
            return rzChiSquared < 341.75;
        }
        else if(layer4 == 5 and layer5 == 6)
        {
            return rzChiSquared < 112.537;
        }
    }
    else if(layer1 == 2 and layer2 == 3 and layer4 == 7)
    {
        if(layer4 == 13 and layer5 == 14)
        {
            return rzChiSquared < 595.545;
        }
        else if(layer4 == 8 and layer5 == 14)
        {
            return rzChiSquared < 74.198;
        }
    }
    else if(layer1 == 2 and layer2 == 7 and layer3 == 8)
    {
        if(layer4 == 14 and layer5 == 15)
        {
            return rzChiSquared < 518.339;
        }
        else if(layer4 == 9 and layer5 == 10)
        {
            return rzChiSquared < 8.046;
        }
        else if(layer4 == 9 and layer5 == 15)
        {
            return rzChiSquared < 451.141;
        }
    }
    else if(layer1 == 3 and layer2 == 7 and layer3 == 8 and layer4 == 14 and layer5 == 15)
    {
        return rzChiSquared < 56.207;
    }
    else if(layer1 == 7 and layer2 == 8 and layer3 == 9)
    {
        if(layer4 == 10 and layer5 == 11)
        {
            return rzChiSquared < 64.578;
        }
        else if(layer4 == 10 and layer5 == 16)
        {
            return rzChiSquared < 85.250;
        }
        else if(layer4 == 15 and layer5 == 16)
        {
            return rzChiSquared < 85.250;
        }
    }
    return true;
}

__device__ float SDL::computePT5RPhiChiSquaredInwards(struct modules& modulesInGPU, struct hits& hitsInGPU, struct quintuplets& quintupletsInGPU, unsigned int quintupletIndex, unsigned int* pixelHits)
{
    /*Using the computed regression center and radius, compute the chi squared for the pixels*/
    float g = quintupletsInGPU.regressionG[quintupletIndex];
    float f = quintupletsInGPU.regressionF[quintupletIndex];
    float r = quintupletsInGPU.regressionRadius[quintupletIndex];
    float x, y;
    float chiSquared = 0;   
    for(size_t i = 0; i < 2; i++)
    {
        x = hitsInGPU.xs[pixelHits[i]];
        y = hitsInGPU.ys[pixelHits[i]];
        float residual = (x - g) * (x -g) + (y - f) * (y - f) - r * r;
        chiSquared += residual * residual;
    }
    chiSquared /= 2;
    return chiSquared;
}

__device__ float SDL::computePT5RZChiSquared(struct modules& modulesInGPU, struct hits& hitsInGPU, unsigned int& pixelAnchorHitIndex1, unsigned int& pixelAnchorHitIndex2, unsigned int* anchorHits, unsigned int* lowerModuleIndices)
{
    //use the two anchor hits of the pixel segment to compute the slope
    //then compute the pseudo chi squared of the five outer hits

    float& rtPix1 = hitsInGPU.rts[pixelAnchorHitIndex1];
    float& rtPix2 = hitsInGPU.rts[pixelAnchorHitIndex2];
    float& zPix1 = hitsInGPU.zs[pixelAnchorHitIndex1];
    float& zPix2 = hitsInGPU.zs[pixelAnchorHitIndex2];
    float slope = (zPix2 - zPix1)/(rtPix2 - rtPix1);
    float rtAnchor, zAnchor;
    float residual = 0;
    float error = 0;
    //hardcoded array indices!!!
    float RMSE = 0;
    float drdz;
    for(size_t i = 0; i < 5; i++)
    {
        unsigned int& anchorHitIndex = anchorHits[i];
        unsigned int& lowerModuleIndex = lowerModuleIndices[i];
        rtAnchor = hitsInGPU.rts[anchorHitIndex];
        zAnchor = hitsInGPU.zs[anchorHitIndex];

        const int moduleType = modulesInGPU.moduleType[lowerModuleIndex];
        const int moduleSide = modulesInGPU.sides[lowerModuleIndex];
        const int moduleLayerType = modulesInGPU.moduleLayerType[lowerModuleIndex];
        const int layer = modulesInGPU.layers[lowerModuleIndex] + 6 * (modulesInGPU.subdets[lowerModuleIndex] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex] == SDL::TwoS);
        
        residual = (layer <= 6) ?  (zAnchor - zPix1) - slope * (rtAnchor - rtPix1) : (rtAnchor - rtPix1) - (zAnchor - zPix1)/slope;
        
        //PS Modules
        if(moduleType == 0)
        {
            error = 0.15;
        }
        else //2S modules
        {
            error = 5.0;
        }

        //special dispensation to tilted PS modules!
        if(moduleType == 0 and layer <= 6 and moduleSide != Center)
        {
            if(moduleLayerType == Strip)
            {
                drdz = modulesInGPU.drdzs[lowerModuleIndex];
            }
            else
            {
                drdz = modulesInGPU.drdzs[modulesInGPU.partnerModuleIndex(lowerModuleIndex)];
            }

            error *= 1/sqrtf(1 + drdz * drdz);
        }
        RMSE += (residual * residual)/(error * error);
    }

    RMSE = sqrtf(0.2 * RMSE);
    return RMSE;
}

