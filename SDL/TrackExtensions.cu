# include "TrackExtensions.cuh"

SDL::trackExtensions::trackExtensions()
{
    constituentTCTypes = nullptr;
    constituentTCIndices = nullptr;
    nTrackExtensions = nullptr;
}

SDL::trackExtensions::~trackExtensions()
{
}

void SDL::trackExtensions::freeMemory()
{
    hipFree(constituentTCTypes);
    hipFree(constituentTCIndices);
    hipFree(nTrackExtensions);
}

void SDL::createTrackExtensionsInUnifiedMemory(struct trackExtensions& trackExtensionsInGPU, unsigned int maxTrackExtensions)
{
    hipMallocManaged(&trackExtensionsInGPU.constituentTCTypes, sizeof(short) * 3 * maxTrackExtensions);
    hipMallocManaged(&trackExtensionsInGPU.constituentTCIndices, sizeof(unsigned int) * 3 * maxTrackExtensions);
    hipMallocManaged(&trackExtensionsInGPU.nLayerOverlaps, sizeof(unsigned int) * 2 * maxTrackExtensions);
    hipMallocManaged(&trackExtensionsInGPU.nHitOverlaps, sizeof(unsigned int) * 2 * maxTrackExtensions);
    hipMallocManaged(&trackExtensionsInGPU.nTrackExtensions, sizeof(unsigned int));

    hipMemset(trackExtensionsInGPU.nTrackExtensions, 0, sizeof(unsigned int));
}

void SDL::createTrackExtensionsInExplicitMemory(struct trackExtensions& trackExtensionsInGPU, unsigned int maxTrackExtensions)
{
    hipMalloc(&trackExtensionsInGPU.constituentTCTypes, sizeof(short) * 3 * maxTrackExtensions);
    hipMalloc(&trackExtensionsInGPU.constituentTCIndices, sizeof(unsigned int) * 3 * maxTrackExtensions);
    hipMalloc(&trackExtensionsInGPU.nLayerOverlaps, sizeof(unsigned int) * 2 * maxTrackExtensions);
    hipMalloc(&trackExtensionsInGPU.nHitOverlaps, sizeof(unsigned int) * 2 * maxTrackExtensions);
    hipMalloc(&trackExtensionsInGPU.nTrackExtensions, sizeof(unsigned int));
    hipMemset(trackExtensionsInGPU.nTrackExtensions, 0, sizeof(unsigned int));

}

__device__ void SDL::addTrackExtensionToMemory(struct trackExtensions& trackExtensionsInGPU, short* constituentTCType, unsigned int* constituentTCIndex, unsigned int* nLayerOverlaps, unsigned int* nHitOverlaps, unsigned int trackExtensionIndex)
{
    
    for(size_t i = 0; i < 3 ; i++)
    {
        trackExtensionsInGPU.constituentTCTypes[3 * trackExtensionIndex + i] = constituentTCType[i];
        trackExtensionsInGPU.constituentTCIndices[3 * trackExtensionIndex + i] = constituentTCIndex[i];
    }
    for(size_t i = 0; i < 2; i++)
    {
        trackExtensionsInGPU.nLayerOverlaps[2 * trackExtensionIndex + i] = nLayerOverlaps[i];
        trackExtensionsInGPU.nHitOverlaps[2 * trackExtensionIndex + i] = nHitOverlaps[i];
    }
}

__device__ bool SDL::runTrackExtensionDefaultAlgo(struct modules& modulesInGPU, struct hits& hitsInGPU, struct miniDoublets& mdsInGPU, struct segments& segmentsInGPU, struct triplets& tripletsInGPU, struct trackCandidates& trackCandidatesInGPU, unsigned int anchorObjectIndex, unsigned int outerObjectIndex, short anchorObjectType, short outerObjectType, unsigned int anchorObjectOuterT3Index, unsigned int layerOverlapTarget, short* constituentTCType, unsigned int* constituentTCIndex, unsigned
        int* nLayerOverlaps, unsigned int* nHitOverlaps)
{
    /*
       Basic premise:
       1. given two objects, get the hit and module indices
       2. check for layer and hit overlap (layer overlap first checked using
       the 2-merge approach)
       3. Additional cuts - rz and rphi chi squared criteria! (TODO) 
    */

    bool pass = true;
    unsigned int* anchorLayerIndices = nullptr;
    unsigned int* anchorHitIndices = nullptr;
    unsigned int* anchorLowerModuleIndices = nullptr;

    unsigned int* outerObjectLayerIndices = nullptr;
    unsigned int* outerObjectHitIndices = nullptr;
    unsigned int* outerObjectLowerModuleIndices = nullptr;

    unsigned int nAnchorLayers = (anchorObjectType == 7) ? 7 : (anchorObjectType == 3 ? 3 : 5);
    if(anchorObjectType != 3) //mostly this
    { 
        anchorLayerIndices = &trackCandidatesInGPU.logicalLayers[7 * anchorObjectIndex];
        anchorHitIndices = &trackCandidatesInGPU.hitIndices[14 * anchorObjectIndex];
        anchorLowerModuleIndices = &trackCandidatesInGPU.lowerModuleIndices[7 * anchorObjectIndex];
    }
    else //outlier
    {
        anchorLayerIndices = &tripletsInGPU.logicalLayers[3 * anchorObjectIndex];
        anchorHitIndices = &tripletsInGPU.hitIndices[6 * anchorObjectIndex];
        anchorLowerModuleIndices = &tripletsInGPU.lowerModuleIndices[3 * anchorObjectIndex];
    }

    unsigned int nOuterLayers =(outerObjectType == 7) ? 7 : (outerObjectType == 3 ? 3 : 5); 

    if(outerObjectType == 3) //mostly this
    {
        outerObjectLayerIndices = &tripletsInGPU.logicalLayers[3 * outerObjectIndex];
        outerObjectHitIndices = &tripletsInGPU.hitIndices[6 * outerObjectIndex];
        outerObjectLowerModuleIndices = &tripletsInGPU.lowerModuleIndices[3 * outerObjectIndex];
    }
    else //outlier
    {
        outerObjectLayerIndices = &trackCandidatesInGPU.logicalLayers[7 * outerObjectIndex];
        outerObjectHitIndices = &trackCandidatesInGPU.hitIndices[14 * outerObjectIndex];
        outerObjectLowerModuleIndices = &tripletsInGPU.lowerModuleIndices[7 * outerObjectIndex];
    }
    
    unsigned int nLayerOverlap(0), nHitOverlap(0);
   
    float zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta; //temp stuff
    float zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ;

    //checks for frivolous cases wherein
    pass = pass &  computeLayerAndHitOverlaps(modulesInGPU, anchorLayerIndices, anchorHitIndices, anchorLowerModuleIndices, outerObjectLayerIndices, outerObjectHitIndices, outerObjectLowerModuleIndices, nAnchorLayers, nOuterLayers, nLayerOverlap, nHitOverlap, layerOverlapTarget);

    pass = pass & runTrackletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, tripletsInGPU.lowerModuleIndices[3 * anchorObjectOuterT3Index], tripletsInGPU.lowerModuleIndices[3 * anchorObjectOuterT3Index + 1], tripletsInGPU.lowerModuleIndices[3 * outerObjectIndex], tripletsInGPU.lowerModuleIndices[3 * outerObjectIndex + 1], tripletsInGPU.segmentIndices[2 * anchorObjectOuterT3Index], tripletsInGPU.segmentIndices[2 * outerObjectIndex], zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta, zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ);

    pass = pass & runTrackletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, tripletsInGPU.lowerModuleIndices[3 * anchorObjectOuterT3Index], tripletsInGPU.lowerModuleIndices[3 * anchorObjectOuterT3Index + 1], tripletsInGPU.lowerModuleIndices[3 * outerObjectIndex + 1], tripletsInGPU.lowerModuleIndices[3 * outerObjectIndex + 2], tripletsInGPU.segmentIndices[2 * anchorObjectOuterT3Index], tripletsInGPU.segmentIndices[2 * outerObjectIndex + 1], zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta, zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ);


    nLayerOverlaps[0] = nLayerOverlap;
    nHitOverlaps[0] = nHitOverlap;

    constituentTCType[0] = anchorObjectType;
    constituentTCType[1] = outerObjectType;

    constituentTCIndex[0] = anchorObjectIndex;
    constituentTCIndex[1] = outerObjectIndex;

   return pass;
}



/*
   If one out of the two hits don't overlap - check if the module indices are identical (case of multiple reco hits)
   If both hits don't overlap - check the above, and check for staggered modules using the staggered module neighbours list

   This function i complicated - computes layer overlaps and checks if layer matches and hit matches are "compatible" i.e., layer overlap = 2 * hit overlap, or if that's not the case, we know why (multiple reco hits/staggered modules)
*/
__device__ bool SDL::computeLayerAndHitOverlaps(SDL::modules& modulesInGPU, unsigned int* anchorLayerIndices, unsigned int* anchorHitIndices, unsigned int* anchorLowerModuleIndices, unsigned int* outerObjectLayerIndices, unsigned int* outerObjectHitIndices, unsigned int* outerObjectLowerModuleIndices, unsigned int nAnchorLayers, unsigned int nOuterLayers, unsigned int& nLayerOverlap, unsigned int& nHitOverlap, unsigned int& layerOverlapTarget)
{
    bool pass = true;
    //merge technique!
    size_t j = 0; //outer object tracker
    unsigned int temp; //container variable
    unsigned int staggeredNeighbours[10];
    for(size_t i = 0; i < nAnchorLayers; i++)
    {
        if(anchorLayerIndices[i] == outerObjectLayerIndices[j])
        {
            //2*i and 2*i + 1 are the hits, similarly 2*j and 2*j+1
            nLayerOverlap++;
            temp = nHitOverlap; //before the hit matching shenanigans

            //FIXME:Assumption, 2*i and 2*i+1 hits are known to be from partner modules!
            if(anchorHitIndices[2 * i] == outerObjectHitIndices[2 * j])
            {
                nHitOverlap++;
            }
            else //check for same module indices
            {
                if(anchorLowerModuleIndices[i] != outerObjectLowerModuleIndices[j])
                {
                    pass = false;
                }
            }
            if(anchorHitIndices[2*i+1] == outerObjectHitIndices[2*j+1])
            {
                nHitOverlap++;
            }
            else //check for same module indices
            {
                if(anchorLowerModuleIndices[i] != outerObjectLowerModuleIndices[j])
                {
                    pass = false;
                }
            }
            
            if(nHitOverlap == temp) //check for staggered modules!
            {
                //this is a redemption case. If both modules did not match in the above case,
                //this case should redeem them!

                //find the neighbours of the anchor lower module, if any of those matches the outer lower module, we're done
                unsigned int nStaggeredModules;                
                findStaggeredNeighbours(modulesInGPU, anchorLowerModuleIndices[i], staggeredNeighbours, nStaggeredModules);
                for(size_t idx = 0; idx < nStaggeredModules; idx++)
                {
                    if(outerObjectLowerModuleIndices[j]  == staggeredNeighbours[idx])
                    {
                        //redeemed!
                        pass = true;
                    }
                }
            }

            j++;
            if(j == nOuterLayers)
            {
                break;
            }
        }
    }
    pass = pass & (nLayerOverlap == layerOverlapTarget); //not really required, because these cases should be handled by the other conditions
    return pass;
}

