#include "hip/hip_runtime.h"
# include "TrackExtensions.cuh"

SDL::trackExtensions::trackExtensions()
{
    constituentTCTypes = nullptr;
    constituentTCIndices = nullptr;
    nHitOverlaps = nullptr;
    nLayerOverlaps = nullptr;
    nTrackExtensions = nullptr;
    rPhiChiSquared = nullptr;
    rzChiSquared = nullptr;
    isDup = nullptr;
}

SDL::trackExtensions::~trackExtensions()
{
}

void SDL::trackExtensions::freeMemory()
{
    hipFree(constituentTCTypes);
    hipFree(constituentTCIndices);
    hipFree(nLayerOverlaps);
    hipFree(nHitOverlaps);
    hipFree(nTrackExtensions);
    hipFree(isDup);
    hipFree(rPhiChiSquared);
    hipFree(rzChiSquared);
}

/*
   Track Extensions memory allocation - 10 slots for each TC (will reduce later)
   Extensions having the same anchor object will be clustered together for easy
   duplicate cleaning
*/

void SDL::createTrackExtensionsInUnifiedMemory(struct trackExtensions& trackExtensionsInGPU, unsigned int maxTrackExtensions, unsigned int nTrackCandidates)
{
    hipMallocManaged(&trackExtensionsInGPU.constituentTCTypes, sizeof(short) * 3 * maxTrackExtensions);
    hipMallocManaged(&trackExtensionsInGPU.constituentTCIndices, sizeof(unsigned int) * 3 * maxTrackExtensions);
    hipMallocManaged(&trackExtensionsInGPU.nLayerOverlaps, sizeof(unsigned int) * 2 * maxTrackExtensions);
    hipMallocManaged(&trackExtensionsInGPU.nHitOverlaps, sizeof(unsigned int) * 2 * maxTrackExtensions);
    hipMallocManaged(&trackExtensionsInGPU.nTrackExtensions, nTrackCandidates * sizeof(unsigned int));
    hipMallocManaged(&trackExtensionsInGPU.rPhiChiSquared, maxTrackExtensions * sizeof(float));
    hipMallocManaged(&trackExtensionsInGPU.rzChiSquared, maxTrackExtensions * sizeof(float));

    hipMallocManaged(&trackExtensionsInGPU.isDup, maxTrackExtensions * sizeof(bool));

    hipMemset(trackExtensionsInGPU.nTrackExtensions, 0, nTrackCandidates * sizeof(unsigned int));

    hipMemset(trackExtensionsInGPU.isDup, true, maxTrackExtensions * sizeof(bool));
}

void SDL::createTrackExtensionsInExplicitMemory(struct trackExtensions& trackExtensionsInGPU, unsigned int maxTrackExtensions, unsigned int nTrackCandidates)
{
    hipMalloc(&trackExtensionsInGPU.constituentTCTypes, sizeof(short) * 3 * maxTrackExtensions);
    hipMalloc(&trackExtensionsInGPU.constituentTCIndices, sizeof(unsigned int) * 3 * maxTrackExtensions);
    hipMalloc(&trackExtensionsInGPU.nLayerOverlaps, sizeof(unsigned int) * 2 * maxTrackExtensions);
    hipMalloc(&trackExtensionsInGPU.nHitOverlaps, sizeof(unsigned int) * 2 * maxTrackExtensions);
    hipMalloc(&trackExtensionsInGPU.nTrackExtensions, nTrackCandidates * sizeof(unsigned int));
    hipMalloc(&trackExtensionsInGPU.rPhiChiSquared, maxTrackExtensions * sizeof(float));
    hipMalloc(&trackExtensionsInGPU.rzChiSquared, maxTrackExtensions * sizeof(float));

    hipMalloc(&trackExtensionsInGPU.isDup, maxTrackExtensions * sizeof(bool));

    hipMemset(trackExtensionsInGPU.nTrackExtensions, 0, nTrackCandidates * sizeof(unsigned int));
    hipMemset(trackExtensionsInGPU.isDup, true, maxTrackExtensions * sizeof(bool));
}

__device__ void SDL::addTrackExtensionToMemory(struct trackExtensions& trackExtensionsInGPU, short* constituentTCType, unsigned int* constituentTCIndex, unsigned int* nLayerOverlaps, unsigned int* nHitOverlaps, float rPhiChiSquared, float rzChiSquared, unsigned int trackExtensionIndex)
{ 
    for(size_t i = 0; i < 3 ; i++)
    {
        trackExtensionsInGPU.constituentTCTypes[3 * trackExtensionIndex + i] = constituentTCType[i];
        trackExtensionsInGPU.constituentTCIndices[3 * trackExtensionIndex + i] = constituentTCIndex[i];
    }
    for(size_t i = 0; i < 2; i++)
    {
        trackExtensionsInGPU.nLayerOverlaps[2 * trackExtensionIndex + i] = nLayerOverlaps[i];
        trackExtensionsInGPU.nHitOverlaps[2 * trackExtensionIndex + i] = nHitOverlaps[i];
    }
    trackExtensionsInGPU.rPhiChiSquared[trackExtensionIndex] = rPhiChiSquared;
    trackExtensionsInGPU.rzChiSquared[trackExtensionIndex] = rzChiSquared;
}

__device__ bool SDL::runTrackExtensionDefaultAlgo(struct modules& modulesInGPU, struct hits& hitsInGPU, struct miniDoublets& mdsInGPU, struct segments& segmentsInGPU, struct triplets& tripletsInGPU, struct quintuplets& quintupletsInGPU, struct pixelTriplets& pixelTripletsInGPU, struct pixelQuintuplets& pixelQuintupletsInGPU, struct trackCandidates& trackCandidatesInGPU, unsigned int anchorObjectIndex, unsigned int outerObjectIndex, short anchorObjectType, short outerObjectType, unsigned int anchorObjectOuterT3Index, unsigned int layerOverlapTarget, short* constituentTCType, unsigned int* constituentTCIndex, unsigned
        int* nLayerOverlaps, unsigned int* nHitOverlaps, float& rPhiChiSquared, float& rzChiSquared)
{
    /*
       Basic premise:
       1. given two objects, get the hit and module indices
       2. check for layer and hit overlap (layer overlap first checked using
       the 2-merge approach)
       3. Additional cuts - rz and rphi chi squared criteria! (TODO) 
    */

    bool pass = true;
    unsigned int* anchorLayerIndices = nullptr;
    unsigned int* anchorHitIndices = nullptr;
    unsigned int* anchorLowerModuleIndices = nullptr;

    unsigned int* outerObjectLayerIndices = nullptr;
    unsigned int* outerObjectHitIndices = nullptr;
    unsigned int* outerObjectLowerModuleIndices = nullptr;

    unsigned int nAnchorLayers = (anchorObjectType == 7) ? 7 : (anchorObjectType == 3 ? 3 : 5);
    unsigned int anchorObjectAnchorHitIndices[7];
    float centerX, centerY, radius;

    if(anchorObjectType != 3) //mostly this
    { 
        anchorLayerIndices = &trackCandidatesInGPU.logicalLayers[7 * anchorObjectIndex];
        anchorHitIndices = &trackCandidatesInGPU.hitIndices[14 * anchorObjectIndex];
        anchorLowerModuleIndices = &trackCandidatesInGPU.lowerModuleIndices[7 * anchorObjectIndex];
        centerX = trackCandidatesInGPU.centerX[anchorObjectIndex];
        centerY = trackCandidatesInGPU.centerY[anchorObjectIndex];
        radius = trackCandidatesInGPU.radius[anchorObjectIndex];
    }
    else //outlier
    {
        anchorLayerIndices = &tripletsInGPU.logicalLayers[3 * anchorObjectIndex];
        anchorHitIndices = &tripletsInGPU.hitIndices[6 * anchorObjectIndex];
        anchorLowerModuleIndices = &tripletsInGPU.lowerModuleIndices[3 * anchorObjectIndex];
    }

    unsigned int layer_binary = 0;
    for(size_t i=0; i<nAnchorLayers;i++)
    {
        if(modulesInGPU.isAnchor[hitsInGPU.moduleIndices[anchorHitIndices[2*i]]] or modulesInGPU.detIds[hitsInGPU.moduleIndices[anchorHitIndices[2*i]]] == 1)
        {
            anchorObjectAnchorHitIndices[i] = anchorHitIndices[2*i];
        }
        else
        {
            anchorObjectAnchorHitIndices[i] = anchorHitIndices[2*i+1];
        }
        layer_binary |= (1 << anchorLayerIndices[i]);
    }

    if(anchorObjectType == 3)
    {
        radius = computeRadiusFromThreeAnchorHits(hitsInGPU.xs[anchorObjectAnchorHitIndices[0]], hitsInGPU.ys[anchorObjectAnchorHitIndices[0]],hitsInGPU.xs[anchorObjectAnchorHitIndices[1]], hitsInGPU.ys[anchorObjectAnchorHitIndices[1]], hitsInGPU.xs[anchorObjectAnchorHitIndices[2]], hitsInGPU.ys[anchorObjectAnchorHitIndices[2]], centerX, centerY);
    }
    unsigned int nOuterLayers =(outerObjectType == 7) ? 7 : (outerObjectType == 3 ? 3 : 5); 

    unsigned int outerObjectAnchorHitIndices[7];
    if(outerObjectType == 3) //mostly this
    {
        outerObjectLayerIndices = &tripletsInGPU.logicalLayers[3 * outerObjectIndex];
        outerObjectHitIndices = &tripletsInGPU.hitIndices[6 * outerObjectIndex];
        outerObjectLowerModuleIndices = &tripletsInGPU.lowerModuleIndices[3 * outerObjectIndex];
    }
    else //outlier
    {
        outerObjectLayerIndices = &trackCandidatesInGPU.logicalLayers[7 * outerObjectIndex];
        outerObjectHitIndices = &trackCandidatesInGPU.hitIndices[14 * outerObjectIndex];
        outerObjectLowerModuleIndices = &tripletsInGPU.lowerModuleIndices[7 * outerObjectIndex];
    }

    for(size_t i=0; i<nOuterLayers;i++)
    {
        if(modulesInGPU.isAnchor[hitsInGPU.moduleIndices[outerObjectHitIndices[2*i]]] or modulesInGPU.detIds[hitsInGPU.moduleIndices[outerObjectHitIndices[2*i]]] == 1)
        {
            outerObjectAnchorHitIndices[i] = outerObjectHitIndices[2*i];
        }
        else
        {
            outerObjectAnchorHitIndices[i] = outerObjectHitIndices[2*i+1];
        }

        layer_binary |= (1 << outerObjectLayerIndices[i]);
    }
    
    unsigned int nLayerOverlap(0), nHitOverlap(0);
   
    float zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta; //temp stuff
    float zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ;

    //checks for frivolous cases wherein
    pass = pass &  computeLayerAndHitOverlaps(modulesInGPU, anchorLayerIndices, anchorHitIndices, anchorLowerModuleIndices, outerObjectLayerIndices, outerObjectHitIndices, outerObjectLowerModuleIndices, nAnchorLayers, nOuterLayers, nLayerOverlap, nHitOverlap, layerOverlapTarget);


    pass = pass & runTrackletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, tripletsInGPU.lowerModuleIndices[3 * anchorObjectOuterT3Index], tripletsInGPU.lowerModuleIndices[3 * anchorObjectOuterT3Index + 1], tripletsInGPU.lowerModuleIndices[3 * outerObjectIndex], tripletsInGPU.lowerModuleIndices[3 * outerObjectIndex + 1], tripletsInGPU.segmentIndices[2 * anchorObjectOuterT3Index], tripletsInGPU.segmentIndices[2 * outerObjectIndex], zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta, zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ);

    pass = pass & runTrackletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, tripletsInGPU.lowerModuleIndices[3 * anchorObjectOuterT3Index], tripletsInGPU.lowerModuleIndices[3 * anchorObjectOuterT3Index + 1], tripletsInGPU.lowerModuleIndices[3 * outerObjectIndex + 1], tripletsInGPU.lowerModuleIndices[3 * outerObjectIndex + 2], tripletsInGPU.segmentIndices[2 * anchorObjectOuterT3Index], tripletsInGPU.segmentIndices[2 * outerObjectIndex + 1], zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta, zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ);

    rPhiChiSquared = computeTERPhiChiSquared(modulesInGPU, hitsInGPU, centerX, centerY, radius, outerObjectAnchorHitIndices, outerObjectLowerModuleIndices);
    
    rzChiSquared = computeTERZChiSquared(modulesInGPU, hitsInGPU, anchorObjectAnchorHitIndices, anchorLowerModuleIndices, outerObjectAnchorHitIndices, outerObjectLowerModuleIndices, anchorObjectType);

//    if(radius < 5.0/(2 * k2Rinv1GeVf))
//    {
        if(anchorObjectType != 3)
        {
            pass = pass and passTERPhiChiSquaredCuts(nLayerOverlap, nHitOverlap, layer_binary, rPhiChiSquared);
            pass = pass and passTERZChiSquaredCuts(nLayerOverlap, nHitOverlap, layer_binary, rzChiSquared);
         }
//    }

    nLayerOverlaps[0] = nLayerOverlap;
    nHitOverlaps[0] = nHitOverlap;

    constituentTCType[0] = anchorObjectType;
    constituentTCType[1] = outerObjectType;

    constituentTCIndex[0] = anchorObjectIndex;
    constituentTCIndex[1] = outerObjectIndex;

    return pass;
}

__device__ bool SDL::passTERZChiSquaredCuts(int nLayerOverlaps, int nHitOverlaps, unsigned int layer_binary, float rzChiSquared)
{
    if(nLayerOverlaps == 2 and nHitOverlaps == 4 and layer_binary == 127)
    {
        return rzChiSquared < 7.003;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 4 and layer_binary == 3971)
    {
        return rzChiSquared < 97.948;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 4 and layer_binary == 191)
    {
        return rzChiSquared < 10.621;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 4 and layer_binary == 1927)
    {
        return rzChiSquared < 97.948;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 4 and layer_binary == 415)
    {
        return rzChiSquared < 10.621;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 4 and layer_binary == 911)
    {
        return rzChiSquared < 97.948;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 4 and layer_binary == 3973)
    {
        return rzChiSquared < 85.250;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 4 and layer_binary == 31)
    {
        return rzChiSquared < 258.885;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 4 and layer_binary == 901)
    {
        return rzChiSquared < 148.559;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 4 and layer_binary == 61)
    {
        return rzChiSquared < 297.446;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 4 and layer_binary == 143)
    {
        return rzChiSquared < 392.655;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 4 and layer_binary == 1921)
    {
        return rzChiSquared < 129.300;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 4 and layer_binary == 157)
    {
        return rzChiSquared < 392.655;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 4 and layer_binary == 391)
    {
        return rzChiSquared < 258.885;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 4 and layer_binary == 3841)
    {
        return rzChiSquared < 112.537;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 4 and layer_binary == 899)
    {
        return rzChiSquared < 129.300;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 4 and layer_binary == 397)
    {
        return rzChiSquared < 518.339;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 4 and layer_binary == 126)
    {
        return rzChiSquared < 28.073;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 4 and layer_binary == 3970)
    {
        return rzChiSquared < 903.274;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 4 and layer_binary == 414)
    {
        return rzChiSquared < 56.207;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 4 and layer_binary == 445)
    {
        return rzChiSquared < 3.044;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 4 and layer_binary == 190)
    {
        return rzChiSquared < 48.920;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 4 and layer_binary == 1926)
    {
        return rzChiSquared < 786.173;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 4 and layer_binary == 910)
    {
        return rzChiSquared < 684.253;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 4 and layer_binary == 3972)
    {
        return rzChiSquared < 595.546;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 4 and layer_binary == 444)
    {
        return rzChiSquared < 56.207;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 4 and layer_binary == 1933)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 3 and layer_binary == 127)
    {
        return rzChiSquared < 8.046;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 3 and layer_binary == 3971)
    {
        return rzChiSquared < 97.948;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 3 and layer_binary == 191)
    {
        return rzChiSquared < 10.621;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 3 and layer_binary == 1927)
    {
        return rzChiSquared < 97.948;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 3 and layer_binary == 415)
    {
        return rzChiSquared < 4.018;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 3 and layer_binary == 911)
    {
        return rzChiSquared < 74.198;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 3 and layer_binary == 3973)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 3 and layer_binary == 31)
    {
        return rzChiSquared < 297.446;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 3 and layer_binary == 901)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 3 and layer_binary == 61)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 3 and layer_binary == 143)
    {
        return rzChiSquared < 56.207;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 3 and layer_binary == 1921)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 3 and layer_binary == 391)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 3 and layer_binary == 3841)
    {
        return rzChiSquared < 18.509;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 3 and layer_binary == 899)
    {
        return rzChiSquared < 64.579;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 3 and layer_binary == 397)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 3 and layer_binary == 126)
    {
        return rzChiSquared < 28.073;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 3 and layer_binary == 1926)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 3 and layer_binary == 910)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 3 and layer_binary == 3972)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 2 and layer_binary == 127)
    {
        return rzChiSquared < 5.305;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 2 and layer_binary == 3971)
    {
        return rzChiSquared < 97.948;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 2 and layer_binary == 191)
    {
        return rzChiSquared < 10.621;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 2 and layer_binary == 1927)
    {
        return rzChiSquared < 85.250;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 2 and layer_binary == 415)
    {
        return rzChiSquared < 7.003;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 2 and layer_binary == 911)
    {
        return rzChiSquared < 48.920;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 2 and layer_binary == 3973)
    {
        return rzChiSquared < 37.058;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 2 and layer_binary == 31)
    {
        return rzChiSquared < 112.537;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 2 and layer_binary == 901)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 2 and layer_binary == 61)
    {
        return rzChiSquared < 32.254;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 2 and layer_binary == 143)
    {
        return rzChiSquared < 225.322;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 2 and layer_binary == 1921)
    {
        return rzChiSquared < 24.433;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 2 and layer_binary == 391)
    {
        return rzChiSquared < 196.111;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 2 and layer_binary == 3841)
    {
        return rzChiSquared < 129.300;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 2 and layer_binary == 899)
    {
        return rzChiSquared < 74.198;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 2 and layer_binary == 397)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 2 and layer_binary == 126)
    {
        return rzChiSquared < 28.073;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 2 and layer_binary == 3970)
    {
        return rzChiSquared < 903.274;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 2 and layer_binary == 414)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 2 and layer_binary == 1926)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 2 and layer_binary == 3972)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 1 and layer_binary == 191)
    {
        return rzChiSquared < 9.244;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 1 and layer_binary == 1927)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 1 and layer_binary == 415)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 1 and layer_binary == 3973)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 1 and layer_binary == 31)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 1 and layer_binary == 143)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 1 and layer_binary == 1921)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 1 and layer_binary == 391)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 1 and layer_binary == 3841)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 1 and layer_binary == 899)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 1 and layer_binary == 397)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 1 and layer_binary == 3970)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 1 and layer_binary == 414)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 1 and layer_binary == 190)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 1 and layer_binary == 1926)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 0 and layer_binary == 127)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 0 and layer_binary == 3971)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 0 and layer_binary == 191)
    {
        return rzChiSquared < 9.244;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 0 and layer_binary == 1927)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 0 and layer_binary == 415)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 0 and layer_binary == 911)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 0 and layer_binary == 3973)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 0 and layer_binary == 31)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 0 and layer_binary == 61)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 0 and layer_binary == 1921)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 0 and layer_binary == 391)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 0 and layer_binary == 3841)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 0 and layer_binary == 899)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 0 and layer_binary == 397)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 0 and layer_binary == 1926)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 0 and layer_binary == 3972)
    {
        return false;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 2 and layer_binary == 3975)
    {
        return rzChiSquared < 64.579;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 2 and layer_binary == 63)
    {
        return rzChiSquared < 258.885;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 2 and layer_binary == 125)
    {
        return rzChiSquared < 21.266;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 2 and layer_binary == 3969)
    {
        return rzChiSquared < 129.300;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 2 and layer_binary == 1925)
    {
        return rzChiSquared < 48.920;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 2 and layer_binary == 413)
    {
        return rzChiSquared < 10.621;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 2 and layer_binary == 903)
    {
        return rzChiSquared < 148.559;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 2 and layer_binary == 399)
    {
        return rzChiSquared < 258.885;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 2 and layer_binary == 189)
    {
        return rzChiSquared < 12.203;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 2 and layer_binary == 1923)
    {
        return rzChiSquared < 148.559;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 2 and layer_binary == 909)
    {
        return rzChiSquared < 97.948;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 2 and layer_binary == 159)
    {
        return rzChiSquared < 451.141;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 2 and layer_binary == 1935)
    {
        return rzChiSquared < 24.433;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 2 and layer_binary == 447)
    {
        return rzChiSquared < 6.095;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 2 and layer_binary == 3974)
    {
        return true;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 1 and layer_binary == 3975)
    {
        return false;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 1 and layer_binary == 63)
    {
        return rzChiSquared < 14.021;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 1 and layer_binary == 3969)
    {
        return false;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 1 and layer_binary == 1925)
    {
        return false;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 1 and layer_binary == 903)
    {
        return false;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 1 and layer_binary == 399)
    {
        return false;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 1 and layer_binary == 1923)
    {
        return rzChiSquared < 14.021;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 1 and layer_binary == 909)
    {
        return false;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 1 and layer_binary == 3974)
    {
        return false;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 0 and layer_binary == 3975)
    {
        return false;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 0 and layer_binary == 63)
    {
        return false;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 0 and layer_binary == 125)
    {
        return false;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 0 and layer_binary == 3969)
    {
        return false;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 0 and layer_binary == 903)
    {
        return false;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 0 and layer_binary == 399)
    {
        return false;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 0 and layer_binary == 1923)
    {
        return false;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 0 and layer_binary == 909)
    {
        return false;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 0 and layer_binary == 159)
    {
        return false;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 0 and layer_binary == 1935)
    {
        return false;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 0 and layer_binary == 446)
    {
        return false;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 0 and layer_binary == 3974)
    {
        return false;
    }
    return true;
}

__device__ bool SDL::passTERPhiChiSquaredCuts(int nLayerOverlaps, int nHitOverlaps, unsigned int layer_binary, float rPhiChiSquared)
{
    if(nLayerOverlaps == 2 and nHitOverlaps == 4 and layer_binary == 127)
    {
        return rPhiChiSquared < 21.266;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 4 and layer_binary == 3971)
    {
        return rPhiChiSquared < 6.095;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 4 and layer_binary == 1927)
    {
        return rPhiChiSquared < 4.617;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 4 and layer_binary == 415)
    {
        return rPhiChiSquared < 9.244;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 4 and layer_binary == 191)
    {
        return rPhiChiSquared < 18.509;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 4 and layer_binary == 3973)
    {
        return rPhiChiSquared < 2.649;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 4 and layer_binary == 911)
    {
        return rPhiChiSquared < 5.305;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 4 and layer_binary == 31)
    {
        return rPhiChiSquared < 12634.215;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 4 and layer_binary == 391)
    {
        return rPhiChiSquared < 16678.281;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 4 and layer_binary == 3841)
    {
        return rPhiChiSquared < 16678.281;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 4 and layer_binary == 61)
    {
        return rPhiChiSquared < 3151.615;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 4 and layer_binary == 143)
    {
        return rPhiChiSquared < 88261.109;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 4 and layer_binary == 901)
    {
        return rPhiChiSquared < 5492.110;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 4 and layer_binary == 1921)
    {
        return rPhiChiSquared < 33393.261;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 4 and layer_binary == 397)
    {
        return rPhiChiSquared < 1037.818;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 4 and layer_binary == 899)
    {
        return rPhiChiSquared < 153806.756;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 4 and layer_binary == 3970)
    {
        return rPhiChiSquared < 37.058;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 4 and layer_binary == 414)
    {
        return rPhiChiSquared < 64.579;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 4 and layer_binary == 126)
    {
        return rPhiChiSquared < 85.250;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 4 and layer_binary == 445)
    {
        return true;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 4 and layer_binary == 1926)
    {
        return rPhiChiSquared < 0.436;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 4 and layer_binary == 190)
    {
        return rPhiChiSquared < 9.244;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 4 and layer_binary == 3972)
    {
        return rPhiChiSquared < 0.010;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 4 and layer_binary == 157)
    {
        return rPhiChiSquared < 297.446;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 4 and layer_binary == 910)
    {
        return rPhiChiSquared < 0.759;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 4 and layer_binary == 444)
    {
        return true;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 2 and layer_binary == 127)
    {
        return rPhiChiSquared < 14.021;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 2 and layer_binary == 3971)
    {
        return rPhiChiSquared < 2.649;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 2 and layer_binary == 1927)
    {
        return rPhiChiSquared < 7.003;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 2 and layer_binary == 415)
    {
        return rPhiChiSquared < 2.007;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 2 and layer_binary == 191)
    {
        return rPhiChiSquared < 3.044;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 2 and layer_binary == 3973)
    {
        return rPhiChiSquared < 0.041;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 2 and layer_binary == 911)
    {
        return rPhiChiSquared < 1.323;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 2 and layer_binary == 31)
    {
        return rPhiChiSquared < 203038.514;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 2 and layer_binary == 391)
    {
        return rPhiChiSquared < 297.446;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 2 and layer_binary == 3841)
    {
        return rPhiChiSquared < 196.111;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 2 and layer_binary == 61)
    {
        return rPhiChiSquared < 8.046;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 2 and layer_binary == 143)
    {
        return rPhiChiSquared < 1370.011;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 2 and layer_binary == 901)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 2 and layer_binary == 1921)
    {
        return rPhiChiSquared < 42.578;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 2 and layer_binary == 397)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 2 and layer_binary == 899)
    {
        return rPhiChiSquared < 595.546;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 2 and layer_binary == 414)
    {
        return rPhiChiSquared < 0.000;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 2 and layer_binary == 126)
    {
        return rPhiChiSquared < 8.046;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 2 and layer_binary == 3840)
    {
        return rPhiChiSquared < 4.617;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 2 and layer_binary == 1920)
    {
        return rPhiChiSquared < 0.872;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 2 and layer_binary == 445)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 2 and layer_binary == 1926)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 2 and layer_binary == 190)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 2 and layer_binary == 3972)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 2 and layer_binary == 157)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 2 and layer_binary == 910)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 2 and layer_binary == 444)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 2 and layer_binary == 1933)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 3 and layer_binary == 127)
    {
        return rPhiChiSquared < 10.621;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 3 and layer_binary == 3971)
    {
        return rPhiChiSquared < 3.044;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 3 and layer_binary == 1927)
    {
        return rPhiChiSquared < 4.018;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 3 and layer_binary == 415)
    {
        return rPhiChiSquared < 4.617;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 3 and layer_binary == 191)
    {
        return rPhiChiSquared < 12.203;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 3 and layer_binary == 3973)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 3 and layer_binary == 911)
    {
        return rPhiChiSquared < 0.872;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 3 and layer_binary == 31)
    {
        return rPhiChiSquared < 935179.568;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 3 and layer_binary == 391)
    {
        return rPhiChiSquared < 0.041;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 3 and layer_binary == 3841)
    {
        return rPhiChiSquared < 8.046;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 3 and layer_binary == 61)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 3 and layer_binary == 143)
    {
        return rPhiChiSquared < 4.018;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 3 and layer_binary == 901)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 3 and layer_binary == 1921)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 3 and layer_binary == 397)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 3 and layer_binary == 899)
    {
        return rPhiChiSquared < 3621.052;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 3 and layer_binary == 3970)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 3 and layer_binary == 414)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 3 and layer_binary == 126)
    {
        return rPhiChiSquared < 6.095;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 3 and layer_binary == 3840)
    {
        return rPhiChiSquared < 4.617;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 3 and layer_binary == 1920)
    {
        return rPhiChiSquared < 0.218;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 3 and layer_binary == 1926)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 3 and layer_binary == 190)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 3 and layer_binary == 157)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 1 and layer_binary == 127)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 1 and layer_binary == 3971)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 1 and layer_binary == 1927)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 1 and layer_binary == 415)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 1 and layer_binary == 3973)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 1 and layer_binary == 911)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 1 and layer_binary == 31)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 1 and layer_binary == 3841)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 1 and layer_binary == 61)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 1 and layer_binary == 143)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 1 and layer_binary == 899)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 1 and layer_binary == 126)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 1 and layer_binary == 3840)
    {
        return rPhiChiSquared < 0.003;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 1 and layer_binary == 1920)
    {
        return rPhiChiSquared < 0.010;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 1 and layer_binary == 157)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 0 and layer_binary == 127)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 0 and layer_binary == 3971)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 0 and layer_binary == 1927)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 0 and layer_binary == 415)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 0 and layer_binary == 191)
    {
        return rPhiChiSquared < 2.649;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 0 and layer_binary == 3973)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 0 and layer_binary == 911)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 0 and layer_binary == 31)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 0 and layer_binary == 61)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 0 and layer_binary == 143)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 0 and layer_binary == 901)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 0 and layer_binary == 397)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 0 and layer_binary == 899)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 0 and layer_binary == 414)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 0 and layer_binary == 126)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 0 and layer_binary == 3840)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 0 and layer_binary == 1920)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 0 and layer_binary == 445)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 0 and layer_binary == 1926)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 0 and layer_binary == 190)
    {
        return false;
    }
    else if(nLayerOverlaps == 2 and nHitOverlaps == 0 and layer_binary == 910)
    {
        return false;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 2 and layer_binary == 3975)
    {
        return rPhiChiSquared < 5.305;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 2 and layer_binary == 1923)
    {
        return rPhiChiSquared < 44082.056;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 2 and layer_binary == 399)
    {
        return rPhiChiSquared < 1574.076;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 2 and layer_binary == 63)
    {
        return rPhiChiSquared < 8329.976;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 2 and layer_binary == 125)
    {
        return rPhiChiSquared < 4780.108;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 2 and layer_binary == 1925)
    {
        return rPhiChiSquared < 170.687;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 2 and layer_binary == 159)
    {
        return rPhiChiSquared < 3621.052;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 2 and layer_binary == 3969)
    {
        return rPhiChiSquared < 268028.788;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 2 and layer_binary == 903)
    {
        return rPhiChiSquared < 12634.215;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 2 and layer_binary == 413)
    {
        return rPhiChiSquared < 684.253;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 2 and layer_binary == 189)
    {
        return rPhiChiSquared < 1370.011;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 2 and layer_binary == 909)
    {
        return rPhiChiSquared < 225.322;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 2 and layer_binary == 1935)
    {
        return rPhiChiSquared < 0.872;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 2 and layer_binary == 447)
    {
        return true;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 2 and layer_binary == 3974)
    {
        return rPhiChiSquared < 0.002;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 2 and layer_binary == 446)
    {
        return false;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 1 and layer_binary == 3975)
    {
        return false;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 1 and layer_binary == 1923)
    {
        return false;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 1 and layer_binary == 399)
    {
        return rPhiChiSquared < 6.095;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 1 and layer_binary == 63)
    {
        return rPhiChiSquared < 4.018;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 1 and layer_binary == 125)
    {
        return false;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 1 and layer_binary == 1925)
    {
        return false;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 1 and layer_binary == 159)
    {
        return false;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 1 and layer_binary == 3969)
    {
        return false;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 1 and layer_binary == 903)
    {
        return false;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 1 and layer_binary == 413)
    {
        return false;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 1 and layer_binary == 189)
    {
        return false;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 0 and layer_binary == 3975)
    {
        return false;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 0 and layer_binary == 1923)
    {
        return false;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 0 and layer_binary == 399)
    {
        return false;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 0 and layer_binary == 63)
    {
        return false;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 0 and layer_binary == 125)
    {
        return false;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 0 and layer_binary == 1925)
    {
        return false;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 0 and layer_binary == 159)
    {
        return false;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 0 and layer_binary == 3969)
    {
        return false;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 0 and layer_binary == 903)
    {
        return false;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 0 and layer_binary == 413)
    {
        return false;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 0 and layer_binary == 189)
    {
        return false;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 0 and layer_binary == 909)
    {
        return false;
    }
    else if(nLayerOverlaps == 1 and nHitOverlaps == 0 and layer_binary == 447)
    {
        return false;
    }

    return true;
}


/*
   If one out of the two hits don't overlap - check if the module indices are identical (case of multiple reco hits)
   If both hits don't overlap - check the above, and check for staggered modules using the staggered module neighbours list

   This function i complicated - computes layer overlaps and checks if layer matches and hit matches are "compatible" i.e., layer overlap = 2 * hit overlap, or if that's not the case, we know why (multiple reco hits/staggered modules)
*/
__device__ bool SDL::computeLayerAndHitOverlaps(SDL::modules& modulesInGPU, unsigned int* anchorLayerIndices, unsigned int* anchorHitIndices, unsigned int* anchorLowerModuleIndices, unsigned int* outerObjectLayerIndices, unsigned int* outerObjectHitIndices, unsigned int* outerObjectLowerModuleIndices, unsigned int nAnchorLayers, unsigned int nOuterLayers, unsigned int& nLayerOverlap, unsigned int& nHitOverlap, unsigned int& layerOverlapTarget)
{
    bool pass = true;
    //merge technique!
    size_t j = 0; //outer object tracker
    unsigned int temp; //container variable
    unsigned int staggeredNeighbours[10];
    for(size_t i = 0; i < nAnchorLayers; i++)
    {
        if(anchorLayerIndices[i] == outerObjectLayerIndices[j])
        {
            //2*i and 2*i + 1 are the hits, similarly 2*j and 2*j+1
            nLayerOverlap++;
            temp = nHitOverlap; //before the hit matching shenanigans

            //FIXME:Assumption, 2*i and 2*i+1 hits are known to be from partner modules!
            if(anchorHitIndices[2 * i] == outerObjectHitIndices[2 * j])
            {
                nHitOverlap++;
            }
            else //check for same module indices
            {
                if(anchorLowerModuleIndices[i] != outerObjectLowerModuleIndices[j])
                {
                    pass = false;
                }
            }
            if(anchorHitIndices[2*i+1] == outerObjectHitIndices[2*j+1])
            {
                nHitOverlap++;
            }
            else //check for same module indices
            {
                if(anchorLowerModuleIndices[i] != outerObjectLowerModuleIndices[j])
                {
                    pass = false;
                }
            }
            
            if(nHitOverlap == temp) //check for staggered modules!
            {
                //this is a redemption case. If both modules did not match in the above case,
                //this case should redeem them!

                //find the neighbours of the anchor lower module, if any of those matches the outer lower module, we're done
                unsigned int nStaggeredModules;                
                findStaggeredNeighbours(modulesInGPU, anchorLowerModuleIndices[i], staggeredNeighbours, nStaggeredModules);
                for(size_t idx = 0; idx < nStaggeredModules; idx++)
                {
                    if(outerObjectLowerModuleIndices[j]  == staggeredNeighbours[idx])
                    {
                        //redeemed!
                        pass = true;
                    }
                }
            }

            j++;
            if(j == nOuterLayers)
            {
                break;
            }
        }
    }
    pass = pass & (nLayerOverlap == layerOverlapTarget); //not really required, because these cases should be handled by the other conditions
    return pass;
}


/* r-z and r-phi chi squared computation*/
__device__ float SDL::computeTERZChiSquared(struct modules& modulesInGPU, struct hits& hitsInGPU, unsigned int* anchorObjectAnchorHitIndices, unsigned int* anchorLowerModuleIndices, unsigned int* outerObjectAnchorHitIndices, unsigned int* outerLowerModuleIndices, short anchorObjectType)
{
    //using the pixel hits to create the slope
    float slope = 0, intercept = 0, RMSE = 0;
    if(anchorObjectType != 4)
    {
        //use the first two anchor object anchor hits (i.e., the pixel hits)
        float& rtPix1 = hitsInGPU.rts[anchorObjectAnchorHitIndices[0]];
        float& rtPix2 = hitsInGPU.rts[anchorObjectAnchorHitIndices[1]];

        float& zPix1 = hitsInGPU.zs[anchorObjectAnchorHitIndices[0]];
        float& zPix2 = hitsInGPU.zs[anchorObjectAnchorHitIndices[1]];

        slope = (zPix2 - zPix1)/(rtPix2 - rtPix1);
        intercept = zPix1 - slope * rtPix1;
    }
    else
    {
        /*only PS modules taken into consideration*/
        float rts[5], zs[5];
        int nPoints = 0;
        for(size_t i =0; i < 5; i++)
        {
            if(modulesInGPU.moduleType[anchorLowerModuleIndices[i]] == SDL::PS)
            {
                rts[nPoints] = hitsInGPU.rts[anchorObjectAnchorHitIndices[i]];
                zs[nPoints] = hitsInGPU.zs[anchorObjectAnchorHitIndices[i]];
                nPoints++;
            }
        }
        if(nPoints <= 1)
        {
            slope = 0;
            intercept = 0;
        }
        else
        {
            fitStraightLine(nPoints, rts, zs, slope, intercept);
        }
    }
    if(slope != 0 and intercept != 0)
    {
        float rtAnchor, zAnchor, residual, error, drdz;
        for(size_t i = 0; i < 3; i++)
        {
            unsigned int& anchorHitIndex = outerObjectAnchorHitIndices[i];
            unsigned int& lowerModuleIndex = outerLowerModuleIndices[i];
            rtAnchor = hitsInGPU.rts[anchorHitIndex];
            zAnchor = hitsInGPU.zs[anchorHitIndex];

            //outerModuleAnchorHitIndices
            const int moduleType = modulesInGPU.moduleType[lowerModuleIndex];
            const int moduleSide = modulesInGPU.sides[lowerModuleIndex];
            const int moduleLayerType = modulesInGPU.moduleLayerType[lowerModuleIndex];
            const int layer = modulesInGPU.layers[lowerModuleIndex] + 6 * (modulesInGPU.subdets[lowerModuleIndex] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex] == SDL::TwoS);
        
            residual = (layer <= 6) ?  zAnchor - (slope * rtAnchor + intercept) : rtAnchor - (zAnchor/slope + intercept/slope);
        
            //PS Modules
            if(moduleType == 0)
            {
                error = 0.15;
            }
            else //2S modules
            {
                error = 5.0;
            }

            //special dispensation to tilted PS modules!
            if(moduleType == 0 and layer <= 6 and moduleSide != Center)
            {
                if(moduleLayerType == Strip)
                {
                    drdz = modulesInGPU.drdzs[lowerModuleIndex];
                }
                else
                {
                    drdz = modulesInGPU.drdzs[modulesInGPU.partnerModuleIndex(lowerModuleIndex)];
                }

                error *= 1/sqrtf(1 + drdz * drdz);
            }
            RMSE += (residual * residual)/(error * error);
        }
        RMSE = sqrtf(0.33 * RMSE);
    }
    return RMSE;
}

__device__ void SDL::fitStraightLine(int nPoints, float* xs, float* ys, float& slope, float& intercept)
{
    float sigmaX2(0), sigmaXY(0), sigmaX(0), sigmaY(0), sigma1(0);
    sigma1 = nPoints;

    for(size_t i=0; i<nPoints; i++)
    {
        sigmaX2 += (xs[i] * xs[i]);
        sigmaXY += (xs[i] * ys[i]);
        sigmaX += xs[i];
        sigmaY += ys[i];
        sigma1 ++;
    }

    float denominator = sigma1 * sigmaX2 - sigmaX * sigmaX;
    intercept = (sigmaX2 * sigmaY - sigmaX * sigmaXY) / denominator;
    slope = (sigmaXY - sigmaX * sigmaY) / denominator;
}

__device__ float SDL::computeTERPhiChiSquared(struct modules& modulesInGPU, struct hits& hitsInGPU, float& g, float& f, float& radius, unsigned int* outerObjectAnchorHits, unsigned int* outerObjectLowerModuleIndices)
{
    //Three cases
    float delta1[3], delta2[3], slopes[3], xs[3], ys[3];
    bool isFlat[3];
    computeSigmasForRegression(modulesInGPU, outerObjectLowerModuleIndices, delta1, delta2, slopes, isFlat, 3, true);

    for(size_t i = 0; i < 3; i++)
    {
        xs[i] = hitsInGPU.xs[outerObjectAnchorHits[i]];
        ys[i] = hitsInGPU.ys[outerObjectAnchorHits[i]];
    }
    float chiSquared = computeChiSquared(3, xs, ys, delta1, delta2, slopes, isFlat, g, f, radius);
    return chiSquared;
}
