# include "TrackExtensions.cuh"

SDL::trackExtensions::trackExtensions()
{
    constituentTCTypes = nullptr;
    constituentTCIndices = nullptr;
    nTrackExtensions = nullptr;
    rPhiChiSquared = nullptr;
    isDup = nullptr;
}

SDL::trackExtensions::~trackExtensions()
{
}

void SDL::trackExtensions::freeMemory()
{
    hipFree(constituentTCTypes);
    hipFree(constituentTCIndices);
    hipFree(nTrackExtensions);
    hipFree(isDup);
    hipFree(rPhiChiSquared);
}

/*
   Track Extensions memory allocation - 10 slots for each TC (will reduce later)
   Extensions having the same anchor object will be clustered together for easy
   duplicate cleaning
*/

void SDL::createTrackExtensionsInUnifiedMemory(struct trackExtensions& trackExtensionsInGPU, unsigned int maxTrackExtensions, unsigned int nTrackCandidates)
{
    hipMallocManaged(&trackExtensionsInGPU.constituentTCTypes, sizeof(short) * 3 * maxTrackExtensions);
    hipMallocManaged(&trackExtensionsInGPU.constituentTCIndices, sizeof(unsigned int) * 3 * maxTrackExtensions);
    hipMallocManaged(&trackExtensionsInGPU.nLayerOverlaps, sizeof(unsigned int) * 2 * maxTrackExtensions);
    hipMallocManaged(&trackExtensionsInGPU.nHitOverlaps, sizeof(unsigned int) * 2 * maxTrackExtensions);
    hipMallocManaged(&trackExtensionsInGPU.nTrackExtensions, nTrackCandidates * sizeof(unsigned int));
    hipMallocManaged(&trackExtensionsInGPU.rPhiChiSquared, maxTrackExtensions * sizeof(float));
    hipMallocManaged(&trackExtensionsInGPU.isDup, maxTrackExtensions * sizeof(bool));

    hipMemset(trackExtensionsInGPU.nTrackExtensions, 0, nTrackCandidates * sizeof(unsigned int));
    hipMemset(trackExtensionsInGPU.isDup, true, maxTrackExtensions * sizeof(bool));
}

void SDL::createTrackExtensionsInExplicitMemory(struct trackExtensions& trackExtensionsInGPU, unsigned int maxTrackExtensions, unsigned int nTrackCandidates)
{
    hipMalloc(&trackExtensionsInGPU.constituentTCTypes, sizeof(short) * 3 * maxTrackExtensions);
    hipMalloc(&trackExtensionsInGPU.constituentTCIndices, sizeof(unsigned int) * 3 * maxTrackExtensions);
    hipMalloc(&trackExtensionsInGPU.nLayerOverlaps, sizeof(unsigned int) * 2 * maxTrackExtensions);
    hipMalloc(&trackExtensionsInGPU.nHitOverlaps, sizeof(unsigned int) * 2 * maxTrackExtensions);
    hipMalloc(&trackExtensionsInGPU.nTrackExtensions, nTrackCandidates * sizeof(unsigned int));
    hipMalloc(&trackExtensionsInGPU.rPhiChiSquared, maxTrackExtensions * sizeof(float));
    hipMalloc(&trackExtensionsInGPU.isDup, maxTrackExtensions * sizeof(bool));

    hipMemset(trackExtensionsInGPU.nTrackExtensions, 0, nTrackCandidates * sizeof(unsigned int));
    hipMemset(trackExtensionsInGPU.isDup, true, maxTrackExtensions * sizeof(bool));
}

__device__ void SDL::addTrackExtensionToMemory(struct trackExtensions& trackExtensionsInGPU, short* constituentTCType, unsigned int* constituentTCIndex, unsigned int* nLayerOverlaps, unsigned int* nHitOverlaps, float rPhiChiSquared, unsigned int trackExtensionIndex)
{ 
    for(size_t i = 0; i < 3 ; i++)
    {
        trackExtensionsInGPU.constituentTCTypes[3 * trackExtensionIndex + i] = constituentTCType[i];
        trackExtensionsInGPU.constituentTCIndices[3 * trackExtensionIndex + i] = constituentTCIndex[i];
    }
    for(size_t i = 0; i < 2; i++)
    {
        trackExtensionsInGPU.nLayerOverlaps[2 * trackExtensionIndex + i] = nLayerOverlaps[i];
        trackExtensionsInGPU.nHitOverlaps[2 * trackExtensionIndex + i] = nHitOverlaps[i];
    }
    trackExtensionsInGPU.rPhiChiSquared[trackExtensionIndex] = rPhiChiSquared;
}

__device__ bool SDL::runTrackExtensionDefaultAlgo(struct modules& modulesInGPU, struct hits& hitsInGPU, struct miniDoublets& mdsInGPU, struct segments& segmentsInGPU, struct triplets& tripletsInGPU, struct quintuplets& quintupletsInGPU, struct pixelTriplets& pixelTripletsInGPU, struct pixelQuintuplets& pixelQuintupletsInGPU, struct trackCandidates& trackCandidatesInGPU, unsigned int anchorObjectIndex, unsigned int outerObjectIndex, short anchorObjectType, short outerObjectType, unsigned int anchorObjectOuterT3Index, unsigned int layerOverlapTarget, short* constituentTCType, unsigned int* constituentTCIndex, unsigned
        int* nLayerOverlaps, unsigned int* nHitOverlaps, float& rPhiChiSquared)
{
    /*
       Basic premise:
       1. given two objects, get the hit and module indices
       2. check for layer and hit overlap (layer overlap first checked using
       the 2-merge approach)
       3. Additional cuts - rz and rphi chi squared criteria! (TODO) 
    */

    bool pass = true;
    unsigned int* anchorLayerIndices = nullptr;
    unsigned int* anchorHitIndices = nullptr;
    unsigned int* anchorLowerModuleIndices = nullptr;

    unsigned int* outerObjectLayerIndices = nullptr;
    unsigned int* outerObjectHitIndices = nullptr;
    unsigned int* outerObjectLowerModuleIndices = nullptr;

    unsigned int nAnchorLayers = (anchorObjectType == 7) ? 7 : (anchorObjectType == 3 ? 3 : 5);
    unsigned int anchorObjectAnchorHitIndices[7];
    float centerX, centerY, radius;

    if(anchorObjectType != 3) //mostly this
    { 
        anchorLayerIndices = &trackCandidatesInGPU.logicalLayers[7 * anchorObjectIndex];
        anchorHitIndices = &trackCandidatesInGPU.hitIndices[14 * anchorObjectIndex];
        anchorLowerModuleIndices = &trackCandidatesInGPU.lowerModuleIndices[7 * anchorObjectIndex];
        centerX = trackCandidatesInGPU.centerX[anchorObjectIndex];
        centerY = trackCandidatesInGPU.centerY[anchorObjectIndex];
        radius = trackCandidatesInGPU.radius[anchorObjectIndex];
    }
    else //outlier
    {
        anchorLayerIndices = &tripletsInGPU.logicalLayers[3 * anchorObjectIndex];
        anchorHitIndices = &tripletsInGPU.hitIndices[6 * anchorObjectIndex];
        anchorLowerModuleIndices = &tripletsInGPU.lowerModuleIndices[3 * anchorObjectIndex];

    }

    for(size_t i=0; i<nAnchorLayers;i++)
    {
        if(modulesInGPU.isAnchor[hitsInGPU.moduleIndices[anchorHitIndices[2*i]]] or modulesInGPU.detIds[hitsInGPU.moduleIndices[anchorHitIndices[2*i]]] == 1)
        {
            anchorObjectAnchorHitIndices[i] = anchorHitIndices[2*i];
        }
        else
        {
            anchorObjectAnchorHitIndices[i] = anchorHitIndices[2*i+1];
        }
    }

    unsigned int nOuterLayers =(outerObjectType == 7) ? 7 : (outerObjectType == 3 ? 3 : 5); 

    unsigned int outerObjectAnchorHitIndices[7];
    if(outerObjectType == 3) //mostly this
    {
        outerObjectLayerIndices = &tripletsInGPU.logicalLayers[3 * outerObjectIndex];
        outerObjectHitIndices = &tripletsInGPU.hitIndices[6 * outerObjectIndex];
        outerObjectLowerModuleIndices = &tripletsInGPU.lowerModuleIndices[3 * outerObjectIndex];
    }
    else //outlier
    {
        outerObjectLayerIndices = &trackCandidatesInGPU.logicalLayers[7 * outerObjectIndex];
        outerObjectHitIndices = &trackCandidatesInGPU.hitIndices[14 * outerObjectIndex];
        outerObjectLowerModuleIndices = &tripletsInGPU.lowerModuleIndices[7 * outerObjectIndex];
    }

    for(size_t i=0; i<nOuterLayers;i++)
    {
        if(modulesInGPU.isAnchor[hitsInGPU.moduleIndices[outerObjectHitIndices[2*i]]] or modulesInGPU.detIds[hitsInGPU.moduleIndices[outerObjectHitIndices[2*i]]] == 1)
        {
            outerObjectAnchorHitIndices[i] = outerObjectHitIndices[2*i];
        }
        else
        {
            outerObjectAnchorHitIndices[i] = outerObjectHitIndices[2*i+1];
        }
    }
    
    unsigned int nLayerOverlap(0), nHitOverlap(0);
   
    float zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta; //temp stuff
    float zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ;

    //checks for frivolous cases wherein
    pass = pass &  computeLayerAndHitOverlaps(modulesInGPU, anchorLayerIndices, anchorHitIndices, anchorLowerModuleIndices, outerObjectLayerIndices, outerObjectHitIndices, outerObjectLowerModuleIndices, nAnchorLayers, nOuterLayers, nLayerOverlap, nHitOverlap, layerOverlapTarget);

    pass = pass & runTrackletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, tripletsInGPU.lowerModuleIndices[3 * anchorObjectOuterT3Index], tripletsInGPU.lowerModuleIndices[3 * anchorObjectOuterT3Index + 1], tripletsInGPU.lowerModuleIndices[3 * outerObjectIndex], tripletsInGPU.lowerModuleIndices[3 * outerObjectIndex + 1], tripletsInGPU.segmentIndices[2 * anchorObjectOuterT3Index], tripletsInGPU.segmentIndices[2 * outerObjectIndex], zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta, zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ);

    pass = pass & runTrackletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, tripletsInGPU.lowerModuleIndices[3 * anchorObjectOuterT3Index], tripletsInGPU.lowerModuleIndices[3 * anchorObjectOuterT3Index + 1], tripletsInGPU.lowerModuleIndices[3 * outerObjectIndex + 1], tripletsInGPU.lowerModuleIndices[3 * outerObjectIndex + 2], tripletsInGPU.segmentIndices[2 * anchorObjectOuterT3Index], tripletsInGPU.segmentIndices[2 * outerObjectIndex + 1], zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta, zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ);

    rPhiChiSquared = computeTERPhiChiSquared(modulesInGPU, hitsInGPU, centerX, centerY, radius, outerObjectAnchorHitIndices, outerObjectLowerModuleIndices);

    nLayerOverlaps[0] = nLayerOverlap;
    nHitOverlaps[0] = nHitOverlap;

    constituentTCType[0] = anchorObjectType;
    constituentTCType[1] = outerObjectType;

    constituentTCIndex[0] = anchorObjectIndex;
    constituentTCIndex[1] = outerObjectIndex;

   return pass;
}



/*
   If one out of the two hits don't overlap - check if the module indices are identical (case of multiple reco hits)
   If both hits don't overlap - check the above, and check for staggered modules using the staggered module neighbours list

   This function i complicated - computes layer overlaps and checks if layer matches and hit matches are "compatible" i.e., layer overlap = 2 * hit overlap, or if that's not the case, we know why (multiple reco hits/staggered modules)
*/
__device__ bool SDL::computeLayerAndHitOverlaps(SDL::modules& modulesInGPU, unsigned int* anchorLayerIndices, unsigned int* anchorHitIndices, unsigned int* anchorLowerModuleIndices, unsigned int* outerObjectLayerIndices, unsigned int* outerObjectHitIndices, unsigned int* outerObjectLowerModuleIndices, unsigned int nAnchorLayers, unsigned int nOuterLayers, unsigned int& nLayerOverlap, unsigned int& nHitOverlap, unsigned int& layerOverlapTarget)
{
    bool pass = true;
    //merge technique!
    size_t j = 0; //outer object tracker
    unsigned int temp; //container variable
    unsigned int staggeredNeighbours[10];
    for(size_t i = 0; i < nAnchorLayers; i++)
    {
        if(anchorLayerIndices[i] == outerObjectLayerIndices[j])
        {
            //2*i and 2*i + 1 are the hits, similarly 2*j and 2*j+1
            nLayerOverlap++;
            temp = nHitOverlap; //before the hit matching shenanigans

            //FIXME:Assumption, 2*i and 2*i+1 hits are known to be from partner modules!
            if(anchorHitIndices[2 * i] == outerObjectHitIndices[2 * j])
            {
                nHitOverlap++;
            }
            else //check for same module indices
            {
                if(anchorLowerModuleIndices[i] != outerObjectLowerModuleIndices[j])
                {
                    pass = false;
                }
            }
            if(anchorHitIndices[2*i+1] == outerObjectHitIndices[2*j+1])
            {
                nHitOverlap++;
            }
            else //check for same module indices
            {
                if(anchorLowerModuleIndices[i] != outerObjectLowerModuleIndices[j])
                {
                    pass = false;
                }
            }
            
            if(nHitOverlap == temp) //check for staggered modules!
            {
                //this is a redemption case. If both modules did not match in the above case,
                //this case should redeem them!

                //find the neighbours of the anchor lower module, if any of those matches the outer lower module, we're done
                unsigned int nStaggeredModules;                
                findStaggeredNeighbours(modulesInGPU, anchorLowerModuleIndices[i], staggeredNeighbours, nStaggeredModules);
                for(size_t idx = 0; idx < nStaggeredModules; idx++)
                {
                    if(outerObjectLowerModuleIndices[j]  == staggeredNeighbours[idx])
                    {
                        //redeemed!
                        pass = true;
                    }
                }
            }

            j++;
            if(j == nOuterLayers)
            {
                break;
            }
        }
    }
    pass = pass & (nLayerOverlap == layerOverlapTarget); //not really required, because these cases should be handled by the other conditions
    return pass;
}


/* r-z and r-phi chi squared computation*/
__device__ float computeTERZChiSquared(struct modules& modulesInGPU, struct hits& hitsInGPU, unsigned int* anchorHitIndices, unsigned int* anchorLowerModuleIndices, unsigned int* outerHitIndices, unsigned int* outerLowerModuleIndices)
{
    //using the pixel hits to create the slope
    
}

__device__ float SDL::computeTERPhiChiSquared(struct modules& modulesInGPU, struct hits& hitsInGPU, float& g, float& f, float& radius, unsigned int* outerObjectAnchorHits, unsigned int* outerObjectLowerModuleIndices)
{
    //Three cases
    float delta1[3], delta2[3], slopes[3], xs[3], ys[3];
    bool isFlat[3];
    computeSigmasForRegression(modulesInGPU, outerObjectLowerModuleIndices, delta1, delta2, slopes, isFlat, 3, true);

    for(size_t i = 0; i < 3; i++)
    {
        xs[i] = hitsInGPU.xs[outerObjectAnchorHits[i]];
        ys[i] = hitsInGPU.ys[outerObjectAnchorHits[i]];
    }
    float chiSquared = computeChiSquared(3, xs, ys, delta1, delta2, slopes, isFlat, g, f, radius);
    return chiSquared;
}
