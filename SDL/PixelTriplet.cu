#include "hip/hip_runtime.h"
# include "PixelTriplet.cuh"
#include "allocate.h"

SDL::pixelTriplets::pixelTriplets()
{
    pixelSegmentIndices = nullptr;
    tripletIndices = nullptr;
    nPixelTriplets = nullptr;
    pixelRadius = nullptr;
    tripletRadius = nullptr;
    pt = nullptr;
    isDup = nullptr;
    partOfPT5 = nullptr;
#ifdef CUT_VALUE_DEBUG
    pixelRadiusError = nullptr;
    rzChiSquared = nullptr;
    rPhiChiSquared = nullptr;
    rPhiChiSquaredInwards = nullptr;
#endif
}

void SDL::pixelTriplets::freeMemory()
{
    hipFree(pixelSegmentIndices);
    hipFree(tripletIndices);
    hipFree(nPixelTriplets);
    hipFree(pixelRadius);
    hipFree(tripletRadius);
    hipFree(pt);
    hipFree(isDup);
    hipFree(partOfPT5);
#ifdef CUT_VALUE_DEBUG
    hipFree(pixelRadiusError);
    hipFree(rPhiChiSquared);
    hipFree(rPhiChiSquaredInwards);
    hipFree(rzChiSquared);
#endif
}

SDL::pixelTriplets::~pixelTriplets()
{
}

void SDL::createPixelTripletsInUnifiedMemory(struct pixelTriplets& pixelTripletsInGPU, unsigned int maxPixelTriplets)
{
    hipMallocManaged(&pixelTripletsInGPU.pixelSegmentIndices, maxPixelTriplets * sizeof(float));
    hipMallocManaged(&pixelTripletsInGPU.tripletIndices, maxPixelTriplets * sizeof(float));
    hipMallocManaged(&pixelTripletsInGPU.nPixelTriplets, sizeof(unsigned int));
    hipMallocManaged(&pixelTripletsInGPU.pixelRadius, maxPixelTriplets * sizeof(float));
    hipMallocManaged(&pixelTripletsInGPU.tripletRadius, maxPixelTriplets * sizeof(float));
    hipMallocManaged(&pixelTripletsInGPU.pt, maxPixelTriplets * 6*sizeof(float));
    hipMallocManaged(&pixelTripletsInGPU.isDup, maxPixelTriplets * sizeof(bool));
    hipMallocManaged(&pixelTripletsInGPU.partOfPT5, maxPixelTriplets * sizeof(bool));
#ifdef CUT_VALUE_DEBUG
    hipMallocManaged(&pixelTripletsInGPU.pixelRadiusError, maxPixelTriplets * sizeof(float));
    hipMallocManaged(&pixelTripletsInGPU.rPhiChiSquared, maxPixelTriplets * sizeof(float));
    hipMallocManaged(&pixelTripletsInGPU.rPhiChiSquaredInwards, maxPixelTriplets * sizeof(float));
    hipMallocManaged(&pixelTripletsInGPU.rzChiSquared, maxPixelTriplets * sizeof(float));
#endif
    pixelTripletsInGPU.eta = pixelTripletsInGPU.pt + maxPixelTriplets;
    pixelTripletsInGPU.phi = pixelTripletsInGPU.pt + maxPixelTriplets * 2;
    pixelTripletsInGPU.eta_pix = pixelTripletsInGPU.pt + maxPixelTriplets *3;
    pixelTripletsInGPU.phi_pix = pixelTripletsInGPU.pt + maxPixelTriplets * 4;
    pixelTripletsInGPU.score = pixelTripletsInGPU.pt + maxPixelTriplets * 5;
    hipMemset(pixelTripletsInGPU.nPixelTriplets, 0, sizeof(unsigned int));
}

void SDL::createPixelTripletsInExplicitMemory(struct pixelTriplets& pixelTripletsInGPU, unsigned int maxPixelTriplets)
{
    hipMalloc(&pixelTripletsInGPU.pixelSegmentIndices, maxPixelTriplets * sizeof(float));
    hipMalloc(&pixelTripletsInGPU.tripletIndices, maxPixelTriplets * sizeof(float));
    hipMalloc(&pixelTripletsInGPU.nPixelTriplets, sizeof(unsigned int));
    hipMalloc(&pixelTripletsInGPU.pixelRadius, maxPixelTriplets * sizeof(float));
    hipMalloc(&pixelTripletsInGPU.tripletRadius, maxPixelTriplets * sizeof(float));
    hipMalloc(&pixelTripletsInGPU.pt, maxPixelTriplets * 6*sizeof(float));
    hipMalloc(&pixelTripletsInGPU.isDup, maxPixelTriplets * sizeof(bool));
    hipMalloc(&pixelTripletsInGPU.partOfPT5, maxPixelTriplets * sizeof(bool));

    pixelTripletsInGPU.eta = pixelTripletsInGPU.pt + maxPixelTriplets;
    pixelTripletsInGPU.phi = pixelTripletsInGPU.pt + maxPixelTriplets * 2;
    pixelTripletsInGPU.eta_pix = pixelTripletsInGPU.pt + maxPixelTriplets *3;
    pixelTripletsInGPU.phi_pix = pixelTripletsInGPU.pt + maxPixelTriplets * 4;
    pixelTripletsInGPU.score = pixelTripletsInGPU.pt + maxPixelTriplets * 5;
    hipMemset(pixelTripletsInGPU.nPixelTriplets, 0, sizeof(unsigned int));

}

#ifdef CUT_VALUE_DEBUG
__device__ void SDL::addPixelTripletToMemory(struct pixelTriplets& pixelTripletsInGPU, unsigned int pixelSegmentIndex, unsigned int tripletIndex, float pixelRadius, float pixelRadiusError, float tripletRadius, float rPhiChiSquared, float rPhiChiSquaredInwards, float rzChiSquared, unsigned int pixelTripletIndex, float pt, float eta, float phi, float eta_pix, float phi_pix, float score)
#else
__device__ void SDL::addPixelTripletToMemory(struct pixelTriplets& pixelTripletsInGPU, unsigned int pixelSegmentIndex, unsigned int tripletIndex, float pixelRadius, float tripletRadius, unsigned int pixelTripletIndex, float pt, float eta, float phi, float eta_pix, float phi_pix,float score)
#endif
{
    pixelTripletsInGPU.pixelSegmentIndices[pixelTripletIndex] = pixelSegmentIndex;
    pixelTripletsInGPU.tripletIndices[pixelTripletIndex] = tripletIndex;
    pixelTripletsInGPU.pixelRadius[pixelTripletIndex] = pixelRadius;
    pixelTripletsInGPU.tripletRadius[pixelTripletIndex] = tripletRadius;
    pixelTripletsInGPU.pt[pixelTripletIndex] = pt;
    pixelTripletsInGPU.eta[pixelTripletIndex] = eta;
    pixelTripletsInGPU.phi[pixelTripletIndex] = phi;
    pixelTripletsInGPU.eta_pix[pixelTripletIndex] = eta_pix;
    pixelTripletsInGPU.phi_pix[pixelTripletIndex] = phi_pix;
    pixelTripletsInGPU.isDup[pixelTripletIndex] = 0;
    pixelTripletsInGPU.score[pixelTripletIndex] = score;

#ifdef CUT_VALUE_DEBUG
    pixelTripletsInGPU.pixelRadiusError[pixelTripletIndex] = pixelRadiusError;
    pixelTripletsInGPU.rPhiChiSquared[pixelTripletIndex] = rPhiChiSquared;
    pixelTripletsInGPU.rPhiChiSquaredInwards[pixelTripletIndex] = rPhiChiSquaredInwards;
    pixelTripletsInGPU.rzChiSquared[pixelTripletIndex] = rzChiSquared;
#endif
}
__device__ void SDL::rmPixelTripletToMemory(struct pixelTriplets& pixelTripletsInGPU,unsigned int pixelTripletIndex)
{
    pixelTripletsInGPU.isDup[pixelTripletIndex] = 1;
}

__device__ bool SDL::runPixelTripletDefaultAlgo(struct modules& modulesInGPU, struct hits& hitsInGPU, struct miniDoublets& mdsInGPU, struct segments& segmentsInGPU, struct triplets& tripletsInGPU, unsigned int& pixelSegmentIndex, unsigned int tripletIndex, float& pixelRadius, float& pixelRadiusError, float& tripletRadius, float& rzChiSquared, float& rPhiChiSquared, float& rPhiChiSquaredInwards, bool runChiSquaredCuts)
{
    bool pass = true;

    //run pT4 compatibility between the pixel segment and inner segment, and between the pixel and outer segment of the triplet


    //placeholder
    float zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta; //temp stuff
    float zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ;

    unsigned int pixelModuleIndex = segmentsInGPU.innerLowerModuleIndices[pixelSegmentIndex];

    unsigned int lowerModuleIndex = tripletsInGPU.lowerModuleIndices[3 * tripletIndex];
    unsigned int middleModuleIndex = tripletsInGPU.lowerModuleIndices[3 * tripletIndex + 1];
    unsigned int upperModuleIndex = tripletsInGPU.lowerModuleIndices[3 * tripletIndex + 2];


    // pixel segment vs inner segment of the triplet
    pass = pass & runPixelTrackletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, pixelModuleIndex, pixelModuleIndex, lowerModuleIndex, middleModuleIndex, pixelSegmentIndex, tripletsInGPU.segmentIndices[2 * tripletIndex], zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta, zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ);

    //pixel segment vs outer segment of triplet
    pass = pass & runPixelTrackletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, pixelModuleIndex, pixelModuleIndex, middleModuleIndex, upperModuleIndex, pixelSegmentIndex, tripletsInGPU.segmentIndices[2 * tripletIndex + 1], zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta, zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ);

    //pt matching between the pixel ptin and the triplet circle pt
    unsigned int pixelSegmentArrayIndex = pixelSegmentIndex - (pixelModuleIndex * 600);
    float pixelSegmentPt = segmentsInGPU.ptIn[pixelSegmentArrayIndex];
    float pixelSegmentPtError = segmentsInGPU.ptErr[pixelSegmentArrayIndex];

    unsigned int pixelInnerMDIndex = segmentsInGPU.mdIndices[2 * pixelSegmentIndex];
    unsigned int pixelOuterMDIndex = segmentsInGPU.mdIndices[2 * pixelSegmentIndex + 1];

    unsigned int pixelAnchorHitIndex1 = mdsInGPU.hitIndices[2 * pixelInnerMDIndex];
    unsigned int pixelNonAnchorHitIndex1 = mdsInGPU.hitIndices[2 * pixelInnerMDIndex + 1];
    unsigned int pixelAnchorHitIndex2 = mdsInGPU.hitIndices[2 * pixelOuterMDIndex];
    unsigned int pixelNonAnchorHitIndex2 = mdsInGPU.hitIndices[2 * pixelOuterMDIndex + 1];

    pixelRadius = pixelSegmentPt/(2 * k2Rinv1GeVf);
    pixelRadiusError = pixelSegmentPtError/(2 * k2Rinv1GeVf);
    unsigned int tripletInnerSegmentIndex = tripletsInGPU.segmentIndices[2 * tripletIndex];
    unsigned int tripletOuterSegmentIndex = tripletsInGPU.segmentIndices[2 * tripletIndex + 1];

    unsigned int innerMDAnchorHitIndex = segmentsInGPU.innerMiniDoubletAnchorHitIndices[tripletInnerSegmentIndex];
    unsigned int middleMDAnchorHitIndex = segmentsInGPU.outerMiniDoubletAnchorHitIndices[tripletInnerSegmentIndex];
    unsigned int outerMDAnchorHitIndex = segmentsInGPU.outerMiniDoubletAnchorHitIndices[tripletOuterSegmentIndex];

    float x1 = hitsInGPU.xs[innerMDAnchorHitIndex];
    float x2 = hitsInGPU.xs[middleMDAnchorHitIndex];
    float x3 = hitsInGPU.xs[outerMDAnchorHitIndex];

    float y1 = hitsInGPU.ys[innerMDAnchorHitIndex];
    float y2 = hitsInGPU.ys[middleMDAnchorHitIndex];
    float y3 = hitsInGPU.ys[outerMDAnchorHitIndex];
    float g,f;
    
    tripletRadius = computeRadiusFromThreeAnchorHits(x1, y1, x2, y2, x3, y3,g,f);
    
    pass = pass & passRadiusCriterion(modulesInGPU, pixelRadius, pixelRadiusError, tripletRadius, lowerModuleIndex, middleModuleIndex, upperModuleIndex);

    unsigned int anchorHits[] = {innerMDAnchorHitIndex, middleMDAnchorHitIndex, outerMDAnchorHitIndex};
    unsigned int pixelAnchorHits[] = {pixelAnchorHitIndex1, pixelAnchorHitIndex2};
    unsigned int lowerModuleIndices[] = {lowerModuleIndex, middleModuleIndex, upperModuleIndex};

    rzChiSquared = computePT3RZChiSquared(modulesInGPU, hitsInGPU, segmentsInGPU, pixelAnchorHitIndex1, pixelAnchorHitIndex2, anchorHits, lowerModuleIndices);

    rPhiChiSquared = computePT3RPhiChiSquared(modulesInGPU, hitsInGPU, segmentsInGPU, pixelSegmentArrayIndex, anchorHits, lowerModuleIndices);

    rPhiChiSquaredInwards = computePT3RPhiChiSquaredInwards(modulesInGPU, hitsInGPU, tripletRadius, g, f, pixelAnchorHits);

    if(runChiSquaredCuts and pixelSegmentPt < 5.0)
    {
        pass = pass & passPT3RZChiSquaredCuts(modulesInGPU, lowerModuleIndex, middleModuleIndex, upperModuleIndex, rzChiSquared);
        pass = pass & passPT3RPhiChiSquaredCuts(modulesInGPU, lowerModuleIndex, middleModuleIndex, upperModuleIndex, rPhiChiSquared);

        pass = pass & passPT3RPhiChiSquaredInwardsCuts(modulesInGPU, lowerModuleIndex, middleModuleIndex, upperModuleIndex, rPhiChiSquaredInwards);
    }


    return pass;

}

__device__ bool SDL::passPT3RPhiChiSquaredInwardsCuts(struct modules& modulesInGPU, unsigned int lowerModuleIndex1, unsigned int lowerModuleIndex2, unsigned int lowerModuleIndex3, float& chiSquared)
{
    const int layer1 = modulesInGPU.layers[lowerModuleIndex1] + 6 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex1] == SDL::TwoS);
    const int layer2 = modulesInGPU.layers[lowerModuleIndex2] + 6 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex2] == SDL::TwoS);
    const int layer3 = modulesInGPU.layers[lowerModuleIndex3] + 6 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex3] == SDL::TwoS);
    
    if(layer1 == 7 and layer2 == 8 and layer3 == 9)
    {
        return chiSquared < 22016.8055;
    }
    else if(layer1 == 7 and layer2 == 8 and layer3 == 14)
    {
        return chiSquared < 935179.56807;
    }
    else if(layer1 == 8 and layer2 == 9 and layer3 == 10)
    {
        return chiSquared < 29064.12959;
    }
    else if(layer1 == 8 and layer2 == 9 and layer3 == 15)
    {
        return chiSquared < 935179.5681;
    }
    else if(layer1 == 1 and layer2 == 2 and layer3 == 3)
    {
        return chiSquared < 1370.0113195101474;
    }
    else if(layer1 == 1 and layer2 == 2 and layer3 == 7)
    {
        return chiSquared < 5492.110048314815;
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 4)
    {
        return chiSquared < 4160.410806470067;
    }
    else if(layer1 == 1 and layer2 == 7 and layer3 == 8)
    {
        return chiSquared < 29064.129591225726;
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 7)
    {
        return chiSquared < 12634.215376250893;
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 12)
    {
        return chiSquared < 353821.69361145404;
    }
    else if(layer1 == 2 and layer2 == 7 and layer3 == 8)
    {
        return chiSquared < 33393.26076341235;
    }
    else if(layer1 == 2 and layer2 == 7 and layer3 == 13)
    {
        return chiSquared < 935179.5680742573;
    }

    return true;
}

__device__ float SDL::computePT3RPhiChiSquaredInwards(struct modules& modulesInGPU, struct hits& hitsInGPU, float& r, float& g, float& f, unsigned int* pixelAnchorHits)
{
    float x,y;
    float chiSquared = 0;
    for(size_t i = 0; i < 2; i++)
    {
        x = hitsInGPU.xs[pixelAnchorHits[i]];
        y = hitsInGPU.ys[pixelAnchorHits[i]];
        float residual = (x - g) * (x -g) + (y - f) * (y - f) - r * r;
        chiSquared += residual * residual;
    }
    chiSquared /= 2;
    return chiSquared;
}

__device__ bool SDL::passPT3RZChiSquaredCuts(struct modules& modulesInGPU, unsigned int lowerModuleIndex1, unsigned int lowerModuleIndex2, unsigned int lowerModuleIndex3, float& rzChiSquared)
{
    const int layer1 = modulesInGPU.layers[lowerModuleIndex1] + 6 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex1] == SDL::TwoS);
    const int layer2 = modulesInGPU.layers[lowerModuleIndex2] + 6 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex2] == SDL::TwoS);
    const int layer3 = modulesInGPU.layers[lowerModuleIndex3] + 6 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex3] == SDL::TwoS);

    if(layer1 == 8 and layer2 == 9 and layer3 == 10)
    {
        return rzChiSquared < 85.2499;
    }
    else if(layer1 == 8 and layer2 == 9 and layer3 == 15)
    {
        return rzChiSquared < 85.2499;
    }

    else if(layer1 == 7 and layer2 == 8 and layer3 == 9)
    {
        return rzChiSquared < 74.19805;
    }
    else if(layer1 == 7 and layer2 == 8 and layer3 == 14)
    {
        return rzChiSquared < 97.9479;
    }

    else if(layer1 == 1 and layer2 == 2 and layer3 == 3)
    {
        return rzChiSquared < 451.1407;;
    }
    else if(layer1 == 1 and layer2 == 2 and layer3 == 7)
    {
        return rzChiSquared < 595.546;
    }

    else if(layer1 == 1 and layer2 == 7 and layer3 == 8)
    {
        return rzChiSquared < 518.339;
    }

    else if(layer1 == 2 and layer2 == 3 and layer3 == 7)
    {
        return rzChiSquared < 684.253;
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 12)
    {
        return rzChiSquared < 684.253;
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 4)
    {
        return rzChiSquared  < 392.654;
    }

    else if(layer1 == 2 and layer2 == 7 and layer3 == 8)
    {
        return rzChiSquared < 518.339;
    }
    else if(layer1 == 2 and layer2 == 7 and layer3 == 13)
    {
        return rzChiSquared < 518.339;
    }

    //default - category not found!
    return true;
}

__device__ float SDL::computePT3RZChiSquared(struct modules& modulesInGPU, struct hits& hitsInGPU, struct segments& segmentsInGPU, unsigned int& pixelAnchorHitIndex1, unsigned int& pixelAnchorHitIndex2, unsigned int* anchorHits, unsigned int* lowerModuleIndices)
{
    float& rtPix1 = hitsInGPU.rts[pixelAnchorHitIndex1];
    float& rtPix2 = hitsInGPU.rts[pixelAnchorHitIndex2];
    float& zPix1 = hitsInGPU.zs[pixelAnchorHitIndex1];
    float& zPix2 = hitsInGPU.zs[pixelAnchorHitIndex2];
    float slope = (zPix2 - zPix1)/(rtPix2 - rtPix1);
    float rtAnchor, zAnchor;
    float residual = 0;
    float error = 0;
    //hardcoded array indices!!!
    float RMSE = 0;
    float drdz;
    for(size_t i = 0; i < 3; i++)
    {
        unsigned int& anchorHitIndex = anchorHits[i];
        unsigned int& lowerModuleIndex = lowerModuleIndices[i];
        rtAnchor = hitsInGPU.rts[anchorHitIndex];
        zAnchor = hitsInGPU.zs[anchorHitIndex];

        const int moduleType = modulesInGPU.moduleType[lowerModuleIndex];
        const int moduleSide = modulesInGPU.sides[lowerModuleIndex];
        const int moduleLayerType = modulesInGPU.moduleLayerType[lowerModuleIndex];
        const int layer = modulesInGPU.layers[lowerModuleIndex] + 6 * (modulesInGPU.subdets[lowerModuleIndex] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex] == SDL::TwoS);
        
        residual = (layer <= 6) ?  (zAnchor - zPix1) - slope * (rtAnchor - rtPix1) : (rtAnchor - rtPix1) - (zAnchor - zPix1)/slope;
        
        //PS Modules
        if(moduleType == 0)
        {
            error = 0.15;
        }
        else //2S modules
        {
            error = 5.0;
        }

        //special dispensation to tilted PS modules!
        if(moduleType == 0 and layer <= 6 and moduleSide != Center)
        {
            if(moduleLayerType == Strip)
            {
                drdz = modulesInGPU.drdzs[lowerModuleIndex];
            }
            else
            {
                drdz = modulesInGPU.drdzs[modulesInGPU.partnerModuleIndex(lowerModuleIndex)];
            }

            error *= 1/sqrtf(1 + drdz * drdz);
        }
        RMSE += (residual * residual)/(error * error);
    }

    RMSE = sqrtf(0.2 * RMSE); //the constant doesn't really matter....
    return RMSE;
}

//TODO: merge this one and the pT5 function later into a single function
__device__ float SDL::computePT3RPhiChiSquared(struct modules& modulesInGPU, struct hits& hitsInGPU, struct segments& segmentsInGPU, unsigned int pixelSegmentArrayIndex, unsigned int* anchorHits, unsigned int* lowerModuleIndices)
{
    float g = segmentsInGPU.circleCenterX[pixelSegmentArrayIndex];
    float f = segmentsInGPU.circleCenterY[pixelSegmentArrayIndex];
    float radius = segmentsInGPU.circleRadius[pixelSegmentArrayIndex];
    float delta1[3], delta2[3], slopes[3];
    bool isFlat[3];
    float xs[3];
    float ys[3];
    float chiSquared = 0;
    for(size_t i = 0; i < 3; i++)
    {
        xs[i] = hitsInGPU.xs[anchorHits[i]];
        ys[i] = hitsInGPU.ys[anchorHits[i]];
    }

    //computeSigmasForRegression(modulesInGPU, lowerModuleIndices, delta1, delta2, slopes, isFlat, 3);
    ModuleType moduleType;
    short moduleSubdet, moduleSide;
    ModuleLayerType moduleLayerType;
    float drdz;
    for(size_t i=0; i<3; i++)
    {
        moduleType = modulesInGPU.moduleType[lowerModuleIndices[i]];
        moduleSubdet = modulesInGPU.subdets[lowerModuleIndices[i]];
        moduleSide = modulesInGPU.sides[lowerModuleIndices[i]];
        moduleLayerType = modulesInGPU.moduleLayerType[lowerModuleIndices[i]];
        //category 1 - barrel PS flat
        if(moduleSubdet == Barrel and moduleType == PS and moduleSide == Center)
        {
            delta1[i] = 1.1111f;//0.01;
            delta2[i] = 1.1111f;//0.01;
            slopes[i] = -999;
            isFlat[i] = true;
        }

        //category 2 - barrel 2S
        else if(moduleSubdet == Barrel and moduleType == TwoS)
        {
            delta1[i] = 1;//0.009;
            delta2[i] = 1;//0.009;
            slopes[i] = -999;
            isFlat[i] = true;
        }

        //category 3 - barrel PS tilted
        else if(moduleSubdet == Barrel and moduleType == PS and moduleSide != Center)
        {

            //get drdz
            if(moduleLayerType == Strip)
            {
                drdz = modulesInGPU.drdzs[lowerModuleIndices[i]];
                slopes[i] = modulesInGPU.slopes[lowerModuleIndices[i]];
            }
            else
            {
                drdz = modulesInGPU.drdzs[modulesInGPU.partnerModuleIndex(lowerModuleIndices[i])];
                slopes[i] = modulesInGPU.slopes[modulesInGPU.partnerModuleIndex(lowerModuleIndices[i])];
            }

            delta1[i] = 1.1111f;//0.01;
            isFlat[i] = false;

            if(anchorHits)
            {
                //delta2[i] = (0.15f * drdz/sqrtf(1 + drdz * drdz))*111.1111f;
                delta2[i] = (0.15f * drdz*rsqrt(1 + drdz * drdz))*111.1111f;
            }
            else
            {
                //delta2[i] = (2.4f * drdz/sqrtf(1 + drdz * drdz))*111.1111f;
                delta2[i] = (2.4f * drdz*rsqrt(1 + drdz * drdz))*111.1111f;
            }
        }

        //category 4 - endcap PS
        else if(moduleSubdet == Endcap and moduleType == PS)
        {
            delta1[i] = 1.1111f;//0.01;
            if(moduleLayerType == Strip)
            {
                slopes[i] = modulesInGPU.slopes[lowerModuleIndices[i]];
            }
            else
            {
                slopes[i] = modulesInGPU.slopes[modulesInGPU.partnerModuleIndex(lowerModuleIndices[i])];

            }
            isFlat[i] = false;

            /*despite the type of the module layer of the lower module index,
            all anchor hits are on the pixel side and all non-anchor hits are
            on the strip side!*/
            if(anchorHits)
            {
                delta2[i] = 16.6666f;//0.15f;
            }
            else
            {
                delta2[i] = 266.666f;//2.4f;
            }
        }

        //category 5 - endcap 2S
        else if(moduleSubdet == Endcap and moduleType == TwoS)
        {
            delta1[i] = 1;//0.009;
            delta2[i] = 555.5555f;//5.f;
            slopes[i] = modulesInGPU.slopes[lowerModuleIndices[i]];
            isFlat[i] = false;
        }
        else
        {
            printf("ERROR!!!!! I SHOULDN'T BE HERE!!!! subdet = %d, type = %d, side = %d\n", moduleSubdet, moduleType, moduleSide);
        }
    }
    chiSquared = computeChiSquared(3, xs, ys, delta1, delta2, slopes, isFlat, g, f, radius);
    
    return chiSquared;
}


//90pc threshold
__device__ bool SDL::passPT3RPhiChiSquaredCuts(struct modules& modulesInGPU, unsigned int lowerModuleIndex1, unsigned int lowerModuleIndex2, unsigned int lowerModuleIndex3, float& chiSquared)
{

    const int layer1 = modulesInGPU.layers[lowerModuleIndex1] + 6 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex1] == SDL::TwoS);
    const int layer2 = modulesInGPU.layers[lowerModuleIndex2] + 6 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex2] == SDL::TwoS);
    const int layer3 = modulesInGPU.layers[lowerModuleIndex3] + 6 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex3] == SDL::TwoS);

    if(layer1 == 8 and layer2 == 9 and layer3 == 10)
    {
        return chiSquared < 7.003;
    }
    else if(layer1 == 8 and layer2 == 9 and layer3 == 15)
    {
        return chiSquared < 0.5;
    }

    else if(layer1 == 7 and layer2 == 8 and layer3 == 9)
    {
        return chiSquared < 8.046;
    }
    else if(layer1 == 7 and layer2 == 8 and layer3 == 14)
    {
        return chiSquared < 0.575;
    }

    else if(layer1 == 1 and layer2 == 2 and layer3 == 7)
    {
        return chiSquared < 5.304;
    }
    else if(layer1 == 1 and layer2 == 2 and layer3 == 3)
    {
        return chiSquared < 10.6211;
    }
    else if(layer1 == 1 and layer2 == 7 and layer3 == 8)
    {
        return chiSquared < 4.617;
    }

    else if(layer1 == 2 and layer2 == 7 and layer3 == 8)
    {
        return chiSquared < 8.046;
    }
    else if(layer1 == 2 and layer2 == 7 and layer3 == 13)
    {
        return chiSquared < 0.435;
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 7)
    {
        return chiSquared < 9.244;
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 12)
    {
        return chiSquared < 0.287;
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 4)
    {
        return chiSquared < 18.509;
    }

    return true;
}

__device__ bool SDL::passRadiusCriterion(struct modules& modulesInGPU, float& pixelRadius, float& pixelRadiusError, float& tripletRadius, unsigned int lowerModuleIndex, unsigned int middleModuleIndex, unsigned int upperModuleIndex)
{
    if(modulesInGPU.subdets[lowerModuleIndex] == SDL::Endcap)
    {
        return passRadiusCriterionEEE(pixelRadius, pixelRadiusError, tripletRadius);
    }
    else if(modulesInGPU.subdets[middleModuleIndex] == SDL::Endcap)
    {
        return passRadiusCriterionBEE(pixelRadius, pixelRadiusError, tripletRadius);
    }
    else if(modulesInGPU.subdets[upperModuleIndex] == SDL::Endcap)
    {
        return passRadiusCriterionBBE(pixelRadius, pixelRadiusError, tripletRadius);
    }
    else
    {
        return passRadiusCriterionBBB(pixelRadius, pixelRadiusError, tripletRadius);
    }

    //return ((modulesInGPU.subdets[lowerModuleIndex] == SDL::Endcap) & (passRadiusCriterionEEE(pixelRadius, pixelRadiusError, tripletRadius))) | ((modulesInGPU.subdets[middleModuleIndex] == SDL::Endcap) & (passRadiusCriterionBEE(pixelRadius, pixelRadiusError, tripletRadius))) | ((modulesInGPU.subdets[upperModuleIndex] == SDL::Endcap) & (passRadiusCriterionBBE(pixelRadius, pixelRadiusError, tripletRadius))) |  (passRadiusCriterionBBB(pixelRadius, pixelRadiusError, tripletRadius));

}

/*bounds for high Pt taken from : http://uaf-10.t2.ucsd.edu/~bsathian/SDL/T5_efficiency/efficiencies/new_efficiencies/efficiencies_20210513_T5_recovering_high_Pt_efficiencies/highE_radius_matching/highE_bounds.txt */
__device__ bool SDL::passRadiusCriterionBBB(float& pixelRadius, float& pixelRadiusError, float& tripletRadius)
{
    float tripletInvRadiusErrorBound = 0.15624;
    float pixelInvRadiusErrorBound = 0.17235;

    if(pixelRadius > 2.0/(2 * k2Rinv1GeVf))
    {
        pixelInvRadiusErrorBound = 0.6375;
        tripletInvRadiusErrorBound = 0.6588;
    }

    float tripletRadiusInvMax = (1 + tripletInvRadiusErrorBound)/tripletRadius;
    float tripletRadiusInvMin = fmaxf((1 - tripletInvRadiusErrorBound)/tripletRadius, 0);

    float pixelRadiusInvMax = fmaxf((1 + pixelInvRadiusErrorBound)/pixelRadius, 1.f/(pixelRadius - pixelRadiusError));
    float pixelRadiusInvMin = fminf((1 - pixelInvRadiusErrorBound)/pixelRadius, 1.f/(pixelRadius + pixelRadiusError));

    return checkIntervalOverlap(tripletRadiusInvMin, tripletRadiusInvMax, pixelRadiusInvMin, pixelRadiusInvMax);
}

__device__ bool SDL::passRadiusCriterionBBE(float& pixelRadius, float& pixelRadiusError, float& tripletRadius)
{
    float tripletInvRadiusErrorBound = 0.45972;
    float pixelInvRadiusErrorBound = 0.19644;

    if(pixelRadius > 2.0/(2 * k2Rinv1GeVf))
    {
        pixelInvRadiusErrorBound = 0.6805;
        tripletInvRadiusErrorBound = 0.8557;
    }

    float tripletRadiusInvMax = (1 + tripletInvRadiusErrorBound)/tripletRadius;
    float tripletRadiusInvMin = fmaxf((1 - tripletInvRadiusErrorBound)/tripletRadius, 0);

    float pixelRadiusInvMax = fmaxf((1 + pixelInvRadiusErrorBound)/pixelRadius, 1.f/(pixelRadius - pixelRadiusError));
    float pixelRadiusInvMin = fminf((1 - pixelInvRadiusErrorBound)/pixelRadius, 1.f/(pixelRadius + pixelRadiusError));

    return checkIntervalOverlap(tripletRadiusInvMin, tripletRadiusInvMax, pixelRadiusInvMin, pixelRadiusInvMax);

}

__device__ bool SDL::passRadiusCriterionBEE(float& pixelRadius, float& pixelRadiusError, float& tripletRadius)
{
    float tripletInvRadiusErrorBound = 1.59294;
    float pixelInvRadiusErrorBound = 0.255181;

    if(pixelRadius > 2.0/(2 * k2Rinv1GeVf)) //as good as not having selections
    {
        pixelInvRadiusErrorBound = 2.2091;
        tripletInvRadiusErrorBound = 2.3548;
    }

    float tripletRadiusInvMax = (1 + tripletInvRadiusErrorBound)/tripletRadius;
    float tripletRadiusInvMin = fmaxf((1 - tripletInvRadiusErrorBound)/tripletRadius, 0);

    float pixelRadiusInvMax = fmaxf((1 + pixelInvRadiusErrorBound)/pixelRadius, 1.f/(pixelRadius - pixelRadiusError));
    float pixelRadiusInvMin = fminf((1 - pixelInvRadiusErrorBound)/pixelRadius, 1.f/(pixelRadius + pixelRadiusError));
    pixelRadiusInvMin = fmaxf(pixelRadiusInvMin, 0);

    return checkIntervalOverlap(tripletRadiusInvMin, tripletRadiusInvMax, pixelRadiusInvMin, pixelRadiusInvMax);

}

__device__ bool SDL::passRadiusCriterionEEE(float& pixelRadius, float& pixelRadiusError, float& tripletRadius)
{
    float tripletInvRadiusErrorBound = 1.7006;
    float pixelInvRadiusErrorBound = 0.26367;

    if(pixelRadius > 2.0/(2 * k2Rinv1GeVf)) //as good as not having selections
    {
        pixelInvRadiusErrorBound = 2.286;
        tripletInvRadiusErrorBound = 2.436;
    }

    float tripletRadiusInvMax = (1 + tripletInvRadiusErrorBound)/tripletRadius;
    float tripletRadiusInvMin = fmaxf((1 - tripletInvRadiusErrorBound)/tripletRadius, 0);

    float pixelRadiusInvMax = fmaxf((1 + pixelInvRadiusErrorBound)/pixelRadius, 1.f/(pixelRadius - pixelRadiusError));
    float pixelRadiusInvMin = fminf((1 - pixelInvRadiusErrorBound)/pixelRadius, 1.f/(pixelRadius + pixelRadiusError));
    pixelRadiusInvMin = fmaxf(0, pixelRadiusInvMin);

    return checkIntervalOverlap(tripletRadiusInvMin, tripletRadiusInvMax, pixelRadiusInvMin, pixelRadiusInvMax);

}
