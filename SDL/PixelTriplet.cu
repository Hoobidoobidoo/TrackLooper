#include "hip/hip_runtime.h"
# include "PixelTriplet.cuh"
#include "allocate.h"

SDL::pixelTriplets::pixelTriplets()
{
    pixelSegmentIndices = nullptr;
    tripletIndices = nullptr;
    nPixelTriplets = nullptr;
}

void SDL::pixelTriplets::freeMemory()
{
    hipFree(pixelSegmentIndices);
    hipFree(tripletIndices);
    hipFree(nPixelTriplets);
}

SDL::pixelTriplets::~pixelTriplets()
{
}

void SDL::createPixelTripletsInUnifiedMemory(struct pixelTriplets& pixelTripletsInGPU, unsigned int maxPixelTriplets)
{
    hipMallocManaged(&pixelTripletsInGPU.pixelSegmentIndices, maxPixelTriplets * sizeof(float));
    hipMallocManaged(&pixelTripletsInGPU.tripletIndices, maxPixelTriplets * sizeof(float));
    hipMallocManaged(&pixelTripletsInGPU.nPixelTriplets, sizeof(unsigned int));
    hipMallocManaged(&pixelTripletsInGPU.pixelRadius, maxPixelTriplets * sizeof(float));
    hipMallocManaged(&pixelTripletsInGPU.pixelRadiusError, maxPixelTriplets * sizeof(float));
    hipMallocManaged(&pixelTripletsInGPU.tripletRadius, maxPixelTriplets * sizeof(float));

    hipMemset(pixelTripletsInGPU.nPixelTriplets, 0, sizeof(unsigned int));
}

void SDL::createPixelTripletsInExplicitMemory(struct pixelTriplets& pixelTripletsInGPU, unsigned int maxPixelTriplets)
{
    hipMalloc(&pixelTripletsInGPU.pixelSegmentIndices, maxPixelTriplets * sizeof(float));
    hipMalloc(&pixelTripletsInGPU.tripletIndices, maxPixelTriplets * sizeof(float));
    hipMalloc(&pixelTripletsInGPU.nPixelTriplets, sizeof(unsigned int));
    hipMalloc(&pixelTripletsInGPU.pixelRadius, maxPixelTriplets * sizeof(float));
    hipMalloc(&pixelTripletsInGPU.pixelRadiusError, maxPixelTriplets * sizeof(float));
    hipMalloc(&pixelTripletsInGPU.tripletRadius, maxPixelTriplets * sizeof(float));

    hipMemset(pixelTripletsInGPU.nPixelTriplets, 0, sizeof(unsigned int));

}

__device__ void SDL::addPixelTripletToMemory(struct pixelTriplets& pixelTripletsInGPU, unsigned int pixelSegmentIndex, unsigned int tripletIndex, float pixelRadius, float pixelRadiusError, float tripletRadius, unsigned int pixelTripletIndex)
{
    pixelTripletsInGPU.pixelSegmentIndices[pixelTripletIndex] = pixelSegmentIndex;
    pixelTripletsInGPU.tripletIndices[pixelTripletIndex] = tripletIndex;
    pixelTripletsInGPU.pixelRadius[pixelTripletIndex] = pixelRadius;
    pixelTripletsInGPU.pixelRadiusError[pixelTripletIndex] = pixelRadiusError;
    pixelTripletsInGPU.tripletRadius[pixelTripletIndex] = tripletRadius;

}

__device__ bool SDL::runPixelTripletDefaultAlgo(struct modules& modulesInGPU, struct hits& hitsInGPU, struct miniDoublets& mdsInGPU, struct segments& segmentsInGPU, struct triplets& tripletsInGPU, unsigned int& pixelSegmentIndex, unsigned int tripletIndex, float& pixelRadius, float& pixelRadiusError, float& tripletRadius)
{
    bool pass = true;

    //run pT4 compatibility between the pixel segment and inner segment, and between the pixel and outer segment of the triplet


    //placeholder
    float zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut;
    unsigned int pixelModuleIndex = segmentsInGPU.innerLowerModuleIndices[pixelSegmentIndex];

    unsigned int lowerModuleIndex = tripletsInGPU.lowerModuleIndices[3 * tripletIndex];
    unsigned int middleModuleIndex = tripletsInGPU.lowerModuleIndices[3 * tripletIndex + 1];
    unsigned int upperModuleIndex = tripletsInGPU.lowerModuleIndices[3 * tripletIndex + 2];


    // pixel segment vs inner segment of the triplet
    pass = pass & runTrackletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, pixelModuleIndex, pixelModuleIndex, lowerModuleIndex, middleModuleIndex, pixelSegmentIndex, tripletsInGPU.segmentIndices[2 * tripletIndex], zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut);

    //pixel segment vs outer segment of triplet
    pass = pass & runTrackletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, pixelModuleIndex, pixelModuleIndex, middleModuleIndex, upperModuleIndex, pixelSegmentIndex, tripletsInGPU.segmentIndices[2 * tripletIndex + 1], zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut);


    //pt matching between the pixel ptin and the triplet circle pt
    float pixelSegmentPt = segmentsInGPU.ptIn[pixelSegmentIndex];
    float pixelSegmentPtError = segmentsInGPU.ptErr[pixelSegmentIndex];

    pixelRadius = pixelSegmentPt/(2 * k2Rinv1GeVf);
    pixelRadiusError = pixelSegmentPtError/(2 * k2Rinv1GeVf);
    
    unsigned int tripletInnerSegmentIndex = tripletsInGPU.segmentIndices[2 * tripletIndex];
    unsigned int tripletOuterSegmentIndex = tripletsInGPU.segmentIndices[2 * tripletIndex + 1];

    unsigned int innerMDAnchorHitIndex = segmentsInGPU.innerMiniDoubletAnchorHitIndices[2 * tripletInnerSegmentIndex];
    unsigned int middleMDAnchorHitIndex = segmentsInGPU.outerMiniDoubletAnchorHitIndices[2 * tripletInnerSegmentIndex + 1];
    unsigned int outerMDAnchorHitIndex = segmentsInGPU.outerMiniDoubletAnchorHitIndices[2 * tripletOuterSegmentIndex + 1];

    float x1 = hitsInGPU.xs[innerMDAnchorHitIndex];
    float x2 = hitsInGPU.xs[middleMDAnchorHitIndex];
    float x3 = hitsInGPU.xs[outerMDAnchorHitIndex];

    float y1 = hitsInGPU.ys[innerMDAnchorHitIndex];
    float y2 = hitsInGPU.ys[middleMDAnchorHitIndex];
    float y3 = hitsInGPU.ys[outerMDAnchorHitIndex];
    
    float g, f;
    tripletRadius = computeRadiusFromThreeAnchorHits(x1, y1, x2, y2, x3, y3, g, f);
    
    pass = pass & passRadiusCriterion(modulesInGPU, pixelRadius, pixelRadiusError, tripletRadius, lowerModuleIndex, middleModuleIndex, upperModuleIndex);

    return pass;
}


__device__ bool SDL::passRadiusCriterion(struct modules& modulesInGPU, float& pixelRadius, float& pixelRadiusError, float& tripletRadius, unsigned int lowerModuleIndex, unsigned int middleModuleIndex, unsigned int upperModuleIndex)
{
    if(modulesInGPU.subdets[lowerModuleIndex] == SDL::Endcap)
    {
        return passRadiusCriterionEEE(pixelRadius, pixelRadiusError, tripletRadius);
    }
    else if(modulesInGPU.subdets[middleModuleIndex] == SDL::Endcap)
    {
        return passRadiusCriterionBEE(pixelRadius, pixelRadiusError, tripletRadius);
    }
    else if(modulesInGPU.subdets[upperModuleIndex] == SDL::Endcap)
    {
        return passRadiusCriterionBBE(pixelRadius, pixelRadiusError, tripletRadius);
    }
    else
    {
        return passRadiusCriterionBBB(pixelRadius, pixelRadiusError, tripletRadius);
    }

    //return ((modulesInGPU.subdets[lowerModuleIndex] == SDL::Endcap) & (passRadiusCriterionEEE(pixelRadius, pixelRadiusError, tripletRadius))) | ((modulesInGPU.subdets[middleModuleIndex] == SDL::Endcap) & (passRadiusCriterionBEE(pixelRadius, pixelRadiusError, tripletRadius))) | ((modulesInGPU.subdets[upperModuleIndex] == SDL::Endcap) & (passRadiusCriterionBBE(pixelRadius, pixelRadiusError, tripletRadius))) |  (passRadiusCriterionBBB(pixelRadius, pixelRadiusError, tripletRadius));

}
__device__ bool SDL::passRadiusCriterionBBB(float& pixelRadius, float& pixelRadiusError, float& tripletRadius)
{
    float tripletInvRadiusErrorBound = 0;
    float tripletRadiusMin = tripletRadius/(1 + tripletInvRadiusErrorBound);
    float tripletRadiusMax = tripletRadius/(1 - tripletInvRadiusErrorBound);
    float pixelRadiusMin = pixelRadius - pixelRadiusError;
    float pixelRadiusMax = pixelRadius + pixelRadiusError;
    
    return true; //checkIntervalOverlap(1.0/tripletRadiusMax, 1.0/tripletRadiusMin, 1.0/pixelRadusMax, 1.0/pixelRadiusMin);
}

__device__ bool SDL::passRadiusCriterionBBE(float& pixelRadius, float& pixelRadiusError, float& tripletRadius)
{
    float tripletInvRadiusErrorBound = 0;
    float tripletRadiusMin = tripletRadius/(1 + tripletInvRadiusErrorBound);
    float tripletRadiusMax = tripletRadius/(1 - tripletInvRadiusErrorBound);
    float pixelRadiusMin = pixelRadius - pixelRadiusError;
    float pixelRadiusMax = pixelRadius + pixelRadiusError;

    return true; //checkIntervalOverlap(1.0/tripletRadiusMax, 1.0/tripletRadiusMin, 1.0/pixelRadusMax, 1.0/pixelRadiusMin);

}

__device__ bool SDL::passRadiusCriterionBEE(float& pixelRadius, float& pixelRadiusError, float& tripletRadius)
{
    float tripletInvRadiusErrorBound = 0;
    float tripletRadiusMin = tripletRadius/(1 + tripletInvRadiusErrorBound);
    float tripletRadiusMax = tripletRadius/(1 - tripletInvRadiusErrorBound);
    float pixelRadiusMin = pixelRadius - pixelRadiusError;
    float pixelRadiusMax = pixelRadius + pixelRadiusError;

    return true; //checkIntervalOverlap(1.0/tripletRadiusMax, 1.0/tripletRadiusMin, 1.0/pixelRadusMax, 1.0/pixelRadiusMin);

}


__device__ bool SDL::passRadiusCriterionEEE(float& pixelRadius, float& pixelRadiusError, float& tripletRadius)
{
    float tripletInvRadiusErrorBound = 0;
    float tripletRadiusMax = tripletRadius/(1 - tripletInvRadiusErrorBound);
    float pixelRadiusMin = pixelRadius - pixelRadiusError;
    float pixelRadiusMax = pixelRadius + pixelRadiusError;

    return true; //checkIntervalOverlap(1.0/tripletRadiusMax, 1.0/tripletRadiusMin, 1.0/pixelRadusMax, 1.0/pixelRadiusMin);

}
