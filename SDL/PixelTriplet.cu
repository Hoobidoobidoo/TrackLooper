#include "hip/hip_runtime.h"
# include "PixelTriplet.cuh"
#include "allocate.h"

SDL::pixelTriplets::pixelTriplets()
{
    pixelSegmentIndices = nullptr;
    tripletIndices = nullptr;
    nPixelTriplets = nullptr;
    pixelRadius = nullptr;
    tripletRadius = nullptr;
    pt = nullptr;
    isDup = nullptr;
    partOfPT5 = nullptr;
#ifdef CUT_VALUE_DEBUG
    pixelRadiusError = nullptr;
#endif
}

void SDL::pixelTriplets::freeMemory()
{
    hipFree(pixelSegmentIndices);
    hipFree(tripletIndices);
    hipFree(nPixelTriplets);
    hipFree(pixelRadius);
    hipFree(tripletRadius);
    hipFree(pt);
    hipFree(isDup);
    hipFree(partOfPT5);
#ifdef CUT_VALUE_DEBUG
    hipFree(pixelRadiusError);
    hipFree(rPhiChiSquared);
#endif
}

SDL::pixelTriplets::~pixelTriplets()
{
}

void SDL::createPixelTripletsInUnifiedMemory(struct pixelTriplets& pixelTripletsInGPU, unsigned int maxPixelTriplets)
{
    hipMallocManaged(&pixelTripletsInGPU.pixelSegmentIndices, maxPixelTriplets * sizeof(float));
    hipMallocManaged(&pixelTripletsInGPU.tripletIndices, maxPixelTriplets * sizeof(float));
    hipMallocManaged(&pixelTripletsInGPU.nPixelTriplets, sizeof(unsigned int));
    hipMallocManaged(&pixelTripletsInGPU.pixelRadius, maxPixelTriplets * sizeof(float));
    hipMallocManaged(&pixelTripletsInGPU.tripletRadius, maxPixelTriplets * sizeof(float));
    hipMallocManaged(&pixelTripletsInGPU.pt, maxPixelTriplets * 6*sizeof(float));
    hipMallocManaged(&pixelTripletsInGPU.isDup, maxPixelTriplets * sizeof(bool));
    hipMallocManaged(&pixelTripletsInGPU.partOfPT5, maxPixelTriplets * sizeof(bool));
#ifdef CUT_VALUE_DEBUG
    hipMallocManaged(&pixelTripletsInGPU.pixelRadiusError, maxPixelTriplets * sizeof(float));
    hipMallocManaged(&pixelTripletsInGPU.rPhiChiSquared, maxPixelTriplets * sizeof(float));
#endif
    pixelTripletsInGPU.eta = pixelTripletsInGPU.pt + maxPixelTriplets;
    pixelTripletsInGPU.phi = pixelTripletsInGPU.pt + maxPixelTriplets * 2;
    pixelTripletsInGPU.eta_pix = pixelTripletsInGPU.pt + maxPixelTriplets *3;
    pixelTripletsInGPU.phi_pix = pixelTripletsInGPU.pt + maxPixelTriplets * 4;
    pixelTripletsInGPU.score = pixelTripletsInGPU.pt + maxPixelTriplets * 5;
    hipMemset(pixelTripletsInGPU.nPixelTriplets, 0, sizeof(unsigned int));
}

void SDL::createPixelTripletsInExplicitMemory(struct pixelTriplets& pixelTripletsInGPU, unsigned int maxPixelTriplets)
{
    hipMalloc(&pixelTripletsInGPU.pixelSegmentIndices, maxPixelTriplets * sizeof(float));
    hipMalloc(&pixelTripletsInGPU.tripletIndices, maxPixelTriplets * sizeof(float));
    hipMalloc(&pixelTripletsInGPU.nPixelTriplets, sizeof(unsigned int));
    hipMalloc(&pixelTripletsInGPU.pixelRadius, maxPixelTriplets * sizeof(float));
    hipMalloc(&pixelTripletsInGPU.tripletRadius, maxPixelTriplets * sizeof(float));
    hipMalloc(&pixelTripletsInGPU.pt, maxPixelTriplets * 6*sizeof(float));
    hipMalloc(&pixelTripletsInGPU.isDup, maxPixelTriplets * sizeof(bool));
    hipMalloc(&pixelTripletsInGPU.partOfPT5, maxPixelTriplets * sizeof(bool));

    pixelTripletsInGPU.eta = pixelTripletsInGPU.pt + maxPixelTriplets;
    pixelTripletsInGPU.phi = pixelTripletsInGPU.pt + maxPixelTriplets * 2;
    pixelTripletsInGPU.eta_pix = pixelTripletsInGPU.pt + maxPixelTriplets *3;
    pixelTripletsInGPU.phi_pix = pixelTripletsInGPU.pt + maxPixelTriplets * 4;
    pixelTripletsInGPU.score = pixelTripletsInGPU.pt + maxPixelTriplets * 5;
    hipMemset(pixelTripletsInGPU.nPixelTriplets, 0, sizeof(unsigned int));

}

#ifdef CUT_VALUE_DEBUG
__device__ void SDL::addPixelTripletToMemory(struct pixelTriplets& pixelTripletsInGPU, unsigned int pixelSegmentIndex, unsigned int tripletIndex, float pixelRadius, float pixelRadiusError, float tripletRadius, float rPhiChiSquared, unsigned int pixelTripletIndex, float pt, float eta, float phi, float eta_pix, float phi_pix, float score)
#else
__device__ void SDL::addPixelTripletToMemory(struct pixelTriplets& pixelTripletsInGPU, unsigned int pixelSegmentIndex, unsigned int tripletIndex, float pixelRadius, float tripletRadius, unsigned int pixelTripletIndex, float pt, float eta, float phi, float eta_pix, float phi_pix,float score)
#endif
{
    pixelTripletsInGPU.pixelSegmentIndices[pixelTripletIndex] = pixelSegmentIndex;
    pixelTripletsInGPU.tripletIndices[pixelTripletIndex] = tripletIndex;
    pixelTripletsInGPU.pixelRadius[pixelTripletIndex] = pixelRadius;
    pixelTripletsInGPU.tripletRadius[pixelTripletIndex] = tripletRadius;
    pixelTripletsInGPU.pt[pixelTripletIndex] = pt;
    pixelTripletsInGPU.eta[pixelTripletIndex] = eta;
    pixelTripletsInGPU.phi[pixelTripletIndex] = phi;
    pixelTripletsInGPU.eta_pix[pixelTripletIndex] = eta_pix;
    pixelTripletsInGPU.phi_pix[pixelTripletIndex] = phi_pix;
    pixelTripletsInGPU.isDup[pixelTripletIndex] = 0;
    pixelTripletsInGPU.score[pixelTripletIndex] = score;

#ifdef CUT_VALUE_DEBUG
    pixelTripletsInGPU.pixelRadiusError[pixelTripletIndex] = pixelRadiusError;
    pixelTripletsInGPU.rPhiChiSquared[pixelTripletIndex] = rPhiChiSquared;
#endif
}
__device__ void SDL::rmPixelTripletToMemory(struct pixelTriplets& pixelTripletsInGPU,unsigned int pixelTripletIndex)
{
    pixelTripletsInGPU.isDup[pixelTripletIndex] = 1;
}

__device__ bool SDL::runPixelTripletDefaultAlgo(struct modules& modulesInGPU, struct hits& hitsInGPU, struct miniDoublets& mdsInGPU, struct segments& segmentsInGPU, struct triplets& tripletsInGPU, unsigned int& pixelSegmentIndex, unsigned int tripletIndex, float& pixelRadius, float& pixelRadiusError, float& tripletRadius, float& rPhiChiSquared)
{
    bool pass = true;

    //run pT4 compatibility between the pixel segment and inner segment, and between the pixel and outer segment of the triplet


    //placeholder
    float zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta; //temp stuff
    float zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ;

    unsigned int pixelModuleIndex = segmentsInGPU.innerLowerModuleIndices[pixelSegmentIndex];

    unsigned int lowerModuleIndex = tripletsInGPU.lowerModuleIndices[3 * tripletIndex];
    unsigned int middleModuleIndex = tripletsInGPU.lowerModuleIndices[3 * tripletIndex + 1];
    unsigned int upperModuleIndex = tripletsInGPU.lowerModuleIndices[3 * tripletIndex + 2];


    // pixel segment vs inner segment of the triplet
    pass = pass & runPixelTrackletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, pixelModuleIndex, pixelModuleIndex, lowerModuleIndex, middleModuleIndex, pixelSegmentIndex, tripletsInGPU.segmentIndices[2 * tripletIndex], zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta, zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ);

    //pixel segment vs outer segment of triplet
    pass = pass & runPixelTrackletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, pixelModuleIndex, pixelModuleIndex, middleModuleIndex, upperModuleIndex, pixelSegmentIndex, tripletsInGPU.segmentIndices[2 * tripletIndex + 1], zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta, zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ);

    //pt matching between the pixel ptin and the triplet circle pt
    unsigned int pixelSegmentArrayIndex = pixelSegmentIndex - (pixelModuleIndex * 600);
    float pixelSegmentPt = segmentsInGPU.ptIn[pixelSegmentArrayIndex];
    float pixelSegmentPtError = segmentsInGPU.ptErr[pixelSegmentArrayIndex];

    pixelRadius = pixelSegmentPt/(2 * k2Rinv1GeVf);
    pixelRadiusError = pixelSegmentPtError/(2 * k2Rinv1GeVf);
    unsigned int tripletInnerSegmentIndex = tripletsInGPU.segmentIndices[2 * tripletIndex];
    unsigned int tripletOuterSegmentIndex = tripletsInGPU.segmentIndices[2 * tripletIndex + 1];

    unsigned int innerMDAnchorHitIndex = segmentsInGPU.innerMiniDoubletAnchorHitIndices[tripletInnerSegmentIndex];
    unsigned int middleMDAnchorHitIndex = segmentsInGPU.outerMiniDoubletAnchorHitIndices[tripletInnerSegmentIndex];
    unsigned int outerMDAnchorHitIndex = segmentsInGPU.outerMiniDoubletAnchorHitIndices[tripletOuterSegmentIndex];

    float x1 = hitsInGPU.xs[innerMDAnchorHitIndex];
    float x2 = hitsInGPU.xs[middleMDAnchorHitIndex];
    float x3 = hitsInGPU.xs[outerMDAnchorHitIndex];

    float y1 = hitsInGPU.ys[innerMDAnchorHitIndex];
    float y2 = hitsInGPU.ys[middleMDAnchorHitIndex];
    float y3 = hitsInGPU.ys[outerMDAnchorHitIndex];
    float g,f;
    
    tripletRadius = computeRadiusFromThreeAnchorHits(x1, y1, x2, y2, x3, y3,g,f);
    
    pass = pass & passRadiusCriterion(modulesInGPU, pixelRadius, pixelRadiusError, tripletRadius, lowerModuleIndex, middleModuleIndex, upperModuleIndex);

    unsigned int anchorHits[] = {innerMDAnchorHitIndex, middleMDAnchorHitIndex, outerMDAnchorHitIndex};
    unsigned int lowerModuleIndices[] = {lowerModuleIndex, middleModuleIndex, upperModuleIndex};

    rPhiChiSquared = computePT3RPhiChiSquared(modulesInGPU, hitsInGPU, segmentsInGPU, pixelSegmentArrayIndex, anchorHits, lowerModuleIndices);

    if(pixelSegmentPt < 5.0)
    {
        pass = pass & passPT3RPhiChiSquaredCuts(modulesInGPU, lowerModuleIndex, middleModuleIndex, upperModuleIndex, rPhiChiSquared);
    }

    return pass;

}


//TODO: merge this one and the pT5 function later into a single function
__device__ float SDL::computePT3RPhiChiSquared(struct modules& modulesInGPU, struct hits& hitsInGPU, struct segments& segmentsInGPU, unsigned int pixelSegmentArrayIndex, unsigned int* anchorHits, unsigned int* lowerModuleIndices)
{
    float g = segmentsInGPU.circleCenterX[pixelSegmentArrayIndex];
    float f = segmentsInGPU.circleCenterY[pixelSegmentArrayIndex];
    float radius = segmentsInGPU.circleRadius[pixelSegmentArrayIndex];
    float delta1[3], delta2[3], slopes[3];
    bool isFlat[3];
    float xs[3];
    float ys[3];
    float chiSquared = 0;
    for(size_t i = 0; i < 3; i++)
    {
        xs[i] = hitsInGPU.xs[anchorHits[i]];
        ys[i] = hitsInGPU.ys[anchorHits[i]];
    }

    computeSigmasForRegression(modulesInGPU, lowerModuleIndices, delta1, delta2, slopes, isFlat, 3);
    chiSquared = computeChiSquared(3, xs, ys, delta1, delta2, slopes, isFlat, g, f, radius);
    
    return chiSquared;
}


__device__ bool SDL::passPT3RPhiChiSquaredCuts(struct modules& modulesInGPU, unsigned int lowerModuleIndex1, unsigned int lowerModuleIndex2, unsigned int lowerModuleIndex3, float& rPhiChiSquared)
{
    const int layer1 = modulesInGPU.layers[lowerModuleIndex1] + 6 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex1] == SDL::TwoS);
    const int layer2 = modulesInGPU.layers[lowerModuleIndex2] + 6 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex2] == SDL::TwoS);
    const int layer3 = modulesInGPU.layers[lowerModuleIndex3] + 6 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex3] == SDL::TwoS);

    if(layer1 == 1 and layer2 == 2 and layer3 == 3)
    {
        return rPhiChiSquared < 21.266;
    }
    else if(layer1 == 1 and layer2 == 2 and layer3 == 7)
    {
        return rPhiChiSquared < 12.203;
    }
    else if(layer1 == 1 and layer2 == 7 and layer3 == 8)
    {
        return rPhiChiSquared < 8.045;
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 12)
    {
        return rPhiChiSquared < 7.002;
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 4)
    {
        return rPhiChiSquared < 37.058;
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 7)
    {
        return rPhiChiSquared < 18.509;
    }
    else if(layer1 == 2 and layer2 == 7 and layer3 == 13)
    {
        return rPhiChiSquared < 5.305;
    }
    else if(layer1 == 2 and layer2 == 7 and layer3 == 8)
    {
        return rPhiChiSquared < 16.109;
    }
    else if(layer1 == 3 and layer2 == 7 and layer3 == 13)
    {   
        return rPhiChiSquared < 5.305;
    }
    else if(layer1 == 3 and layer2 == 7 and layer3 == 8)
    {
        return rPhiChiSquared < 37.058;
    }
    else if(layer1 == 7 and layer2 == 8 and layer3 == 14)
    {
        return rPhiChiSquared < 7.003;
    }
    else if(layer1 == 7 and layer2 == 8 and layer3 == 9)
    {
        return rPhiChiSquared < 12.203;
    }
    else if(layer1 == 8 and layer2 == 9 and layer3 == 10)
    {
        return rPhiChiSquared < 14.021;
    }
    else if(layer1 == 8 and layer2 == 9 and layer3 == 15)
    {
        return rPhiChiSquared < 4.018;
    }
    else if(layer1 == 9 and layer2 == 10 and layer3 == 11)
    {
        return rPhiChiSquared < 24.433;
    }
    else if(layer1 == 9 and layer2 == 10 and layer3 == 16)
    {
        return rPhiChiSquared < 4.617;
    }
    return true;
}


__device__ bool SDL::passRadiusCriterion(struct modules& modulesInGPU, float& pixelRadius, float& pixelRadiusError, float& tripletRadius, unsigned int lowerModuleIndex, unsigned int middleModuleIndex, unsigned int upperModuleIndex)
{
    if(modulesInGPU.subdets[lowerModuleIndex] == SDL::Endcap)
    {
        return passRadiusCriterionEEE(pixelRadius, pixelRadiusError, tripletRadius);
    }
    else if(modulesInGPU.subdets[middleModuleIndex] == SDL::Endcap)
    {
        return passRadiusCriterionBEE(pixelRadius, pixelRadiusError, tripletRadius);
    }
    else if(modulesInGPU.subdets[upperModuleIndex] == SDL::Endcap)
    {
        return passRadiusCriterionBBE(pixelRadius, pixelRadiusError, tripletRadius);
    }
    else
    {
        return passRadiusCriterionBBB(pixelRadius, pixelRadiusError, tripletRadius);
    }

    //return ((modulesInGPU.subdets[lowerModuleIndex] == SDL::Endcap) & (passRadiusCriterionEEE(pixelRadius, pixelRadiusError, tripletRadius))) | ((modulesInGPU.subdets[middleModuleIndex] == SDL::Endcap) & (passRadiusCriterionBEE(pixelRadius, pixelRadiusError, tripletRadius))) | ((modulesInGPU.subdets[upperModuleIndex] == SDL::Endcap) & (passRadiusCriterionBBE(pixelRadius, pixelRadiusError, tripletRadius))) |  (passRadiusCriterionBBB(pixelRadius, pixelRadiusError, tripletRadius));

}

/*bounds for high Pt taken from : http://uaf-10.t2.ucsd.edu/~bsathian/SDL/T5_efficiency/efficiencies/new_efficiencies/efficiencies_20210513_T5_recovering_high_Pt_efficiencies/highE_radius_matching/highE_bounds.txt */
__device__ bool SDL::passRadiusCriterionBBB(float& pixelRadius, float& pixelRadiusError, float& tripletRadius)
{
    float tripletInvRadiusErrorBound = 0.15624;
    float pixelInvRadiusErrorBound = 0.17235;

    if(pixelRadius > 2.0/(2 * k2Rinv1GeVf))
    {
        pixelInvRadiusErrorBound = 0.6375;
        tripletInvRadiusErrorBound = 0.6588;
    }

    float tripletRadiusInvMax = (1 + tripletInvRadiusErrorBound)/tripletRadius;
    float tripletRadiusInvMin = fmaxf((1 - tripletInvRadiusErrorBound)/tripletRadius, 0);

    float pixelRadiusInvMax = fmaxf((1 + pixelInvRadiusErrorBound)/pixelRadius, 1.f/(pixelRadius - pixelRadiusError));
    float pixelRadiusInvMin = fminf((1 - pixelInvRadiusErrorBound)/pixelRadius, 1.f/(pixelRadius + pixelRadiusError));

    return checkIntervalOverlap(tripletRadiusInvMin, tripletRadiusInvMax, pixelRadiusInvMin, pixelRadiusInvMax);
}

__device__ bool SDL::passRadiusCriterionBBE(float& pixelRadius, float& pixelRadiusError, float& tripletRadius)
{
    float tripletInvRadiusErrorBound = 0.45972;
    float pixelInvRadiusErrorBound = 0.19644;

    if(pixelRadius > 2.0/(2 * k2Rinv1GeVf))
    {
        pixelInvRadiusErrorBound = 0.6805;
        tripletInvRadiusErrorBound = 0.8557;
    }

    float tripletRadiusInvMax = (1 + tripletInvRadiusErrorBound)/tripletRadius;
    float tripletRadiusInvMin = fmaxf((1 - tripletInvRadiusErrorBound)/tripletRadius, 0);

    float pixelRadiusInvMax = fmaxf((1 + pixelInvRadiusErrorBound)/pixelRadius, 1.f/(pixelRadius - pixelRadiusError));
    float pixelRadiusInvMin = fminf((1 - pixelInvRadiusErrorBound)/pixelRadius, 1.f/(pixelRadius + pixelRadiusError));

    return checkIntervalOverlap(tripletRadiusInvMin, tripletRadiusInvMax, pixelRadiusInvMin, pixelRadiusInvMax);

}

__device__ bool SDL::passRadiusCriterionBEE(float& pixelRadius, float& pixelRadiusError, float& tripletRadius)
{
    float tripletInvRadiusErrorBound = 1.59294;
    float pixelInvRadiusErrorBound = 0.255181;

    if(pixelRadius > 2.0/(2 * k2Rinv1GeVf)) //as good as not having selections
    {
        pixelInvRadiusErrorBound = 2.2091;
        tripletInvRadiusErrorBound = 2.3548;
    }

    float tripletRadiusInvMax = (1 + tripletInvRadiusErrorBound)/tripletRadius;
    float tripletRadiusInvMin = fmaxf((1 - tripletInvRadiusErrorBound)/tripletRadius, 0);

    float pixelRadiusInvMax = fmaxf((1 + pixelInvRadiusErrorBound)/pixelRadius, 1.f/(pixelRadius - pixelRadiusError));
    float pixelRadiusInvMin = fminf((1 - pixelInvRadiusErrorBound)/pixelRadius, 1.f/(pixelRadius + pixelRadiusError));
    pixelRadiusInvMin = fmaxf(pixelRadiusInvMin, 0);

    return checkIntervalOverlap(tripletRadiusInvMin, tripletRadiusInvMax, pixelRadiusInvMin, pixelRadiusInvMax);

}

__device__ bool SDL::passRadiusCriterionEEE(float& pixelRadius, float& pixelRadiusError, float& tripletRadius)
{
    float tripletInvRadiusErrorBound = 1.7006;
    float pixelInvRadiusErrorBound = 0.26367;

    if(pixelRadius > 2.0/(2 * k2Rinv1GeVf)) //as good as not having selections
    {
        pixelInvRadiusErrorBound = 2.286;
        tripletInvRadiusErrorBound = 2.436;
    }

    float tripletRadiusInvMax = (1 + tripletInvRadiusErrorBound)/tripletRadius;
    float tripletRadiusInvMin = fmaxf((1 - tripletInvRadiusErrorBound)/tripletRadius, 0);

    float pixelRadiusInvMax = fmaxf((1 + pixelInvRadiusErrorBound)/pixelRadius, 1.f/(pixelRadius - pixelRadiusError));
    float pixelRadiusInvMin = fminf((1 - pixelInvRadiusErrorBound)/pixelRadius, 1.f/(pixelRadius + pixelRadiusError));
    pixelRadiusInvMin = fmaxf(0, pixelRadiusInvMin);

    return checkIntervalOverlap(tripletRadiusInvMin, tripletRadiusInvMax, pixelRadiusInvMin, pixelRadiusInvMax);

}
