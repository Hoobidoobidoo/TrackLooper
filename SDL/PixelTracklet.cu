#include "hip/hip_runtime.h"
# include "PixelTracklet.cuh"

#ifdef __HIPCC__
#define CUDA_CONST_VAR __device__
#endif

//#ifdef CACHE_ALLOC
#include "allocate.h"
//#endif

void SDL::createPixelTrackletsInUnifiedMemory(struct pixelTracklets& pixelTrackletsInGPU, unsigned int maxPixelTracklets)
{
#ifdef CACHE_ALLOC
    hipStream_t stream =0;
    pixelTrackletsInGPU.segmentIndices = (unsigned int*)cms::cuda::allocate_managed(maxPixelTracklets * sizeof(unsigned int) * 2,stream);
    pixelTrackletsInGPU.lowerModuleIndices = (unsigned int*)cms::cuda::allocate_managed(maxPixelTracklets * sizeof(unsigned int) * 2,stream);//split up to avoid runtime error of exceeding max byte allocation at a time
    pixelTrackletsInGPU.nPixelTracklets = (unsigned int*)cms::cuda::allocate_managed(sizeof(unsigned int),stream);
    pixelTrackletsInGPU.zOut = (float*)cms::cuda::allocate_managed(maxPixelTracklets * sizeof(float) * 4,stream);
    pixelTrackletsInGPU.betaIn = (float*)cms::cuda::allocate_managed(maxPixelTracklets * sizeof(float) * 3,stream);
#else
    hipMallocManaged(&pixelTrackletsInGPU.segmentIndices, 2 * maxPixelTracklets * sizeof(unsigned int));
    hipMallocManaged(&pixelTrackletsInGPU.lowerModuleIndices, 2 * maxPixelTracklets * sizeof(unsigned int));
    hipMallocManaged(&pixelTrackletsInGPU.nPixelTracklets, sizeof(unsigned int));
    hipMallocManaged(&pixelTrackletsInGPU.zOut, maxPixelTracklets *4* sizeof(float));
    hipMallocManaged(&pixelTrackletsInGPU.betaIn, maxPixelTracklets *3* sizeof(float));

#ifdef CUT_VALUE_DEBUG
    hipMallocManaged(&pixelTrackletsInGPU.zLo, maxPixelTracklets * sizeof(float));
    hipMallocManaged(&pixelTrackletsInGPU.zHi, maxPixelTracklets * sizeof(float));
    hipMallocManaged(&pixelTrackletsInGPU.zLoPointed, maxPixelTracklets * sizeof(float));
    hipMallocManaged(&pixelTrackletsInGPU.zHiPointed, maxPixelTracklets * sizeof(float));
    hipMallocManaged(&pixelTrackletsInGPU.sdlCut, maxPixelTracklets * sizeof(float));
    hipMallocManaged(&pixelTrackletsInGPU.betaInCut, maxPixelTracklets * sizeof(float));
    hipMallocManaged(&pixelTrackletsInGPU.betaOutCut, maxPixelTracklets * sizeof(float));
    hipMallocManaged(&pixelTrackletsInGPU.deltaBetaCut, maxPixelTracklets * sizeof(float));
    hipMallocManaged(&pixelTrackletsInGPU.rtLo, maxPixelTracklets * sizeof(float));
    hipMallocManaged(&pixelTrackletsInGPU.rtHi, maxPixelTracklets * sizeof(float));
    hipMallocManaged(&pixelTrackletsInGPU.kZ, maxPixelTracklets * sizeof(float));

#endif
#endif
    pixelTrackletsInGPU.rtOut = pixelTrackletsInGPU.zOut + maxPixelTracklets;
    pixelTrackletsInGPU.deltaPhiPos = pixelTrackletsInGPU.zOut + maxPixelTracklets * 2;
    pixelTrackletsInGPU.deltaPhi = pixelTrackletsInGPU.zOut + maxPixelTracklets * 3;
    pixelTrackletsInGPU.betaOut = pixelTrackletsInGPU.betaIn + maxPixelTracklets;
    pixelTrackletsInGPU.pt_beta = pixelTrackletsInGPU.betaIn + maxPixelTracklets * 2;

    hipMemset(pixelTrackletsInGPU.nPixelTracklets, 0, sizeof(unsigned int));
}

void SDL::createPixelTrackletsInExplicitMemory(struct pixelTracklets& pixelTrackletsInGPU, unsigned int maxPixelTracklets)
{
#ifdef CACHE_ALLOC
    hipStream_t stream = 0;
    int dev;
    hipGetDevice(&dev);

    pixelTrackletsInGPU.segmentIndices = (unsigned int*)cms::cuda::allocate_device(dev, maxPixelTracklets * sizeof(unsigned int) * 2,stream);
    pixelTrackletsInGPU.lowerModuleIndices = (unsigned int*)cms::cuda::allocate_device(dev, maxPixelTracklets * sizeof(unsigned int) * 2,stream);//split up to avoid runtime error of exceeding max byte allocation at a time
    pixelTrackletsInGPU.nPixelTracklets = (unsigned int*)cms::cuda::allocate_device(dev, sizeof(unsigned int),stream);
    pixelTrackletsInGPU.zOut = (float*)cms::cuda::allocate_device(dev, maxPixelTracklets * sizeof(float) * 4,stream);
    pixelTrackletsInGPU.betaIn = (float*)cms::cuda::allocate_device(dev, maxPixelTracklets * sizeof(float) * 3,stream);

#else
    hipMalloc(&pixelTrackletsInGPU.segmentIndices, 2 * maxPixelTracklets * sizeof(unsigned int));
    hipMalloc(&pixelTrackletsInGPU.lowerModuleIndices, 2 * maxPixelTracklets * sizeof(unsigned int));
    hipMalloc(&pixelTrackletsInGPU.nPixelTracklets, sizeof(unsigned int));
    hipMalloc(&pixelTrackletsInGPU.zOut, maxPixelTracklets *4* sizeof(float));
    hipMalloc(&pixelTrackletsInGPU.betaIn, maxPixelTracklets *3* sizeof(float));
#endif
    pixelTrackletsInGPU.rtOut = pixelTrackletsInGPU.zOut + maxPixelTracklets;
    pixelTrackletsInGPU.deltaPhiPos = pixelTrackletsInGPU.zOut + maxPixelTracklets * 2;
    pixelTrackletsInGPU.deltaPhi = pixelTrackletsInGPU.zOut + maxPixelTracklets * 3;
    pixelTrackletsInGPU.betaOut = pixelTrackletsInGPU.betaIn + maxPixelTracklets;
    pixelTrackletsInGPU.pt_beta = pixelTrackletsInGPU.betaIn + maxPixelTracklets * 2;

    hipMemset(pixelTrackletsInGPU.nPixelTracklets, 0, sizeof(unsigned int));
}

#ifdef CUT_VALUE_DEBUG
__device__ void SDL::addPixelTrackletToMemory(struct pixelTracklets& pixelTrackletsInGPU, unsigned int innerSegmentIndex, unsigned int outerSegmentIndex, unsigned int innerInnerLowerModuleIndex, unsigned int innerOuterLowerModuleIndex, unsigned int outerInnerLowerModuleIndex, unsigned int outerOuterLowerModuleIndex, float& zOut, float& rtOut, float& deltaPhiPos, float& deltaPhi, float& betaIn, float& betaOut, float pt_beta, float& zLo, float& zHi, float& rtLo, float& rtHi, float& zLoPointed, float&
        zHiPointed, float& sdlCut, float& betaInCut, float& betaOutCut, float& deltaBetaCut, float& kZ, unsigned int pixelTrackletIndex)
#else
__device__ void SDL::addPixelTrackletToMemory(struct pixelTracklets& pixelTrackletsInGPU, unsigned int innerSegmentIndex, unsigned int outerSegmentIndex, unsigned int innerInnerLowerModuleIndex, unsigned int innerOuterLowerModuleIndex, unsigned int outerInnerLowerModuleIndex, unsigned int outerOuterLowerModuleIndex, float& zOut, float& rtOut, float& deltaPhiPos, float& deltaPhi, float& betaIn, float& betaOut, float pt_beta, unsigned int pixelTrackletIndex)
#endif
{
    pixelTrackletsInGPU.segmentIndices[2 * pixelTrackletIndex] = innerSegmentIndex;
    pixelTrackletsInGPU.segmentIndices[2 * pixelTrackletIndex + 1] = outerSegmentIndex;
    pixelTrackletsInGPU.lowerModuleIndices[2 * pixelTrackletIndex] = outerInnerLowerModuleIndex;
    pixelTrackletsInGPU.lowerModuleIndices[2 * pixelTrackletIndex + 1] = outerOuterLowerModuleIndex;

    pixelTrackletsInGPU.zOut[pixelTrackletIndex] = zOut;
    pixelTrackletsInGPU.rtOut[pixelTrackletIndex] = rtOut;
    pixelTrackletsInGPU.deltaPhiPos[pixelTrackletIndex] = deltaPhiPos;
    pixelTrackletsInGPU.deltaPhi[pixelTrackletIndex] = deltaPhi;

    pixelTrackletsInGPU.betaIn[pixelTrackletIndex] = betaIn;
    pixelTrackletsInGPU.betaOut[pixelTrackletIndex] = betaOut;
    pixelTrackletsInGPU.pt_beta[pixelTrackletIndex] = pt_beta;

#ifdef CUT_VALUE_DEBUG
    pixelTrackletsInGPU.zLo[pixelTrackletIndex] = zLo;
    pixelTrackletsInGPU.zHi[pixelTrackletIndex] = zHi;
    pixelTrackletsInGPU.rtLo[pixelTrackletIndex] = rtLo;
    pixelTrackletsInGPU.rtHi[pixelTrackletIndex] = rtHi;
    pixelTrackletsInGPU.zLoPointed[pixelTrackletIndex] = zLoPointed;
    pixelTrackletsInGPU.zHiPointed[pixelTrackletIndex] = zHiPointed;
    pixelTrackletsInGPU.sdlCut[pixelTrackletIndex] = sdlCut;
    pixelTrackletsInGPU.betaInCut[pixelTrackletIndex] = betaInCut;
    pixelTrackletsInGPU.betaOutCut[pixelTrackletIndex] = betaOutCut;
    pixelTrackletsInGPU.deltaBetaCut[pixelTrackletIndex] = deltaBetaCut;
    pixelTrackletsInGPU.kZ[pixelTrackletIndex] = kZ;
#endif

}

void SDL::pixelTracklets::freeMemoryCache()
{
#ifdef Explicit_Tracklet
    int dev;
    hipGetDevice(&dev);
    cms::cuda::free_device(dev,segmentIndices);
    cms::cuda::free_device(dev,lowerModuleIndices);
    cms::cuda::free_device(dev,zOut);
    cms::cuda::free_device(dev,betaIn);
    cms::cuda::free_device(dev,nPixelTracklets);
#else
    cms::cuda::free_managed(segmentIndices);
    cms::cuda::free_managed(lowerModuleIndices);
    cms::cuda::free_managed(zOut);
    cms::cuda::free_managed(betaIn);
    cms::cuda::free_managed(nPixelTracklets);
#endif
}

void SDL::pixelTracklets::freeMemory()
{
    hipFree(segmentIndices);
    hipFree(lowerModuleIndices);
    hipFree(nPixelTracklets);
    hipFree(zOut);
    hipFree(betaIn);
#ifdef CUT_VALUE_DEBUG
    hipFree(zLo);
    hipFree(zHi);
    hipFree(rtLo);
    hipFree(rtHi);
    hipFree(zLoPointed);
    hipFree(zHiPointed);
    hipFree(sdlCut);
    hipFree(betaInCut);
    hipFree(betaOutCut);
    hipFree(deltaBetaCut);
    hipFree(kZ);
#endif
}

SDL::pixelTracklets::pixelTracklets()
{
    segmentIndices = nullptr;
    lowerModuleIndices = nullptr;
    nPixelTracklets = nullptr;
    zOut = nullptr;
    rtOut = nullptr;

    deltaPhiPos = nullptr;
    deltaPhi = nullptr;
    betaIn = nullptr;
    betaOut = nullptr;
    pt_beta = nullptr;
#ifdef CUT_VALUE_DEBUG
    zLo = nullptr;
    zHi = nullptr;
    rtLo = nullptr;
    rtHi = nullptr;
    zLoPointed = nullptr;
    zHiPointed = nullptr;
    sdlCut = nullptr;
    betaInCut = nullptr;
    betaOutCut = nullptr;
    deltaBetaCut = nullptr;
    kZ = nullptr;
#endif

}

SDL::pixelTracklets::~pixelTracklets()
{

}

