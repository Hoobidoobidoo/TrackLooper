#include "TrackCandidate.cuh"

#include "allocate.h"


void SDL::createEligibleModulesListForTrackCandidates(struct modules& modulesInGPU, unsigned int& nEligibleModules, unsigned int maxTrackCandidates)
{
    //an extra array in the modulesInGPU struct that will provide us with starting indices for the memory locations. If a
    //module is not supposed to have any memory, it gets a -1

    //the array will be filled in createTrackCandidatesInUnfiedMemory

    unsigned int nLowerModules = *modulesInGPU.nLowerModules;
    hipMemset(modulesInGPU.trackCandidateModuleIndices, -1, sizeof(int) * (nLowerModules + 1));

    //start filling
    for(size_t i = 0; i <= nLowerModules; i++)
    {
        //condition for a track candidate to exist for a module
        //TCs don't exist for layers 5 and 6 barrel, and layers 2,3,4,5 endcap
        unsigned int idx = modulesInGPU.lowerModuleIndices[i];
        if((modulesInGPU.subdets[idx] == SDL::Barrel and modulesInGPU.layers[idx] < 5) or (modulesInGPU.subdets[idx] == SDL::Endcap and modulesInGPU.layers[idx] == 1) or modulesInGPU.subdets[idx] == SDL::InnerPixel)
        {
            modulesInGPU.trackCandidateModuleIndices[i] = nEligibleModules * maxTrackCandidates;
            nEligibleModules++;
        }
    }
}


void SDL::createTrackCandidatesInUnifiedMemory(struct trackCandidates& trackCandidatesInGPU, unsigned int maxTrackCandidates, unsigned int maxPixelTrackCandidates, unsigned int nLowerModules, unsigned int nEligibleModules)
{
    unsigned int nMemoryLocations = maxTrackCandidates * (nEligibleModules-1) + maxPixelTrackCandidates;
    std::cout<<"Number of eligible modules = "<<nEligibleModules<<std::endl;
#ifdef CACHE_ALLOC
    hipStream_t stream=0;
    trackCandidatesInGPU.trackCandidateType = (short*)cms::cuda::allocate_managed(nMemoryLocations * sizeof(short),stream);
    //trackCandidatesInGPU.objectIndices = (unsigned int*)cms::cuda::allocate_managed(2 * nMemoryLocations * sizeof(unsigned int),stream);
    hipMallocManaged(&trackCandidatesInGPU.objectIndices, 2 * nMemoryLocations * sizeof(unsigned int));
    trackCandidatesInGPU.nTrackCandidates= (unsigned int*)cms::cuda::allocate_managed( nLowerModules * sizeof(unsigned int),stream);
    trackCandidatesInGPU.nTrackCandidatesT4T4= (unsigned int*)cms::cuda::allocate_managed( nLowerModules * sizeof(unsigned int),stream);
    trackCandidatesInGPU.nTrackCandidatesT4T3= (unsigned int*)cms::cuda::allocate_managed( nLowerModules * sizeof(unsigned int),stream);
    trackCandidatesInGPU.nTrackCandidatesT3T4= (unsigned int*)cms::cuda::allocate_managed( nLowerModules * sizeof(unsigned int),stream);
#else
    hipMallocManaged(&trackCandidatesInGPU.trackCandidateType, nMemoryLocations * sizeof(short));
    hipMallocManaged(&trackCandidatesInGPU.objectIndices, 2 * nMemoryLocations * sizeof(unsigned int));
    hipMallocManaged(&trackCandidatesInGPU.nTrackCandidates, nLowerModules * sizeof(unsigned int));
    hipMallocManaged(&trackCandidatesInGPU.nTrackCandidatesT4T4, nLowerModules * sizeof(unsigned int));
    hipMallocManaged(&trackCandidatesInGPU.nTrackCandidatesT4T3, nLowerModules * sizeof(unsigned int));
    hipMallocManaged(&trackCandidatesInGPU.nTrackCandidatesT3T4, nLowerModules * sizeof(unsigned int));
#endif

#pragma omp parallel for
    for(size_t i = 0; i<nLowerModules;i++)
    {
        trackCandidatesInGPU.nTrackCandidates[i] = 0;
        trackCandidatesInGPU.nTrackCandidatesT4T4[i] = 0;
        trackCandidatesInGPU.nTrackCandidatesT4T3[i] = 0;
        trackCandidatesInGPU.nTrackCandidatesT3T4[i] = 0;
    }
}
void SDL::createTrackCandidatesInExplicitMemory(struct trackCandidates& trackCandidatesInGPU, unsigned int maxTrackCandidates, unsigned int maxPixelTrackCandidates, unsigned int nLowerModules ,unsigned int nEligibleModules)
{
    unsigned int nMemoryLocations = maxTrackCandidates * (nEligibleModules-1) + maxPixelTrackCandidates;
    std::cout<<"Number of eligible modules = "<<nEligibleModules<<std::endl;
#ifdef CACHE_ALLOC
    hipStream_t stream=0;
    int dev;
    hipGetDevice(&dev);
    //trackCandidatesInGPU.trackCandidateType = (short*)cms::cuda::allocate_device(dev,nMemoryLocations * sizeof(short),stream);
    hipMalloc(&trackCandidatesInGPU.trackCandidateType, nMemoryLocations * sizeof(short));
    //trackCandidatesInGPU.objectIndices = (unsigned int*)cms::cuda::allocate_device(dev,2 * nMemoryLocations * sizeof(unsigned int),stream); // too big to cache
    hipMalloc(&trackCandidatesInGPU.objectIndices, 2 * nMemoryLocations * sizeof(unsigned int));
    trackCandidatesInGPU.nTrackCandidates= (unsigned int*)cms::cuda::allocate_device(dev, nLowerModules * sizeof(unsigned int),stream);
    trackCandidatesInGPU.nTrackCandidatesT4T4= (unsigned int*)cms::cuda::allocate_device(dev, nLowerModules * sizeof(unsigned int),stream);
    trackCandidatesInGPU.nTrackCandidatesT4T3= (unsigned int*)cms::cuda::allocate_device(dev, nLowerModules * sizeof(unsigned int),stream);
    trackCandidatesInGPU.nTrackCandidatesT3T4= (unsigned int*)cms::cuda::allocate_device(dev, nLowerModules * sizeof(unsigned int),stream);

#else
    hipMalloc(&trackCandidatesInGPU.trackCandidateType, nMemoryLocations * sizeof(short));
    hipMalloc(&trackCandidatesInGPU.objectIndices, 2 * nMemoryLocations * sizeof(unsigned int));
    hipMalloc(&trackCandidatesInGPU.nTrackCandidates, nLowerModules * sizeof(unsigned int));
    hipMalloc(&trackCandidatesInGPU.nTrackCandidatesT4T4, nLowerModules * sizeof(unsigned int));
    hipMalloc(&trackCandidatesInGPU.nTrackCandidatesT4T3, nLowerModules * sizeof(unsigned int));
    hipMalloc(&trackCandidatesInGPU.nTrackCandidatesT3T4, nLowerModules * sizeof(unsigned int));
#endif
    hipMemset(trackCandidatesInGPU.nTrackCandidates,0, nLowerModules * sizeof(unsigned int));
    hipMemset(trackCandidatesInGPU.nTrackCandidatesT4T4,0, nLowerModules * sizeof(unsigned int));
    hipMemset(trackCandidatesInGPU.nTrackCandidatesT4T3,0, nLowerModules * sizeof(unsigned int));
    hipMemset(trackCandidatesInGPU.nTrackCandidatesT3T4,0, nLowerModules * sizeof(unsigned int));

}

__device__ void SDL::addTrackCandidateToMemory(struct trackCandidates& trackCandidatesInGPU, short trackCandidateType, unsigned int innerTrackletIndex, unsigned int outerTrackletIndex, unsigned int trackCandidateIndex)
{
    trackCandidatesInGPU.trackCandidateType[trackCandidateIndex] = trackCandidateType;
    trackCandidatesInGPU.objectIndices[2 * trackCandidateIndex] = innerTrackletIndex;
    trackCandidatesInGPU.objectIndices[2 * trackCandidateIndex + 1] = outerTrackletIndex;
}

__device__ bool SDL::runTrackCandidateDefaultAlgoTwoTracklets(struct tracklets& trackletsInGPU, struct triplets& tripletsInGPU, unsigned int innerTrackletIndex, unsigned int outerTrackletIndex, short& trackCandidateType)
{
    bool pass = true;
    trackCandidateType = 0;
    if(not hasCommonSegment(trackletsInGPU, tripletsInGPU, innerTrackletIndex, outerTrackletIndex, trackCandidateType))
    {
        pass = false;
    }
    return pass;
}

SDL::trackCandidates::trackCandidates()
{
    trackCandidateType = nullptr;
    objectIndices = nullptr;
    nTrackCandidates = nullptr;
}

SDL::trackCandidates::~trackCandidates()
{
}

void SDL::trackCandidates::freeMemoryCache()
{
#ifdef Explicit_Track
    int dev;
    hipGetDevice(&dev);
    //cms::cuda::free_device(dev,trackCandidateType);
    hipFree(trackCandidateType);
    //cms::cuda::free_device(dev,objectIndices);
    cms::cuda::free_device(dev,nTrackCandidates);
    cms::cuda::free_device(dev,nTrackCandidatesT4T4);
    cms::cuda::free_device(dev,nTrackCandidatesT4T3);
    cms::cuda::free_device(dev,nTrackCandidatesT3T4);
#else
    cms::cuda::free_managed(trackCandidateType);
    //cms::cuda::free_managed(objectIndices);
    cms::cuda::free_managed(nTrackCandidates);
    cms::cuda::free_managed(nTrackCandidatesT4T4);
    cms::cuda::free_managed(nTrackCandidatesT4T3);
    cms::cuda::free_managed(nTrackCandidatesT3T4);

#endif
    hipFree(objectIndices);

}
void SDL::trackCandidates::freeMemory()
{
    hipFree(trackCandidateType);
    hipFree(objectIndices);
    hipFree(nTrackCandidates);
    hipFree(nTrackCandidatesT4T4);
    hipFree(nTrackCandidatesT4T3);
    hipFree(nTrackCandidatesT3T4);
}

__device__ bool SDL::runTrackCandidateDefaultAlgoTrackletToTriplet(struct tracklets& trackletsInGPU, struct triplets& tripletsInGPU, unsigned int innerTrackletIndex, unsigned int outerTripletIndex, short& trackCandidateType)
{
    bool pass = true;
    trackCandidateType = 1;
    if(not hasCommonSegment(trackletsInGPU, tripletsInGPU, innerTrackletIndex, outerTripletIndex, trackCandidateType))
    {
        pass = false;
    }

    return pass;
}

__device__ bool SDL::runTrackCandidateDefaultAlgoTripletToTracklet(struct tracklets& trackletsInGPU, struct triplets& tripletsInGPU, unsigned int innerTripletIndex, unsigned int outerTrackletIndex, short& trackCandidateType)
{
    bool pass = true;
    trackCandidateType = 2;
    if(not hasCommonSegment(trackletsInGPU, tripletsInGPU, innerTripletIndex, outerTrackletIndex, trackCandidateType))
    {
        pass = false;
    }
    return pass;
}

__device__ bool SDL::hasCommonSegment(struct tracklets& trackletsInGPU, struct triplets& tripletsInGPU, unsigned int innerObjectIndex, unsigned int outerObjectIndex, short trackCandidateType)
{
    unsigned int innerObjectOuterSegmentIndex, outerObjectInnerSegmentIndex;

    if(trackCandidateType == 0)
    {
        //2 tracklets
        innerObjectOuterSegmentIndex = trackletsInGPU.segmentIndices[2 * innerObjectIndex + 1];
        outerObjectInnerSegmentIndex = trackletsInGPU.segmentIndices[2 * outerObjectIndex];
    }
    else if(trackCandidateType == 1)
    {
        //T4T3
        innerObjectOuterSegmentIndex = trackletsInGPU.segmentIndices[2 * innerObjectIndex + 1];
        outerObjectInnerSegmentIndex = tripletsInGPU.segmentIndices[2 * outerObjectIndex];
    }
    else if(trackCandidateType == 2)
    {
        //T3T4
        innerObjectOuterSegmentIndex = tripletsInGPU.segmentIndices[2 * innerObjectIndex + 1];
        outerObjectInnerSegmentIndex = trackletsInGPU.segmentIndices[2 * outerObjectIndex];
    }

    return (innerObjectOuterSegmentIndex == outerObjectInnerSegmentIndex);
}


