#include "TrackCandidate.cuh"

#include "allocate.h"


void SDL::createTrackCandidatesInUnifiedMemory(struct trackCandidates& trackCandidatesInGPU, unsigned int maxTrackCandidates)
{
#ifdef CACHE_ALLOC
    hipStream_t stream=0;
    trackCandidatesInGPU.trackCandidateType = (short*)cms::cuda::allocate_managed(maxTrackCandidates * sizeof(short),stream);
    hipMallocManaged(&trackCandidatesInGPU.objectIndices, 2 * maxTrackCandidates * sizeof(unsigned int));
    trackCandidatesInGPU.nTrackCandidates= (unsigned int*)cms::cuda::allocate_managed( sizeof(unsigned int),stream);
    trackCandidatesInGPU.nTrackCandidatespT3= (unsigned int*)cms::cuda::allocate_managed(sizeof(unsigned int),stream);
    trackCandidatesInGPU.nTrackCandidatesT5= (unsigned int*)cms::cuda::allocate_managed( sizeof(unsigned int),stream);
    trackCandidatesInGPU.nTrackCandidatespT5= (unsigned int*)cms::cuda::allocate_managed( sizeof(unsigned int),stream);
    trackCandidatesInGPU.nTrackCandidatespLS= (unsigned int*)cms::cuda::allocate_managed( sizeof(unsigned int),stream);

#else
    hipMallocManaged(&trackCandidatesInGPU.trackCandidateType, maxTrackCandidates * sizeof(short));
    hipMallocManaged(&trackCandidatesInGPU.objectIndices, 2 * maxTrackCandidates * sizeof(unsigned int));
    hipMallocManaged(&trackCandidatesInGPU.nTrackCandidates, sizeof(unsigned int));
    hipMallocManaged(&trackCandidatesInGPU.nTrackCandidatespT3, sizeof(unsigned int));
    hipMallocManaged(&trackCandidatesInGPU.nTrackCandidatesT5, sizeof(unsigned int));
    hipMallocManaged(&trackCandidatesInGPU.nTrackCandidatespT5, sizeof(unsigned int));

    hipMallocManaged(&trackCandidatesInGPU.nTrackCandidatespLS, sizeof(unsigned int));
#endif

    hipMemset(trackCandidatesInGPU.nTrackCandidates,0, sizeof(unsigned int));
    hipMemset(trackCandidatesInGPU.nTrackCandidatesT5,0, sizeof(unsigned int));
    hipMemset(trackCandidatesInGPU.nTrackCandidatespT3,0, sizeof(unsigned int));
    hipMemset(trackCandidatesInGPU.nTrackCandidatespT5,0, sizeof(unsigned int));
    hipMemset(trackCandidatesInGPU.nTrackCandidatespLS,0, sizeof(unsigned int));
}
void SDL::createTrackCandidatesInExplicitMemory(struct trackCandidates& trackCandidatesInGPU, unsigned int maxTrackCandidates)
{
#ifdef CACHE_ALLOC
    hipStream_t stream=0;
    int dev;
    hipGetDevice(&dev);
    //TODO 
    hipMalloc(&trackCandidatesInGPU.trackCandidateType, maxTrackCandidates * sizeof(short));
    hipMalloc(&trackCandidatesInGPU.objectIndices, 2 * maxTrackCandidates * sizeof(unsigned int));
    trackCandidatesInGPU.nTrackCandidates= (unsigned int*)cms::cuda::allocate_device(dev, sizeof(unsigned int),stream);
    trackCandidatesInGPU.nTrackCandidatespT3= (unsigned int*)cms::cuda::allocate_device(dev, sizeof(unsigned int),stream);
    trackCandidatesInGPU.nTrackCandidatesT5= (unsigned int*)cms::cuda::allocate_device(dev, sizeof(unsigned int),stream);
    trackCandidatesInGPU.nTrackCandidatespT5= (unsigned int*)cms::cuda::allocate_device(dev, sizeof(unsigned int),stream);
    trackCandidatesInGPU.nTrackCandidatespLS= (unsigned int*)cms::cuda::allocate_device(dev, sizeof(unsigned int),stream);
    trackCandidatesInGPU.partOfExtension = (bool*)cms::cuda::allocate_device(dev, maxTrackCandidates * sizeof(bool), stream);

#else
    hipMalloc(&trackCandidatesInGPU.trackCandidateType, maxTrackCandidates * sizeof(short));
    hipMalloc(&trackCandidatesInGPU.objectIndices, 2 * maxTrackCandidates * sizeof(unsigned int));
    hipMalloc(&trackCandidatesInGPU.nTrackCandidates, sizeof(unsigned int));
    hipMalloc(&trackCandidatesInGPU.nTrackCandidatespT3, sizeof(unsigned int));
    hipMalloc(&trackCandidatesInGPU.nTrackCandidatesT5, sizeof(unsigned int));
    hipMalloc(&trackCandidatesInGPU.nTrackCandidatespT5, sizeof(unsigned int));
    hipMalloc(&trackCandidatesInGPU.nTrackCandidatespLS, sizeof(unsigned int));
#endif
    hipMemset(trackCandidatesInGPU.nTrackCandidates,0, sizeof(unsigned int));
    hipMemset(trackCandidatesInGPU.nTrackCandidatesT5,0, sizeof(unsigned int));
    hipMemset(trackCandidatesInGPU.nTrackCandidatespT3,0, sizeof(unsigned int));
    hipMemset(trackCandidatesInGPU.nTrackCandidatespT5,0, sizeof(unsigned int));
    hipMemset(trackCandidatesInGPU.nTrackCandidatespLS,0, sizeof(unsigned int));
}

__device__ void SDL::addTrackCandidateToMemory(struct trackCandidates& trackCandidatesInGPU, short trackCandidateType, unsigned int innerTrackletIndex, unsigned int outerTrackletIndex, unsigned int trackCandidateIndex)
{
    trackCandidatesInGPU.trackCandidateType[trackCandidateIndex] = trackCandidateType;
    trackCandidatesInGPU.objectIndices[2 * trackCandidateIndex] = innerTrackletIndex;
    trackCandidatesInGPU.objectIndices[2 * trackCandidateIndex + 1] = outerTrackletIndex;
}

SDL::trackCandidates::trackCandidates()
{
    trackCandidateType = nullptr;
    objectIndices = nullptr;
    nTrackCandidates = nullptr;
    nTrackCandidatesT5 = nullptr;
    nTrackCandidatespT3 = nullptr;
    nTrackCandidatespT5 = nullptr;
    nTrackCandidatespLS = nullptr;
}

SDL::trackCandidates::~trackCandidates()
{
}

void SDL::trackCandidates::freeMemoryCache()
{
#ifdef Explicit_Track
    int dev;
    hipGetDevice(&dev);
    //FIXME
    hipFree(trackCandidateType);
    cms::cuda::free_device(dev,nTrackCandidates);
    cms::cuda::free_device(dev,nTrackCandidatespT3);
    cms::cuda::free_device(dev,nTrackCandidatesT5);
    cms::cuda::free_device(dev,nTrackCandidatespT5);


    cms::cuda::free_device(dev,nTrackCandidatespLS);
#else
    cms::cuda::free_managed(trackCandidateType);
    cms::cuda::free_managed(nTrackCandidates);
    cms::cuda::free_managed(nTrackCandidatespT3);
    cms::cuda::free_managed(nTrackCandidatesT5);
    cms::cuda::free_managed(nTrackCandidatespT5);
    cms::cuda::free_managed(nTrackCandidatespLS);
#endif
    hipFree(objectIndices);

}
void SDL::trackCandidates::freeMemory()
{
    hipFree(trackCandidateType);
    hipFree(objectIndices);
    hipFree(nTrackCandidates);
    hipFree(nTrackCandidatespT3);
    hipFree(nTrackCandidatesT5);
    hipFree(nTrackCandidatespT5);
    hipFree(nTrackCandidatespLS);
}

