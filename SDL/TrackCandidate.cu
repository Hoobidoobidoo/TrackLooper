#include "TrackCandidate.cuh"

#include "allocate.h"


void SDL::createTrackCandidatesInUnifiedMemory(struct trackCandidates& trackCandidatesInGPU, unsigned int maxTrackCandidates)
{
#ifdef CACHE_ALLOC
    hipStream_t stream=0;
    trackCandidatesInGPU.trackCandidateType = (short*)cms::cuda::allocate_managed(maxTrackCandidates * sizeof(short),stream);
    hipMallocManaged(&trackCandidatesInGPU.objectIndices, 2 * maxTrackCandidates * sizeof(unsigned int));
    trackCandidatesInGPU.nTrackCandidates= (unsigned int*)cms::cuda::allocate_managed( sizeof(unsigned int),stream);
    trackCandidatesInGPU.nTrackCandidatespT3= (unsigned int*)cms::cuda::allocate_managed(sizeof(unsigned int),stream);
    trackCandidatesInGPU.nTrackCandidatesT5= (unsigned int*)cms::cuda::allocate_managed( sizeof(unsigned int),stream);
    trackCandidatesInGPU.nTrackCandidatespT5= (unsigned int*)cms::cuda::allocate_managed( sizeof(unsigned int),stream);
    trackCandidatesInGPU.logicalLayers = (unsigned int*)cms::cuda::allocate_managed(7 * maxTrackCandidates * sizeof(unsigned int), stream);
    trackCandidatesInGPU.lowerModuleIndices = (unsigned int*)cms::cuda::allocate_managed(7 * maxTrackCandidates * sizeof(unsigned int), stream);
    trackCandidatesInGPU.hitIndices = (unsigned int*)cms::cuda::allocate_managed(14 * maxTrackCandidates * sizeof(unsigned int), stream);

    trackCandidatesInGPU.centerX = (float*)cms::cuda::allocate_managed(maxTrackCandidates * sizeof(float), stream);
    trackCandidatesInGPU.centerY = (float*)cms::cuda::allocate_managed(maxTrackCandidates * sizeof(float), stream);
    trackCandidatesInGPU.radius = (float*)cms::cuda::allocate_managed(maxTrackCandidates * sizeof(float), stream);

    trackCandidatesInGPU.partOfExtension = (bool*)cms::cuda::allocate_managed(maxTrackCandidates * sizeof(bool), stream);
    trackCandidatesInGPU.nTrackCandidatespLS= (unsigned int*)cms::cuda::allocate_managed( sizeof(unsigned int),stream);

#else
    hipMallocManaged(&trackCandidatesInGPU.trackCandidateType, maxTrackCandidates * sizeof(short));
    hipMallocManaged(&trackCandidatesInGPU.objectIndices, 2 * maxTrackCandidates * sizeof(unsigned int));
    hipMallocManaged(&trackCandidatesInGPU.nTrackCandidates, sizeof(unsigned int));
    hipMallocManaged(&trackCandidatesInGPU.nTrackCandidatespT3, sizeof(unsigned int));
    hipMallocManaged(&trackCandidatesInGPU.nTrackCandidatesT5, sizeof(unsigned int));
    hipMallocManaged(&trackCandidatesInGPU.nTrackCandidatespT5, sizeof(unsigned int));
    hipMallocManaged(&trackCandidatesInGPU.nTrackCandidatespLS, sizeof(unsigned int));

    hipMallocManaged(&trackCandidatesInGPU.logicalLayers, maxTrackCandidates * 7 * sizeof(unsigned int));
    hipMallocManaged(&trackCandidatesInGPU.lowerModuleIndices, maxTrackCandidates * 7 * sizeof(unsigned int));
    hipMallocManaged(&trackCandidatesInGPU.hitIndices, maxTrackCandidates * 14 * sizeof(unsigned int));
    hipMallocManaged(&trackCandidatesInGPU.partOfExtension, maxTrackCandidates*sizeof(bool));

    hipMallocManaged(&trackCandidatesInGPU.centerX, maxTrackCandidates * sizeof(float));
    hipMallocManaged(&trackCandidatesInGPU.centerY, maxTrackCandidates * sizeof(float));
    hipMallocManaged(&trackCandidatesInGPU.radius, maxTrackCandidates * sizeof(float));

#endif

    hipMemset(trackCandidatesInGPU.nTrackCandidates,0, sizeof(unsigned int));
    hipMemset(trackCandidatesInGPU.nTrackCandidatesT5,0, sizeof(unsigned int));
    hipMemset(trackCandidatesInGPU.nTrackCandidatespT3,0, sizeof(unsigned int));
    hipMemset(trackCandidatesInGPU.nTrackCandidatespT5,0, sizeof(unsigned int));
    hipMemset(trackCandidatesInGPU.nTrackCandidatespLS,0, sizeof(unsigned int));
    hipMemset(trackCandidatesInGPU.partOfExtension, false, maxTrackCandidates * sizeof(bool));
}
void SDL::createTrackCandidatesInExplicitMemory(struct trackCandidates& trackCandidatesInGPU, unsigned int maxTrackCandidates)
{
#ifdef CACHE_ALLOC
    hipStream_t stream=0;
    int dev;
    hipGetDevice(&dev);
    //TODO 
    hipMalloc(&trackCandidatesInGPU.trackCandidateType, maxTrackCandidates * sizeof(short));
    hipMalloc(&trackCandidatesInGPU.objectIndices, 2 * maxTrackCandidates * sizeof(unsigned int));
    trackCandidatesInGPU.nTrackCandidates= (unsigned int*)cms::cuda::allocate_device(dev, sizeof(unsigned int),stream);
    trackCandidatesInGPU.nTrackCandidatespT3= (unsigned int*)cms::cuda::allocate_device(dev, sizeof(unsigned int),stream);
    trackCandidatesInGPU.nTrackCandidatesT5= (unsigned int*)cms::cuda::allocate_device(dev, sizeof(unsigned int),stream);
    trackCandidatesInGPU.nTrackCandidatespT5= (unsigned int*)cms::cuda::allocate_device(dev, sizeof(unsigned int),stream);
    trackCandidatesInGPU.nTrackCandidatespLS= (unsigned int*)cms::cuda::allocate_device(dev, sizeof(unsigned int),stream);
    trackCandidatesInGPU.partOfExtension = (bool*)cms::cuda::allocate_device(dev, maxTrackCandidates * sizeof(bool), stream);

    trackCandidatesInGPU.logicalLayers = (unsigned int*)cms::cuda::allocate_device(dev, 7 * maxTrackCandidates * sizeof(unsigned int), stream);
    trackCandidatesInGPU.lowerModuleIndices = (unsigned int*)cms::cuda::allocate_device(dev, 7 * maxTrackCandidates * sizeof(unsigned int), stream);
    trackCandidatesInGPU.hitIndices = (unsigned int*)cms::cuda::allocate_device(dev, 14 * maxTrackCandidates * sizeof(unsigned int), stream);

    trackCandidatesInGPU.centerX = (float*)cms::cuda::allocate_device(dev, maxTrackCandidates * sizeof(float), stream);
    trackCandidatesInGPU.centerY = (float*)cms::cuda::allocate_device(dev, maxTrackCandidates * sizeof(float), stream);
    trackCandidatesInGPU.radius = (float*)cms::cuda::allocate_device(dev, maxTrackCandidates * sizeof(float), stream);

#else
    hipMalloc(&trackCandidatesInGPU.trackCandidateType, maxTrackCandidates * sizeof(short));
    hipMalloc(&trackCandidatesInGPU.objectIndices, 2 * maxTrackCandidates * sizeof(unsigned int));
    hipMalloc(&trackCandidatesInGPU.nTrackCandidates, sizeof(unsigned int));
    hipMalloc(&trackCandidatesInGPU.nTrackCandidatespT3, sizeof(unsigned int));
    hipMalloc(&trackCandidatesInGPU.nTrackCandidatesT5, sizeof(unsigned int));
    hipMalloc(&trackCandidatesInGPU.nTrackCandidatespT5, sizeof(unsigned int));
    hipMalloc(&trackCandidatesInGPU.nTrackCandidatespLS, sizeof(unsigned int));
    hipMalloc(&trackCandidatesInGPU.partOfExtension, maxTrackCandidates * sizeof(bool));
    hipMalloc(&trackCandidatesInGPU.logicalLayers, 7 * maxTrackCandidates * sizeof(unsigned int));
    hipMalloc(&trackCandidatesInGPU.lowerModuleIndices, 7 * maxTrackCandidates * sizeof(unsigned int));
    hipMalloc(&trackCandidatesInGPU.hitIndices, 14 * maxTrackCandidates * sizeof(unsigned int));
    hipMalloc(&trackCandidatesInGPU.centerX, maxTrackCandidates * sizeof(float));
    hipMalloc(&trackCandidatesInGPU.centerY, maxTrackCandidates * sizeof(float));
    hipMalloc(&trackCandidatesInGPU.radius, maxTrackCandidates * sizeof(float));
#endif
    hipMemset(trackCandidatesInGPU.nTrackCandidates,0, sizeof(unsigned int));
    hipMemset(trackCandidatesInGPU.nTrackCandidatesT5,0, sizeof(unsigned int));
    hipMemset(trackCandidatesInGPU.nTrackCandidatespT3,0, sizeof(unsigned int));
    hipMemset(trackCandidatesInGPU.nTrackCandidatespT5,0, sizeof(unsigned int));
    hipMemset(trackCandidatesInGPU.nTrackCandidatespLS,0, sizeof(unsigned int));
    hipMemset(trackCandidatesInGPU.partOfExtension, false, maxTrackCandidates * sizeof(bool));
}

__device__ void SDL::addTrackCandidateToMemory(struct trackCandidates& trackCandidatesInGPU, short trackCandidateType, unsigned int innerTrackletIndex, unsigned int outerTrackletIndex, unsigned int trackCandidateIndex)
{
    trackCandidatesInGPU.trackCandidateType[trackCandidateIndex] = trackCandidateType;
    trackCandidatesInGPU.objectIndices[2 * trackCandidateIndex] = innerTrackletIndex;
    trackCandidatesInGPU.objectIndices[2 * trackCandidateIndex + 1] = outerTrackletIndex;
}

__device__ void SDL::addTrackCandidateToMemory(struct trackCandidates& trackCandidatesInGPU, short trackCandidateType, unsigned int innerTrackletIndex, unsigned int outerTrackletIndex, unsigned int* logicalLayerIndices, unsigned int* lowerModuleIndices, unsigned int* hitIndices, float& centerX, float& centerY, float& radius, unsigned int trackCandidateIndex)
{
    trackCandidatesInGPU.trackCandidateType[trackCandidateIndex] = trackCandidateType;
    trackCandidatesInGPU.objectIndices[2 * trackCandidateIndex] = innerTrackletIndex;
    trackCandidatesInGPU.objectIndices[2 * trackCandidateIndex + 1] = outerTrackletIndex;
    
    size_t limits = trackCandidateType == 7 ? 7 : 5;

    //send the starting pointer to the logicalLayer and hitIndices
    for(size_t i = 0; i < limits; i++)
    {
        trackCandidatesInGPU.logicalLayers[7 * trackCandidateIndex + i] = logicalLayerIndices[i];
        trackCandidatesInGPU.lowerModuleIndices[7 * trackCandidateIndex + i] = lowerModuleIndices[i];
    }
    for(size_t i = 0; i < 2 * limits; i++)
    {
        trackCandidatesInGPU.hitIndices[14 * trackCandidateIndex + i] = hitIndices[i];
    }
    trackCandidatesInGPU.centerX[trackCandidateIndex] = centerX;
    trackCandidatesInGPU.centerY[trackCandidateIndex] = centerY;
    trackCandidatesInGPU.radius[trackCandidateIndex] = radius;
}

SDL::trackCandidates::trackCandidates()
{
    trackCandidateType = nullptr;
    objectIndices = nullptr;
    nTrackCandidates = nullptr;
    nTrackCandidatesT5 = nullptr;
    nTrackCandidatespT3 = nullptr;
    logicalLayers = nullptr;
    hitIndices = nullptr;
    lowerModuleIndices = nullptr;
    partOfExtension = nullptr;

    centerX = nullptr;
    centerY = nullptr;
    radius = nullptr;
    nTrackCandidatespT5 = nullptr;
    nTrackCandidatespLS = nullptr;
}

SDL::trackCandidates::~trackCandidates()
{
}

void SDL::trackCandidates::freeMemoryCache()
{
#ifdef Explicit_Track
    int dev;
    hipGetDevice(&dev);
    //FIXME
    hipFree(trackCandidateType);
    cms::cuda::free_device(dev,nTrackCandidates);
    cms::cuda::free_device(dev,nTrackCandidatespT3);
    cms::cuda::free_device(dev,nTrackCandidatesT5);
    cms::cuda::free_device(dev,nTrackCandidatespT5);

    cms::cuda::free_device(dev, logicalLayers);
    cms::cuda::free_device(dev, hitIndices);
    cms::cuda::free_device(dev, lowerModuleIndices);

    cms::cuda::free_device(dev,nTrackCandidatespLS);
    cms::cuda::free_device(dev, centerX);
    cms::cuda::free_device(dev, centerY);
    cms::cuda::free_device(dev, radius);
    cms::cuda::free_device(dev, partOfExtension);
#else
    cms::cuda::free_managed(trackCandidateType);
    cms::cuda::free_managed(nTrackCandidates);
    cms::cuda::free_managed(nTrackCandidatespT3);
    cms::cuda::free_managed(nTrackCandidatesT5);
    cms::cuda::free_managed(nTrackCandidatespT5);
    cms::cuda::free_managed(logicalLayers);
    cms::cuda::free_managed(hitIndices);
    cms::cuda::free_managed(lowerModuleIndices);
    cms::cuda::free_managed(centerX);
    cms::cuda::free_managed(centerY);
    cms::cuda::free_managed(radius);
    cms::cuda::free_managed(partOfExtension);
    cms::cuda::free_managed(nTrackCandidatespLS);
#endif
    hipFree(objectIndices);

}
void SDL::trackCandidates::freeMemory()
{
    hipFree(trackCandidateType);
    hipFree(objectIndices);
    hipFree(nTrackCandidates);
    hipFree(nTrackCandidatespT3);
    hipFree(nTrackCandidatesT5);
    hipFree(nTrackCandidatespT5);

    hipFree(logicalLayers);
    hipFree(hitIndices);
    hipFree(lowerModuleIndices);
    hipFree(partOfExtension);

    hipFree(centerX);
    hipFree(centerY);
    hipFree(radius);
    
    hipFree(nTrackCandidatespLS);
}
