#include "TrackCandidate.cuh"

#include "allocate.h"


void SDL::trackCandidates::resetMemory(unsigned int maxTrackCandidates,hipStream_t stream)
{
    hipMemsetAsync(trackCandidateType,0, maxTrackCandidates * sizeof(short),stream);
    hipMemsetAsync(objectIndices, 0,2 * maxTrackCandidates * sizeof(unsigned int),stream);
    hipMemsetAsync(nTrackCandidates, 0,sizeof(unsigned int),stream);
    hipMemsetAsync(nTrackCandidatespT3, 0,sizeof(unsigned int),stream);
    hipMemsetAsync(nTrackCandidatesT5, 0,sizeof(unsigned int),stream);
    hipMemsetAsync(nTrackCandidatespT5,0, sizeof(unsigned int),stream);
    hipMemsetAsync(nTrackCandidatespLS, 0,sizeof(unsigned int),stream);

#ifdef TRACK_EXTENSIONS
    hipMemsetAsync(logicalLayers, 0, 7 * maxTrackCandidates * sizeof(unsigned int), stream);
    hipMemsetAsync(lowerModuleIndices, 0, 7 * maxTrackCandidates * sizeof(unsigned int), stream);
    hipMemsetAsync(hitIndices, 0, 14 * maxTrackCandidates * sizeof(unsigned int), stream);
    hipMemsetAsync(centerX, 0, maxTrackCandidates * sizeof(float), stream);
    hipMemsetAsync(centerY, 0, maxTrackCandidates * sizeof(float), stream);
    hipMemsetAsync(radius, 0, maxTrackCandidates * sizeof(float), stream);
    hipMemsetAsync(partOfExtension, 0, maxTrackCandidates * sizeof(bool), stream);
#endif
}
void SDL::createTrackCandidatesInUnifiedMemory(struct trackCandidates& trackCandidatesInGPU, unsigned int maxTrackCandidates,hipStream_t stream)
{
#ifdef CACHE_ALLOC
    trackCandidatesInGPU.trackCandidateType = (short*)cms::cuda::allocate_managed(maxTrackCandidates * sizeof(short),stream);
    trackCandidatesInGPU.objectIndices = (unsigned int*)cms::cuda::allocate_managed(maxTrackCandidates * 2*sizeof(unsigned int),stream);
    trackCandidatesInGPU.nTrackCandidates= (unsigned int*)cms::cuda::allocate_managed( sizeof(unsigned int),stream);
    trackCandidatesInGPU.nTrackCandidatespT3= (unsigned int*)cms::cuda::allocate_managed(sizeof(unsigned int),stream);
    trackCandidatesInGPU.nTrackCandidatesT5= (unsigned int*)cms::cuda::allocate_managed( sizeof(unsigned int),stream);
    trackCandidatesInGPU.nTrackCandidatespT5= (unsigned int*)cms::cuda::allocate_managed( sizeof(unsigned int),stream);

#ifdef TRACK_EXTENSIONS
    trackCandidatesInGPU.logicalLayers = (unsigned int*)cms::cuda::allocate_managed(7 * maxTrackCandidates * sizeof(unsigned int), stream);
    trackCandidatesInGPU.lowerModuleIndices = (unsigned int*)cms::cuda::allocate_managed(7 * maxTrackCandidates * sizeof(unsigned int), stream);
    trackCandidatesInGPU.hitIndices = (unsigned int*)cms::cuda::allocate_managed(14 * maxTrackCandidates * sizeof(unsigned int), stream);
    trackCandidatesInGPU.centerX = (float*)cms::cuda::allocate_managed(maxTrackCandidates * sizeof(float), stream);
    trackCandidatesInGPU.centerY = (float*)cms::cuda::allocate_managed(maxTrackCandidates * sizeof(float), stream);
    trackCandidatesInGPU.radius = (float*)cms::cuda::allocate_managed(maxTrackCandidates * sizeof(float), stream);
    trackCandidatesInGPU.partOfExtension = (bool*)cms::cuda::allocate_managed(maxTrackCandidates * sizeof(bool), stream);
#endif

    trackCandidatesInGPU.nTrackCandidatespLS= (unsigned int*)cms::cuda::allocate_managed( sizeof(unsigned int),stream);
#else
    hipMallocManaged(&trackCandidatesInGPU.trackCandidateType, maxTrackCandidates * sizeof(short));
    hipMallocManaged(&trackCandidatesInGPU.objectIndices, 2 * maxTrackCandidates * sizeof(unsigned int));
    hipMallocManaged(&trackCandidatesInGPU.nTrackCandidates, sizeof(unsigned int));
    hipMallocManaged(&trackCandidatesInGPU.nTrackCandidatespT3, sizeof(unsigned int));
    hipMallocManaged(&trackCandidatesInGPU.nTrackCandidatesT5, sizeof(unsigned int));
    hipMallocManaged(&trackCandidatesInGPU.nTrackCandidatespT5, sizeof(unsigned int));

    hipMallocManaged(&trackCandidatesInGPU.nTrackCandidatespLS, sizeof(unsigned int));
#ifdef TRACK_EXTENSIONS
    hipMallocManaged(&trackCandidatesInGPU.logicalLayers, maxTrackCandidates * 7 * sizeof(unsigned int));
    hipMallocManaged(&trackCandidatesInGPU.lowerModuleIndices, maxTrackCandidates * 7 * sizeof(unsigned int));
    hipMallocManaged(&trackCandidatesInGPU.hitIndices, maxTrackCandidates * 14 * sizeof(unsigned int));
    hipMallocManaged(&trackCandidatesInGPU.partOfExtension, maxTrackCandidates*sizeof(bool));
    hipMallocManaged(&trackCandidatesInGPU.centerX, maxTrackCandidates * sizeof(float));
    hipMallocManaged(&trackCandidatesInGPU.centerY, maxTrackCandidates * sizeof(float));
    hipMallocManaged(&trackCandidatesInGPU.radius, maxTrackCandidates * sizeof(float));
#endif
#endif
    hipMemsetAsync(trackCandidatesInGPU.nTrackCandidates,0, sizeof(unsigned int),stream);
    hipMemsetAsync(trackCandidatesInGPU.nTrackCandidatesT5,0, sizeof(unsigned int),stream);
    hipMemsetAsync(trackCandidatesInGPU.nTrackCandidatespT3,0, sizeof(unsigned int),stream);
    hipMemsetAsync(trackCandidatesInGPU.nTrackCandidatespT5,0, sizeof(unsigned int),stream);
    hipMemsetAsync(trackCandidatesInGPU.nTrackCandidatespLS,0, sizeof(unsigned int),stream);
#ifdef TRACK_EXTENSIONS
    hipMemsetAsync(trackCandidatesInGPU.partOfExtension, false, maxTrackCandidates * sizeof(bool));
#endif
}
void SDL::createTrackCandidatesInExplicitMemory(struct trackCandidates& trackCandidatesInGPU, unsigned int maxTrackCandidates,hipStream_t stream)
{
#ifdef CACHE_ALLOC
    int dev;
    hipGetDevice(&dev);
    trackCandidatesInGPU.trackCandidateType = (short*)cms::cuda::allocate_device(dev,maxTrackCandidates * sizeof(short),stream);
    trackCandidatesInGPU.objectIndices = (unsigned int*)cms::cuda::allocate_device(dev,maxTrackCandidates * 2*sizeof(unsigned int),stream);
    trackCandidatesInGPU.nTrackCandidates= (unsigned int*)cms::cuda::allocate_device(dev, sizeof(unsigned int),stream);
    trackCandidatesInGPU.nTrackCandidatespT3= (unsigned int*)cms::cuda::allocate_device(dev, sizeof(unsigned int),stream);
    trackCandidatesInGPU.nTrackCandidatesT5= (unsigned int*)cms::cuda::allocate_device(dev, sizeof(unsigned int),stream);
    trackCandidatesInGPU.nTrackCandidatespT5= (unsigned int*)cms::cuda::allocate_device(dev, sizeof(unsigned int),stream);
    trackCandidatesInGPU.nTrackCandidatespLS= (unsigned int*)cms::cuda::allocate_device(dev, sizeof(unsigned int),stream);

#ifdef TRACK_EXTENSIONS
    trackCandidatesInGPU.partOfExtension = (bool*)cms::cuda::allocate_device(dev, maxTrackCandidates * sizeof(bool), stream);
    trackCandidatesInGPU.logicalLayers = (unsigned int*)cms::cuda::allocate_device(dev, 7 * maxTrackCandidates * sizeof(unsigned int), stream);
    trackCandidatesInGPU.lowerModuleIndices = (unsigned int*)cms::cuda::allocate_device(dev, 7 * maxTrackCandidates * sizeof(unsigned int), stream);
    trackCandidatesInGPU.hitIndices = (unsigned int*)cms::cuda::allocate_device(dev, 14 * maxTrackCandidates * sizeof(unsigned int), stream);
    trackCandidatesInGPU.centerX = (float*)cms::cuda::allocate_device(dev, maxTrackCandidates * sizeof(float), stream);
    trackCandidatesInGPU.centerY = (float*)cms::cuda::allocate_device(dev, maxTrackCandidates * sizeof(float), stream);
    trackCandidatesInGPU.radius = (float*)cms::cuda::allocate_device(dev, maxTrackCandidates * sizeof(float), stream);
#endif

#else
    hipMalloc(&trackCandidatesInGPU.trackCandidateType, maxTrackCandidates * sizeof(short));
    hipMalloc(&trackCandidatesInGPU.objectIndices, 2 * maxTrackCandidates * sizeof(unsigned int));
    hipMalloc(&trackCandidatesInGPU.nTrackCandidates, sizeof(unsigned int));
    hipMalloc(&trackCandidatesInGPU.nTrackCandidatespT3, sizeof(unsigned int));
    hipMalloc(&trackCandidatesInGPU.nTrackCandidatesT5, sizeof(unsigned int));
    hipMalloc(&trackCandidatesInGPU.nTrackCandidatespT5, sizeof(unsigned int));
    hipMalloc(&trackCandidatesInGPU.nTrackCandidatespLS, sizeof(unsigned int));

#ifdef TRACK_EXTENSIONS
    hipMalloc(&trackCandidatesInGPU.partOfExtension, maxTrackCandidates * sizeof(bool));
    hipMalloc(&trackCandidatesInGPU.logicalLayers, 7 * maxTrackCandidates * sizeof(unsigned int));
    hipMalloc(&trackCandidatesInGPU.lowerModuleIndices, 7 * maxTrackCandidates * sizeof(unsigned int));
    hipMalloc(&trackCandidatesInGPU.hitIndices, 14 * maxTrackCandidates * sizeof(unsigned int));
    hipMalloc(&trackCandidatesInGPU.centerX, maxTrackCandidates * sizeof(float));
    hipMalloc(&trackCandidatesInGPU.centerY, maxTrackCandidates * sizeof(float));
    hipMalloc(&trackCandidatesInGPU.radius, maxTrackCandidates * sizeof(float));
#endif
#endif
    hipMemsetAsync(trackCandidatesInGPU.nTrackCandidates,0, sizeof(unsigned int));
    hipMemsetAsync(trackCandidatesInGPU.nTrackCandidatesT5,0, sizeof(unsigned int));
    hipMemsetAsync(trackCandidatesInGPU.nTrackCandidatespT3,0, sizeof(unsigned int));
    hipMemsetAsync(trackCandidatesInGPU.nTrackCandidatespT5,0, sizeof(unsigned int));
    hipMemsetAsync(trackCandidatesInGPU.nTrackCandidatespLS,0, sizeof(unsigned int));
#ifdef TRACK_EXTENSIONS
    hipMemsetAsync(trackCandidatesInGPU.partOfExtension, false, maxTrackCandidates * sizeof(bool));
#endif
    hipStreamSynchronize(stream);
}

__device__ void SDL::addTrackCandidateToMemory(struct trackCandidates& trackCandidatesInGPU, short trackCandidateType, unsigned int innerTrackletIndex, unsigned int outerTrackletIndex, unsigned int trackCandidateIndex)
{
    trackCandidatesInGPU.trackCandidateType[trackCandidateIndex] = trackCandidateType;
    trackCandidatesInGPU.objectIndices[2 * trackCandidateIndex] = innerTrackletIndex;
    trackCandidatesInGPU.objectIndices[2 * trackCandidateIndex + 1] = outerTrackletIndex;
}

#ifdef TRACK_EXTENSIONS
__device__ void SDL::addTrackCandidateToMemory(struct trackCandidates& trackCandidatesInGPU, short trackCandidateType, unsigned int innerTrackletIndex, unsigned int outerTrackletIndex, unsigned int* logicalLayerIndices, unsigned int* lowerModuleIndices, unsigned int* hitIndices, float centerX, float centerY, float radius, unsigned int trackCandidateIndex)
{
    trackCandidatesInGPU.trackCandidateType[trackCandidateIndex] = trackCandidateType;
    trackCandidatesInGPU.objectIndices[2 * trackCandidateIndex] = innerTrackletIndex;
    trackCandidatesInGPU.objectIndices[2 * trackCandidateIndex + 1] = outerTrackletIndex;
    
    size_t limits = trackCandidateType == 7 ? 7 : 5;

    //send the starting pointer to the logicalLayer and hitIndices
    for(size_t i = 0; i < limits; i++)
    {
        trackCandidatesInGPU.logicalLayers[7 * trackCandidateIndex + i] = logicalLayerIndices[i];
        trackCandidatesInGPU.lowerModuleIndices[7 * trackCandidateIndex + i] = lowerModuleIndices[i];
    }
    for(size_t i = 0; i < 2 * limits; i++)
    {
        trackCandidatesInGPU.hitIndices[14 * trackCandidateIndex + i] = hitIndices[i];
    }
    trackCandidatesInGPU.centerX[trackCandidateIndex] = centerX;
    trackCandidatesInGPU.centerY[trackCandidateIndex] = centerY;
    trackCandidatesInGPU.radius[trackCandidateIndex] = radius;
}
#endif

SDL::trackCandidates::trackCandidates()
{
    trackCandidateType = nullptr;
    objectIndices = nullptr;
    nTrackCandidates = nullptr;
    nTrackCandidatesT5 = nullptr;
    nTrackCandidatespT3 = nullptr;
    nTrackCandidatespT5 = nullptr;
    nTrackCandidatespLS = nullptr;

#ifdef TRACK_EXTENSIONS
    logicalLayers = nullptr;
    hitIndices = nullptr;
    lowerModuleIndices = nullptr;
    partOfExtension = nullptr;
    centerX = nullptr;
    centerY = nullptr;
    radius = nullptr;
#endif
}

SDL::trackCandidates::~trackCandidates()
{
}

void SDL::trackCandidates::freeMemoryCache()
{
#ifdef Explicit_Track
    int dev;
    hipGetDevice(&dev);
    //FIXME
    //hipFree(trackCandidateType);
    cms::cuda::free_device(dev,objectIndices);
    cms::cuda::free_device(dev,trackCandidateType);
    cms::cuda::free_device(dev,nTrackCandidates);
    cms::cuda::free_device(dev,nTrackCandidatespT3);
    cms::cuda::free_device(dev,nTrackCandidatesT5);
    cms::cuda::free_device(dev,nTrackCandidatespT5);
    cms::cuda::free_device(dev,nTrackCandidatespLS);

#ifdef TRACK_EXTENSIONS
    cms::cuda::free_device(dev, logicalLayers);
    cms::cuda::free_device(dev, hitIndices);
    cms::cuda::free_device(dev, lowerModuleIndices);
    cms::cuda::free_device(dev, centerX);
    cms::cuda::free_device(dev, centerY);
    cms::cuda::free_device(dev, radius);
    cms::cuda::free_device(dev, partOfExtension);
#endif
#else
    cms::cuda::free_managed(objectIndices);
    cms::cuda::free_managed(trackCandidateType);
    cms::cuda::free_managed(nTrackCandidates);
    cms::cuda::free_managed(nTrackCandidatespT3);
    cms::cuda::free_managed(nTrackCandidatesT5);
    cms::cuda::free_managed(nTrackCandidatespT5);
    cms::cuda::free_managed(nTrackCandidatespLS);

#ifdef TRACK_EXTENSIONS
    cms::cuda::free_managed(logicalLayers);
    cms::cuda::free_managed(hitIndices);
    cms::cuda::free_managed(lowerModuleIndices);
    cms::cuda::free_managed(centerX);
    cms::cuda::free_managed(centerY);
    cms::cuda::free_managed(radius);
    cms::cuda::free_managed(partOfExtension);
#endif
#endif
}
void SDL::trackCandidates::freeMemory(hipStream_t stream)
{
    hipFree(trackCandidateType);
    hipFree(objectIndices);
    hipFree(nTrackCandidates);
    hipFree(nTrackCandidatespT3);
    hipFree(nTrackCandidatesT5);
    hipFree(nTrackCandidatespT5);
    hipFree(nTrackCandidatespLS);

#ifdef TRACK_EXTENSIONS
    hipFree(logicalLayers);
    hipFree(hitIndices);
    hipFree(lowerModuleIndices);
    hipFree(partOfExtension);
    hipFree(centerX);
    hipFree(centerY);
    hipFree(radius);
#endif
    
    hipStreamSynchronize(stream);
}
