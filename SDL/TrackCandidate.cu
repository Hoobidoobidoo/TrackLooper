#include "TrackCandidate.cuh"

#include "allocate.h"


void SDL::createTrackCandidatesInUnifiedMemory(struct trackCandidates& trackCandidatesInGPU, unsigned int maxTrackCandidates,hipStream_t stream)
{
#ifdef CACHE_ALLOC
//    hipStream_t stream=0;
    trackCandidatesInGPU.trackCandidateType = (short*)cms::cuda::allocate_managed(maxTrackCandidates * sizeof(short),stream);
//    hipMallocManaged(&trackCandidatesInGPU.objectIndices, 2 * maxTrackCandidates * sizeof(unsigned int));
    trackCandidatesInGPU.objectIndices = (unsigned int*)cms::cuda::allocate_managed(maxTrackCandidates * 2*sizeof(unsigned int),stream);
    trackCandidatesInGPU.nTrackCandidates= (unsigned int*)cms::cuda::allocate_managed( sizeof(unsigned int),stream);
    trackCandidatesInGPU.nTrackCandidatespT3= (unsigned int*)cms::cuda::allocate_managed(sizeof(unsigned int),stream);
    trackCandidatesInGPU.nTrackCandidatesT5= (unsigned int*)cms::cuda::allocate_managed( sizeof(unsigned int),stream);
    trackCandidatesInGPU.nTrackCandidatespT5= (unsigned int*)cms::cuda::allocate_managed( sizeof(unsigned int),stream);
    trackCandidatesInGPU.nTrackCandidatespLS= (unsigned int*)cms::cuda::allocate_managed( sizeof(unsigned int),stream);

#else
    hipMallocManaged(&trackCandidatesInGPU.trackCandidateType, maxTrackCandidates * sizeof(short));
    hipMallocManaged(&trackCandidatesInGPU.objectIndices, 2 * maxTrackCandidates * sizeof(unsigned int));
    hipMallocManaged(&trackCandidatesInGPU.nTrackCandidates, sizeof(unsigned int));
    hipMallocManaged(&trackCandidatesInGPU.nTrackCandidatespT3, sizeof(unsigned int));
    hipMallocManaged(&trackCandidatesInGPU.nTrackCandidatesT5, sizeof(unsigned int));
    hipMallocManaged(&trackCandidatesInGPU.nTrackCandidatespT5, sizeof(unsigned int));

    hipMallocManaged(&trackCandidatesInGPU.nTrackCandidatespLS, sizeof(unsigned int));
#endif

    hipMemsetAsync(trackCandidatesInGPU.nTrackCandidates,0, sizeof(unsigned int),stream);
    hipMemsetAsync(trackCandidatesInGPU.nTrackCandidatesT5,0, sizeof(unsigned int),stream);
    hipMemsetAsync(trackCandidatesInGPU.nTrackCandidatespT3,0, sizeof(unsigned int),stream);
    hipMemsetAsync(trackCandidatesInGPU.nTrackCandidatespT5,0, sizeof(unsigned int),stream);
    hipMemsetAsync(trackCandidatesInGPU.nTrackCandidatespLS,0, sizeof(unsigned int),stream);
}
void SDL::createTrackCandidatesInExplicitMemory(struct trackCandidates& trackCandidatesInGPU, unsigned int maxTrackCandidates,hipStream_t stream)
{
#ifdef CACHE_ALLOC
//    hipStream_t stream=0;
    int dev;
    hipGetDevice(&dev);
    //TODO 
    //hipMalloc(&trackCandidatesInGPU.trackCandidateType, maxTrackCandidates * sizeof(short));
    //hipMalloc(&trackCandidatesInGPU.objectIndices, 2 * maxTrackCandidates * sizeof(unsigned int));
    trackCandidatesInGPU.trackCandidateType = (short*)cms::cuda::allocate_device(dev,maxTrackCandidates * sizeof(short),stream);
    trackCandidatesInGPU.objectIndices = (unsigned int*)cms::cuda::allocate_device(dev,maxTrackCandidates * 2*sizeof(unsigned int),stream);
    trackCandidatesInGPU.nTrackCandidates= (unsigned int*)cms::cuda::allocate_device(dev, sizeof(unsigned int),stream);
    trackCandidatesInGPU.nTrackCandidatespT3= (unsigned int*)cms::cuda::allocate_device(dev, sizeof(unsigned int),stream);
    trackCandidatesInGPU.nTrackCandidatesT5= (unsigned int*)cms::cuda::allocate_device(dev, sizeof(unsigned int),stream);
    trackCandidatesInGPU.nTrackCandidatespT5= (unsigned int*)cms::cuda::allocate_device(dev, sizeof(unsigned int),stream);
    trackCandidatesInGPU.nTrackCandidatespLS= (unsigned int*)cms::cuda::allocate_device(dev, sizeof(unsigned int),stream);
    //trackCandidatesInGPU.partOfExtension = (bool*)cms::cuda::allocate_device(dev, maxTrackCandidates * sizeof(bool), stream);

#else
    hipMallocAsync(&trackCandidatesInGPU.trackCandidateType, maxTrackCandidates * sizeof(short),stream);
    hipMallocAsync(&trackCandidatesInGPU.objectIndices, 2 * maxTrackCandidates * sizeof(unsigned int),stream);
    hipMallocAsync(&trackCandidatesInGPU.nTrackCandidates, sizeof(unsigned int),stream);
    hipMallocAsync(&trackCandidatesInGPU.nTrackCandidatespT3, sizeof(unsigned int),stream);
    hipMallocAsync(&trackCandidatesInGPU.nTrackCandidatesT5, sizeof(unsigned int),stream);
    hipMallocAsync(&trackCandidatesInGPU.nTrackCandidatespT5, sizeof(unsigned int),stream);
    hipMallocAsync(&trackCandidatesInGPU.nTrackCandidatespLS, sizeof(unsigned int),stream);
#endif
    hipMemsetAsync(trackCandidatesInGPU.nTrackCandidates,0, sizeof(unsigned int),stream);
    hipMemsetAsync(trackCandidatesInGPU.nTrackCandidatesT5,0, sizeof(unsigned int),stream);
    hipMemsetAsync(trackCandidatesInGPU.nTrackCandidatespT3,0, sizeof(unsigned int),stream);
    hipMemsetAsync(trackCandidatesInGPU.nTrackCandidatespT5,0, sizeof(unsigned int),stream);
    hipMemsetAsync(trackCandidatesInGPU.nTrackCandidatespLS,0, sizeof(unsigned int),stream);
  hipStreamSynchronize(stream);
}

__device__ void SDL::addTrackCandidateToMemory(struct trackCandidates& trackCandidatesInGPU, short trackCandidateType, unsigned int innerTrackletIndex, unsigned int outerTrackletIndex, unsigned int trackCandidateIndex)
{
    trackCandidatesInGPU.trackCandidateType[trackCandidateIndex] = trackCandidateType;
    trackCandidatesInGPU.objectIndices[2 * trackCandidateIndex] = innerTrackletIndex;
    trackCandidatesInGPU.objectIndices[2 * trackCandidateIndex + 1] = outerTrackletIndex;
}

SDL::trackCandidates::trackCandidates()
{
    trackCandidateType = nullptr;
    objectIndices = nullptr;
    nTrackCandidates = nullptr;
    nTrackCandidatesT5 = nullptr;
    nTrackCandidatespT3 = nullptr;
    nTrackCandidatespT5 = nullptr;
    nTrackCandidatespLS = nullptr;
}

SDL::trackCandidates::~trackCandidates()
{
}

void SDL::trackCandidates::freeMemoryCache()
{
#ifdef Explicit_Track
    int dev;
    hipGetDevice(&dev);
    //FIXME
    //hipFree(trackCandidateType);
    cms::cuda::free_device(dev,objectIndices);
    cms::cuda::free_device(dev,trackCandidateType);
    cms::cuda::free_device(dev,nTrackCandidates);
    cms::cuda::free_device(dev,nTrackCandidatespT3);
    cms::cuda::free_device(dev,nTrackCandidatesT5);
    cms::cuda::free_device(dev,nTrackCandidatespT5);


    cms::cuda::free_device(dev,nTrackCandidatespLS);
#else
    cms::cuda::free_managed(objectIndices);
    cms::cuda::free_managed(trackCandidateType);
    cms::cuda::free_managed(nTrackCandidates);
    cms::cuda::free_managed(nTrackCandidatespT3);
    cms::cuda::free_managed(nTrackCandidatesT5);
    cms::cuda::free_managed(nTrackCandidatespT5);
    cms::cuda::free_managed(nTrackCandidatespLS);
#endif
//    hipFree(objectIndices);

}
void SDL::trackCandidates::freeMemory(hipStream_t stream)
{
    hipFreeAsync(trackCandidateType,stream);
    hipFreeAsync(objectIndices,stream);
    hipFreeAsync(nTrackCandidates,stream);
    hipFreeAsync(nTrackCandidatespT3,stream);
    hipFreeAsync(nTrackCandidatesT5,stream);
    hipFreeAsync(nTrackCandidatespT5,stream);
    hipFreeAsync(nTrackCandidatespLS,stream);
hipStreamSynchronize(stream);
}

